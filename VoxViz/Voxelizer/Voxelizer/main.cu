#include "hip/hip_runtime.h"
#include "Voxelizer.h"

#include "VoxelBrickWriter.h"

#include <cmath>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <map>

#include <osg/ArgumentParser>
#include <osg/Geometry>
#include <osg/PrimitiveSet>
#include <osg/Node>
#include <osg/Matrix>
#include <osg/MatrixTransform>
#include <osg/TriangleFunctor>
#include <osg/ValueObject>
#include <osgDB/FileUtils>
#include <osgDB/FileNameUtils>
#include <osgDB/ReadFile>
#include <osg/NodeVisitor>
#include <osg/Geode>
#include <osg/Texture2D>
#include <osg/PagedLOD>
#include <osg/ProxyNode>
#include <osg/ref_ptr>
#include <osgDB/Registry>
#include <osgDB/ReaderWriter>
#include <osgUtil/SmoothingVisitor>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

class Primitive
{
public:
    
    Primitive() : m_worldTransformIsIdentity(true),  m_faceNormalDirty(true){}
    
    ~Primitive() {}
        
    osg::Geometry* getGeometry() { return m_spGeometry.get(); }
    const osg::Geometry* getGeometry() const { return m_spGeometry.get(); }
        
    osg::PrimitiveSet* getPrimitiveSet()
    {
        return m_spPrimSet.get();
    }
        
    void setGeometry(osg::Geometry* newGeometry)
    {
        m_spGeometry = newGeometry;
    }
        
    void setPrimitiveSet(osg::PrimitiveSet* primSet)
    {
        m_spPrimSet = primSet;
    }
    
    void setIndices(unsigned int index1,
                    unsigned int index2,
                    unsigned int index3)
    {
        m_indices.clear();
        m_indices.push_back(index1);
        m_indices.push_back(index2);
        m_indices.push_back(index3);
    }

    unsigned int getVertexIndex(unsigned int pos) const
    {
        return m_indices[pos];
    }
    
    void setIndices(unsigned int index1,
                    unsigned int index2,
                    unsigned int index3,
                    unsigned int index4)
    {
        m_indices.clear();
        m_indices.push_back(index1);
        m_indices.push_back(index2);
        m_indices.push_back(index3);
        m_indices.push_back(index4);
    }

    void setWorldTransform(osg::RefMatrix* pWorldTransform)
    {
        if(!(m_worldTransformIsIdentity = pWorldTransform->isIdentity()))
        {
            m_spWorldTransform = pWorldTransform;
            m_faceNormalDirty = true;
        }
    }

    void setWorldTransform(const osg::Matrix& worldTransform)
    {
        if(!(m_worldTransformIsIdentity = worldTransform.isIdentity()))
        {
            m_spWorldTransform = new osg::RefMatrix(worldTransform);
            m_faceNormalDirty = true;
        }
    }

    const osg::RefMatrix* getWorldTransform() const { return m_spWorldTransform.get(); }
    
    void setPrimSetIndex(int primSetIndex) { m_primSetIndex = primSetIndex; }
        
    unsigned int getPrimSetIndex() const { return m_primSetIndex; }
    
    bool getVertex(unsigned int index, osg::Vec3& vtx) const;
    osg::Vec3 getVertex(unsigned int index) const;
    
    unsigned int getVtxCount() const { return m_indices.size(); }
    
    bool hasColorData() const
    {
        if(!m_spGeometry.valid())
            return false;
    
        return m_spGeometry->getColorArray() != NULL &&
                m_spGeometry->getColorArray()->getNumElements() > 0;
    }
    
    osg::Geometry::AttributeBinding getColorBinding() const
    {
        if(!m_spGeometry.valid())
            return osg::Geometry::BIND_OFF;
    
        return m_spGeometry->getColorBinding();
    }
    
    bool getColor(unsigned int index, osg::Vec4& color) const;
    osg::Vec4ub getColor(unsigned int index) const
    {
        osg::Vec4ub ret(UCHAR_MAX, 
                        UCHAR_MAX, 
                        UCHAR_MAX, 
                        UCHAR_MAX);
        osg::Vec4 rgba;
        if(getColor(index, rgba))
        {
            ret.r() = static_cast<unsigned char>(rgba.r() * 255.0f);
            ret.g() = static_cast<unsigned char>(rgba.g() * 255.0f);
            ret.b() = static_cast<unsigned char>(rgba.b() * 255.0f);
            ret.a() = static_cast<unsigned char>(rgba.a() * 255.0f);               
        }

        return ret;
    }

    bool hasNormalData() const
    {
        if(!m_spGeometry.valid())
            return false;
    
        return m_spGeometry->getNormalArray() != NULL &&
                m_spGeometry->getNormalArray()->getNumElements() > 0;
    }
    
    osg::Geometry::AttributeBinding getNormalBinding() const
    {
        if(!m_spGeometry.valid())
            return osg::Geometry::BIND_OFF;
    
        return m_spGeometry->getNormalBinding();
    }
    
    bool getNormal(unsigned int index, osg::Vec3& normal) const;
    osg::Vec3 getNormal(unsigned int index) const
    {
        osg::Vec3 ret(0.0f, 0.0f, 1.0f);
        getNormal(index, ret);

        return ret;
    }
    
    bool hasTextureData() const
    {
        if(!m_spGeometry.valid())
            return false;
    
        return m_spGeometry->getNumTexCoordArrays() > 0 &&
                m_spGeometry->getTexCoordArray(0) != NULL && 
                m_spGeometry->getTexCoordArray(0)->getNumElements() > 0;
    }

    unsigned int getNumTextureCoordArrays() const
    {
        return m_spGeometry->getNumTexCoordArrays();
    }
    
    bool getTexCoord(unsigned int texCoordArrayIndex, unsigned int index, osg::Vec2& texCoord) const;
    osg::Vec2 getTexCoord(unsigned int texCoordArrayIndex, unsigned int index) const
    {
        osg::Vec2 ret;
        getTexCoord(texCoordArrayIndex, index, ret);

        return ret;
    }
    
    const osg::Vec3& getFaceNormal() const
    {
        if(m_faceNormalDirty)
        {
            calcFaceNormal();
            m_faceNormalDirty = false;
        }
        return m_faceNormal;
    }
    
    void deleteMe();
     
private:
    
    void calcFaceNormal() const
    {
        osg::Vec3 vtx0;
        getVertex(0, vtx0);
            
        osg::Vec3 vtx1;
        getVertex(1, vtx1);
            
        osg::Vec3 vtx2;
        getVertex(2, vtx2);

        m_faceNormal = calcNormal(vtx0, vtx1, vtx2);
    }
    
    osg::Vec3 calcNormal(const osg::Vec3& v0, const osg::Vec3& v1, const osg::Vec3& v2) const
    {
        float ux, uy, uz, vx, vy, vz;
    
        ux = v1.x() - v0.x();
        uy = v1.y() - v0.y();
        uz = v1.z() - v0.z();
        vx = v2.x() - v0.x();
        vy = v2.y() - v0.y();
        vz = v2.z() - v0.z();
    
        osg::Vec3 norm(uy*vz - uz*vy,
                        uz*vx - ux*vz,
                        ux*vy - uy*vx);
            
        float oneOverLen = 1.0f/norm.length();
    
        norm.x() *= oneOverLen;           
        norm.y() *= oneOverLen;           
        norm.z() *= oneOverLen;           
    
        return norm;
    }
    
    osg::ref_ptr<osg::Geometry> m_spGeometry;
    osg::ref_ptr<osg::PrimitiveSet> m_spPrimSet;
    osg::ref_ptr<osg::RefMatrix> m_spWorldTransform;
    
    std::vector<unsigned int> m_indices;
        
    unsigned int m_primSetIndex;           //index of polygon in primitive set
    
    bool m_worldTransformIsIdentity;
    mutable bool m_faceNormalDirty;
    mutable osg::Vec3 m_faceNormal;
};
    
class PrimitiveSetIterator : public osg::Referenced
{
public:
    
    PrimitiveSetIterator();
    
    virtual ~PrimitiveSetIterator();
    
    virtual void init(osg::Geometry* searchGeometry, osg::PrimitiveSet* searchPrimSet)=0;
    void reset()
    {
        m_spGeometry = NULL;
        m_spPrimSet = NULL;
        m_numberOfPolys = 0;
    }
    /* virtual method implemented by individual instances that inherit 
        this class.Returns the last created RequestedPoly data structure */
    virtual bool getPoly(unsigned int index, Primitive& poly) = 0;
    
    unsigned int getNumberOfPolys() const
    {
        return m_numberOfPolys;
    }

    unsigned int getNumberOfVerticesPerPoly() const
    {
        Primitive poly;
        const_cast<PrimitiveSetIterator*>(this)->getPoly(0, poly);

        return poly.getVtxCount();
    }

    static PrimitiveSetIterator* getIterator(osg::PrimitiveSet* primitiveSet, osg::Geometry* geometry);
    static unsigned int getNumberOfSupportedModes() { return 5; }
    static unsigned int getNumberOfSupportedTypes() { return 5; }
    
    const osg::Geometry* getGeometry() const { return m_spGeometry.get(); }
    const osg::PrimitiveSet* getPrimitiveSet() const { return m_spPrimSet.get(); }
protected:
    
    //osg::Geometry* geometry;
    osg::ref_ptr<osg::Geometry> m_spGeometry;
    osg::ref_ptr<osg::PrimitiveSet> m_spPrimSet;
    unsigned int m_numberOfPolys;
};

class TriangleDrawArraysIterator : public PrimitiveSetIterator
{
  public:

    TriangleDrawArraysIterator() {};
    
    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static TriangleDrawArraysIterator* TriangleDrawArraysIteratorInit(osg::Geometry* searchGeometry, osg::PrimitiveSet* primSet)
{
    TriangleDrawArraysIterator* triDrawArraysItr = new TriangleDrawArraysIterator();
    triDrawArraysItr->init(searchGeometry, primSet);
    return triDrawArraysItr;
}


class TriangleDrawArrayLengthsIterator : public PrimitiveSetIterator
{
  public:

    TriangleDrawArrayLengthsIterator() {};
    
    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);

};

static TriangleDrawArrayLengthsIterator* TriangleDrawArrayLengthsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleDrawArrayLengthsIterator* triDrawArrayLengthsItr = new TriangleDrawArrayLengthsIterator();
    triDrawArrayLengthsItr->init(geom, primSet);
    return triDrawArrayLengthsItr;
}

class TriangleDrawElementsIterator : public PrimitiveSetIterator
{
  public:

    TriangleDrawElementsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};


static TriangleDrawElementsIterator* TriangleDrawElementsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleDrawElementsIterator* triDrawElemsItr = new TriangleDrawElementsIterator();
    triDrawElemsItr->init(geom, primSet);
    return triDrawElemsItr;
}

class TriangleStripDrawArraysIterator : public PrimitiveSetIterator
{
  public:

    TriangleStripDrawArraysIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static TriangleStripDrawArraysIterator* TriangleStripDrawArraysIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleStripDrawArraysIterator* triStripDrawArraysItr = new TriangleStripDrawArraysIterator();
    triStripDrawArraysItr->init(geom, primSet);
    return triStripDrawArraysItr;
}

class TriangleStripDrawArrayLengthsIterator : public PrimitiveSetIterator
{
  public:

    TriangleStripDrawArrayLengthsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static TriangleStripDrawArrayLengthsIterator* TriangleStripDrawArrayLengthsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleStripDrawArrayLengthsIterator* triStripDrawArrayLengthsItr = new TriangleStripDrawArrayLengthsIterator();
    triStripDrawArrayLengthsItr->init(geom, primSet);
    return triStripDrawArrayLengthsItr;
}

class TriangleStripDrawElementsIterator : public PrimitiveSetIterator
{
  public:

    TriangleStripDrawElementsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};


static TriangleStripDrawElementsIterator* TriangleStripDrawElementsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleStripDrawElementsIterator* triStripDrawElemsItr = new TriangleStripDrawElementsIterator();
    triStripDrawElemsItr->init(geom, primSet);
    return triStripDrawElemsItr;
}

class TriangleFanDrawArraysIterator : public PrimitiveSetIterator
{
  public:

    TriangleFanDrawArraysIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);

};

static TriangleFanDrawArraysIterator* TriangleFanDrawArraysIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleFanDrawArraysIterator* triFanDrawArraysItr = new TriangleFanDrawArraysIterator();
    triFanDrawArraysItr->init(geom, primSet);
    return triFanDrawArraysItr;
}

class TriangleFanDrawArrayLengthsIterator : public PrimitiveSetIterator
{
  public:

    TriangleFanDrawArrayLengthsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static TriangleFanDrawArrayLengthsIterator* TriangleFanDrawArrayLengthsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleFanDrawArrayLengthsIterator* triFanDrawArrayLengthsItr = new TriangleFanDrawArrayLengthsIterator();
    triFanDrawArrayLengthsItr->init(geom, primSet);
    return triFanDrawArrayLengthsItr;
}

class TriangleFanDrawElementsIterator : public PrimitiveSetIterator
{
  public:

    TriangleFanDrawElementsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static TriangleFanDrawElementsIterator* TriangleFanDrawElementsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    TriangleFanDrawElementsIterator* triFanDrawElemsItr = new TriangleFanDrawElementsIterator();
    triFanDrawElemsItr->init(geom, primSet);
    return triFanDrawElemsItr;
}

class QuadDrawArraysIterator : public PrimitiveSetIterator
{
  public:

    QuadDrawArraysIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static QuadDrawArraysIterator* QuadDrawArraysIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    QuadDrawArraysIterator* quadDrawArraysItr = new QuadDrawArraysIterator();
    quadDrawArraysItr->init(geom, primSet);
    return quadDrawArraysItr;
}

class QuadDrawArrayLengthsIterator : public PrimitiveSetIterator
{
  public:

    QuadDrawArrayLengthsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static QuadDrawArrayLengthsIterator* QuadDrawArrayLengthsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    QuadDrawArrayLengthsIterator* quadDrawArrayLengthsItr = new QuadDrawArrayLengthsIterator();
    quadDrawArrayLengthsItr->init(geom, primSet);
    return quadDrawArrayLengthsItr;
}

class QuadDrawElementsIterator : public PrimitiveSetIterator
{
  public:

    QuadDrawElementsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static QuadDrawElementsIterator* QuadDrawElementsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    QuadDrawElementsIterator* quadDrawElemsItr = new QuadDrawElementsIterator();
    quadDrawElemsItr->init(geom, primSet);
    return quadDrawElemsItr;
}

class QuadStripDrawArraysIterator : public PrimitiveSetIterator
{
  public:

    QuadStripDrawArraysIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static QuadStripDrawArraysIterator* QuadStripDrawArraysIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    QuadStripDrawArraysIterator* quadStripDrawArraysItr = new QuadStripDrawArraysIterator();
    quadStripDrawArraysItr->init(geom, primSet);
    return quadStripDrawArraysItr;
}

class QuadStripDrawArrayLengthsIterator : public PrimitiveSetIterator
{
  public:

    QuadStripDrawArrayLengthsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static QuadStripDrawArrayLengthsIterator* QuadStripDrawArrayLengthsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    QuadStripDrawArrayLengthsIterator* quadStripDrawArrayLengthsItr = new QuadStripDrawArrayLengthsIterator();
    quadStripDrawArrayLengthsItr->init(geom, primSet);
    return quadStripDrawArrayLengthsItr;
}

class QuadStripDrawElementsIterator : public PrimitiveSetIterator
{
  public:

    QuadStripDrawElementsIterator() {};

    virtual void init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet);

    virtual bool getPoly(unsigned int index, Primitive& poly);
};

static QuadStripDrawElementsIterator* QuadStripDrawElementsIteratorInit(osg::Geometry* geom, osg::PrimitiveSet* primSet)
{
    QuadStripDrawElementsIterator* quadStripDrawElemsItr = new QuadStripDrawElementsIterator();
    quadStripDrawElemsItr->init(geom, primSet);
    return quadStripDrawElemsItr;
}

/*
  Method which returns the right primitiveSetIterator class instance
*/
PrimitiveSetIterator* PrimitiveSetIterator::getIterator(osg::PrimitiveSet* primitiveSet, osg::Geometry* geometry)
{
    if (primitiveSet->getMode() == osg::PrimitiveSet::QUADS)
    {
        switch(primitiveSet->getType())
        {
        case osg::PrimitiveSet::DrawArraysPrimitiveType:
            return QuadDrawArraysIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType:
            return QuadDrawArrayLengthsIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            return QuadDrawElementsIteratorInit(geometry, primitiveSet);

        default:
            return NULL;
        }
    }
    else if (primitiveSet->getMode() == osg::PrimitiveSet::QUAD_STRIP)
    {
        switch(primitiveSet->getType())
        {
        case osg::PrimitiveSet::DrawArraysPrimitiveType:
            return QuadStripDrawArraysIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType:
            return QuadStripDrawArrayLengthsIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            return QuadStripDrawElementsIteratorInit(geometry, primitiveSet);

        default:
            return NULL;
        }
    }
    else if (primitiveSet->getMode() == osg::PrimitiveSet::TRIANGLES)
    {
        switch(primitiveSet->getType())
        {
        case osg::PrimitiveSet::DrawArraysPrimitiveType:
            return TriangleDrawArraysIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType:
            return TriangleDrawArrayLengthsIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            return TriangleDrawElementsIteratorInit(geometry, primitiveSet);

        default:
            return NULL;
        }
    }
    else if (primitiveSet->getMode() == osg::PrimitiveSet::TRIANGLE_STRIP)
    {
        switch(primitiveSet->getType())
        {
        case osg::PrimitiveSet::DrawArraysPrimitiveType:
            return TriangleStripDrawArraysIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType:
            return TriangleStripDrawArrayLengthsIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            return TriangleStripDrawElementsIteratorInit(geometry, primitiveSet);

        default:
            return NULL;
        }
    }
    else if (primitiveSet->getMode() == osg::PrimitiveSet::TRIANGLE_FAN)
    {
        switch(primitiveSet->getType())
        {
        case osg::PrimitiveSet::DrawArraysPrimitiveType:
            return TriangleFanDrawArraysIteratorInit(geometry, primitiveSet);

        case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType:
            return TriangleFanDrawArrayLengthsIteratorInit(geometry, primitiveSet);
        
        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            return TriangleFanDrawElementsIteratorInit(geometry, primitiveSet);

        default:
            return NULL;
        }
    }
    
    return NULL;
}

class DrawElementsAnyType
{
  public:
    DrawElementsAnyType(osg::PrimitiveSet* primSet) :
        m_spPrimSet(primSet)
    {
        
    }
    
    DrawElementsAnyType(const DrawElementsAnyType& copy) :
        m_spPrimSet(NULL)
    {
        const osg::PrimitiveSet* primSet = copy.getPrimitiveSet();
        switch(primSet->getType())
        {
        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            {
                m_spPrimSet = new osg::DrawElementsUByte(static_cast<const osg::DrawElementsUByte&>(*primSet), 0); //0 for shallow copy
                break;
            }
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
            {
                m_spPrimSet = new osg::DrawElementsUShort(static_cast<const osg::DrawElementsUShort&>(*primSet), 0);
                break;
            }
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            {
                m_spPrimSet = new osg::DrawElementsUInt(static_cast<const osg::DrawElementsUInt&>(*primSet), 0);
                break;
            }
        default:
            {
                throw std::string("DrawElementsAnyType(): invalid type");
            }
        }
    }

    const osg::PrimitiveSet* getPrimitiveSet() const { return m_spPrimSet.get(); }
    osg::PrimitiveSet* getPrimitiveSet() { return m_spPrimSet.get(); }

    unsigned int at(unsigned int index) const
    {
        switch(m_spPrimSet->getType())
        {
        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            return static_cast<osg::DrawElementsUByte*>(m_spPrimSet.get())->at(index);
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
            return static_cast<osg::DrawElementsUShort*>(m_spPrimSet.get())->at(index);
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            return static_cast<osg::DrawElementsUInt*>(m_spPrimSet.get())->at(index);
        default:
            {
                throw std::string("DrawElementsAnyType(): invalid type");
            }
        }
    }

    unsigned int size() const
    {
        switch(m_spPrimSet->getType())
        {
        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            return static_cast<osg::DrawElementsUByte*>(m_spPrimSet.get())->size();
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
            return static_cast<osg::DrawElementsUShort*>(m_spPrimSet.get())->size();
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            return static_cast<osg::DrawElementsUInt*>(m_spPrimSet.get())->size();
        default:
            {
                throw std::string("DrawElementsAnyType(): invalid type");
            }
        }
    }
    
    unsigned int begin() const
    {
        return 0;
    }

    unsigned int end()
    {
        return size();
    }
    
    void erase(unsigned int index)
    {
        switch(m_spPrimSet->getType())
        {
        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            {
                osg::DrawElementsUByte* primSet = static_cast<osg::DrawElementsUByte*>(m_spPrimSet.get());
                primSet->erase(primSet->begin() + index);
                break;
            }
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
            {
                osg::DrawElementsUShort* primSet = static_cast<osg::DrawElementsUShort*>(m_spPrimSet.get());
                primSet->erase(primSet->begin() + index);
                break;
            }
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            {
                osg::DrawElementsUInt* primSet = static_cast<osg::DrawElementsUInt*>(m_spPrimSet.get());
                primSet->erase(primSet->begin() + index);
                break;
            }
        default:
            {
                throw std::string("DrawElementsAnyType(): invalid type");
            }
        }
    }

    void erase(unsigned int start, unsigned int end)
    {
        switch(m_spPrimSet->getType())
        {
        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            {
                osg::DrawElementsUByte* primSet = static_cast<osg::DrawElementsUByte*>(m_spPrimSet.get());
                primSet->erase(primSet->begin() + start, primSet->begin() + end);
                break;
            }
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
            {
                osg::DrawElementsUShort* primSet = static_cast<osg::DrawElementsUShort*>(m_spPrimSet.get());
                primSet->erase(primSet->begin() + start, primSet->begin() + end);
                break;
            }
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            {
                osg::DrawElementsUInt* primSet = static_cast<osg::DrawElementsUInt*>(m_spPrimSet.get());
                primSet->erase(primSet->begin() + start, primSet->begin() + end);
                break;
            }
        default:
            {
                throw std::string("DrawElementsAnyType(): invalid type");
            }
        }
    }
    
    void insert(unsigned int index, unsigned int value)
    {
        switch(m_spPrimSet->getType())
        {
        case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            {
                osg::DrawElementsUByte* primSet = static_cast<osg::DrawElementsUByte*>(m_spPrimSet.get());
                primSet->insert(primSet->begin() + index, (osg::DrawElementsUByte::value_type)value);
                break;
            }
        case osg::PrimitiveSet::DrawElementsUShortPrimitiveType:
            {
                osg::DrawElementsUShort* primSet = static_cast<osg::DrawElementsUShort*>(m_spPrimSet.get());
                primSet->insert(primSet->begin() + index, (osg::DrawElementsUShort::value_type)value);
                break;
            }
        case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
            {
                osg::DrawElementsUInt* primSet = static_cast<osg::DrawElementsUInt*>(m_spPrimSet.get());
                primSet->insert(primSet->begin() + index, (osg::DrawElementsUInt::value_type)value);
                break;
            }
        default:
            {
                throw std::string("DrawElementsAnyType(): invalid type");
            }
        }
    }

  private:
    osg::ref_ptr<osg::PrimitiveSet> m_spPrimSet;
};


class PrimitiveSetEraser
{
  public:
    PrimitiveSetEraser() {}
    virtual ~PrimitiveSetEraser() {}

    virtual void deletePoly(Primitive& poly)=0;
    static PrimitiveSetEraser* getPrimitiveSetEraser(osg::PrimitiveSet& primSet);
  protected:
    void updateColorsAndNormals(Primitive& poly, int primSetsAdded);
};

class TriangleDrawArraysEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleDrawArraysEraser g_triangleDrawArraysEraser;

class TriangleDrawArrayLengthsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleDrawArrayLengthsEraser g_triangleDrawArrayLengthsEraser;

class TriangleDrawElementsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleDrawElementsEraser g_triangleDrawElementsEraser;

class TriangleStripDrawArraysEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleStripDrawArraysEraser g_triangleStripDrawArraysEraser;

class TriangleStripDrawArrayLengthsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleStripDrawArrayLengthsEraser g_triangleStripDrawArrayLengthsEraser;

class TriangleStripDrawElementsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleStripDrawElementsEraser g_triangleStripDrawElementsEraser;

class TriangleFanDrawArraysEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleFanDrawArraysEraser g_triangleFanDrawArraysEraser;

class TriangleFanDrawArrayLengthsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleFanDrawArrayLengthsEraser g_triangleFanDrawArrayLengthsEraser;

class TriangleFanDrawElementsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static TriangleFanDrawElementsEraser g_triangleFanDrawElementsEraser;

class QuadDrawArraysEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static QuadDrawArraysEraser g_quadDrawArraysEraser;

class QuadDrawArrayLengthsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static QuadDrawArrayLengthsEraser g_quadDrawArrayLengthsEraser;

class QuadDrawElementsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static QuadDrawElementsEraser g_quadDrawElementsEraser;

class QuadStripDrawArraysEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static QuadStripDrawArraysEraser g_quadStripDrawArraysEraser;

class QuadStripDrawArrayLengthsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static QuadStripDrawArrayLengthsEraser g_quadStripDrawArrayLengthsEraser;

class QuadStripDrawElementsEraser : public PrimitiveSetEraser
{
  public:
    virtual void deletePoly(Primitive& poly);
};

static QuadStripDrawElementsEraser g_quadStripDrawElementsEraser;

PrimitiveSetEraser* PrimitiveSetEraser::getPrimitiveSetEraser(osg::PrimitiveSet& primSet)
{
    switch(primSet.getMode())
    {
    case osg::PrimitiveSet::TRIANGLES:
        {
            switch(primSet.getType())
            {
            case osg::PrimitiveSet::DrawArraysPrimitiveType:
                return &g_triangleDrawArraysEraser;

            case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType: 
                return &g_triangleDrawArrayLengthsEraser;

            case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            case osg::PrimitiveSet::DrawElementsUShortPrimitiveType: 
            case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
                return &g_triangleDrawElementsEraser;
            
            default:
                return NULL;
            }
            break;
        }
    case osg::PrimitiveSet::TRIANGLE_STRIP:
        {
            switch(primSet.getType())
            {
            case osg::PrimitiveSet::DrawArraysPrimitiveType:
                return &g_triangleStripDrawArraysEraser;

            case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType: 
                return &g_triangleStripDrawArrayLengthsEraser;

            case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            case osg::PrimitiveSet::DrawElementsUShortPrimitiveType: 
            case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
                return &g_triangleStripDrawElementsEraser;
            
            default:
                return NULL;

            }
            break;
        }
    case osg::PrimitiveSet::TRIANGLE_FAN:
        {
            switch(primSet.getType())
            {
            case osg::PrimitiveSet::DrawArraysPrimitiveType:
                return &g_triangleFanDrawArraysEraser;

            case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType: 
                return &g_triangleFanDrawArrayLengthsEraser;

            case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            case osg::PrimitiveSet::DrawElementsUShortPrimitiveType: 
            case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
                return &g_triangleFanDrawElementsEraser;
            
            default:
                return NULL;

            }
            break;
        }
    case osg::PrimitiveSet::QUADS:
        {
            switch(primSet.getType())
            {
            case osg::PrimitiveSet::DrawArraysPrimitiveType:
                return &g_quadDrawArraysEraser;

            case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType: 
                return &g_quadDrawArrayLengthsEraser;

            case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            case osg::PrimitiveSet::DrawElementsUShortPrimitiveType: 
            case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
                return &g_quadDrawElementsEraser;
            
            default:
                return NULL;

            }
            break;
        }
    case osg::PrimitiveSet::QUAD_STRIP:
        {
            switch(primSet.getType())
            {
            case osg::PrimitiveSet::DrawArraysPrimitiveType:
                return &g_quadStripDrawArraysEraser;

            case osg::PrimitiveSet::DrawArrayLengthsPrimitiveType: 
                return &g_quadStripDrawArrayLengthsEraser;

            case osg::PrimitiveSet::DrawElementsUBytePrimitiveType:
            case osg::PrimitiveSet::DrawElementsUShortPrimitiveType: 
            case osg::PrimitiveSet::DrawElementsUIntPrimitiveType:
                return &g_quadStripDrawElementsEraser;
            
            default:
                return NULL;

            }
            break;
        }
    default:
        {
            //do nothing
            break;
        }
    }
    
    return NULL;
}

/*
  ----------------------
  Primitive CLASS:
  ----------------------
*/

void Primitive::deleteMe()
{
    if(!m_spGeometry.valid() || !m_spPrimSet.valid())
        return;

    PrimitiveSetEraser* eraser = PrimitiveSetEraser::getPrimitiveSetEraser(*m_spPrimSet.get());
    if(eraser == NULL)
        return;

    eraser->deletePoly(*this);

    //set these to null so now you can't try to delete this poly again
    m_spGeometry = NULL;
    m_spPrimSet = NULL;
}

inline unsigned int getIndex(unsigned int pos, const osg::IndexArray* indices)
{
    switch(indices->getType())
    {
        case(osg::Array::ByteArrayType): return (*static_cast<const osg::ByteArray*>(indices))[pos];
        case(osg::Array::ShortArrayType): return (*static_cast<const osg::ShortArray*>(indices))[pos];
        case(osg::Array::IntArrayType): return (*static_cast<const osg::IntArray*>(indices))[pos];
        case(osg::Array::UByteArrayType): return (*static_cast<const osg::UByteArray*>(indices))[pos];
        case(osg::Array::UShortArrayType): return (*static_cast<const osg::UShortArray*>(indices))[pos];
        case(osg::Array::UIntArrayType): return (*static_cast<const osg::UIntArray*>(indices))[pos];
        default: return 0;
    }
}

inline void pushBackIndex(unsigned int value, osg::IndexArray* indices)
{
    switch(indices->getType())
    {
        case(osg::Array::ByteArrayType): 
        {
            osg::ByteArray& array = (*static_cast<osg::ByteArray*>(indices));
            array.push_back(value);
            break;
        }
        case(osg::Array::ShortArrayType):
        {
            osg::ShortArray& array = (*static_cast<osg::ShortArray*>(indices));
            array.push_back(value);
            break;
        }
        case(osg::Array::IntArrayType):
        {
            osg::IntArray& array = (*static_cast<osg::IntArray*>(indices));
            array.push_back(value);
            break;
        }
        case(osg::Array::UByteArrayType):
        {
            osg::UByteArray& array = (*static_cast<osg::UByteArray*>(indices));
            array.push_back(value);
            break;
        }
        case(osg::Array::UShortArrayType):
        {
            osg::UShortArray& array = (*static_cast<osg::UShortArray*>(indices));
            array.push_back(value);
            break;
        }
        case(osg::Array::UIntArrayType):
        {
            osg::UIntArray& array = (*static_cast<osg::UIntArray*>(indices));
            array.push_back(value);
            break;
        }
        default:
        {
            //do nothing
            break;
        }
    }
}

inline void eraseIndex(unsigned int pos, osg::IndexArray* indices)
{
    switch(indices->getType())
    {
        case(osg::Array::ByteArrayType): 
        {
            osg::ByteArray& array = (*static_cast<osg::ByteArray*>(indices));
            array.erase(array.begin()+pos);
            break;
        }
        case(osg::Array::ShortArrayType):
        {
            osg::ShortArray& array = (*static_cast<osg::ShortArray*>(indices));
            array.erase(array.begin()+pos);
            break;
        }
        case(osg::Array::IntArrayType):
        {
            osg::IntArray& array = (*static_cast<osg::IntArray*>(indices));
            array.erase(array.begin()+pos);
            break;
        }
        case(osg::Array::UByteArrayType):
        {
            osg::UByteArray& array = (*static_cast<osg::UByteArray*>(indices));
            array.erase(array.begin()+pos);
            break;
        }
        case(osg::Array::UShortArrayType):
        {
            osg::UShortArray& array = (*static_cast<osg::UShortArray*>(indices));
            array.erase(array.begin()+pos);
            break;
        }
        case(osg::Array::UIntArrayType):
        {
            osg::UIntArray& array = (*static_cast<osg::UIntArray*>(indices));
            array.erase(array.begin()+pos);
            break;
        }
        default:
        {
            //do nothing
            break;
        }
    }
}

osg::Vec3 Primitive::getVertex(unsigned int index) const
{
    osg::Vec3 vtx;
    getVertex(index, vtx);
    return vtx;
}

bool Primitive::getVertex(unsigned int index, osg::Vec3& vtx) const
{
    class translator : public osg::ConstValueVisitor
    {
    public:
        osg::Vec3 translate;
        bool translated;
        translator() : translated(false) {}
        virtual void apply(const osg::Vec2& v) { translate.set(v.x(), v.y(), 0.0f);  translated = true; }
        virtual void apply(const osg::Vec3& v) { translate.set(v.x(), v.y(), v.z()); translated = true; }
        virtual void apply(const osg::Vec4& v) { translate.set(v.x(), v.y(), v.z()); translated = true; }
    };

    const osg::IndexArray* vtxIndices = m_spGeometry->getVertexIndices();
    const osg::Array* polyVertices = m_spGeometry->getVertexArray();

    translator trans;

    unsigned int pos = m_indices[index];
    if(vtxIndices)
        pos = getIndex(m_indices[index], vtxIndices);

    if(pos >= polyVertices->getNumElements())
        return false;

    polyVertices->accept(pos, trans);

    if(trans.translated)
    {
        if(!m_worldTransformIsIdentity)
            vtx = m_spWorldTransform->preMult(trans.translate);
        else
            vtx = trans.translate;

        return true;
    }
    else
        return false;
}

bool Primitive::getColor(unsigned int index, osg::Vec4& color) const
{
    if(m_spGeometry->getColorArray() == NULL)
        return false;

    const osg::IndexArray* colorIndices = m_spGeometry->getColorIndices();

    unsigned int vindex = index;
    if(m_spGeometry->getColorBinding() == osg::Geometry::BIND_PER_VERTEX)
    {
        if(index >= m_indices.size())
            return false;

        vindex = m_indices[index];
    }

    unsigned int pos = vindex;
    if(colorIndices != NULL)
        pos = getIndex(vindex, colorIndices);

    const osg::Array* polyColors = m_spGeometry->getColorArray();

    if(pos >= polyColors->getNumElements())
        return false;
    
    switch(polyColors->getType())
    {
    case(osg::Array::Vec4ArrayType):
        {
            color = (*static_cast<const osg::Vec4Array*>(polyColors))[pos];
            break;
        }
    case(osg::Array::Vec4ubArrayType):
        {
            const osg::Vec4ub& colorUB =(*static_cast<const osg::Vec4ubArray*>(polyColors))[pos];
            color.set((float)colorUB.r() / 255.0f,
                      (float)colorUB.g() / 255.0f,
                      (float)colorUB.b() / 255.0f,
                      (float)colorUB.a() / 255.0f);
            break;
        }
    case(osg::Array::Vec3ArrayType):
        {   
            const osg::Vec3& color3f = (*static_cast<const osg::Vec3Array*>(polyColors))[pos];
            color.set((float)color3f.x() / 255.0f,
                      (float)color3f.y() / 255.0f,
                      (float)color3f.z() / 255.0f,
                      1.0f);
            break;
        }
    default:
    //OSG by default only supports the above three array types for normals
        return false;
    }

    return true;
}

bool Primitive::getNormal(unsigned int index, osg::Vec3& normal) const
{
    if(m_spGeometry->getNormalArray() == NULL)
        return false;

    const osg::IndexArray* normalIndices = m_spGeometry->getNormalIndices();

    unsigned int vindex = index;
    if(m_spGeometry->getNormalBinding() == osg::Geometry::BIND_PER_VERTEX)
    {
        if(index >= m_indices.size())
            return false;

        vindex = m_indices[index];
    }

    unsigned int pos = vindex;
    if(normalIndices != NULL)
        pos = getIndex(vindex, normalIndices);

    const osg::Array* polyNormals = m_spGeometry->getNormalArray();

    if(pos >= polyNormals->getNumElements())
        return false;
    
    switch(polyNormals->getType())
    {
    case(osg::Array::Vec3ArrayType):
        {
            normal = (*static_cast<const osg::Vec3Array*>(polyNormals))[pos];
            break;
        }
    case(osg::Array::Vec3sArrayType):
        {
            const osg::Vec3s& normal3s =(*static_cast<const osg::Vec3sArray*>(polyNormals))[pos];
            normal.set((float)normal3s.x(),
                       (float)normal3s.y(),
                       (float)normal3s.z());
            break;
        }
    case(osg::Array::Vec3bArrayType):
        {   
            const osg::Vec3b& normal3b = (*static_cast<const osg::Vec3bArray*>(polyNormals))[pos];
            normal.set((float)normal3b.x(),
                      (float)normal3b.y(),
                      (float)normal3b.z());
            break;
        }
    case(osg::Array::Vec4sArrayType):
        {
            const osg::Vec4s& normal4s =(*static_cast<const osg::Vec4sArray*>(polyNormals))[pos];
            normal.set((float)normal4s.x(),
                       (float)normal4s.y(),
                       (float)normal4s.z());
            break;
        }
    case(osg::Array::Vec4bArrayType):
        {   
            const osg::Vec4b& normal4b = (*static_cast<const osg::Vec4bArray*>(polyNormals))[pos];
            normal.set((float)normal4b.x(),
                      (float)normal4b.y(),
                      (float)normal4b.z());
            break;
        }
    default:
    //OSG by default only supports the types above for normals
        return false;
    }

    return true;
}

bool Primitive::getTexCoord(unsigned int texCoordArrayIndex, unsigned int index, osg::Vec2& texCoord) const
{
    if(texCoordArrayIndex >= m_spGeometry->getNumTexCoordArrays())
        texCoordArrayIndex = m_spGeometry->getNumTexCoordArrays()-1;

    if(index >= m_indices.size())
        index = m_indices.size()-1;

    const osg::IndexArray* texIndices = m_spGeometry->getTexCoordIndices(texCoordArrayIndex);

    unsigned int pos = m_indices[index];
    if(texIndices)
        pos = getIndex(m_indices[index], texIndices);

    //we only support 2 component texture coords so need to translate to Vec2
    class translator : public osg::ConstValueVisitor
    {
    public:
        osg::Vec2& translate;
        bool translated;
        translator(osg::Vec2& ref) : translate(ref), translated(false) {}
        virtual void apply(const GLfloat& v){ translate.set(v, v); translated = true; }
        virtual void apply(const osg::Vec2& v)   { translate.set(v.x(), v.y()); translated = true; }
        virtual void apply(const osg::Vec3& v)   { translate.set(v.x(), v.y()); translated = true; }
        virtual void apply(const osg::Vec4& v)   { translate.set(v.x(), v.y()); translated = true; }
    };
    
    osg::Array* texCoords = m_spGeometry->getTexCoordArray(texCoordArrayIndex);
    translator trans(texCoord);

    if(pos < texCoords->getNumElements())
    {
        texCoords->accept(pos, trans);

        return trans.translated;
    }
    
    texCoord = osg::Vec2(0.0f, 0.0f);
    return false;
}

/*
  --------------------------
  PRIMITIVESETITERATOR CLASS:
  --------------------------
*/

PrimitiveSetIterator::PrimitiveSetIterator()
{
    m_spGeometry = NULL;
    m_spPrimSet = NULL;
    m_numberOfPolys = 0;
}

PrimitiveSetIterator::~PrimitiveSetIterator()
{
}

void copyNormalToEndOfNormalArray(osg::Geometry* pGeometry, unsigned int indexToCopy)
{
    osg::Array* normalArray = pGeometry->getNormalArray();
    osg::IndexArray* indexArray = pGeometry->getNormalIndices();
    switch(normalArray->getType())
    {
        case (osg::Array::Vec3ArrayType):
        {
            osg::Vec3Array& normals = *static_cast<osg::Vec3Array*>(normalArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(normals.size(), indexArray);
            }
            const osg::Vec3& copyNormal = normals[indexToCopy];
            normals.push_back(copyNormal);
            break;
        }
        case (osg::Array::Vec3sArrayType):
        {
            osg::Vec3sArray& normals = *static_cast<osg::Vec3sArray*>(normalArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(normals.size(), indexArray);
            }
            const osg::Vec3s& copyNormal = normals[indexToCopy];
            normals.push_back(copyNormal);
            break;
        }
        case (osg::Array::Vec4sArrayType):
        {
            osg::Vec4sArray& normals = *static_cast<osg::Vec4sArray*>(normalArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(normals.size(), indexArray);
            }
            const osg::Vec4s& copyNormal = normals[indexToCopy];
            normals.push_back(copyNormal);
            break;
        }
        case (osg::Array::Vec3bArrayType):
        {
            osg::Vec3bArray& normals = *static_cast<osg::Vec3bArray*>(normalArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(normals.size(), indexArray);
            }
            const osg::Vec3b& copyNormal = normals[indexToCopy];
            normals.push_back(copyNormal);
            break;
        }
        case (osg::Array::Vec4bArrayType):
        {
            osg::Vec4bArray& normals = *static_cast<osg::Vec4bArray*>(normalArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(normals.size(), indexArray);
            }
            const osg::Vec4b& copyNormal = normals[indexToCopy];
            normals.push_back(copyNormal);
            break;
        }
        default:
            break;
    }
}

void eraseNormal(osg::Geometry* pGeometry, unsigned int indexToErase)
{
    osg::Array* normalArray = pGeometry->getNormalArray();
    osg::IndexArray* indexArray = pGeometry->getNormalIndices();
    switch(normalArray->getType())
    {
        case (osg::Array::Vec3ArrayType):
        {
            osg::Vec3Array& normals = *static_cast<osg::Vec3Array*>(normalArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            normals.erase(normals.begin()+indexToErase);
            break;
        }
        case (osg::Array::Vec3sArrayType):
        {
            osg::Vec3sArray& normals = *static_cast<osg::Vec3sArray*>(normalArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            normals.erase(normals.begin()+indexToErase);
            break;
        }
        case (osg::Array::Vec4sArrayType):
        {
            osg::Vec4sArray& normals = *static_cast<osg::Vec4sArray*>(normalArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            normals.erase(normals.begin()+indexToErase);
            break;
        }
        case (osg::Array::Vec3bArrayType):
        {
            osg::Vec3bArray& normals = *static_cast<osg::Vec3bArray*>(normalArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            normals.erase(normals.begin()+indexToErase);
            break;
        }
        case (osg::Array::Vec4bArrayType):
        {
            osg::Vec4bArray& normals = *static_cast<osg::Vec4bArray*>(normalArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            normals.erase(normals.begin()+indexToErase);
        }
        default:
        {
            break;
        }
    }
}

void copyColorToEndOfColorArray(osg::Geometry* pGeometry, unsigned int indexToCopy)
{
    osg::Array* colorArray = pGeometry->getNormalArray();
    osg::IndexArray* indexArray = pGeometry->getColorIndices();
    switch(colorArray->getType())
    {
        case (osg::Array::Vec3ArrayType):
        {
            osg::Vec3Array& colors = *static_cast<osg::Vec3Array*>(colorArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(colors.size(), indexArray);
            }
            const osg::Vec3& copyColor = colors[indexToCopy];
            colors.push_back(copyColor);
            break;
        }
        case (osg::Array::Vec4ArrayType):
        {
            osg::Vec4Array& colors = *static_cast<osg::Vec4Array*>(colorArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(colors.size(), indexArray);
            }
            const osg::Vec4& copyColor = colors[indexToCopy];
            colors.push_back(copyColor);
            break;
        }
        case (osg::Array::Vec4ubArrayType):
        {
            osg::Vec4ubArray& colors = *static_cast<osg::Vec4ubArray*>(colorArray);
            if(indexArray != NULL)
            {
                indexToCopy = getIndex(indexToCopy, indexArray);
                pushBackIndex(colors.size(), indexArray);
            }
            const osg::Vec4ub& copyColor = colors[indexToCopy];
            colors.push_back(copyColor);
            break;
        }
        default:
            break;
    }
}

void eraseColor(osg::Geometry* pGeometry, unsigned int indexToErase)
{
    osg::Array* colorArray = pGeometry->getColorArray();
    osg::IndexArray* indexArray = pGeometry->getColorIndices();
    switch(colorArray->getType())
    {
        case (osg::Array::Vec3ArrayType):
        {
            osg::Vec3Array& colors = *static_cast<osg::Vec3Array*>(colorArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            colors.erase(colors.begin()+indexToErase);
            break;
        }
        case (osg::Array::Vec4ArrayType):
        {
            osg::Vec4Array& colors = *static_cast<osg::Vec4Array*>(colorArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            colors.erase(colors.begin()+indexToErase);
            break;
        }
        case (osg::Array::Vec4ubArrayType):
        {
            osg::Vec4ubArray& colors = *static_cast<osg::Vec4ubArray*>(colorArray);
            if(indexArray != NULL)
            {
                unsigned int saveIndex = indexToErase;
                indexToErase = getIndex(indexToErase, indexArray);
                eraseIndex(saveIndex, indexArray);
            }
            colors.erase(colors.begin()+indexToErase);
            break;
        }
        default:
        {
            break;
        }
    }
}

void PrimitiveSetEraser::updateColorsAndNormals(Primitive& poly, int primSetsAdded)
{
    osg::Geometry* pGeometry = poly.getGeometry();
    //normal array 
    if (pGeometry->getNormalBinding() == osg::Geometry::BIND_PER_PRIMITIVE_SET)
    {
        osg::PrimitiveSet* pPrimSet = poly.getPrimitiveSet();
        osg::Array* normals = pGeometry->getNormalArray();
        if(normals->referenceCount() > 1)//if this normal array is shared then copy it
        {
            osg::CopyOp copyOp(osg::CopyOp::DEEP_COPY_DRAWABLES);
            normals = copyOp(normals);
            pGeometry->setNormalArray(normals);

            osg::Array* indexArray = pGeometry->getNormalIndices();
            if(indexArray != NULL && indexArray->referenceCount() > 1)
            {
                //copy the index array too
                pGeometry->setNormalIndices(static_cast<osg::IndexArray*>(copyOp(indexArray)));
            }
        }
            
        unsigned int primSetIndex = pGeometry->getPrimitiveSetIndex(pPrimSet);
        for (int i = 0; i < primSetsAdded; ++i)
        {
            //copy the normal at index 'primSetIndex' to the end of this
            //geom's normal array because the added primitive sets should 
            //use the same normal vector as the poly's primitive set
            copyNormalToEndOfNormalArray(pGeometry, primSetIndex);
        }

    } 
    else if (pGeometry->getNormalBinding() == osg::Geometry::BIND_PER_PRIMITIVE)
    {
        osg::Array* normals = pGeometry->getNormalArray();
        if(normals->referenceCount() > 1)
        {
            osg::CopyOp copyOp(osg::CopyOp::DEEP_COPY_DRAWABLES);
            normals = copyOp(normals);
            pGeometry->setNormalArray(normals);
            
            osg::Array* indexArray = pGeometry->getNormalIndices();
            if(indexArray != NULL && indexArray->referenceCount() > 1)
            {
                //copy the index array too
                pGeometry->setNormalIndices(static_cast<osg::IndexArray*>(copyOp(indexArray)));
            }
        }
        
        //this poly was deleted so need to delete the normal
        //that goes with it
        int polyIndex = poly.getPrimSetIndex();
        eraseNormal(pGeometry, polyIndex);
    }

    //color array
    if (pGeometry->getColorBinding() == osg::Geometry::BIND_PER_PRIMITIVE_SET)
    {
        osg::PrimitiveSet* pPrimSet = poly.getPrimitiveSet();
        osg::Array* colors = pGeometry->getColorArray();
        if(colors->referenceCount() > 1)//if this normal array is shared then copy it
        {
            osg::CopyOp copyOp(osg::CopyOp::DEEP_COPY_DRAWABLES);
            colors = copyOp(colors);
            pGeometry->setColorArray(colors);

            osg::Array* indexArray = pGeometry->getColorIndices();
            if(indexArray != NULL && indexArray->referenceCount() > 1)
            {
                //copy the index array too
                pGeometry->setColorIndices(static_cast<osg::IndexArray*>(copyOp(indexArray)));
            }
        }
            
        unsigned int primSetIndex = pGeometry->getPrimitiveSetIndex(pPrimSet);
        for (int i = 0; i < primSetsAdded; ++i)
        {
            //copy the color at index 'primSetIndex' to the end of this
            //geom's color array because the added primitive sets should 
            //use the same color as the poly's primitive set
            copyColorToEndOfColorArray(pGeometry, primSetIndex);
        }
    }
    else if (pGeometry->getColorBinding() == osg::Geometry::BIND_PER_PRIMITIVE)
    {
        osg::Array* colors = pGeometry->getColorArray();
        
        if(colors->referenceCount() > 1)//if this normal array is shared then copy it
        {
            osg::CopyOp copyOp(osg::CopyOp::DEEP_COPY_DRAWABLES);
            colors = copyOp(colors);
            pGeometry->setColorArray(colors);

            osg::Array* indexArray = pGeometry->getColorIndices();
            if(indexArray != NULL && indexArray->referenceCount() > 1)
            {
                //copy the index array too
                pGeometry->setColorIndices(static_cast<osg::IndexArray*>(copyOp(indexArray)));
            }
        }
        
        //this poly was deleted so need to delete the color
        //that goes with it
        int polyIndex = poly.getPrimSetIndex();
        eraseColor(pGeometry, polyIndex);
    }
}


/*
  ----------------------------------------------------
  DIFFERENT TYPES OF POLYGON CLASSES:
  essentially how they work is that they are used 
  to iterate through a primitive and in each iteration 
  the properties of the current polygon overwrites that 
  of the previous then if the current polygon is the 
  right one its properties are saved within the 
  instance of the class.
  ---------------------------------------------------
*/

void TriangleDrawArraysIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet = primitiveSet;
    m_numberOfPolys = dynamic_cast<osg::DrawArrays*>(primitiveSet)->getCount() / 3;
}

bool TriangleDrawArraysIterator::getPoly(unsigned int index, Primitive& poly)
{

    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        osg::DrawArrays * primSet = (osg::DrawArrays*)m_spPrimSet.get();
        unsigned int vtxArrayIndex = primSet->getFirst() + (index*3);
        poly.setIndices(vtxArrayIndex, vtxArrayIndex + 1, vtxArrayIndex + 2);

        return true;
    }
}

void TriangleDrawArraysEraser::deletePoly(Primitive& poly)
{

    //next part will find if it is middle,end or begining triangle and thus how to delete
    int index = poly.getPrimSetIndex()*3; //this is then the last point of the triangle
    osg::DrawArrays * primSet = (osg::DrawArrays*)poly.getPrimitiveSet();

    unsigned int primitiveSetsAdded = 0;
    if (index == 0)
    {
        //if the triangle is the first one in this primitive set
        //then change count and first
        primSet->setCount(primSet->getCount() - 3);
        primSet->setFirst(primSet->getFirst() + 3);
    }
    else if (index < primSet->getCount() - 3)
    {
        //if the primitive is not the first and not the last then
        //break up into two primitives 
        //break up primitive into two at the triangle to be deleted and thus exclude it

        osg::DrawArrays * secondPrimSet = new osg::DrawArrays(*primSet, 0); //0 for shallow copy
        secondPrimSet->setFirst(primSet->getFirst() + index + 3);
        secondPrimSet->setCount(primSet->getCount() - (index + 3));
        poly.getGeometry()->addPrimitiveSet(secondPrimSet);
        
        primSet->setCount(index); 

        primitiveSetsAdded = 1;
    }
    else
    {
        //if the triangle is the last in this primitive set
        //then change the count
        primSet->setCount(primSet->getCount() - 3);
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();
}

void TriangleDrawArrayLengthsIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    m_numberOfPolys = primitiveSet->getNumIndices() / 3;
}

bool TriangleDrawArrayLengthsIterator::getPoly(unsigned int index, Primitive& poly)
{

    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
        unsigned int vtxArrayIndex = primSet->getFirst() + (index*3);
        poly.setIndices(vtxArrayIndex, vtxArrayIndex + 1, vtxArrayIndex + 2);

        return true;
    }
}

void TriangleDrawArrayLengthsEraser::deletePoly(Primitive& poly)
{

    //next part will find if it is middle,end or begining triangle and thus how to delete
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)poly.getPrimitiveSet();

    //this is the index of the first vertex in the triangle
    unsigned int polyIndex = primSet->getFirst() + (poly.getPrimSetIndex()*3);
    //if this is the first triangle then we can skip all the complicated crap below

    if (polyIndex == static_cast<unsigned int>(primSet->getFirst()))
    {
        //just change the first and the count of the first element
        primSet->setFirst(polyIndex + 3);
        //we could potentially be changing the count here to zero,
        //but its not a huge deal if we do it will just start at the 
        //next length
        (*primSet)[0] = primSet->at(0)-3;
        
        updateColorsAndNormals(poly, 0/*primitiveSetsAdded*/);
        
        //must be called or deletion will not occur
        poly.getGeometry()->dirtyBound();
        poly.getGeometry()->releaseGLObjects();
        return;
    }

    unsigned int currentVtxStart = primSet->getFirst();

    osg::DrawArrayLengths::vector_type::iterator itr = primSet->begin();
    unsigned int itrIndex = 0;
    for (; itr != primSet->end(); ++itr, ++itrIndex)
    {
        unsigned int currentVtxEnd = currentVtxStart + *itr;

        //figure out if the current triangle is part of the current
        //draw array length
        if (polyIndex >= currentVtxStart && polyIndex < currentVtxEnd)
        {
            //just break into two primitive sets and exclude the triangle to be deleted
            osg::DrawArrayLengths * secondPrimSet = new osg::DrawArrayLengths(*primSet, 0); //0 for shallow copy
            //delete from the beginning up to, but not including, the current itr
            secondPrimSet->erase(secondPrimSet->begin(), secondPrimSet->begin() + itrIndex);
            //change the starting point of second prim set to be just beyond the end
            //of the current triangle
            secondPrimSet->setFirst(polyIndex+3);
            (*secondPrimSet)[0] = secondPrimSet->at(0)-3;

            poly.getGeometry()->addPrimitiveSet(secondPrimSet);
            //delete everything after the current itr
            primSet->erase(itr+1, primSet->end());
            //then modify the value (i.e. the length) of the current itr
            (*primSet)[itrIndex] = polyIndex - currentVtxStart;
            
            updateColorsAndNormals(poly, 1/*primitiveSetsAdded*/);
            break;
        }

        currentVtxStart = currentVtxEnd;
    }

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();
}

void TriangleDrawElementsIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    m_numberOfPolys = primitiveSet->getNumIndices() / 3;

}

bool TriangleDrawElementsIterator::getPoly(unsigned int index, Primitive& poly)
{

    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        DrawElementsAnyType primSet(m_spPrimSet.get());
        unsigned int vtxIndex = index*3;
        poly.setIndices(primSet.at(vtxIndex), primSet.at(vtxIndex + 1), primSet.at(vtxIndex + 2));

        return true;
    }
}


void TriangleDrawElementsEraser::deletePoly(Primitive& poly)
{
    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = poly.getPrimSetIndex()*3; //this is then the last point of the triangle
    DrawElementsAnyType primSet(poly.getPrimitiveSet());

    unsigned int primitiveSetsAdded = 0;
    if (index == 0)
    {   
        //if this is the first poly then
        //delete first three elements
        primSet.erase(0, 3);

    }
    else if (index < primSet.size() - 3)
    {
        //if this poly is not the first and not the last then
        //break up into two primitives,assuming sequential and no repeats 
        //break up primitive into two at the triangle to be deleted and thus exclude it  
        DrawElementsAnyType secondPrimSet(primSet);//new osg::DrawElementsUByte(*primSet, 0);   //0 for shallow copy
        secondPrimSet.erase(secondPrimSet.begin(), secondPrimSet.begin() + index + 3);
        poly.getGeometry()->addPrimitiveSet(secondPrimSet.getPrimitiveSet());

        primSet.erase(primSet.begin() + index, primSet.end());

        primitiveSetsAdded = 1;
    }
    else
    {
        //if this poly is the last one then
        //delete last three elements
        primSet.erase(index, primSet.end());
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();
}


void TriangleStripDrawArraysIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    m_numberOfPolys = (primitiveSet->getNumIndices() - 2);
}

bool TriangleStripDrawArraysIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {

        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);
        /*
           for a triangleStripDrawArrays the second one triangle has its 
           first point at vertex array index 1
         */
        osg::DrawArrays* primSet = (osg::DrawArrays*)m_spPrimSet.get();
        index += primSet->getFirst();
        //order is important
        if(index & 1)//if the number is odd
            poly.setIndices(index, index + 2, index + 1);
        else
            poly.setIndices(index, index + 1, index + 2);

        return true;
    }
}



void TriangleStripDrawArraysEraser::deletePoly(Primitive& poly)
{

    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = poly.getPrimSetIndex(); //this is then the last point of the triangle
    osg::DrawArrays * primSet = (osg::DrawArrays*)poly.getPrimitiveSet();

    unsigned int primitiveSetsAdded = 0;

    if (index == 0)
    {
        //if poly is the first one then move the first to the next vertex
        //and shorten the count by one
        //NOTE: sort of works but second and third points do not print out in the same order after deletion
        primSet->setCount(primSet->getCount() - 1);
        primSet->setFirst(primSet->getFirst() + 1);
    }

    else if (index < static_cast<unsigned int>(primSet->getCount()) - 3)
    {
        //NOTE:same problem with secondPrimSet here
        //if the poly is a middle polygon then 
        //break up primitive into two at the triangle to be deleted and thus exclude it
        osg::DrawArrays * secondPrimSet = new osg::DrawArrays(*primSet, 0); //0 for shallow copy
        secondPrimSet->setFirst(primSet->getFirst() + index + 1); //exclude first point of triangle to be deleted
        secondPrimSet->setCount(primSet->getCount() - (index + 1));
        poly.getGeometry()->addPrimitiveSet(secondPrimSet);
        
        primSet->setCount(index+2);   //end right before triangle to be deleted

        primitiveSetsAdded = 1;
    }
    else
    {
        //if the poly is the last one then shorten the count
        primSet->setCount(index+2);
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();
}

void TriangleStripDrawArrayLengthsIterator::init(osg::Geometry * searchGeometry,
                                                 osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;

    m_numberOfPolys = 0;
    //find the numberOfPrimitives in this primitive set
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
    for (osg::DrawArrayLengths::vector_type::const_iterator itr = primSet->begin(); 
         itr != primSet->end(); 
         ++itr)
    {
        m_numberOfPolys += (*itr - 2);
    }

}

bool TriangleStripDrawArrayLengthsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setPrimSetIndex(index);
        /*
           basic algorithm here is that for each element value
           there are (element value - 2) triangles.
         */
        osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
        osg::DrawArrayLengths::vector_type::const_iterator itr = primSet->begin();
        unsigned int polysTraversed = 0;
        unsigned int verticesTraversed = 0;
        for (polysTraversed = 0, verticesTraversed = 0; itr != primSet->end(); ++itr)
        {
            //if the polygon is in the current strip
            if (index <= (polysTraversed + (*itr - 2)))
            {
                poly.setGeometry(m_spGeometry.get());
                poly.setPrimitiveSet(m_spPrimSet.get());

                index += primSet->getFirst();
                //+2 becasue dont count the first 2 points
                //look at the remaining number of polys that are in this strip
                if(index & 1)//if this index is odd
                    poly.setIndices((index - polysTraversed) + verticesTraversed,
                                ((index - polysTraversed) + verticesTraversed) + 2,
                                ((index - polysTraversed) + verticesTraversed) + 1);
                else
                    poly.setIndices((index - polysTraversed) + verticesTraversed,
                                ((index - polysTraversed) + verticesTraversed) + 1,
                                ((index - polysTraversed) + verticesTraversed) + 2);


                return true;
            }

            polysTraversed += (*itr - 2);
            verticesTraversed += *itr;
        }

        return false;
    }
}


void TriangleStripDrawArrayLengthsEraser::deletePoly(Primitive& poly)
{

    //next part will find if it is middle,end or begining triangle and thus how to delete
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)poly.getPrimitiveSet();
    unsigned int polyIndex = poly.getPrimSetIndex();

    //if this is the first triangle in the strip
    //then can skip the complicated crap below
    if(polyIndex == 0)
    {
        //change the first
        primSet->setFirst(primSet->getFirst()+1);
        (*primSet)[0] = primSet->at(0)-1;
        
        updateColorsAndNormals(poly, 0/*primitiveSetsAdded*/);

        //must be called or deletion will not occur
        poly.getGeometry()->dirtyBound();
        poly.getGeometry()->releaseGLObjects();
        return;
    }

    osg::DrawArrayLengths::vector_type::iterator itr = primSet->begin();
    unsigned int itrIndex = 0;
    unsigned int curPolyStartIndex = 0;
    unsigned int curFirst = primSet->getFirst();
    
    for (; itr != primSet->end(); ++itr, ++itrIndex)
    {
        unsigned int curVtxCount = *itr;
        unsigned int curPolyCount = curVtxCount - 2;
        
        if(polyIndex >= curPolyStartIndex && polyIndex < (curPolyStartIndex + curPolyCount))
        {
            osg::DrawArrayLengths * secondPrimSet = new osg::DrawArrayLengths(*primSet, 0); //0 for shallow copy
            //erase everything up to, but not including, the current itr
            secondPrimSet->erase(secondPrimSet->begin(), secondPrimSet->begin() + itrIndex);
            secondPrimSet->setFirst(curFirst + (polyIndex - curPolyStartIndex + 1));
            //update the count
            (*secondPrimSet)[0] = secondPrimSet->at(0) - (secondPrimSet->getFirst() - curFirst);
            poly.getGeometry()->addPrimitiveSet(secondPrimSet);

            //now erase from itr+1 to end
            primSet->erase(itr+1, primSet->end());
            //change the count for itr
            (*primSet)[itrIndex] = primSet->at(itrIndex) - secondPrimSet->at(0) + 1;

            updateColorsAndNormals(poly, 1/*primitiveSetsAdded*/);
            break;
        }

        curPolyStartIndex += curPolyCount;
        curFirst += curVtxCount;

    }

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();

}

void TriangleStripDrawElementsIterator::init(osg::Geometry * searchGeometry,
                                             osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    m_numberOfPolys = (primitiveSet->getNumIndices() - 2);
}

bool TriangleStripDrawElementsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        DrawElementsAnyType primSet(m_spPrimSet.get());
        if(index + 2 >= m_spPrimSet->getNumIndices())
            index = index - 2;
        if(index & 1)//if odd
            poly.setIndices(primSet.at(index), primSet.at(index + 2), primSet.at(index + 1));
        else
            poly.setIndices(primSet.at(index), primSet.at(index + 1), primSet.at(index + 2));

        return true;
    }
}



void TriangleStripDrawElementsEraser::deletePoly(Primitive& poly)
{

    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = poly.getPrimSetIndex(); //this is then the last point of the triangle
    DrawElementsAnyType primSet(poly.getPrimitiveSet());

    //in this case I assume that you cannot have Triangle_strips of DrawElements where two of the elements are the same.

    unsigned int primitiveSetsAdded = 0;
    if (index == 0)
    {
        //if this poly is the first one then
        //erase this index so that the first 
        //index becomes the index of the second poly
        primSet.erase(primSet.begin());
    }
    else if (index < primSet.size()-2)
    {
        DrawElementsAnyType secondPrimSet(primSet);
        //break up primitive into two at the triangle to be deleted and thus exclude it
        secondPrimSet.erase(secondPrimSet.begin(), secondPrimSet.begin() + index + 1);
        poly.getGeometry()->addPrimitiveSet(secondPrimSet.getPrimitiveSet());

        primSet.erase(index+1, primSet.end());

        primitiveSetsAdded = 1;
    }
    else
    {
        primSet.erase(primSet.end()-1);
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();

}

void TriangleFanDrawArraysIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    osg::DrawArrays* primSet = (osg::DrawArrays*)m_spPrimSet.get();
    m_numberOfPolys = (primSet->getCount() - 1);
}

bool TriangleFanDrawArraysIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        osg::DrawArrays* primSet = (osg::DrawArrays*)m_spPrimSet.get();
        int primSetStart = primSet->getFirst();
        index += primSetStart;
        poly.setIndices(primSetStart, index + 1, index + 2);

        return true;
    }
}

void TriangleFanDrawArraysEraser::deletePoly(Primitive& poly)
{

    osg::DrawArrays* primSet = (osg::DrawArrays*)poly.getPrimitiveSet();
    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = poly.getPrimSetIndex(); //this is then the last point of the triangle

    unsigned int primitiveSetsAdded = 0;
    if (index == 0)
    {
        //in this case you have to convert the entire primitive to a DrawElements
        osg::DrawElementsUShort * deus = new osg::DrawElementsUShort(osg::PrimitiveSet::TRIANGLE_FAN, 0);   //(mode,vector size)
        deus->push_back(primSet->getFirst());

        //deusElement used to skip over the first two points which are of the deleted triangle
        for (GLsizei deusElement = 2; deusElement < primSet->getCount(); ++deusElement)
        {
            deus->push_back(primSet->getFirst() + deusElement);
        }

        poly.getGeometry()->removePrimitiveSet(poly.getGeometry()->getPrimitiveSetIndex(primSet));
        poly.getGeometry()->addPrimitiveSet(deus);
    }
    else if (index < static_cast<unsigned int>(primSet->getCount()) - 2)
    {
        //only second half has to be a DrawElements, first half can be made into triangle_strips
        osg::DrawElementsUShort * deus = new osg::DrawElementsUShort(osg::PrimitiveSet::TRIANGLE_FAN, 0);
        deus->push_back(primSet->getFirst());

        //deusElement used to skip over the first two points which are of the deleted triangle
        for (GLsizei deusElement = index+2; deusElement < primSet->getCount(); ++deusElement)
        {
            deus->push_back(primSet->getFirst() + deusElement);
        }

        poly.getGeometry()->addPrimitiveSet(deus);
        
        primSet->setCount(index+2);

        primitiveSetsAdded = 1;
    }
    else
    {
        primSet->setCount(primSet->getCount() - 1);
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();

}

void TriangleFanDrawArrayLengthsIterator::init(osg::Geometry * searchGeometry,
                                               osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;

    m_numberOfPolys = 0;
    //find the numberOfPrimitives in this primitive set
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
    for (osg::DrawArrayLengths::vector_type::const_iterator itr = primSet->begin(); itr != primSet->end(); ++itr)
    {
        m_numberOfPolys += (*itr - 2);
    }
}

bool TriangleFanDrawArrayLengthsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setPrimSetIndex(index);

        osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
        osg::DrawArrayLengths::vector_type::const_iterator itr = primSet->begin();
        unsigned int polysTraversed;
        unsigned int verticesTraversed;
        unsigned int primSetStart = primSet->getFirst();
        for (polysTraversed = 0, verticesTraversed = primSetStart; itr != primSet->end(); ++itr)
        {
            unsigned int curPolyCount = (*itr - 2);
            //if in this fan
            if (index <= (polysTraversed + curPolyCount))
            {
                poly.setGeometry(m_spGeometry.get());
                poly.setPrimitiveSet(m_spPrimSet.get());

                index -= polysTraversed;
                poly.setIndices(verticesTraversed, index + verticesTraversed + 1, index + verticesTraversed + 2);

                return true;

            }

            verticesTraversed += *itr;
            polysTraversed += curPolyCount;

        }

    }

    return false;
}



void TriangleFanDrawArrayLengthsEraser::deletePoly(Primitive& poly)
{
    //next part will find if it is middle,end or begining triangle and thus how to delete
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)poly.getPrimitiveSet();
    unsigned int polyIndex = poly.getPrimSetIndex();

    unsigned int curFirstVtx = primSet->getFirst();
    unsigned int curFirstPoly = 0;
    osg::DrawArrayLengths::vector_type::iterator itr = primSet->begin();
    unsigned int itrIndex = 0;
    for (; itr != primSet->end(); ++itr, ++itrIndex)
    {
        unsigned int curVtxCount = *itr;
        unsigned int curPolyCount = curVtxCount - 2;
        
        if(polyIndex >= curFirstPoly && polyIndex < (curFirstPoly + curPolyCount))
        {
            osg::DrawElementsUInt* secondPrimSet = new osg::DrawElementsUInt();
            secondPrimSet->push_back(curFirstVtx);

            unsigned int startVtxIndex = curFirstVtx + ((polyIndex - curFirstPoly) + 2);
            for(unsigned int index = startVtxIndex; index < (curFirstVtx + curVtxCount); ++index)
            {
                secondPrimSet->push_back(index);
            }

            poly.getGeometry()->addPrimitiveSet(secondPrimSet);

            osg::DrawArrayLengths* thirdPrimSet = new osg::DrawArrayLengths(*primSet, 0);
            //erase from beginning up to and including the itr
            thirdPrimSet->erase(thirdPrimSet->begin(), thirdPrimSet->begin() + (itrIndex + 1));
            thirdPrimSet->setFirst(curFirstVtx + curVtxCount);

            poly.getGeometry()->addPrimitiveSet(thirdPrimSet);
            
            //erase everything from the itr to the end
            primSet->erase(itr, primSet->end());

            updateColorsAndNormals(poly, 2/*primitiveSetsAdded*/);
    
            //must be called or deletion will not occur
            poly.getGeometry()->dirtyBound();
            poly.getGeometry()->releaseGLObjects();
            break;
        }

        curFirstVtx += curVtxCount;
        curFirstPoly += curPolyCount;
    }
}


void TriangleFanDrawElementsIterator::init(osg::Geometry * searchGeometry,
                                           osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    m_numberOfPolys = (primitiveSet->getNumIndices() - 2);    //starts at 0
}

bool TriangleFanDrawElementsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        DrawElementsAnyType primSet(m_spPrimSet.get());

        poly.setIndices(primSet.at(0), primSet.at(index + 1), primSet.at(index + 2));

        return true;
    }
}


void TriangleFanDrawElementsEraser::deletePoly(Primitive& poly)
{
    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = poly.getPrimSetIndex(); //this is then the last point of the triangle
    DrawElementsAnyType primSet(poly.getPrimitiveSet());

    unsigned int primitiveSetsAdded = 0;
    //begining one
    if (index == 0)
    { 
        //erase the first vertice's index, so we jump right to the second one
        primSet.erase(1);
        //primSet.insert(1, primSet.at(0));
        //primSet.erase(0);
    }
    //middle one
    //there is one vertex to specify each triangle except the first, last, and the fan vertex
    else if (index < primSet.size()-3) 
    {   //break up into two primitive sets, the second one starts at index 
        primSet.erase(index+2, primSet.end());
        
        DrawElementsAnyType secondPrimSet(primSet);
        //don't erase the fan vertex
        secondPrimSet.erase(secondPrimSet.begin()+1, index+2);
        poly.getGeometry()->addPrimitiveSet(secondPrimSet.getPrimitiveSet());

        primitiveSetsAdded = 1;
    }
    //end one
    else
    {   //delete last point
        primSet.erase(primSet.end() - 1, primSet.end());
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();

}

void QuadDrawArraysIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    osg::DrawArrays * primSet = (osg::DrawArrays*)m_spPrimSet.get();
    m_numberOfPolys = primSet->getCount() / 4;
}

bool QuadDrawArraysIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        osg::DrawArrays * primSet = (osg::DrawArrays*)m_spPrimSet.get();
        unsigned int vtxArrayIndex = primSet->getFirst() + (index*4);
        poly.setIndices(vtxArrayIndex, vtxArrayIndex + 1, vtxArrayIndex + 2, vtxArrayIndex + 3);

        return true;
    }
}


void QuadDrawArraysEraser::deletePoly(Primitive& poly)
{

    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = poly.getPrimSetIndex()*4;
    osg::DrawArrays * primSet = (osg::DrawArrays*)poly.getPrimitiveSet();
    osg::Geometry* geometry = poly.getGeometry();

    unsigned int primitiveSetsAdded = 0;

    if (index == 0)
    {                           //change count and first
        primSet->setCount(primSet->getCount() - 4);
        primSet->setFirst(primSet->getFirst() + 4);
    }
    else if (index < static_cast<unsigned int>(primSet->getCount()) - 4)
    {                           //break up primitive into two at the triangle to be deleted and thus exclude it
        osg::DrawArrays * secondPrimSet = new osg::DrawArrays(*primSet, 0); //0 for shallow copy
        secondPrimSet->setFirst(primSet->getFirst() + index + 4);
        secondPrimSet->setCount(primSet->getCount() - (index + 4));
        poly.getGeometry()->addPrimitiveSet(secondPrimSet);
        
        primSet->setCount(index); 

        primitiveSetsAdded = 1;
    }
    else
    {
        primSet->setCount(primSet->getCount() - 4);
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    geometry->dirtyBound();
    geometry->releaseGLObjects();
}



void QuadDrawArrayLengthsIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    m_numberOfPolys = primitiveSet->getNumIndices() / 4;
}

bool QuadDrawArrayLengthsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
        unsigned int vtxArrayIndex = primSet->getFirst() + (index*4);
        poly.setIndices(vtxArrayIndex, vtxArrayIndex + 1, vtxArrayIndex + 2, vtxArrayIndex + 3);

        return true;

    }
}

void QuadDrawArrayLengthsEraser::deletePoly(Primitive& poly)
{
    //next part will find if it is middle,end or begining triangle and thus how to delete
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)poly.getPrimitiveSet();

    //this is the index of the first vertex in the triangle
    unsigned int polyIndex = static_cast<unsigned int>(primSet->getFirst()) + (poly.getPrimSetIndex()*4);
    //if this is the first triangle then we can skip all the complicated crap below
    if (polyIndex == static_cast<unsigned int>(primSet->getFirst()))
    {
        //just change the first and the count of the first element
        primSet->setFirst(polyIndex + 4);
        //we could potentially be changing the count here to zero,
        //but its not a huge deal if we do it will just start at the 
        //next length
        (*primSet)[0] = primSet->at(0)-4;

        updateColorsAndNormals(poly, 0/*primitiveSetsAdded*/);

        //must be called or deletion will not occur
        poly.getGeometry()->dirtyBound();
        poly.getGeometry()->releaseGLObjects();
        return;
    }

    unsigned int currentVtxStart = static_cast<unsigned int>(primSet->getFirst());

    osg::DrawArrayLengths::vector_type::iterator itr = primSet->begin();
    unsigned int itrIndex = 0;
    for (; itr != primSet->end(); ++itr, ++itrIndex)
    {
        unsigned int currentVtxEnd = currentVtxStart + *itr;

        //figure out if the current triangle is part of the current
        //draw array length
        if (polyIndex >= currentVtxStart && polyIndex < currentVtxEnd)
        {
            //just break into two primitive sets and exclude the triangle to be deleted
            osg::DrawArrayLengths * secondPrimSet = new osg::DrawArrayLengths(*primSet, 0); //0 for shallow copy
            //delete from the beginning up to, but not including, the current itr
            secondPrimSet->erase(secondPrimSet->begin(), secondPrimSet->begin() + itrIndex);
            //change the starting point of second prim set to be just beyond the end
            //of the current triangle
            secondPrimSet->setFirst(polyIndex+4);
            (*secondPrimSet)[0] = secondPrimSet->at(0)-4;

            poly.getGeometry()->addPrimitiveSet(secondPrimSet);
            //delete everything after the current itr
            primSet->erase(itr+1, primSet->end());
            //then modify the value (i.e. the length) of the current itr
            (*primSet)[itrIndex] = polyIndex - currentVtxStart;

            updateColorsAndNormals(poly, 1/*primitiveSetsAdded*/);

            //must be called or deletion will not occur
            poly.getGeometry()->dirtyBound();
            poly.getGeometry()->releaseGLObjects();
            break;
        }

        currentVtxStart = currentVtxEnd;
    }

}

void QuadDrawElementsIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    DrawElementsAnyType primSet(m_spPrimSet.get());
    m_numberOfPolys = primSet.size() / 4;
}

bool QuadDrawElementsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);
        DrawElementsAnyType primSet(m_spPrimSet.get());
        unsigned int vtxIndex = index*4;
        poly.setIndices(primSet.at(vtxIndex), primSet.at(vtxIndex + 1), primSet.at(vtxIndex + 2), primSet.at(vtxIndex + 3));

        return true;
    }
}



void QuadDrawElementsEraser::deletePoly(Primitive& poly)
{

    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = poly.getPrimSetIndex()*4; //this is then the last point of the triangle
    DrawElementsAnyType primSet(poly.getPrimitiveSet());

    unsigned int primitiveSetsAdded = 0;

    if (index == 0)
    {                           //delete first three elements
        primSet.erase(primSet.begin(), primSet.begin() + 4);
    }
    else if (index < primSet.size() - 4)
    {                           //break up into two primitives,assuming sequential and no repeats 
        //break up primitive into two at the triangle to be deleted and thus exclude it  
        DrawElementsAnyType secondPrimSet(primSet);
        secondPrimSet.erase(secondPrimSet.begin(), secondPrimSet.begin() + index + 4);
        poly.getGeometry()->addPrimitiveSet(secondPrimSet.getPrimitiveSet());
        
        primSet.erase(primSet.begin() + index, primSet.end());

        primitiveSetsAdded = 1;
    }
    else
    {                           //delete last four elements
        primSet.erase(index, primSet.end());
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();
}

void QuadStripDrawArraysIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    osg::DrawArrays * primSet = (osg::DrawArrays*)m_spPrimSet.get();
    m_numberOfPolys = (primSet->getCount() - 2) / 2;
}

bool QuadStripDrawArraysIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);
        /*
           for a QuadStripDrawArrays the second quad has its 
           first point at vertex array index 2
         */
        osg::DrawArrays * primSet = (osg::DrawArrays*)m_spPrimSet.get();
        index *= 2;
        index += primSet->getFirst();
        poly.setIndices(index, index + 1, index + 2, index + 3);

        return true;
    }
}

void QuadStripDrawArraysEraser::deletePoly(Primitive& poly)
{
    //next part will find if it is middle,end or begining triangle and thus how to delete
    //this is the index of the first vertex in the poly
    unsigned int index = (poly.getPrimSetIndex() << 1);
    osg::DrawArrays * primSet = (osg::DrawArrays*)poly.getPrimitiveSet();
    osg::Geometry* geometry = poly.getGeometry();

    unsigned int primitiveSetsAdded = 0;

    //first one
    if (index == 0)
    {                           //change first and count
        primSet->setCount(primSet->getCount() - 2);
        primSet->setFirst(primSet->getFirst() + 2);
    }
    //middle one
    else if (index > ((unsigned int)primSet->getFirst() +3) && index < (unsigned int)(primSet->getFirst() + primSet->getCount() - 1))
    {                           //break up into two primitive sets
        //break up primitive into two at the triangle to be deleted and thus exclude it
        osg::DrawArrays * secondPrimSet = new osg::DrawArrays(*primSet, 0); //0 for shallow copy
        secondPrimSet->setFirst(primSet->getFirst() + (index + 2));
        secondPrimSet->setCount(primSet->getCount() - (index + 2));
        
        geometry->addPrimitiveSet(secondPrimSet);
        
        primSet->setCount(index + 2);

        primitiveSetsAdded = 1;
    }
    //end one
    else
    {                           //change count
        primSet->setCount(primSet->getCount() - 2);
    }

    updateColorsAndNormals(poly, primitiveSetsAdded);
    //must be called or deletion will not occur
    geometry->dirtyBound();
    geometry->releaseGLObjects();

}

void QuadStripDrawArrayLengthsIterator::init(osg::Geometry * searchGeometry, 
                                             osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet = primitiveSet;

    m_numberOfPolys = 0;
    //find the numberOfPrimitives in this primitive set
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
    for (osg::DrawArrayLengths::vector_type::const_iterator itr = primSet->begin(); 
         itr != primSet->end(); 
         ++itr)
    {
        m_numberOfPolys += (*itr - 2) / 2;
    }

}

bool QuadStripDrawArrayLengthsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        /*
           basic algorithm here is that for each element value
           there are (element value - 2)/2 quads.
         */
        osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)m_spPrimSet.get();
        osg::DrawArrayLengths::vector_type::const_iterator itr = primSet->begin();
        unsigned int polysTraversed;
        for (polysTraversed = 0; itr != primSet->end(); ++itr)
        {
            unsigned int curPolyCount = (unsigned int)((*itr-2)*0.5f);
            //if the polygon is in the current strip
            if (index <= (polysTraversed + curPolyCount))
            {
                poly.setGeometry(m_spGeometry.get());
                poly.setPrimitiveSet(m_spPrimSet.get());

                poly.setPrimSetIndex(index);

                index *= 2;
                index += primSet->getFirst();
                poly.setIndices(index, index + 1, index + 2, index + 3);

                return true;
            }

            polysTraversed += curPolyCount;
        }

        return false;
    }
}

void QuadStripDrawArrayLengthsEraser::deletePoly(Primitive& poly)
{
    //next part will find if it is middle,end or begining triangle and thus how to delete
    osg::DrawArrayLengths* primSet = (osg::DrawArrayLengths*)poly.getPrimitiveSet();
    unsigned int polyIndex = poly.getPrimSetIndex();

    //if this is the first triangle in the strip
    //then can skip the complicated crap below
    if(polyIndex == 0)
    {
        //change the first
        primSet->setFirst(primSet->getFirst()+2);
        (*primSet)[0] = primSet->at(0)-2;

        updateColorsAndNormals(poly, 1/*primSetsAdded*/);

        //must be called or deletion will not occur
        poly.getGeometry()->dirtyBound();
        poly.getGeometry()->releaseGLObjects();
        return;
    }

    osg::DrawArrayLengths::vector_type::iterator itr = primSet->begin();
    unsigned int itrIndex = 0;
    unsigned int curPolyStartIndex = 0;
    unsigned int curFirst = primSet->getFirst();
    
    for (; itr != primSet->end(); ++itr, ++itrIndex)
    {
        unsigned int curVtxCount = *itr;
        //vertex count minus two then divided by 2
        unsigned int curPolyCount = ((curVtxCount - 2) >> 1);
        
        if(polyIndex >= curPolyStartIndex && polyIndex < (curPolyStartIndex + curPolyCount))
        {
            osg::DrawArrayLengths * secondPrimSet = new osg::DrawArrayLengths(*primSet, 0); //0 for shallow copy
            //erase everything up to, but not including, the current itr
            secondPrimSet->erase(secondPrimSet->begin(), secondPrimSet->begin() + itrIndex);

            unsigned int polyFirstIndex = curFirst + (polyIndex << 1);
            secondPrimSet->setFirst(curFirst + (polyFirstIndex + 2));
            //update the count
            (*secondPrimSet)[0] = secondPrimSet->at(0) - (secondPrimSet->getFirst() - curFirst);
            poly.getGeometry()->addPrimitiveSet(secondPrimSet);

            //now erase from itr+1 to end
            primSet->erase(itr+1, primSet->end());
            //change the count for itr
            (*primSet)[itrIndex] = primSet->at(itrIndex) - secondPrimSet->at(0);
            
            updateColorsAndNormals(poly, 1/*primSetsAdded*/);
    
            //must be called or deletion will not occur
            poly.getGeometry()->dirtyBound();
            poly.getGeometry()->releaseGLObjects();
            break;
        }

        curPolyStartIndex += curPolyCount;
        curFirst += curVtxCount;
    }
}

void QuadStripDrawElementsIterator::init(osg::Geometry * searchGeometry, osg::PrimitiveSet * primitiveSet)
{
    m_spGeometry = searchGeometry;
    m_spPrimSet =  primitiveSet;
    m_numberOfPolys = (primitiveSet->getNumIndices() - 2) / 2;
}

bool QuadStripDrawElementsIterator::getPoly(unsigned int index, Primitive& poly)
{
    if (index >= m_numberOfPolys)
        return false;
    else
    {
        poly.setGeometry(m_spGeometry.get());
        poly.setPrimitiveSet(m_spPrimSet.get());

        poly.setPrimSetIndex(index);

        DrawElementsAnyType primSet(m_spPrimSet.get());
        index *= 2;
        poly.setIndices(primSet.at(index), primSet.at(index + 1), primSet.at(index + 2), primSet.at(index + 3));

        return true;
    }
}

void QuadStripDrawElementsEraser::deletePoly(Primitive& poly)
{


    //next part will find if it is middle,end or begining triangle and thus how to delete
    unsigned int index = (poly.getPrimSetIndex() << 1);
    DrawElementsAnyType primSet(poly.getPrimitiveSet());

    unsigned int primSetsAdded = 0;
    //begining one
    if (index == 0)
    {
        primSet.erase(primSet.begin(), primSet.begin() + 2);
    }
    //middle one
    else if (index < primSet.size() - 4)
    {
        //break up primitive into two at the triangle to be deleted and thus exclude it
        DrawElementsAnyType secondPrimSet(primSet);
        secondPrimSet.erase(secondPrimSet.begin(), secondPrimSet.begin() + index + 2);
        poly.getGeometry()->addPrimitiveSet(secondPrimSet.getPrimitiveSet());

        primSet.erase(primSet.begin() + index + 2, primSet.end());

        primSetsAdded = 1;
    }
    //end one
    else
    {
        primSet.erase(primSet.end() - 2, primSet.end());
    }

    updateColorsAndNormals(poly, primSetsAdded);

    //must be called or deletion will not occur
    poly.getGeometry()->dirtyBound();
    poly.getGeometry()->releaseGLObjects();
}

class OsgFileReader : public osgDB::ReadFileCallback
{
public:
    bool loadImageData;
    std::map<std::string, osg::ref_ptr<osg::Image> > cachedImages;
    osg::ref_ptr<osgDB::Options> spImageLoadOptions;
    OsgFileReader() : loadImageData(false), spImageLoadOptions(new osgDB::Options())
    {
        spImageLoadOptions->setObjectCacheHint(osgDB::Options::CACHE_NONE);
    }

    virtual osgDB::ReaderWriter::ReadResult readImage(const std::string& filename, const osgDB::Options* options)
    {
        if(this->loadImageData)
        {
            std::string newFilename = filename;
            if(osgDB::getFileExtension(newFilename) != "tga")
            {
                newFilename = osgDB::getNameLessExtension(newFilename);
                newFilename += ".tga";
                std::string findFilename = osgDB::findDataFile(newFilename);
                if(findFilename.length() == 0)
                {
                    newFilename = osgDB::getNameLessExtension(newFilename);
                    newFilename += ".bmp";
                }
            }
            
            newFilename = osgDB::findDataFile(newFilename);
            newFilename = osgDB::getRealPath(newFilename);
                
            std::map<std::string, osg::ref_ptr<osg::Image> >::iterator findIt = cachedImages.find(newFilename);
            if(findIt == cachedImages.end())
            {
                osgDB::ReaderWriter::ReadResult result = osgDB::ReadFileCallback::readImage(newFilename,
                                                                                            spImageLoadOptions.get());
                if(result.getImage() != nullptr)
                {
                    if(cachedImages.size() == 5)
                        cachedImages.clear();
                    /*if(result.getImage()->getPixelFormat() != GL_RGBA ||
                       result.getImage()->getDataType() != GL_UNSIGNED_BYTE)
                    {
                        if(!cuda::VoxelBrickWriter::ConvertImageToRGBA8(result.getImage()))
                            return nullptr;
                    }*/
                    if(result.getImage()->getPixelFormat() == GL_RGB)
                    {
                        int imageWidth = result.getImage()->s();
                        int imageHeight = result.getImage()->t();
                        osg::Vec4ub* pRGBA = new osg::Vec4ub[imageWidth * imageHeight];
                        for(int row = 0; row < imageHeight; ++row)
                        {
                            for(int col = 0; col < imageWidth; ++col)
                            {
                                const unsigned char* color = result.getImage()->data(col, row);
                                pRGBA[(row*imageWidth) + col] = osg::Vec4ub(color[0], color[1], color[2], 255);
                            }
                        }
                        result.getImage()->setImage(imageWidth,
                                                    imageHeight, 
                                                    1,
                                                    result.getImage()->getInternalTextureFormat(),
                                                    GL_RGBA,
                                                    result.getImage()->getDataType(),
                                                    (unsigned char*)pRGBA, 
                                                    osg::Image::USE_NEW_DELETE);
                    }
                    cachedImages[newFilename] = result.getImage();
                }
                return result;
            }
            return findIt->second.get();
        }
        else
        {
            osg::Image* pImage = new osg::Image();
            pImage->setFileName(filename);
            return pImage;
        }
    }
};

static OsgFileReader* s_pOsgFileReader = nullptr;

typedef std::vector<glm::vec3> TriangleVerts;
typedef std::vector<glm::vec3> VertexNormals;
typedef std::vector<glm::vec2> TriangleTexCoords;
struct TriData
{
    TriangleVerts triangles;
    VertexNormals normals;
    TriangleTexCoords triangleTexCoords;
    osg::BoundingBox triBBox;
};
typedef std::map< osg::ref_ptr<osg::Texture2D>, TriData> TriangleData;
typedef std::set<std::string> TerrainTextureSet;

static hipTextureAddressMode OpenGLModeToCudaMode(osg::Texture::WrapMode wrapMode)
{
    switch(wrapMode)
    {
    case osg::Texture::CLAMP_TO_BORDER:
        return hipAddressModeBorder;
    case osg::Texture::REPEAT:
        return hipAddressModeWrap;
    case osg::Texture::MIRROR:
        return hipAddressModeMirror;
    case osg::Texture::CLAMP:
    case osg::Texture::CLAMP_TO_EDGE:
    default:
        return hipAddressModeClamp;
    
    }
}

int Voxelize(cuda::Voxelizer& voxelizer,
             const glm::vec3& p,
             const glm::vec3& deltaP,
             TriangleData& triData,
             const std::string& terrainTexturePrefix,
             const TerrainTextureSet& terrainTextures,
             bool hasNormals,
             const std::string& outputDir,
             bool outputBinary,
             bool outputCompressed)
{
    voxelizer.setVoxelizationParams(p, deltaP);

    //load images and triangle data into voxelizer
    std::vector< osg::ref_ptr<osg::Image> > images;
    //load images now
    s_pOsgFileReader->loadImageData = true;

    for(TriangleData::iterator itr = triData.begin();
        itr != triData.end();
        ++itr)
    {
        osg::Texture2D* pTexture = itr->first;
        unsigned char* pImageData = nullptr;
        int imageWidth = 0;
        int imageHeight = 0;
        bool isTerrain = pTexture != nullptr;
        if(pTexture != nullptr)
        {
            osg::ref_ptr<osg::Image> spImage = pTexture->getImage();
            if(spImage->data() == nullptr)
            {
                std::string filePath;
                if(pTexture->getUserValue("TexImagePath", filePath))
                {
                    filePath += "/";
                    filePath += spImage->getFileName();
                }
                else
                    filePath = spImage->getFileName();
                spImage = osgDB::readImageFile(filePath);
            }
            //float u = 0.006932;
            //float v = 0.006932;
            //osg::Vec4 color = spImage->getColor(u, v);
            if(spImage.get() != nullptr)
            {
                pImageData = spImage->data();
                images.push_back(spImage.get());
                imageWidth = spImage->s();
                imageHeight = spImage->t();
                
                std::string filename = osgDB::getSimpleFileName(spImage->getFileName());
                if(filename.find(terrainTexturePrefix) != std::string::npos ||
                    terrainTextures.find(filename) != terrainTextures.end())
                {
                    isTerrain = true;
                }
                else
                    isTerrain = false;
            }
            else
            {
                std::cerr << "Failed to load image " << spImage->getFileName() << std::endl;
            }
        }

        hipTextureAddressMode texAddressMode0 = OpenGLModeToCudaMode(pTexture->getWrap(osg::Texture::WRAP_S));
        hipTextureAddressMode texAddressMode1 = OpenGLModeToCudaMode(pTexture->getWrap(osg::Texture::WRAP_T));

        voxelizer.addTriangleGroup(&itr->second.triangles, 
                                   hasNormals ? &itr->second.normals : nullptr,
                                   &itr->second.triangleTexCoords,
                                   glm::vec3(itr->second.triBBox.xMin(),
                                             itr->second.triBBox.yMin(),
                                             itr->second.triBBox.zMin()),
                                   glm::vec3(itr->second.triBBox.xMax(),
                                             itr->second.triBBox.yMax(),
                                             itr->second.triBBox.zMax()),
                                   isTerrain,
                                   pImageData,
                                   imageWidth, imageHeight,
                                   texAddressMode0,
                                   texAddressMode1);
    }

    s_pOsgFileReader->loadImageData = false;

    if(!voxelizer.allocateTriangleMemory())
    {
        std::cerr << "allocateTriangleMemory() failed." << std::endl;
        std::cerr << voxelizer.getErrorMessage() << std::endl;
        return 1;
    }

    if(!voxelizer.computeEdgesFaceNormalsAndBounds())
    {
        std::cerr << "computeEdgesFaceNormalsAndBounds() failed." << std::endl;
        std::cerr << voxelizer.getErrorMessage() << std::endl;
        return 1;
    }

    int retStatus = voxelizer.generateVoxelsAndOctTree(outputDir, outputBinary, outputCompressed);
    if(retStatus == 0)
    {
        std::cerr << "generateVoxelsAndOctTree failed." << std::endl;
        std::cerr << voxelizer.getErrorMessage() << std::endl;
        return 1;
    }
    else if(retStatus == -1)
    {
        std::cerr << "No triangles overlapped with voxel grid." << std::endl;
        return -1;
    }

    return 0;
}


void cpuTest(const glm::vec3* verts, 
             int numVerts,
             const glm::vec3& p,
             const glm::vec3& deltaP,
             int *voxels,
             const glm::uvec3& voxDim);

class PagedGigaVoxelOctTreeGenerator
{
private:
    glm::vec3 _voxelSizeMeters;
    glm::uvec3 _brickDimensions;
    glm::uvec3 _maxVoxelMipMapDimensions;
    glm::vec3 _maxVoxelMipMapSizeMeters;

    osg::ref_ptr<osg::Node> _spInputNode;
    std::string _inputDir;
    std::string _outputDir;
    std::string _progressFile;

public:
    PagedGigaVoxelOctTreeGenerator() {}
    ~PagedGigaVoxelOctTreeGenerator() {}

    void setVoxelizationParams(const glm::vec3& voxelSizeMeters, 
                               const glm::uvec3& brickDimensions, 
                               const glm::uvec3& maxVoxelMipMapDimension)
    {
        _voxelSizeMeters = voxelSizeMeters;
        _brickDimensions = brickDimensions;
        _maxVoxelMipMapDimensions = maxVoxelMipMapDimension;

        _maxVoxelMipMapSizeMeters = static_cast<glm::vec3>(_maxVoxelMipMapDimensions) * _voxelSizeMeters;
    }

    void setInput(const std::string& inputFileName, osg::Node* pNode)
    {
        _inputDir = osgDB::getFilePath(inputFileName);
        osgDB::Registry::instance()->getDataFilePathList().push_back(_inputDir);
        _spInputNode = pNode;
        _spInputNode->getBound();//initialize bounds
    }

    void setOutputDirectory(const std::string& outputDir)
    {
        _outputDir = outputDir;
        _progressFile = _outputDir;
        _progressFile += "/";
        _progressFile += "pgvot_progress.txt";
    }

    struct Triangle
    {
        osg::Vec3f v0;
        osg::Vec3f v1;
        osg::Vec3f v2;
        osg::Vec3f n0;
        osg::Vec3f n1;
        osg::Vec3f n2;
        osg::Vec2f uv0;
        osg::Vec2f uv1;
        osg::Vec2f uv2;
        bool hasNormals;
        Triangle() {}
        Triangle(const osg::Vec3& _v0,const osg::Vec3& _v1,const osg::Vec3& _v2) :
        v0(_v0), v1(_v1), v2(_v2), hasNormals(false) {}
        void setUVs(const osg::Vec2& _uv0, const osg::Vec2& _uv1, const osg::Vec2& _uv2)
        {
            uv0 = _uv0;
            uv1 = _uv1;
            uv2 = _uv2;
        }
        void setNormals(const osg::Vec3& _n0,const osg::Vec3& _n1,const osg::Vec3& _n2)
        {
            n0 = _n0;
            n1 = _n1;
            n2 = _n2;
            hasNormals = true;
        }
    };

    typedef std::vector<Triangle> Triangles;
    typedef std::map< osg::ref_ptr<osg::Texture2D> , Triangles> TrianglesPerTexture;
    typedef std::string PagedFileName;
    typedef std::pair< osg::ref_ptr<osg::MatrixTransform>, PagedFileName> PagedFilePair;
    typedef std::vector< PagedFilePair > PagedFiles;
        
    class TriangleCollector : public osg::NodeVisitor
    {
   public:
        TriangleCollector() : 
            osg::NodeVisitor(osg::NodeVisitor::TRAVERSE_ALL_CHILDREN) {}

        osg::ref_ptr<osg::RefMatrixd> curWorldTransform;
        osg::BoundingBox filterBBox;
        TrianglesPerTexture trianglesPerTexture;
        //PagedFiles pagedFiles;
        typedef std::list<std::string> PagedFileUsageQueue;
        PagedFileUsageQueue pagedFileUsageQueue;
        typedef std::map<std::string, std::pair<osg::ref_ptr<osg::Node>, PagedFileUsageQueue::iterator> > PagedFileCache;
        PagedFileCache pagedFileCache;
        std::string curFilePath;

        class TriangleAttributeFunctor : public osg::Drawable::AttributeFunctor
        {
        private:
            osg::MatrixList worldMats;
            osg::MatrixList worldRotMats;
        public:
            osg::BoundingBox filterBBox;
            TrianglesPerTexture& trianglesPerTexture;
            void setWorldMats(const osg::MatrixList& mlist)
            {
                worldMats = mlist;
                for(size_t matIndex = 0; 
                        matIndex < worldMats.size(); 
                        ++matIndex)
                {
                    const osg::Matrix& worldMat = worldMats.at(matIndex);
                    worldRotMats.resize(worldRotMats.size()+1);
                    worldRotMats.back() = osg::Matrix(worldMat.getRotate());
                }
            }
            TriangleAttributeFunctor(TrianglesPerTexture& tpt) : 
                trianglesPerTexture(tpt) {}

            void assignTrianglesToTexture(osg::Texture2D* pTexture, PrimitiveSetIterator* pItr)
            {
                TrianglesPerTexture::iterator findIt = trianglesPerTexture.find(pTexture);

                if(findIt == trianglesPerTexture.end())
                    findIt = trianglesPerTexture.insert(std::make_pair(pTexture, Triangles())).first;

                for(size_t i = 0; i < pItr->getNumberOfPolys(); ++i)
                {
                    Primitive poly;
                    pItr->getPoly(i, poly);
                    if(poly.getVtxCount() != 3)
                        continue;
                    this->insert(poly,
                                 findIt->second);
                }
            }

            void insert(Primitive& poly, 
                        Triangles& triangles)
            {
                osg::Vec3d v1 = poly.getVertex(0);
                osg::Vec3d v2 = poly.getVertex(1);
                osg::Vec3d v3 = poly.getVertex(2);

                if(worldMats.size() == 0)
                {
                    osg::BoundingBox triBBox;
                    triBBox.expandBy(v1);
                    triBBox.expandBy(v2);
                    triBBox.expandBy(v3);
                    if(!filterBBox.intersects(triBBox))
                    {
                        //if triangle completely outside then throw out
                        return;
                    }

                    Triangle tri(v1, v2, v3);
                    if(poly.hasTextureData())
                        tri.setUVs(poly.getTexCoord(0, 0),
                                   poly.getTexCoord(0, 1),
                                   poly.getTexCoord(0, 2));
                    if(poly.hasNormalData() &&
                       poly.getNormalBinding() == osg::Geometry::AttributeBinding::BIND_PER_VERTEX)
                    {
                        tri.setNormals(poly.getNormal(0),
                                       poly.getNormal(1),
                                       poly.getNormal(2));
                    }
                    triangles.push_back(tri);
                }
                else
                {
                    //transform to world space
                    for(size_t matIndex = 0; 
                        matIndex < worldMats.size(); 
                        ++matIndex)
                    {
                        const osg::Matrix& worldMat = worldMats.at(matIndex);
                        const osg::Matrix& rotMat = worldRotMats.at(matIndex);
                        osg::Vec3d worldVtx1 = worldMat.preMult(v1);
                        osg::Vec3d worldVtx2 = worldMat.preMult(v2);
                        osg::Vec3d worldVtx3 = worldMat.preMult(v3);
                        osg::BoundingBox triBBox;
                        triBBox.expandBy(worldVtx1);
                        triBBox.expandBy(worldVtx2);
                        triBBox.expandBy(worldVtx3);
                        if(!filterBBox.intersects(triBBox))
                        {
                            return;
                        }
                        
                        triangles.resize(triangles.size()+1);
                        Triangle& tri = triangles.at(triangles.size()-1);
                        tri.v0 = worldVtx1;
                        tri.v1 = worldVtx2;
                        tri.v2 = worldVtx3;
                        if(poly.hasTextureData())
                            tri.setUVs(poly.getTexCoord(0, 0),
                                       poly.getTexCoord(0, 1),
                                       poly.getTexCoord(0, 2));

                        if(poly.hasNormalData() &&
                           poly.getNormalBinding() == osg::Geometry::AttributeBinding::BIND_PER_VERTEX)
                        {
                            osg::Vec3d n0 = poly.getNormal(0);
                            //osg::Vec3d t = v1 + n0;
                            n0 = rotMat.preMult(n0);
                            //t = worldMat.preMult(t);
                            //osg::Vec3d t2 = tri.v0 + n0;
                            
                            osg::Vec3d n1 = poly.getNormal(1);
                            n1 = rotMat.preMult(n1);
                            
                            osg::Vec3d n2 = poly.getNormal(2);
                            n2 = rotMat.preMult(n2);

                            tri.setNormals(n0, n1, n2);
                        }
                    }
                }
            }
        };

        osg::BoundingBox transformBBox(const osg::BoundingBox& bbox) const
        {
            if(this->curWorldTransform.get() != nullptr)
            {
                osg::BoundingBox xformedBBox;
                for(size_t i = 0; i < 8; ++i)
                {
                    osg::Vec3d corner = bbox.corner(i);
                    osg::Vec3d xformed = this->curWorldTransform->preMult(corner);
                    xformedBBox.expandBy(xformed);
                }
                return xformedBBox;
            }
            return bbox;
        }

        osg::BoundingSphere transformBSphere(const osg::BoundingSphere& bsphere) const
        {
            if(this->curWorldTransform.get() != nullptr)
            {
                osg::Vec3d center = bsphere.center();
                //going to assume that the transform has no scaling factor
                center = this->curWorldTransform->preMult(center);
                osg::BoundingSphere newBSphere(center, bsphere.radius());
                return newBSphere;
            }

            return bsphere;
        }

        virtual void apply(osg::MatrixTransform& node) override
        {
            const osg::Matrixd& matrix = node.getMatrix();
            osg::ref_ptr<osg::RefMatrixd> saveCur = this->curWorldTransform;
            if(this->curWorldTransform.get() != nullptr)
            {
                osg::Matrixd combined = *this->curWorldTransform.get();
                combined *= matrix;
                this->curWorldTransform = new osg::RefMatrixd(combined);
            }
            else
                this->curWorldTransform = new osg::RefMatrixd(matrix);

            traverse(node);

            this->curWorldTransform = saveCur;
        }

        virtual void apply(osg::Geode& geode) override
        {
            if(this->filterBBox.intersects(transformBBox(geode.getBoundingBox())) == false)
            {
                traverse(geode);
                return;
            }

            TriangleAttributeFunctor tf(this->trianglesPerTexture);
            tf.filterBBox = this->filterBBox;
            tf.setWorldMats(geode.getWorldMatrices());
            //TODO test normal transformation
            for(size_t i = 0; i < geode.getNumDrawables(); ++i)
            {
                osg::Drawable* pDrawable = geode.getDrawable(i);
                osg::Geometry* pGeometry = dynamic_cast<osg::Geometry*>(pDrawable);
                if(pGeometry == nullptr)
                    continue;

                osg::Geometry::AttributeBinding normalBinding = 
                                            pGeometry->getNormalBinding();
                if(normalBinding != osg::Geometry::BIND_PER_VERTEX &&
                   normalBinding != osg::Geometry::BIND_PER_PRIMITIVE)
                {
                    osgUtil::SmoothingVisitor::smooth(*pGeometry);
                }

                for(size_t j = 0; j < pGeometry->getNumPrimitiveSets(); ++j)
                {
                    osg::PrimitiveSet* pPrimSet = pGeometry->getPrimitiveSet(j);
                    osg::ref_ptr<PrimitiveSetIterator> spItr = PrimitiveSetIterator::getIterator(pPrimSet, pGeometry);
                    osg::StateSet* pStateSet = pDrawable->getStateSet();
                    if(pStateSet == nullptr)
                        pStateSet = geode.getStateSet();
                    osg::Texture2D* pTexture = nullptr;
                    if(pStateSet != nullptr)
                    {
                        pTexture = dynamic_cast<osg::Texture2D*>(pStateSet->getTextureAttribute(0, osg::StateAttribute::TEXTURE));
                        if(pTexture)
                            pTexture->setUserValue("TexImagePath", this->curFilePath);
                    }
                    
                    tf.assignTrianglesToTexture(pTexture, spItr.get());
                }
            }

            traverse(geode);
        }

        virtual void apply(osg::ProxyNode& node) override
        {
            traverse(node);
        }

        virtual void apply(osg::PagedLOD& pagedLOD) override
        {
            osg::BoundingSphere bsphere = transformBSphere(pagedLOD.getBound());
            if(this->filterBBox.contains(bsphere.center()) == false)
            {
                osg::Vec2 bsphereXYCenter = osg::Vec2(bsphere.center().x(), bsphere.center().y());
                osg::Vec2 bboxXYCenter = osg::Vec2(this->filterBBox.center().x(), this->filterBBox.center().y());
                osg::Vec2 diffVector = bsphereXYCenter - bboxXYCenter;
                float dist2 = diffVector.length2();
                float bboxHalfWidth = (this->filterBBox.xMax() - this->filterBBox.xMin()) * 0.5f;
                float bboxHalfHeight = (this->filterBBox.yMax() - this->filterBBox.yMin()) * 0.5f;
                float bboxXYRadius2 = (bboxHalfWidth * bboxHalfWidth) + (bboxHalfHeight * bboxHalfHeight);
                if(dist2 > bsphere.radius2() + bboxXYRadius2)
                {
                    //traverse(pagedLOD);
                    return;
                }
            }

            if(pagedLOD.getNumFileNames() == 0)
            {
                traverse(pagedLOD);
                return;
            }

            //we only want the highest resolution
            for(size_t fileIndex = 0; fileIndex < pagedLOD.getNumFileNames(); ++fileIndex)
            {
                float minRange = pagedLOD.getMinRange(fileIndex);
                if(minRange > 0.0f)
                    continue;
                const std::string& pagedFile = pagedLOD.getFileName(fileIndex);

                std::string relativeFilePath = pagedLOD.getDatabasePath() + "/" + pagedFile;
                relativeFilePath = osgDB::findDataFile(relativeFilePath);
                std::string pagedFilePath = osgDB::getRealPath(relativeFilePath);

                PagedFileCache::iterator findIt = pagedFileCache.find(pagedFilePath);
                osg::ref_ptr<osg::Node> spNode;
                if(findIt == pagedFileCache.end())
                {
                    spNode = osgDB::readNodeFile(pagedFilePath);
                    this->pagedFileUsageQueue.push_front(pagedFilePath);
                    this->pagedFileCache[pagedFilePath] = std::make_pair(spNode, this->pagedFileUsageQueue.begin());
                    if(this->pagedFileCache.size() > 10)
                    {
                        const std::string& lruPagedFilePath = this->pagedFileUsageQueue.back();
                        this->pagedFileCache.erase(lruPagedFilePath);
                        this->pagedFileUsageQueue.pop_back();
                    }
                }
                else
                {
                    spNode = findIt->second.first;

                    PagedFileUsageQueue::iterator itr = findIt->second.second;
                    this->pagedFileUsageQueue.splice(this->pagedFileUsageQueue.begin(), 
                                                     this->pagedFileUsageQueue, 
                                                     itr);
                    findIt->second.second = this->pagedFileUsageQueue.begin();
                }

                if(spNode.get() == nullptr)
                    std::cerr << "ERROR: failed to load external file " << pagedFilePath << std::endl;
                else
                {
                    osg::MatrixList worldMats = pagedLOD.getWorldMatrices();
                    for(size_t matIndex = 0; 
                        matIndex < worldMats.size(); 
                        ++matIndex)
                    {
                        const osg::Matrix& worldMat = worldMats.at(matIndex);
                        
                        osg::ref_ptr<osg::MatrixTransform> spMatrixTransform = 
                                                                new osg::MatrixTransform();

                        spMatrixTransform->setMatrix(worldMat);
                        
                        spMatrixTransform->addChild(spNode.get());

                        //this->pagedFiles.push_back(std::make_pair(spMatrixTransform.get(), 
                                                                  //pagedFilePath));

                        //spMatrixTransform->removeChild(spNode.get());
                        spMatrixTransform->getBound();//initialize bounds
                        osg::ref_ptr<osg::RefMatrixd> saveCur = this->curWorldTransform;
                        std::string saveCurFilePath = this->curFilePath;
                        this->curWorldTransform = nullptr;
                        this->curFilePath = osgDB::getFilePath(pagedFilePath);
                        spMatrixTransform->accept(*this);
                        this->curWorldTransform = saveCur.get();
                        this->curFilePath = saveCurFilePath;
                    }
                }
            }
        }

        virtual void apply(osg::LOD& lod) override
        {
            osg::BoundingSphere bsphere = transformBSphere(lod.getBound());
            if(this->filterBBox.contains(bsphere.center()) == false)
            {
                osg::Vec2 bsphereXYCenter = osg::Vec2(bsphere.center().x(), bsphere.center().y());
                osg::Vec2 bboxXYCenter = osg::Vec2(this->filterBBox.center().x(), this->filterBBox.center().y());
                osg::Vec2 diffVector = bsphereXYCenter - bboxXYCenter;
                float dist2 = diffVector.length2();
                float bboxHalfWidth = (this->filterBBox.xMax() - this->filterBBox.xMin()) * 0.5f;
                float bboxHalfHeight = (this->filterBBox.yMax() - this->filterBBox.yMin()) * 0.5f;
                float bboxXYRadius2 = (bboxHalfWidth * bboxHalfWidth) + (bboxHalfHeight * bboxHalfHeight);
                if(dist2 > bsphere.radius2() + bboxXYRadius2)
                {
                    //traverse(lod);
                    return;
                }
            }

            if(lod.getNumChildren() == 0)
            {
                //traverse(lod);
                return;
            }

            //we only want the highest resolution
            for(size_t cIndex = 0; cIndex < lod.getNumChildren(); ++cIndex)
            {
                float minRange = lod.getMinRange(cIndex);
                if(minRange > 0.0f)
                    continue;
                osg::Node* pChild = lod.getChild(cIndex);

                pChild->accept(*this);
            }
        }
    };

    class TriangleOctTreeGrid
    {
    public:
        struct GridTree
        {
            glm::vec3 origin;
            osg::BoundingBox bbox;
            TriangleData triangleData;
            bool hasNormals;
            GridTree* pNeighbor;
            GridTree() : hasNormals(false), pNeighbor(NULL) {}
            ~GridTree() { if(pNeighbor != NULL) delete pNeighbor; }
            GridTree* addNeighborTree()
            {
                pNeighbor = new GridTree();
                pNeighbor->hasNormals = this->hasNormals;
                pNeighbor->origin = this->origin;
                pNeighbor->bbox = this->bbox;
                return pNeighbor;
            }

            bool floatsEqual(float a, float b) const
            {
                float diff = a - b;
                static float epsilon = 0.001f;
                return diff <= epsilon && diff >= -epsilon;
            }

            bool vertsEqual(const glm::vec3& lhs, const glm::vec3& rhs) const
            {
                if(floatsEqual(lhs.x, rhs.x) == false)
                    return false;
                if(floatsEqual(lhs.y, rhs.y) == false)
                    return false;
                return floatsEqual(lhs.z, rhs.z);
            }

            void add(osg::Texture2D* pTexture, const Triangle& tri)
            {
                TriangleData::iterator findIt = triangleData.find(pTexture);
                if(findIt == triangleData.end())
                    findIt = triangleData.insert(std::make_pair(pTexture, TriData())).first;
                TriData& triData = findIt->second;
                /*for(size_t itr = 0; itr < triData.triangles.size(); itr+=3)
                {
                    glm::vec3& v0 = triData.triangles.at(itr);
                    if(vertsEqual(v0, (glm::vec3&)tri.v0) == false)
                        break;
                    glm::vec3& v1 = triData.triangles.at(itr+1);
                    if(vertsEqual(v1, (glm::vec3&)tri.v1) == false)
                        break;
                    glm::vec3& v2 = triData.triangles.at(itr+2);
                    if(vertsEqual(v2, (glm::vec3&)tri.v2) == false)
                        break;
                    return;
                }*/

                triData.triBBox.expandBy(tri.v0);
                triData.triBBox.expandBy(tri.v1);
                triData.triBBox.expandBy(tri.v2);
                triData.triangles.push_back((glm::vec3&)tri.v0);
                triData.triangles.push_back((glm::vec3&)tri.v1);
                triData.triangles.push_back((glm::vec3&)tri.v2);
                triData.triangleTexCoords.push_back((glm::vec2&)tri.uv0);
                triData.triangleTexCoords.push_back((glm::vec2&)tri.uv1);
                triData.triangleTexCoords.push_back((glm::vec2&)tri.uv2);
                if(tri.hasNormals)
                {
                    triData.normals.push_back((glm::vec3&)tri.n0);
                    triData.normals.push_back((glm::vec3&)tri.n1);
                    triData.normals.push_back((glm::vec3&)tri.n2);
                    hasNormals = true;
                }
                else if(hasNormals)
                {
                    std::cerr << "Triangle has no per vertex normals, but it is grouped with other triangles that have per vertex normals!" << std::endl;
                    glm::vec3 up(0.0f, 0.0f, 1.0f);
                    triData.normals.push_back(up);
                    triData.normals.push_back(up);
                    triData.normals.push_back(up);
                }
                
                if(bbox.zMin() > tri.v0.z())
                    bbox.zMin() = tri.v0.z();
                if(bbox.zMax() < tri.v0.z())
                    bbox.zMax() = tri.v0.z();
                
                if(bbox.zMin() > tri.v1.z())
                    bbox.zMin() = tri.v1.z();
                if(bbox.zMax() < tri.v1.z())
                    bbox.zMax() = tri.v1.z();

                if(bbox.zMin() > tri.v2.z())
                    bbox.zMin() = tri.v2.z();
                if(bbox.zMax() < tri.v2.z())
                    bbox.zMax() = tri.v2.z();
            }
            //PagedFiles pagedFiles;
        };
        typedef std::vector<GridTree> GridTrees;
    private:
        GridTrees _grid;

        glm::uvec2 _gridDim;
        glm::vec3 _voxelMipMapSizeInMeters;
        glm::vec3 _voxelBorderSizeInMeters;
        glm::vec3 _triOrigin;
    public:
        TriangleOctTreeGrid(const glm::uvec2& gridDim, 
                            const osg::BoundingBox& inputBBox,
                            const glm::vec3& voxelMipMapSizeInMeters,
                            const glm::vec3& voxelBorderSizeInMeters) :
            _gridDim(gridDim),
            _voxelMipMapSizeInMeters(voxelMipMapSizeInMeters),
            _voxelBorderSizeInMeters(voxelBorderSizeInMeters),
            _triOrigin(inputBBox.xMin(), inputBBox.yMin(), inputBBox.zMin())
        {
            glm::vec3 origin;
            origin.z = inputBBox.zMin();
            //for(size_t z = 0; z < gridDim.z; ++z, origin.z += _voxelMipMapSizeInMeters.z)
            {
                origin.y = inputBBox.yMin();
                for(size_t y = 0; y < gridDim.y; ++y, origin.y += _voxelMipMapSizeInMeters.y)
                {
                    origin.x = inputBBox.xMin();
                    for(size_t x = 0; x < gridDim.x; ++x, origin.x += _voxelMipMapSizeInMeters.x)
                    {
                        GridTree gridTree;
                        gridTree.origin = origin;
                        gridTree.bbox.set(origin.x, origin.y, FLT_MAX,
                                          origin.x + _voxelMipMapSizeInMeters.x,
                                          origin.y + _voxelMipMapSizeInMeters.y,
                                          -FLT_MAX);

                        _grid.push_back(gridTree);
                    }
                }
            }
        }

    public:
        const GridTrees& getGrid() const { return _grid; }
        const glm::uvec2& getGridDim() const { return _gridDim; }

        void assignTriangles(TrianglesPerTexture& trianglesPerTexture)
        {
            //TODO make this CUDA
            for(TrianglesPerTexture::iterator itr = trianglesPerTexture.begin();
                itr != trianglesPerTexture.end();
                ++itr)
            {
                osg::ref_ptr<osg::Texture2D> spTexture = itr->first;
                Triangles& triangles = itr->second;
                for(size_t i = 0; i < triangles.size(); ++i)
                {                
                    glm::uvec2 minXY;
                    glm::uvec2 maxXY;
                    glm::uvec2 min(_gridDim);
                    glm::uvec2 max(0u);

                    const Triangle& triangle = triangles.at(i);
                    getVertGridTreeXY(triangle.v0, 
                                      minXY.x, maxXY.x,
                                      minXY.y, maxXY.y);
                    min = minXY;
                    max = maxXY;

                    getVertGridTreeXY(triangle.v1, 
                                      minXY.x, maxXY.x,
                                      minXY.y, maxXY.y);
                    min = glm::min(min, minXY);
                    max = glm::max(max, maxXY);

                    getVertGridTreeXY(triangle.v2, 
                                      minXY.x, maxXY.x,
                                      minXY.y, maxXY.y);
                    min = glm::min(min, minXY);
                    max = glm::max(max, maxXY);

                    //insert into each tree that this triangle is potentially
                    //contained inside
                    //for(glm::uint z = min.z; z <= max.z; ++z)
                    {
                        for(glm::uint y = min.y; y <= max.y; ++y)
                        {
                            for(glm::uint x = min.x; x <= max.x; ++x)
                            {
                                GridTree& gridTree = getGridTree(x, y);
                            
                                gridTree.add(spTexture.get(), triangle);
                            }
                        }
                    }
                }
            }
        }

        const TriangleData& getTriangles(size_t x, size_t y) const
        {
            const GridTree& gridTree = _grid.at((y * _gridDim.x) + x);
            const TriangleData& triData = gridTree.triangleData;

            return triData;
        }

        TriangleData& getTriangles(size_t x, size_t y)
        {
            GridTree& gridTree = _grid.at((y * _gridDim.x) + x);
            TriangleData& triData = gridTree.triangleData;
            
            return triData;
        }

        const glm::vec3& getOrigin(size_t x, size_t y) const
        {
            return _grid.at((y * _gridDim.x) + x).origin;
        }

        GridTree& getGridTree(size_t x, size_t y)
        {
             return _grid.at((y * _gridDim.x) + x);
        }

        const GridTree& getGridTree(size_t x, size_t y) const
        {
             return _grid.at((y * _gridDim.x) + x);
        }

    private:

        void computeMinMaxGridCell(float vert, 
                                   float triOrigin, 
                                   float voxelMipMapSizeInMeters, 
                                   float voxelBorderSizeInMeters, 
                                   unsigned int gridDim,
                                   glm::uint& min, glm::uint& max)
        {
            float distFromOrigin = vert - triOrigin;
            min = max = static_cast<glm::uint>(distFromOrigin / voxelMipMapSizeInMeters);

            //if the x is within the distance of the voxel border from the edge 
            //then add it to both grids so that we can compute voxelization overlaps for 
            //interpolation
            float maxBorder = (min+1) * voxelMipMapSizeInMeters;

            float dist = maxBorder - distFromOrigin;
            if(dist <= voxelBorderSizeInMeters)
                max = min + 1;
            else if(min != 0u)
            {
                float minBorder = maxBorder - voxelMipMapSizeInMeters;
                dist = distFromOrigin - minBorder;
                if(dist <= voxelBorderSizeInMeters)
                {
                    max = min;
                    min = min - 1;
                }
            }
                
            if(min >= gridDim)
                min = gridDim - 1;

            if(max >= gridDim)
                max = gridDim - 1;
        }

        void getVertGridTreeXY(const osg::Vec3& vert,
                                glm::uint& minX,
                                glm::uint& maxX,
                                glm::uint& minY,
                                glm::uint& maxY)
        {
            if(vert.x() < _triOrigin.x)
                minX = maxX = 0u;
            else
            {
                computeMinMaxGridCell(vert.x(), 
                                      _triOrigin.x, 
                                      _voxelMipMapSizeInMeters.x, 
                                      _voxelBorderSizeInMeters.x, 
                                      _gridDim.x, 
                                      minX, maxX);
            }

            if(vert.y() < _triOrigin.y)
                minY = maxY = 0u;
            else
            {
                computeMinMaxGridCell(vert.y(), 
                                      _triOrigin.y, 
                                      _voxelMipMapSizeInMeters.y, 
                                      _voxelBorderSizeInMeters.y, 
                                      _gridDim.y, 
                                      minY, maxY);
            }
        }
    };

    void removeProgressFile()
    {
        int error = remove(_progressFile.c_str());
        if(error != 0)
        {
            const char* errorString = strerror(errno);
            std::cerr << "Failed to remove " 
                      << _progressFile 
                      << ", error: " 
                      << errorString 
                      << std::endl;
        }
    }

    void updateProgressFile(size_t x, size_t y, size_t z)
    {
        std::fstream fileOut;
        fileOut.open(_progressFile, std::fstream::out | std::fstream::trunc);
        if(fileOut.is_open())
            fileOut << x << " " << y << " " << z << std::endl;
        else
        {
            std::cerr << "Failed to open " << _progressFile << "." << std::endl;
        }
    }

    bool generate(size_t startGridX, size_t startGridY, size_t startGridZ,
                  bool generateRootFileOnly,
                  const osg::BoundingBox& filterBBox,
                  const std::string& terrainTexturePrefix,
                  const TerrainTextureSet& terrainTextures,
                  bool binaryOutput, bool compressedOutput,
                  int numOctTrees)
    {
        if(_spInputNode.get() == nullptr ||
           osgDB::fileExists(_outputDir) == false)
        {
            return false;
        }

        if(compressedOutput)
            binaryOutput = true;

        //compute number of separate gigavoxel trees we'll need
        osg::BoundingBox inputBBox;
        if(filterBBox.valid())
            inputBBox = filterBBox;
        else
            inputBBox.expandBy(_spInputNode->getBound());

        float inputWidth = (inputBBox.xMax() - inputBBox.xMin());
        float inputDepth = (inputBBox.yMax() - inputBBox.yMin());
        //float inputHeight = (inputBBox.zMax() - inputBBox.zMin());
        glm::vec2 octTreeGridFDim;
        octTreeGridFDim.x = inputWidth / _maxVoxelMipMapSizeMeters.x;
        octTreeGridFDim.y = inputDepth / _maxVoxelMipMapSizeMeters.y;
        //octTreeGridFDim.z = inputHeight / _maxVoxelMipMapSizeMeters.z;

        glm::uvec2 octTreeGridUDim = static_cast<glm::uvec2>(glm::ceil(octTreeGridFDim));
        
        cuda::Voxelizer voxelizer(_maxVoxelMipMapDimensions, 
                                  _brickDimensions);
        //compute actual voxel size after adding a border to each edge
        glm::vec3 voxelBorderSizeInMeters = _voxelSizeMeters *
            static_cast<glm::vec3>(voxelizer.getExtraVoxChunk());//16.0f;// * static_cast<float>((glm::max(glm::max(_maxVoxelMipMapDimensions.x,
                                                             //                    _maxVoxelMipMapDimensions.y),
                                                             //                    _maxVoxelMipMapDimensions.z) >> 1u));

        TriangleOctTreeGrid octTreeGrid(octTreeGridUDim, 
                                        inputBBox, 
                                        _maxVoxelMipMapSizeMeters, 
                                        voxelBorderSizeInMeters);

        

        std::cout << "Generating grid "
                  << octTreeGridUDim.x 
                  << "x" 
                  << octTreeGridUDim.y 
                  << std::endl;

        if(!voxelizer.initCuda()
           || !voxelizer.allocateVoxelMemory())
        {
            std::cerr << voxelizer.getErrorMessage() << std::endl;
            return false;
        }

        std::cout << "Starting at "
                  << startGridX
                  << " "
                  << startGridY
                  << " "
                  << startGridZ
                  << std::endl;

        TriangleCollector bboxTriVis;
        //octTreeGrid.assignPagedFiles(bboxTriVis.pagedFiles);
        //build one oct-tree per grid cell
        //for(size_t gridZ = 0;
        //    gridZ < octTreeGridUDim.z;
        //    ++gridZ)
        {
            size_t gridY = startGridY;
            size_t gridX = startGridX;
            size_t gridZ = startGridZ;
            for( ;
                gridY < octTreeGridUDim.y;
                ++gridY)
            {
                for( ;
                    gridX < octTreeGridUDim.x;
                    ++gridX)
                {
                    osg::BoundingBox gridTreeBBox = octTreeGrid.getGridTree(gridX, gridY).bbox;
                    
                    osg::Vec3 newMin(gridTreeBBox.xMin() - voxelBorderSizeInMeters.x,
                                     gridTreeBBox.yMin() - voxelBorderSizeInMeters.y,
                                     -FLT_MAX);// - voxelBorderSizeInMeters.z);
                    osg::Vec3 newMax(gridTreeBBox.xMax() + voxelBorderSizeInMeters.x,
                                     gridTreeBBox.yMax() + voxelBorderSizeInMeters.y,
                                     FLT_MAX);// + voxelBorderSizeInMeters.z);
                    gridTreeBBox.expandBy(newMin);
                    gridTreeBBox.expandBy(newMax);

                    bboxTriVis.filterBBox = gridTreeBBox;

                    _spInputNode->accept(bboxTriVis);

                    TriangleOctTreeGrid::GridTree& gridTree = octTreeGrid.getGridTree(gridX, gridY);
                    TriangleData& triData = gridTree.triangleData;
                    triData.clear();

                    octTreeGrid.assignTriangles(bboxTriVis.trianglesPerTexture);

                    //reset the visitor
                    bboxTriVis.trianglesPerTexture.clear();
    
                    if(triData.size() == 0)
                        continue;

                    float inputHeight = gridTree.bbox.zMax() - gridTree.bbox.zMin();
                    int numZGrids = static_cast<int>(std::ceilf(inputHeight / _maxVoxelMipMapSizeMeters.z));
                    float gridHeight = numZGrids * _maxVoxelMipMapSizeMeters.z;
                    if(gridHeight > inputHeight)
                    {
                        float diff = (gridHeight - inputHeight) * 0.5f;
                        gridTree.bbox.zMin() -= diff;
                        gridTree.bbox.zMax() = gridTree.bbox.zMin() + _maxVoxelMipMapSizeMeters.z;
                    }
                    gridTree.origin.z = std::floorf(gridTree.bbox.zMin());
                    TriangleOctTreeGrid::GridTree* pCurGridTree = &gridTree;
                    for( ;
                        gridZ < numZGrids;
                        ++gridZ)
                    {
                        std::stringstream outputDirStr;
                        outputDirStr << _outputDir 
                                     << "/grid_" 
                                     << gridX << "_"
                                     << gridY << "_"
                                     << gridZ;

                        std::string outputDir = osgDB::getRealPath(outputDirStr.str());
                        if(!osgDB::makeDirectory(outputDir))
                        {
                            std::cerr << "ERROR: failed to create output directory " << outputDir << std::endl;
                            return false;
                        }

                        updateProgressFile(gridX, gridY, gridZ);

                        if(numOctTrees == 0)
                        {
                            std::cout << "Generated max number of oct-trees. Quiting now." << std::endl;
                            return false;
                        }

                        std::cout << "Generating GigaVoxel Octree: " 
                                  << osgDB::getSimpleFileName(outputDir)
                                  << std::endl;

                        const glm::vec3& octTreeOrigin = pCurGridTree->origin;

                        bool hasNormals = pCurGridTree->hasNormals;

                        bool trisVoxelized = true;
                        bool doVoxelization = generateRootFileOnly == false;//(gridX == 2);
                        int status = 0;
                        if(doVoxelization)
                        {
                            for(size_t tries = 0; tries < 3; ++tries)
                            {
                                status = Voxelize(voxelizer,
                                                octTreeOrigin,
                                                _voxelSizeMeters,
                                                triData,
                                                terrainTexturePrefix,
                                                terrainTextures,
                                                hasNormals,
                                                outputDir, 
                                                binaryOutput,
                                                compressedOutput);

                                voxelizer.deallocateTriangleMemory();

                                if(status == 0 || status == -1)
                                {
                                    trisVoxelized = (status == 0);
                                    break;
                                }
                                else
                                    voxelizer.resetCuda();
                            }

                            if(status == 0 || status == -1)
                                --numOctTrees;
                        }

                        std::string emptyFile = outputDir;
                        emptyFile += "/";
                        emptyFile += "GigaVoxelOctreeIsEmpty.txt";
                        bool emptyMarkerExists = osgDB::fileExists(emptyFile);
                        if(trisVoxelized == false)
                        {
                            //mark this grid as empty
                            if(emptyMarkerExists == false)
                            {
                                std::fstream emptyMarker(emptyFile, std::fstream::out);
                                emptyMarker << "This octree voxelized no triangles." << std::endl;
                            }
                        }
                        else
                        {
                            //make sure this grid is not marked as empty
                            if(emptyMarkerExists)
                            {
                                status = remove(emptyFile.c_str());
                                if(status != 0)
                                {
                                    std::cerr << "Failed to remove empty tree marker on non-empty octree: " 
                                              << osgDB::getSimpleFileName(outputDir)
                                              << std::endl;
                                }
                            }
                        }
                        
                        if(status == 0 || status == -1)
                            std::cout << "Generation done" << (trisVoxelized ? "." : " - no triangles voxelized.") << std::endl;
                        else//error ocurred, so quit
                        {
                            if(gridX == startGridX && gridY == startGridY && gridZ == startGridZ)
                            {
                                std::cerr << "Error ocurred on starting grid cell, skipping it for now." << std::endl;
                                //if this is the grid we started on then skip it and go to next one
                                //otherwise we'll try again with this grid
                                gridZ += 1;
                                if(gridZ == numZGrids)
                                {
                                    gridZ = startGridZ;
                                    gridX += 1;
                                    if(gridX == octTreeGridUDim.x)
                                    {
                                        gridX = startGridX;
                                        gridY += 1;
                                        if(gridY == octTreeGridUDim.y)
                                        {
                                            std::cerr << "Starting grid cell is last grid cell... Finished." << std::endl;
                                            removeProgressFile();//remove this so script doesn't loop
                                            return false;//this is the last grid so just return
                                        }
                                    }
                                }
                            }
                            updateProgressFile(gridX, gridY, gridZ);
                            return false;
                        }

                        if(gridZ+1 != numZGrids)
                        {
                            pCurGridTree = pCurGridTree->addNeighborTree();
                            pCurGridTree->origin.z += _maxVoxelMipMapSizeMeters.z;
                            pCurGridTree->bbox.zMin() = pCurGridTree->bbox.zMax();
                            pCurGridTree->bbox.zMax() = pCurGridTree->bbox.zMin() + _maxVoxelMipMapSizeMeters.z;
                        }
                    }
                    gridZ = 0;

                    triData.clear();
                }
                gridX = 0;
            }
        }

        voxelizer.deallocateVoxelMemory();

        removeProgressFile();//don't leave behind a progress file
        
        if(generateRootFileOnly || (startGridX == 0 && startGridY == 0 && startGridZ == 0))
            return writeRootFile(octTreeGrid,
                                 octTreeGridUDim, 
                                 compressedOutput);
        else
            return true;
    }

private:
    bool writeRootFile(const TriangleOctTreeGrid& grid,
                       const glm::uvec2& gridDim,
                       bool compressed)
    {
        std::string rootFile = _outputDir + "/";
        rootFile += "root.gvp";

        rootFile = osgDB::getRealPath(rootFile);
        std::fstream fileOut(rootFile, std::fstream::out);
        if(!fileOut.is_open())
        {
            std::cerr << "ERROR: failed to open root file for writing: "
                      << rootFile
                      << std::endl;
            return false;
        }

        std::cout << "Writing root file: " << rootFile << std::endl;

        fileOut << "<OctTrees "
                << "Compressed='" << (compressed ? "YES" : "NO") << "' "
                << "BrickXSize='" << _brickDimensions.x << "' "
                << "BrickYSize='" << _brickDimensions.y << "' "
                << "BrickZSize='" << _brickDimensions.z << "' "
                << ">" << std::endl;

        for(size_t x = 0; x < gridDim.x; ++x)
        {
            for(size_t y = 0; y < gridDim.y; ++y)
            {
                const TriangleOctTreeGrid::GridTree& rootGridTree = grid.getGridTree(x, y);
                size_t z = 0;
                for(const TriangleOctTreeGrid::GridTree* pCurGridTree = &rootGridTree;
                    pCurGridTree != NULL;
                    pCurGridTree = pCurGridTree->pNeighbor,
                    ++z)
                {
                    const TriangleOctTreeGrid::GridTree& gridTree = *pCurGridTree;
                    
                    std::stringstream outputDir;
                    outputDir << "grid_" 
                             << x << "_"
                             << y << "_"
                             << z;
                    std::string emptyFile = _outputDir + "/" + outputDir.str();
                    emptyFile += "/";
                    emptyFile += "GigaVoxelOctreeIsEmpty.txt";
                    if(osgDB::fileExists(emptyFile))
                    {
                        std::cout << "Skipping empty GigaVoxelOctree at " << emptyFile << std::endl;
                        continue;//this gvo was generated, but no tris were voxelized
                    }
                        
                    std::string treeFileRelPath = outputDir.str() + "/tree.gvx";
                    std::string fullPath = _outputDir + "/" + treeFileRelPath;
                    if(osgDB::fileExists(fullPath) == false)
                    {
                        std::cout << "Skipping non-existent GigaVoxelOctree at " << fullPath << std::endl;
                        continue;
                    }

                    std::cout << "Adding GigaVoxelOctree at " << treeFileRelPath << std::endl;

                    fileOut << "    "
                            << "<OctTree "
                            << "CenterX=\""
                            << gridTree.bbox.center().x() << "\" "
                            << "CenterY=\""
                            << gridTree.bbox.center().y() << "\" "
                            << "CenterZ=\""
                            << gridTree.bbox.center().z() << "\" "
                            << "Radius=\""
                            << gridTree.bbox.radius() << "\" "
                            << "SizeXMeters=\"" 
                            << _maxVoxelMipMapSizeMeters.x << "\" "
                            << "SizeYMeters=\""
                            << _maxVoxelMipMapSizeMeters.y << "\" "
                            << "SizeZMeters=\"" 
                            << _maxVoxelMipMapSizeMeters.z << "\" "
                            << "TreeFile=\""
                            << treeFileRelPath
                            << "\" />"
                            << std::endl;
                }
            }
        }
        fileOut << "</OctTrees>" << std::endl;

        std::cout << "Done writing root file." << std::endl;

        return true;
    }
};

static void OnExit()
{
    char* debuggerEnv = getenv("VOXELIZER_DEBUGGER_ENV");
    if(debuggerEnv != NULL)
    {
        std::cout << "Hit Enter to exit..." << std::endl;
        char key;
        std::cin.get(key);
        std::cin.clear();
    }
}
     
int main(int argc, char* argv[])
{
    //std::cout << "Sleeping" << std::endl;
    //_sleep(10000);
    //std::cout << "Done Sleeping" << std::endl;

    osg::ArgumentParser arguments(&argc,argv);

    arguments.getApplicationUsage()->setApplicationName(arguments.getApplicationName());
    arguments.getApplicationUsage()->setDescription(arguments.getApplicationName() +
                                                    " is a application for generating a GigaVoxel Oct-Tree "
                                                    "from polygonal data.");
    arguments.getApplicationUsage()->setCommandLineUsage(arguments.getApplicationName() +
                                                         " --input <filename> "
                                                         "--output <directory> [ "
                                                         "--voxel-size <width centimeters> <height centimeters> <depth centimeters> " 
                                                         "--brick-size <width> <height> <depth> "
                                                         "--filter-bbox <min-x> <min-y> <min-z> <max-x> <max-y> <max-z>] "
                                                         "--grid-start-x <start-x> --grid-start-y <start-y> --grid-start-z <start-z> "
                                                         "--output-count <number of oct-trees to output before quiting> "
                                                         "--geocentric]");
    arguments.getApplicationUsage()->addCommandLineOption("-h or --help","Display command line parameters");

    if(arguments.read("-h") || arguments.read("--help"))
    {
        arguments.getApplicationUsage()->write(std::cerr);
        OnExit();
        return 1;
    }

    std::string outputDir;
    if(!arguments.read("--output", outputDir))
    {
        std::cerr << "ERROR: Please specify an output directory with --output <directory>." << std::endl;
        OnExit();
        return 1;
    }
    else
    {
        outputDir = osgDB::getRealPath(outputDir);

        osgDB::FileType fileType = osgDB::fileType(outputDir);
        if(fileType == osgDB::FILE_NOT_FOUND)
        {
            if(osgDB::makeDirectory(outputDir) == false)
            {
                std::cerr << "ERROR: Unable to create directory for output at " << outputDir << std::endl;
                OnExit();
                return 1;
            }
        }
        else if(fileType != osgDB::DIRECTORY)
        {
            std::cerr << "ERROR: The specified output directory is not a directory." << std::endl;
            OnExit();
            return 1;
        }
    }

    std::string inputFileName;
    if(!arguments.read("--input", inputFileName))
    {
        std::cerr << "ERROR: Please specify an input file with --input <filename>." << std::endl;
        OnExit();
        return 1;
    }
    else if(!osgDB::fileExists(inputFileName))
    {
        std::cerr << "ERROR: The specified input file " << inputFileName << " does not exist." << std::endl;
        OnExit();
        return 1;
    }
    
    inputFileName = osgDB::getRealPath(inputFileName);

    osgDB::Options* pOptions = osgDB::Registry::instance()->getOptions();
    if(pOptions == nullptr)
        pOptions = new osgDB::Options();
    pOptions->setObjectCacheHint(osgDB::Options::CACHE_ALL);
    osgDB::Registry::instance()->setOptions(pOptions);
    s_pOsgFileReader = new OsgFileReader();
    osgDB::Registry::instance()->setReadFileCallback(s_pOsgFileReader);

    //initially don't load externally referenced files
    pOptions->setOptionString("noLoadExternalReferenceFiles");
    osg::ref_ptr<osg::Node> spRootNode = osgDB::readNodeFile(inputFileName, pOptions);
    if(spRootNode.get() == nullptr)
    {
        std::cerr << "ERROR: Failed to read " << inputFileName << std::endl;
        OnExit();
        return 1;
    }
    
    bool isGeocentric = arguments.read("--geocentric");

    if(isGeocentric)
    {
        osg::Matrixd gccInvXForm;
        osg::Vec3d center = spRootNode->getBound().center();
        osg::Matrixd inverseTranslate;
        inverseTranslate.makeTranslate(-center);
        osg::Matrixd inverseRotation;
        osg::Vec3d normal = center;
        normal.normalize(); 
        inverseRotation.makeRotate(normal, osg::Vec3d(0.0, 0.0, 1.0));
        //{This is wrong?
        //gccInvXForm *= inverseRotation;
        //gccInvXForm *= inverseTranslate;
        //osg::Vec3d test1 = gccInvXForm * (center + normal);
        //osg::Vec3d test2 = (center + normal) * gccInvXForm;
        //osg::Vec3d normal2 = osg::Matrixd::transform3x3(gccInvXForm, normal);
        //osg::Vec3d normal3 = inverseRotation * normal;
        //osg::Vec3d normal4 = normal * inverseRotation;
        //osg::Vec3d normal5;
        //}
        //{This is right!
        //gccInvXForm.makeIdentity();
        gccInvXForm *= inverseTranslate;
        gccInvXForm *= inverseRotation;
        //osg::Vec3d test1 = gccInvXForm * (center + normal);
        //osg::Vec3d test2 = (center + normal) * gccInvXForm;
        //osg::Vec3d normal2 = osg::Matrixd::transform3x3(gccInvXForm, normal);
        //osg::Matrixd rot(gccInvXForm.getRotate());
        //osg::Vec3d normal3 = rot * normal;
        //osg::Vec3d normal4 = normal * rot;
        //osg::Vec3d normal5;
        //}
        osg::ref_ptr<osg::MatrixTransform> spGccInvXForm = new osg::MatrixTransform(gccInvXForm);
        spGccInvXForm->addChild(spRootNode.get());
        spRootNode = spGccInvXForm.get();
        spRootNode->getBound();
    }

    pOptions->setOptionString("");//reset this now so that paged terrain loads external files (if any)

    glm::vec3 voxelSize(10.0f, 10.0f, 10.0f);//width, height, depth of a single voxel in centimeters
    arguments.read("--voxel-size", 
                   voxelSize.x,
                   voxelSize.y,
                   voxelSize.z);

    //convert to meters
    voxelSize /= 100.0f;

    glm::uvec3 brickDimensions(30u, 30u, 30u);//width, height, depth of a voxel brick
    arguments.read("--brick-size",
                   brickDimensions.x,
                   brickDimensions.y,
                   brickDimensions.z);

    //glm::uvec3 maxVoxelMipMapDimension(480, 480, 480);//max resolution of highest detail voxel mip map
    //glm::uvec3 maxVoxelMipMapDimension(960, 960, 960);//max resolution of highest detail voxel mip map
    glm::uvec3 maxVoxelMipMapDimension(1920, 1920, 1920);
    
    arguments.read("--max-voxels-size", 
                    maxVoxelMipMapDimension.x,
                    maxVoxelMipMapDimension.y,
                    maxVoxelMipMapDimension.z);
    
    osg::BoundingBox filterBBox;
    
    if(arguments.read("--filter-bbox",
                   filterBBox.xMin(),
                   filterBBox.yMin(),
                   filterBBox.xMax(),
                   filterBBox.yMax()))
    {
        filterBBox.zMin() = -FLT_MAX;
        filterBBox.zMax() = FLT_MAX;

        std::cout << "Filter BBox: "
                  << filterBBox.xMin()
                  << " "
                  << filterBBox.yMin()
                  << " "
                  << filterBBox.xMax()
                  << " "
                  << filterBBox.yMax()
                  << std::endl;
    }

    std::string terrainTexturePrefix = "";
    arguments.read("--terrain-texture-prefix", terrainTexturePrefix);

    TerrainTextureSet terrainTextures;
    std::string terrainTexture;
    while(arguments.read("--terrain-texture", terrainTexture))
        terrainTextures.insert(terrainTexture);

    unsigned int gridStartX = 0;
    arguments.read("--grid-start-x", gridStartX);

    unsigned int gridStartY = 0;
    arguments.read("--grid-start-y", gridStartY);
    
    unsigned int gridStartZ = 0;
    arguments.read("--grid-start-z", gridStartZ);

    int numOctTrees = -1;
    arguments.read("--output-count", numOctTrees);

    bool generateRootFileOnly = arguments.read("--generate-root-file-only");

    PagedGigaVoxelOctTreeGenerator generator;
    
    generator.setVoxelizationParams(voxelSize, brickDimensions, maxVoxelMipMapDimension);

    generator.setInput(inputFileName, spRootNode.get());

    generator.setOutputDirectory(outputDir);

    bool outputCompressed = true;
    bool outputBinary = true;
    generator.generate(gridStartX, gridStartY, gridStartZ,
                       generateRootFileOnly,
                       filterBBox,
                       terrainTexturePrefix,
                       terrainTextures,
                       outputBinary, 
                       outputCompressed,
                       numOctTrees);
    OnExit();
#if 0
    int numVerts = 6;    
    glm::vec3 vertArray[] = { 
        //front face
        //glm::vec3(0.5f, 15.5f, 7.5f),
        //glm::vec3(7.5f, 15.5f, 7.5f),
        //glm::vec3(7.5f, 15.5f, 0.5f),
        glm::vec3(0.5f, 15.5f, 7.5f),
        glm::vec3(7.5f, 15.5f, 7.5f),
        glm::vec3(7.5f, 23.5f, 7.5f),
        //top face 1
        /*glm::vec3(0.5f, 0.5f, 2.5f),
        glm::vec3(2.5f, 0.5f, 2.5f),
        glm::vec3(2.5f, 2.5f, 2.5f),
        //top face 2
        glm::vec3(2.5f, 2.5f, 2.5f),
        glm::vec3(0.5f, 2.5f, 2.5f),
        glm::vec3(0.5f, 0.5f, 2.5f),
        //left face 1
        glm::vec3(0.5f, 0.5f, 0.5f),
        glm::vec3(0.5f, 2.5f, 2.5f),
        glm::vec3(0.5f, 2.5f, 0.5f),
        //left face 2
        glm::vec3(0.5f, 0.5f, 0.5f),
        glm::vec3(0.5f, 0.5f, 2.5f),
        glm::vec3(0.5f, 2.5f, 2.5f),
        //right face 1
        glm::vec3(0.5f, 0.5f, 0.5f),
        glm::vec3(0.5f, 2.5, 0.5f),
        glm::vec3(2.5f, 0.5f, 2.5f),
        //right face 2
        glm::vec3(2.5f, 0.5f, 2.5f),
        glm::vec3(0.5f, 2.5f, 0.5f),
        glm::vec3(2.5f, 2.5f, 2.5f),*/
        //back face
        glm::vec3(2.5f, 0.5f, 7.5f),
        glm::vec3(2.5f, 2.5f, 7.5f),
        glm::vec3(0.5f, 2.5f, 7.5f)
    };

    std::vector<glm::vec3> verts(&vertArray[0], &vertArray[numVerts]);

    glm::vec3 p(0.0f, 0.0f, 0.0f);
    glm::vec3 deltaP(1.0f, 1.0f, 1.0f);
    glm::uvec3 voxDim(64u, 64u, 64u);
    glm::uvec3 brickDim(8u, 8u, 8u);
    std::string outputDir = ".";

    int *voxels = new int[voxDim.x * voxDim.y * voxDim.z];
    memset(voxels, 0, sizeof(int) * voxDim.x * voxDim.y * voxDim.z);
    //cpuTest(&verts.front(), verts.size(), p, deltaP, voxels, voxDim);

    Voxelize(verts, voxDim, brickDim, p, deltaP, outputDir);
    
    std::cout << "Hit Enter to exit..." << std::endl;
    char key;
    std::cin.get(key);
    std::cin.clear();
#endif

    return 0;
}
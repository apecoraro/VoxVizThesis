#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "Voxelizer.h"

#include <fstream>
#include <hip/hip_runtime.h>
#include <>

#include <cmath>
#include <stdio.h>
#include <iostream>
#include <string>
#include <vector>
#include <set>
#include <list>

using namespace cuda;

//kernel declaration
__global__ void ComputeEdgesFaceNormalsAndBounds(const glm::vec3* pVerts,
                                                 size_t numVerts, 
                                                 glm::vec3 p,
                                                 glm::vec3 deltaP,
                                                 glm::uvec3 voxDim,
                                                 glm::vec3* pEdges, 
                                                 cuda::VoxNorm* pNormals, 
                                                 cuda::VoxelBBox* pBounds);

__global__ void ComputeVoxelization(const glm::vec3* pVerts,
                                    size_t triOffset, 
                                    const glm::vec3* pEdges, 
                                    const cuda::VoxNorm* pNormals, 
                                    const cuda::VoxelBBox* pBounds,
                                    glm::vec3 p,
                                    glm::vec3 deltaP,
                                    glm::uvec3 minVoxChunk,
                                    glm::uvec3 maxVoxChunk,
                                    glm::uint zOffset,
                                    hipPitchedPtr pdevVoxelTriCounts,
                                    hipPitchedPtr pdevVoxelTriIndices);

//__global__ void ComputeVoxelizationAverages(hipPitchedPtr pvoxelTriCountsDevPtr,
//                                            hipPitchedPtr voxelColorsDevPtr,
//                                            glm::uvec3 voxDim);

bool BindTextureToArray(hipArray* pgImageArray,
                        hipChannelFormatDesc imageDesc,
                        hipTextureAddressMode addressMode0,
                        hipTextureAddressMode addressMode1);

__global__ void ComputeColorsAndNormals(size_t triGrpOffset,
                                        const glm::vec3* pVerts,
                                        const glm::vec3* pVtxNormals,
                                        const glm::vec3* pFaceNormals,
                                        const glm::vec2* pUVs,
                                        bool isTerrain,
                                        glm::vec3 voxOrigin,
                                        glm::vec3 voxSizeMeters,
                                        hipPitchedPtr voxelTriCountsDevPtr,
                                        hipPitchedPtr voxelTriIndicesDevPtr,
                                        hipPitchedPtr voxelColorsDevMipMapPtr,
                                        hipPitchedPtr voxelNormalsDevMipMapPtr,
                                        glm::uvec3 voxWriteDim);

__global__ void ComputeColorsAsNormals(size_t triGrpOffset,
                                       const glm::vec3* pVerts,
                                       const glm::vec3* pVtxNormals,
                                       const glm::vec3* pFaceNormals,
                                       glm::vec3 voxOrigin,
                                       glm::vec3 voxSizeMeters,
                                       hipPitchedPtr voxelTriCountsDevPtr,
                                       hipPitchedPtr voxelTriIndicesDevPtr,
                                       hipPitchedPtr voxelColorsDevMipMapPtr,
                                       hipPitchedPtr voxelNormalsDevMipMapPtr,
                                       glm::uvec3 voxWriteDim);


__global__ void ComputeVoxelMipMap64(
                        glm::uint voxInputDimX, glm::uint voxInputDimY, glm::uint voxInputDimZ,//input size
                        hipPitchedPtr inputVoxelColors,//input colors
                        hipPitchedPtr inputVoxelNormals,//input normals
                        glm::uint voxDimX, glm::uint voxDimY, glm::uint voxDimZ,//output size
                        hipPitchedPtr mipMapColors,//output colors
                        hipPitchedPtr mipMapNormals);//output normals

__global__ void ComputeVoxelMipMap8(
                        glm::uint voxInputDimX, glm::uint voxInputDimY, glm::uint voxInputDimZ,//input size
                        hipPitchedPtr inputVoxelColors,//input colors
                        hipPitchedPtr inputVoxelNormals,//input normals
                        glm::uint voxDimX, glm::uint voxDimY, glm::uint voxDimZ,//output size
                        hipPitchedPtr mipMapColors,//output colors
                        hipPitchedPtr mipMapNormals);//output normals

__global__ void ComputeOctTreeNodeConstColor(hipPitchedPtr pVoxelColors, 
                                             cuda::VoxColor* pOctTreeNodesConstColorPtr,
                                             glm::ivec3 voxOffset,
                                             glm::ivec3 offsetToBrickBorder,
                                             glm::ivec3 voxDim,
                                             glm::uvec3 octTreeChunkDim,
                                             glm::uvec3 octTreeDim,
                                             glm::ivec3 nodeOffset,
                                             glm::uvec3 brickDim);

__global__ void ComputeOctTreeNodeType(hipPitchedPtr pVoxelColors, 
                                       glm::uint* pOctTreeNodesDevPtr,
                                       cuda::VoxColor* pOctTreeNodesConstColorPtr,
                                       glm::ivec3 voxOffset,
                                       glm::ivec3 offsetToBrickBorder,
                                       //glm::ivec3 brickSampleOffset,
                                       glm::ivec3 voxDim,
                                       glm::ivec3 fullVoxDim,
                                       glm::uvec3 octTreeDim,
                                       glm::uvec3 brickDim,
                                       int xOffset);

static size_t s_totalAllocatedDeviceMemory = 0u;
static hipError_t s_cudaStatus = hipSuccess;
#ifdef __DEBUG__
static bool s_debug_svg = false;
static bool s_debug_mipmaps = false;
static bool s_debug_output = false;
static bool s_debug_voxels = false;
static bool s_debug_scalars = false;
#endif

Voxelizer::Voxelizer(const glm::uvec3& voxDim,
                     const glm::uvec3& brickDim,
                     const glm::uvec3& voxChunkDim) :
    _pgVerts(0),
    _pgEdges(0),
    _pgFaceNormals(0),
    _pgVtxNormals(0),
    _pgBounds(0),
    _pgUVs(0),
    _totalVertCount(0),
    _totalVertNormalsCount(0),
    _totalUVCount(0),
    _voxDim(voxDim),
    _voxChunkDim(voxChunkDim),
    _brickDim(brickDim),
    _octTreeNodesWriteIndex(0u)
{
    //compute ratio full res voxels to lowest res voxels
    _extraVoxChunk = (_voxDim / _brickDim);
    _extraVoxChunk >>= 1u;
    //move the origin to new edge of voxels
    _numMipMapLevels = 1;
    glm::uvec3 mipMapDim = _brickDim;
    while(mipMapDim != voxDim)
    {
        mipMapDim <<= 1;
        ++_numMipMapLevels;
    }
 }

void Voxelizer::setVoxelizationParams(const glm::vec3& p,
                                      const glm::vec3& deltaP)
{
    _p = p;
    _deltaP = deltaP;
    _offsetP = _p - (static_cast<glm::vec3>(_extraVoxChunk) * _deltaP);
}   
 
bool Voxelizer::allocateVoxelMemory()
{
    glm::uvec3 voxChunkDim = _voxChunkDim;
    VoxelColorMipMaps& voxelColorMipMaps = _voxelColorMipMaps;
    voxelColorMipMaps.resize(_numMipMapLevels);
    VoxelNormalMipMaps& voxelNormalMipMaps = _voxelNormalMipMaps;
    voxelNormalMipMaps.resize(_numMipMapLevels);
    allocateHostMipMaps(voxelColorMipMaps, voxelNormalMipMaps,
                        voxChunkDim.x, //allocate nearest multiple of four
                        voxChunkDim.y, 
                        voxChunkDim.z);//allocate with 1 voxel border

    //allocate memory for voxel mip maps
    if(!allocateDeviceVoxelizationMemory(voxChunkDim,
                                         _voxelTriIndicesDevPtr,
                                         _voxelTriCountsDevPtr))
    {
        return false;
    }

    if(!allocateDeviceMipMaps(voxChunkDim,
                              _voxelColorsDevMipMapPtr,
                              _voxelNormalsDevMipMapPtr,
                              _voxelDeviceMipMaps))
    {
        return false;
    }

    _octTreeDeviceBuffers.resize(voxelColorMipMaps.size());
    if(!allocateOctTreeDeviceBuffers(_octTreeDeviceBuffers,
                                     voxelColorMipMaps.size()-1u))
    {
        return false;
    }

    size_t totalNodeCount = 0;
    _octTreeNodesWriteIndex = 0;
    for(size_t i = 0; i < voxelColorMipMaps.size(); ++i)
    {
        if(i + 1 == voxelColorMipMaps.size())
            _octTreeNodesWriteIndex = totalNodeCount;//offset to start writing leaf nodes

        totalNodeCount += static_cast<size_t>(glm::pow(8.0f, static_cast<float>(i)));
    }

    _octTreeNodes.resize(totalNodeCount, 0u);
    cuda::VoxColor zeros;
    zeros.x = zeros.y = zeros.z = zeros.w = 0;
    _octTreeConstColors.resize(totalNodeCount, zeros);

    return true;
}

void Voxelizer::deallocateVoxelMemory()
{
    hipFree(_voxelTriIndicesDevPtr.ptr);
    hipFree(_voxelTriCountsDevPtr.ptr);

    freeVoxelDeviceMipMaps();

    //unsigned int* pTest = new unsigned int[ octTreeNodesDim.x * octTreeNodesDim.y * octTreeNodesDim.z];
    //hipMemcpy(pTest, pOctTreeNodesDevPtr, octTreeSize, hipMemcpyDeviceToHost);
    for(size_t i = 0; i < _octTreeDeviceBuffers.size(); ++i)
    {
        hipFree(_octTreeDeviceBuffers[i].pTypes);
        hipFree(_octTreeDeviceBuffers[i].pConstColors);
    }
    _octTreeDeviceBuffers.clear();

    s_totalAllocatedDeviceMemory = 0u;

    freeVoxelMipMapsAndOctTree();
}

void Voxelizer::deallocateTriangleMemory()
{
    freeTriangleHostMemory();

    freeTriangleDeviceMemory();
}

Voxelizer::~Voxelizer()
{
    deallocateTriangleMemory();

    deallocateVoxelMemory();
}

void Voxelizer::freeTriangleHostMemory()
{
    _triangleGroups.clear();
    TriangleGroups empty;
    empty.swap(_triangleGroups);
    _totalVertCount = 0;
    _totalVertNormalsCount = 0;
    _totalUVCount = 0;
}

void Voxelizer::freeTriangleDeviceMemory()
{
    if(_pgVerts)
        hipFree(_pgVerts);
    _pgVerts = nullptr;

    if(_pgEdges)
        hipFree(_pgEdges);
    _pgEdges = nullptr;

    if(_pgFaceNormals)
        hipFree(_pgFaceNormals);
    _pgFaceNormals = nullptr;

    if(_pgVtxNormals)
        hipFree(_pgVtxNormals);
    _pgVtxNormals = nullptr;

    if(_pgBounds)
        hipFree(_pgBounds);
    _pgBounds = nullptr;

    if(_pgUVs)
        hipFree(_pgUVs);
    _pgUVs = nullptr;

    for(ImageMap::iterator itr = _imageMap.begin();
        itr != _imageMap.end();
        ++itr)
    {
        if(itr->second)
            hipFreeArray(itr->second);
    }
    _imageMap.clear();
}

bool Voxelizer::initCuda()
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    s_cudaStatus = hipSetDevice(0);
    if(s_cudaStatus != hipSuccess) 
    {
        _error << "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" << std::endl;
        return false;
    }

    hipDeviceGetAttribute(&_maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, 0);
    //hipDeviceGetAttribute(&_maxBlockDim.x, hipDeviceAttributeMaxBlockDimX, 0);
    //hipDeviceGetAttribute(&_maxBlockDim.y, hipDeviceAttributeMaxBlockDimY, 0);
    //hipDeviceGetAttribute(&_maxBlockDim.z, hipDeviceAttributeMaxBlockDimZ, 0);
    
    int maxBlockDim = static_cast<int>(glm::pow(static_cast<float>(_maxThreadsPerBlock), 1.0f / 3.0f));
    //need largest multiple of four that is less than cube root
    maxBlockDim -= (maxBlockDim % 4);

    _maxBlockDim = glm::ivec3(maxBlockDim);

    //for some reason I can't run a 1D kernel with the max amount of threads of 1024
    //the largest I've been able to get is 960, so cutting this in half to make it work
    _maxThreadsPerBlock >>= 1;
    return true;
}

bool Voxelizer::resetCuda()
{
    deallocateTriangleMemory();

    deallocateVoxelMemory();

    s_cudaStatus = hipDeviceReset();
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipDeviceReset failed, error code=" << s_cudaStatus << std::endl;
        return false;
    }

    if(!allocateVoxelMemory())
    {
        _error << "allocateVoxelMemory failed." << std::endl;
        return false;
    }

    return true;
}

bool Voxelizer::allocateTriangleMemory()
{
    size_t vertSize = sizeof(glm::vec3) * _totalVertCount;
    s_cudaStatus = hipMalloc(&_pgVerts, vertSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMalloc failed to allocate vertex buffer " << vertSize << " bytes." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += vertSize;

    size_t numTris = _totalVertCount / 3;

    size_t normalsSize = (numTris * sizeof(glm::vec3));
    s_cudaStatus = hipMalloc(&_pgFaceNormals, normalsSize);
    if(s_cudaStatus != hipSuccess) 
    {
        _error << "hipMalloc failed to allocate normals buffer " << normalsSize << " bytes." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += normalsSize;

    if(_totalVertNormalsCount > 0u)
    {
        //size_t numTrisWithVtxNormals = _totalVertNormalsCount / 3;
        size_t vtxNormalsSize = sizeof(glm::vec3) * _totalVertNormalsCount;
        s_cudaStatus = hipMalloc(&_pgVtxNormals, vtxNormalsSize);
        if(s_cudaStatus != hipSuccess) 
        {
            _error << "hipMalloc failed to allocate vertex normals buffer " << vtxNormalsSize << " bytes." << std::endl;
            return false;
        }
        s_totalAllocatedDeviceMemory += vtxNormalsSize;
    }

    size_t uvSize = 0u;
    if(_totalUVCount > 0u)
    {
        uvSize = sizeof(glm::vec2) * _totalUVCount;
        s_cudaStatus = hipMalloc(&_pgUVs, uvSize);
        if(s_cudaStatus != hipSuccess)
        {
            _error << "hipMalloc failed to allocate uv buffer " << uvSize << " bytes." << std::endl;
            return false;
        }
        s_totalAllocatedDeviceMemory += uvSize;
    }

    size_t vertOffset = 0u;
    size_t normalsOffset = 0u;
    size_t uvOffset = 0u;
    for(TriangleGroups::iterator itr = _triangleGroups.begin();
        itr != _triangleGroups.end();
        ++itr)
    {
        size_t grpVertSize = itr->pVerts->size() * sizeof(glm::vec3);
        s_cudaStatus = hipMemcpy(&_pgVerts[vertOffset], 
                                  &itr->pVerts->front(),
                                  grpVertSize, 
                                  hipMemcpyHostToDevice);
        if(s_cudaStatus != hipSuccess)
        {
            _error << "hipMemcpy failed for vertex buffer error=" << s_cudaStatus << "." << std::endl;
            return false;
        }
        vertOffset += itr->pVerts->size();

        if(itr->pVertNormals != nullptr)
        {
            size_t grpVertNormalsSize = itr->pVertNormals->size() * sizeof(glm::vec3);
            s_cudaStatus = hipMemcpy(&_pgVtxNormals[normalsOffset],
                                      &itr->pVertNormals->front(),
                                      grpVertNormalsSize,
                                      hipMemcpyHostToDevice);
            normalsOffset += itr->pVertNormals->size();
        }

        if(itr->pImageData != nullptr && itr->pUVs != nullptr)
        {
            size_t grpUVSize = itr->pUVs->size() * sizeof(glm::vec2);
            s_cudaStatus = hipMemcpy(&_pgUVs[uvOffset], 
                                      &itr->pUVs->front(),
                                      grpUVSize, 
                                      hipMemcpyHostToDevice);
            if(s_cudaStatus != hipSuccess)
            {
                _error << "hipMemcpy failed for uv buffer error=" << s_cudaStatus << "." << std::endl;
                return false;
            }
            uvOffset += itr->pUVs->size();
        }
    }
    
    s_cudaStatus = hipMalloc(&_pgEdges, vertSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMalloc failed to allocate edges buffer " << vertSize << " bytes." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += vertSize;

    size_t boundsSize = numTris * sizeof(VoxelBBox);
    s_cudaStatus = hipMalloc(&_pgBounds, boundsSize);
    if(s_cudaStatus != hipSuccess) 
    {
        _error << "hipMalloc failed to allocate bounds buffer " << boundsSize << " bytes." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += boundsSize;

    return true;
}

bool Voxelizer::computeEdgesFaceNormalsAndBounds()
{
    size_t numTris = _totalVertCount / 3;
    dim3 threadsPerBlock;
    dim3 numBlocks;
    if(static_cast<size_t>(_maxThreadsPerBlock) > numTris)
        threadsPerBlock.x = static_cast<glm::uint>(numTris);
    else
    {
        threadsPerBlock.x = _maxThreadsPerBlock;
        numBlocks.x = static_cast<size_t>(glm::ceil((static_cast<float>(numTris) / static_cast<float>(_maxThreadsPerBlock))));
    }

    glm::uvec3 voxDimPlusExtra = _voxDim + (_extraVoxChunk << glm::uvec3(2u));

    ComputeEdgesFaceNormalsAndBounds<<<numBlocks, threadsPerBlock>>>(_pgVerts, _totalVertCount, 
                                                          _offsetP, _deltaP, 
                                                          voxDimPlusExtra, 
                                                          _pgEdges,
                                                          _pgFaceNormals,
                                                          _pgBounds);

    s_cudaStatus = hipDeviceSynchronize();
    if (s_cudaStatus != hipSuccess)
    {
        _error << "ComputeEdgesFaceNormalsAndBounds() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }

    return true;
}

bool Voxelizer::computeMipMaps(size_t xSize, size_t ySize, size_t zSize, //dimension of current chunk
                               size_t mmXSize, size_t mmYSize, size_t mmZSize,
                               VoxelDeviceMipMaps& voxelDeviceMipMaps,
                               VoxelColorMipMaps& voxelColorMipMaps,
                               VoxelNormalMipMaps& voxelNormalMipMaps,
                               size_t depth/*=1*/)
{
    //figure out the kernel dimensions, need one thread per voxel of mip map
    dim3 blocksPerGrid(1u, 1u, 1u);
    dim3 threadsPerBlock(static_cast<glm::uint>(mmXSize), 
                         static_cast<glm::uint>(mmYSize), 
                         static_cast<glm::uint>(mmZSize));
    if(mmXSize > static_cast<glm::uint>(_maxBlockDim.x) || 
       mmYSize > static_cast<glm::uint>(_maxBlockDim.y) || 
       mmZSize > static_cast<glm::uint>(_maxBlockDim.z))
    {
        threadsPerBlock.x = glm::min(static_cast<glm::uint>(mmXSize), static_cast<glm::uint>(_maxBlockDim.x));
        threadsPerBlock.y = glm::min(static_cast<glm::uint>(mmYSize), static_cast<glm::uint>(_maxBlockDim.y));
        threadsPerBlock.z = glm::min(static_cast<glm::uint>(mmZSize), static_cast<glm::uint>(_maxBlockDim.z));

        blocksPerGrid.x = 
            static_cast<size_t>(glm::ceil((static_cast<float>(mmXSize) / static_cast<float>(threadsPerBlock.x))));
        blocksPerGrid.y = 
            static_cast<size_t>(glm::ceil((static_cast<float>(mmYSize) / static_cast<float>(threadsPerBlock.y))));
        blocksPerGrid.z = 
            static_cast<size_t>(glm::ceil((static_cast<float>(mmZSize) / static_cast<float>(threadsPerBlock.z))));
    }

    size_t inputDepth;
    if(depth == 1)
        inputDepth = depth - 1;
    else
        inputDepth = depth - 2;
    //input voxel mip maps
    const VoxelDataPtrs& inputVoxelsDevPtrs = voxelDeviceMipMaps[inputDepth];

    //output voxel mip maps
    VoxelDataPtrs& mipMapPtrs = voxelDeviceMipMaps[depth];
    hipPitchedPtr& outputColorsDevPtr = mipMapPtrs.first;
    hipPitchedPtr& outputNormalsDevPtr = mipMapPtrs.second;

    size_t xScale = static_cast<size_t>(glm::ceil(static_cast<float>(xSize) / static_cast<float>(mmXSize)));
    size_t yScale = static_cast<size_t>(glm::ceil(static_cast<float>(ySize) / static_cast<float>(mmYSize)));
    size_t zScale = static_cast<size_t>(glm::ceil(static_cast<float>(zSize) / static_cast<float>(mmZSize)));
    //if(xScale == 2)
    {
        ComputeVoxelMipMap8<<<blocksPerGrid, threadsPerBlock>>>(
                                                               static_cast<glm::uint>(xSize), 
                                                               static_cast<glm::uint>(ySize), 
                                                               static_cast<glm::uint>(zSize),
                                                               inputVoxelsDevPtrs.first,//input colors
                                                               inputVoxelsDevPtrs.second,//input normals
                                                               static_cast<glm::uint>(mmXSize), 
                                                               static_cast<glm::uint>(mmYSize), 
                                                               static_cast<glm::uint>(mmZSize),//output size
                                                               outputColorsDevPtr,//output colors
                                                               outputNormalsDevPtr);//output normals
    }
    /*else
    {
            ComputeVoxelMipMap64<<<blocksPerGrid, threadsPerBlock>>>(
                                                               static_cast<glm::uint>(xSize), 
                                                               static_cast<glm::uint>(ySize), 
                                                               static_cast<glm::uint>(zSize),
                                                               inputVoxelsDevPtrs.first,//input colors
                                                               inputVoxelsDevPtrs.second,//input normals
                                                               static_cast<glm::uint>(mmXSize), 
                                                               static_cast<glm::uint>(mmYSize), 
                                                               static_cast<glm::uint>(mmZSize),//output size
                                                               outputColorsDevPtr,//output colors
                                                               outputNormalsDevPtr);//output normals
    }*/
    s_cudaStatus = hipDeviceSynchronize();
    if(s_cudaStatus != hipSuccess)
    {
        _error << "computeMipMaps() error: " << s_cudaStatus << " returned from hipDeviceSynchronize." << std::endl;
        return false;
    }

    if(depth != _numMipMapLevels-1)
    {
        if(!computeMipMaps(depth == 1 ? xSize : xSize>>1, 
                           depth == 1 ? ySize : ySize>>1,
                           depth == 1 ? zSize : zSize>>1,
                           mmXSize>>1, mmYSize>>1, mmZSize>>1,
                           voxelDeviceMipMaps,
                           voxelColorMipMaps,
                           voxelNormalMipMaps,
                           depth+1))
        {
            return false;
        }
    }

    VoxelColorMipMap& voxelColors = voxelColorMipMaps.at(depth);
    VoxelNormalMipMap& voxelNormals = voxelNormalMipMaps.at(depth);
    //copy this chunk of the mip map to the host
    copyDeviceChunkToHostMipMapChunk(voxelColors.colors,
                                     voxelNormals.normals,
                                     mmXSize, mmYSize, mmZSize,
                                     outputColorsDevPtr,
                                     outputNormalsDevPtr);
    return true;
}

bool Voxelizer::allocateDeviceMipMaps(size_t xSize, size_t ySize, size_t zSize, //dimension of current chunk
                                VoxelDeviceMipMaps& voxelDeviceMipMaps)
{
    //compute size of mip map chunk
    size_t mmXSize = static_cast<size_t>(glm::ceil(xSize * 0.5f));

    size_t mmYSize = static_cast<size_t>(glm::ceil(ySize * 0.5f));
    
    size_t mmZSize = static_cast<size_t>(glm::ceil(zSize * 0.5f));

    hipExtent ext3D = make_hipExtent(mmXSize * sizeof(cuda::VoxColor),
                                       mmYSize,
                                       mmZSize);
    hipPitchedPtr mipMapColors;
    s_cudaStatus = hipMalloc3D(&mipMapColors, 
                                ext3D);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceMipMaps() error: " << s_cudaStatus << " returned from hipMalloc3D." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += (ext3D.width * ext3D.height * ext3D.depth);

    ext3D = make_hipExtent(mmXSize * sizeof(cuda::VoxNorm),
                            mmYSize,
                            mmZSize);

    hipPitchedPtr mipMapNormals;
    s_cudaStatus = hipMalloc3D(&mipMapNormals,
                                ext3D);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceMipMaps() error: " << s_cudaStatus << " returned from hipMalloc3D." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += (ext3D.width * ext3D.height * ext3D.depth);

    voxelDeviceMipMaps.push_back(VoxelDataPtrs(mipMapColors, mipMapNormals));

    if(voxelDeviceMipMaps.size() != _numMipMapLevels)
    {
        return allocateDeviceMipMaps(mmXSize, mmYSize, mmZSize,
                                     voxelDeviceMipMaps);
    }

    return true;
}

void Voxelizer::allocateHostMipMaps(VoxelColorMipMaps& voxelColorMipMaps,
                                    VoxelNormalMipMaps& voxelNormalMipMaps,
                                    size_t xSize, size_t ySize, size_t zSize,
                                    size_t depth/*=0*/)
{    
    VoxelColorMipMap& voxelColors = voxelColorMipMaps.at(depth);
    voxelColors.dim.x = xSize;
    voxelColors.dim.y = ySize;
    voxelColors.dim.z = zSize;
    voxelColors.colors.resize(xSize * ySize * zSize);

    VoxelNormalMipMap& voxelNormals = voxelNormalMipMaps.at(depth);
    voxelNormals.dim.x = xSize;
    voxelNormals.dim.y = ySize;
    voxelNormals.dim.z = zSize;
    voxelNormals.normals.resize(xSize * ySize * zSize);

    if((depth + 1) != _numMipMapLevels)
    {
        size_t mmXSize = static_cast<size_t>(glm::ceil(xSize * 0.5f));
        size_t mmYSize = static_cast<size_t>(glm::ceil(ySize * 0.5f));
        size_t mmZSize = static_cast<size_t>(glm::ceil(zSize * 0.5f));
        allocateHostMipMaps(voxelColorMipMaps, voxelNormalMipMaps,
                            mmXSize, mmYSize, mmZSize,
                            depth+1);
    }
}

//static void LoadDeviceMemory(size_t xOffset, size_t yOffset, size_t zOffset,
//                             size_t xSize, size_t ySize, size_t zSize,
//                             cuda::VoxColor* pVoxelsDevPtr,
//                             const VoxelColors& voxelColors,
//                             const glm::uvec3& voxDim)
//{
//    for(size_t z = zOffset; z < (zOffset + zSize); ++z)
//    {
//        for(size_t y = yOffset; y < (yOffset + ySize); ++y)
//        {
//            //TODO use this
//            /*hipMemcpyAsync(pVoxelsDevPtr, 
//                            &voxelColors[(z * voxDim.x * voxDim.y) + (y * voxDim.x) + xOffset], 
//                            sizeof(cuda::VoxColor) * xSize,
//                            hipMemcpyHostToDevice);*/
//            size_t startIndex = (z * voxDim.x * voxDim.y) + (y * voxDim.x) + xOffset;
//
//            hipMemcpy(pVoxelsDevPtr, 
//                       &voxelColors[startIndex], 
//                       sizeof(cuda::VoxColor) * xSize,
//                       hipMemcpyHostToDevice);
//
//            pVoxelsDevPtr += xSize;
//        }
//    }
//
//    hipDeviceSynchronize();
//}

static bool CollapseConstantBranches(cuda::Voxelizer::OctTreeNodes& octTreeNodes,
                                     size_t curX=0,
                                     size_t curY=0,
                                     size_t curZ=0,
                                     size_t curXSize=1,
                                     size_t curYSize=1,
                                     size_t curZSize=1,
                                     size_t curStart=0,
                                     size_t childX=0,
                                     size_t childY=0,
                                     size_t childZ=0,
                                     size_t childXSize=2,
                                     size_t childYSize=2,
                                     size_t childZSize=2,
                                     size_t nextStartIndex=1)
{
    size_t childStartIndex = nextStartIndex + 
                             ((childZ * childYSize * childXSize) + (childY * childXSize) + childX);
    size_t curIndex = curStart + 
                      ((curZ * curYSize * curXSize) + (curY * curXSize) + curX);
    if(childStartIndex >= octTreeNodes.size())//if we are at a leaf node
       return octTreeNodes[curIndex] == 0;

    size_t gChildXSize = childXSize << 1;
    size_t gChildYSize = childYSize << 1;
    size_t gChildZSize = childZSize << 1;
    size_t gChildStartIndex = nextStartIndex + (childXSize * childYSize * childZSize);

    bool allChildrenAreConst = true;
    size_t endX = childX + 2;
    size_t endY = childY + 2;
    size_t endZ = childZ + 2;
    for(size_t zChild = childZ ; zChild < endZ; ++zChild)
    {
        size_t gChildZ = zChild << 1;
        for(size_t yChild = childY; yChild < endY; ++yChild)
        {
            size_t gChildY = yChild << 1;
            for(size_t xChild = childX; xChild < endX; ++xChild)
            {
                size_t gChildX = xChild << 1;
                if(!CollapseConstantBranches(octTreeNodes,
                                             xChild,
                                             yChild,
                                             zChild,
                                             childXSize,
                                             childYSize,
                                             childZSize,
                                             nextStartIndex,
                                             gChildX,
                                             gChildY,
                                             gChildZ,
                                             gChildXSize,
                                             gChildYSize,
                                             gChildZSize,
                                             gChildStartIndex))
                {
                    allChildrenAreConst = false;
                }
            }
        }
    }

    //if i am const (0) and all my children are too
    //then ok to collapse
    if(octTreeNodes[curIndex] == 0 &&
       allChildrenAreConst)
    {
        octTreeNodes[curIndex] = 2;
        return true;
    }

    return false;
}

static inline bool VEC4_EQUAL(const glm::vec4& v1, const glm::vec4& v2);
static inline bool VEC4_EQUAL(const uchar4& v1, const uchar4& v2);

static bool ValidateNode(size_t xOffset, size_t yOffset, size_t zOffset,
                    size_t xSize, size_t ySize, size_t zSize,
                    const glm::uvec3& brickDim,
                    const VoxelColors& voxelColors,
                    glm::uint type)
{
    size_t startX = xOffset;
    if(xOffset != 0)
        startX -= 1;

    size_t endX = xOffset + brickDim.x;
    if(endX != xSize)
        endX += 1;

    size_t startY = yOffset;
    if(yOffset != 0)
        startY -= 1;

    size_t endY = yOffset + brickDim.y;
    if(endY != ySize)
        endY += 1;

    size_t startZ = zOffset;
    if(zOffset != 0)
        startZ -= 1;

    size_t endZ = zOffset + brickDim.z;
    if(endZ != zSize)
        endZ += 1;

    const cuda::VoxColor& constColor = voxelColors[(zOffset * xSize * ySize) + (yOffset * xSize) + xOffset];
    for(size_t z = startZ; z < endZ; ++z)
    {
        for(size_t y = startY; y < endY; ++y)
        {
            for(size_t x = startX; x < endX; ++x)
            {
                const cuda::VoxColor& curColor = voxelColors[(z * xSize * ySize) + (y * xSize) + x];
                if(VEC4_EQUAL(curColor, constColor) == false)
                {
                    if(type == 1u)
                        return true;
                    else
                        return false;
                }
            }
        }
    }

    if(type == 0u || type == 2u)
        return true;
    else
        return false;
}

bool Voxelizer::allocateOctTreeDeviceBuffers(OctTreeDeviceBuffers& octTreeDevBuffers,
                                             size_t depth)
{
    size_t numNodes = static_cast<size_t>(glm::pow(8.0f, static_cast<float>(depth)));
    
    size_t octTreeSize = sizeof(glm::uint) * numNodes;

    glm::uint* pOctTreeNodesDevPtr = 0u;
    s_cudaStatus = hipMalloc(&pOctTreeNodesDevPtr, 
                              octTreeSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMalloc failed to allocate buffer " << octTreeSize << " bytes." << std::endl;
        return false;
    }

    s_totalAllocatedDeviceMemory += octTreeSize;

    //initialize to zero, which is code for const-node, use 1 for non-const-node
    s_cudaStatus = hipMemset(pOctTreeNodesDevPtr, 0, octTreeSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMemset failed "
               << s_cudaStatus
               << std::endl;
        return false;
    }

    octTreeDevBuffers[depth].pTypes = pOctTreeNodesDevPtr;

    octTreeSize = sizeof(cuda::VoxColor) * numNodes;

    cuda::VoxColor* pOctTreeNodesColorsDevPtr = 0u;
    s_cudaStatus = hipMalloc(&pOctTreeNodesColorsDevPtr, 
                              octTreeSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMalloc failed to allocate buffer " << octTreeSize << " bytes." << std::endl;
        return false;
    }

    s_totalAllocatedDeviceMemory += octTreeSize;

    s_cudaStatus = hipMemset(pOctTreeNodesColorsDevPtr, 0, octTreeSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMemset failed "
               << s_cudaStatus
               << std::endl;
        return false;
    }

    octTreeDevBuffers[depth].pConstColors = pOctTreeNodesColorsDevPtr;

    if(depth == 0)
        return true;

    return allocateOctTreeDeviceBuffers(octTreeDevBuffers, depth - 1);
}

bool Voxelizer::initOctTreeDeviceBuffers(OctTreeDeviceBuffers& octTreeDevBuffers,
                                         size_t depth)
{
    size_t numNodes = static_cast<size_t>(glm::pow(8.0f, static_cast<float>(depth)));
    
    size_t octTreeSize = sizeof(glm::uint) * numNodes;

    glm::uint* pOctTreeNodesDevPtr = octTreeDevBuffers[depth].pTypes;
    //initialize to zero, which is code for const-node, use 1 for non-const-node
    s_cudaStatus = hipMemset(pOctTreeNodesDevPtr, 0, octTreeSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMemset failed "
               << s_cudaStatus
               << std::endl;
        return false;
    }

    octTreeSize = sizeof(cuda::VoxColor) * numNodes;

    cuda::VoxColor* pOctTreeNodesColorsDevPtr = octTreeDevBuffers[depth].pConstColors;
    
    s_cudaStatus = hipMemset(pOctTreeNodesColorsDevPtr, 0, octTreeSize);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "hipMemset failed "
               << s_cudaStatus
               << std::endl;
        return false;
    }

    if(depth == 0)
        return true;

    return initOctTreeDeviceBuffers(octTreeDevBuffers, depth - 1);
}

void Voxelizer::computeOctTreeNodes(size_t xOffset, size_t yOffset, size_t zOffset,
                                    VoxelDeviceMipMaps& voxelDeviceMipMaps,
                                    VoxelColorMipMaps& voxelColorMipMaps,
                                    VoxelNormalMipMaps& voxelNormalMipMaps,
                                    VoxelBrickWriters& voxelBrickWriters,
                                    const glm::uvec3& mipMapChunkDim, //dimension of the chunk of mip map that we are processing
                                    const glm::uvec3& offsetToBrickBorder,
                                    OctTreeDeviceBuffers& octTreeDevBuffers,
                                    OctTreeNodes& octTreeNodes,
                                    OctTreeConstColors& octTreeConstColors,
                                    size_t writeIndex,
                                    size_t depth)
{
    size_t numNodes = static_cast<size_t>(glm::pow(8.0f, static_cast<float>(depth)));
    size_t cubeRoot = static_cast<size_t>(glm::pow(static_cast<float>(numNodes), 1.0f / 3.0f));
    glm::uvec3 octTreeNodesDim(cubeRoot, cubeRoot, cubeRoot);

    const hipPitchedPtr& voxelColors = voxelDeviceMipMaps.at(voxelDeviceMipMaps.size() - depth - 1u).first;

    glm::uint* pOctTreeNodesDevPtr = octTreeDevBuffers[depth].pTypes;
    cuda::VoxColor* pOctTreeNodesConstColorPtr = octTreeDevBuffers[depth].pConstColors;

    glm::ivec3 voxOffset(xOffset, 
                         yOffset, 
                         zOffset);
    //where to sample from the brick to do the matching against
    //glm::ivec3 brickSampleOffset(xOffset == 0u ? 0 : static_cast<int>(_brickDim.x - 1u),
    //                             yOffset == 0u ? 0 : static_cast<int>(_brickDim.y - 1u),
    //                             zOffset == 0u ? 0 : static_cast<int>(_brickDim.z - 1u));
    //this is the xyz of the starting offset without the extra border
    glm::ivec3 startVoxXYZ(xOffset - offsetToBrickBorder.x,
                           yOffset - offsetToBrickBorder.y,
                           zOffset - offsetToBrickBorder.z);

    size_t nodeXStart = 0u;
    if(startVoxXYZ.x >= 0)
        nodeXStart = static_cast<size_t>(static_cast<float>(glm::max(startVoxXYZ.x - 1, 0)) / static_cast<float>(_brickDim.x));
    size_t nodeYStart = 0u;
    if(startVoxXYZ.y >= 0)
        nodeYStart = static_cast<size_t>(static_cast<float>(glm::max(startVoxXYZ.y - 1, 0)) / static_cast<float>(_brickDim.y));
    size_t nodeZStart = 0u;
    if(startVoxXYZ.z >= 0)
        nodeZStart = static_cast<size_t>(static_cast<float>(glm::max(startVoxXYZ.z - 1, 0)) / static_cast<float>(_brickDim.z));

    //this is the xyz of the starting offset without the extra border
    glm::ivec3 endVoxXYZ((xOffset + mipMapChunkDim.x - 1) - offsetToBrickBorder.x,
                         (yOffset + mipMapChunkDim.y - 1) - offsetToBrickBorder.y,
                         (zOffset + mipMapChunkDim.z - 1) - offsetToBrickBorder.z);

    size_t nodeXEnd = static_cast<size_t>(static_cast<float>(endVoxXYZ.x + 1) / static_cast<float>(_brickDim.x));
    if(nodeXEnd >= octTreeNodesDim.x)
        nodeXEnd = octTreeNodesDim.x - 1u;
    size_t nodeYEnd = static_cast<size_t>(static_cast<float>(endVoxXYZ.y + 1) / static_cast<float>(_brickDim.y));
    if(nodeYEnd >= octTreeNodesDim.y)
        nodeYEnd = octTreeNodesDim.y - 1u;
    size_t nodeZEnd = static_cast<size_t>(static_cast<float>(endVoxXYZ.z + 1) / static_cast<float>(_brickDim.z));
    if(nodeZEnd >= octTreeNodesDim.z)
        nodeZEnd = octTreeNodesDim.z - 1u;

    glm::uvec3 nodesChunkDim(nodeXEnd - nodeXStart + 1u,
                       nodeYEnd - nodeYStart + 1u,
                       nodeZEnd - nodeZStart + 1u);
    //one block per node
    //need one thread per voxel
    //if my voxel grid is larger than the amount of device memory then we'll have to break it
    //up into multiple executions
    dim3 numBlocksPerGrid(1u, 1u, 1u);
    dim3 numThreadsPerBlock(nodesChunkDim.x, nodesChunkDim.y, nodesChunkDim.z);
    //dim3 numThreadsPerBlock(128u, 128u, 128u);
    if(numThreadsPerBlock.x > static_cast<glm::uint>(_maxBlockDim.x) || 
       numThreadsPerBlock.y > static_cast<glm::uint>(_maxBlockDim.y) || 
       numThreadsPerBlock.z > static_cast<glm::uint>(_maxBlockDim.z))
    {
        numThreadsPerBlock.x = glm::min(numThreadsPerBlock.x, static_cast<glm::uint>(_maxBlockDim.x));
        numThreadsPerBlock.y = glm::min(numThreadsPerBlock.y, static_cast<glm::uint>(_maxBlockDim.y));
        numThreadsPerBlock.z = glm::min(numThreadsPerBlock.z, static_cast<glm::uint>(_maxBlockDim.z));

        numBlocksPerGrid.x = 
            static_cast<size_t>(glm::ceil((static_cast<float>(nodesChunkDim.x) / static_cast<float>(numThreadsPerBlock.x))));
            //static_cast<size_t>(glm::ceil((static_cast<float>(128u) / static_cast<float>(numThreadsPerBlock.x))));
        numBlocksPerGrid.y = 
            static_cast<size_t>(glm::ceil((static_cast<float>(nodesChunkDim.y) / static_cast<float>(numThreadsPerBlock.y))));
            //static_cast<size_t>(glm::ceil((static_cast<float>(128u) / static_cast<float>(numThreadsPerBlock.y))));
        numBlocksPerGrid.z = 
            static_cast<size_t>(glm::ceil((static_cast<float>(nodesChunkDim.z) / static_cast<float>(numThreadsPerBlock.z))));
            //static_cast<size_t>(glm::ceil((static_cast<float>(128u) / static_cast<float>(numThreadsPerBlock.z))));
    }

    ComputeOctTreeNodeConstColor<<<numBlocksPerGrid, numThreadsPerBlock>>>(voxelColors, 
                                                                           pOctTreeNodesConstColorPtr,
                                                                           voxOffset,
                                                                           glm::ivec3(offsetToBrickBorder.x,
                                                                                      offsetToBrickBorder.y,
                                                                                      offsetToBrickBorder.z),
                                                                           glm::ivec3(mipMapChunkDim.x,
                                                                                      mipMapChunkDim.y,
                                                                                      mipMapChunkDim.z),
                                                                           nodesChunkDim,
                                                                           octTreeNodesDim,
                                                                           glm::ivec3(nodeXStart, 
                                                                                      nodeYStart,
                                                                                      nodeZStart),
                                                                           _brickDim);
    s_cudaStatus = hipDeviceSynchronize();
    if(s_cudaStatus != hipSuccess)
    {
        _error << "computeOctTreeNodes() error: " << s_cudaStatus << "." << std::endl;
        return;
    }


    glm::ivec3 fullVoxDim(octTreeNodesDim.x * _brickDim.x,
                        octTreeNodesDim.y * _brickDim.y,
                        octTreeNodesDim.z * _brickDim.z);
    //one block per node
    //need one thread per voxel
    //if my voxel grid is larger than the amount of device memory then we'll have to break it
    //up into multiple executions
    numBlocksPerGrid = dim3(1u, 1u, 1u);
    glm::uvec3 tpb(mipMapChunkDim.x, mipMapChunkDim.y, mipMapChunkDim.z);
    glm::uvec3 fullVoxOffset(tpb.x + voxOffset.x - offsetToBrickBorder.x,
                             tpb.y + voxOffset.y - offsetToBrickBorder.y,
                             tpb.z + voxOffset.z - offsetToBrickBorder.z);
    if(fullVoxOffset.x > fullVoxDim.x)
        tpb.x = fullVoxDim.x - (voxOffset.x - offsetToBrickBorder.x) + 2u;
    if(fullVoxOffset.y > fullVoxDim.y)
        tpb.y = fullVoxDim.y - (voxOffset.y - offsetToBrickBorder.y) + 2u;
    if(fullVoxOffset.z > fullVoxDim.z)
        tpb.z =  fullVoxDim.z - (voxOffset.z - offsetToBrickBorder.z) + 2u;
    numThreadsPerBlock = dim3(tpb.x, tpb.y, tpb.z);
    //dim3 numThreadsPerBlock(128u, 128u, 128u);
    if(numThreadsPerBlock.x > static_cast<glm::uint>(_maxBlockDim.x) || 
       numThreadsPerBlock.y > static_cast<glm::uint>(_maxBlockDim.y) || 
       numThreadsPerBlock.z > static_cast<glm::uint>(_maxBlockDim.z))
    {
        numThreadsPerBlock.x = glm::min(numThreadsPerBlock.x, static_cast<glm::uint>(_maxBlockDim.x));
        numThreadsPerBlock.y = glm::min(numThreadsPerBlock.y, static_cast<glm::uint>(_maxBlockDim.y));
        numThreadsPerBlock.z = glm::min(numThreadsPerBlock.z, static_cast<glm::uint>(_maxBlockDim.z));

        numBlocksPerGrid.x = 
            static_cast<size_t>(glm::ceil((static_cast<float>(tpb.x) / static_cast<float>(numThreadsPerBlock.x))));
            //static_cast<size_t>(glm::ceil((static_cast<float>(128u) / static_cast<float>(numThreadsPerBlock.x))));
        numBlocksPerGrid.y = 
            static_cast<size_t>(glm::ceil((static_cast<float>(tpb.y) / static_cast<float>(numThreadsPerBlock.y))));
            //static_cast<size_t>(glm::ceil((static_cast<float>(128u) / static_cast<float>(numThreadsPerBlock.y))));
        numBlocksPerGrid.z = 
            static_cast<size_t>(glm::ceil((static_cast<float>(tpb.z) / static_cast<float>(numThreadsPerBlock.z))));
            //static_cast<size_t>(glm::ceil((static_cast<float>(128u) / static_cast<float>(numThreadsPerBlock.z))));
    }

    size_t numKernExecutions = 1;
    int kernXOffset = 0;
    if(tpb.x > 128u)
    {
        numBlocksPerGrid.x >>= 1;
        if(numBlocksPerGrid.x & 1 != 0)
            numBlocksPerGrid.x += 1;
        //numBlocksPerGrid.y >>= 1;
        //numBlocksPerGrid.z >>= 1;
        numKernExecutions = 2;
    }

    //had issue with launch timeout, so having to break this up
    //into two executions to get work around
    for(size_t kernExeIdx = 0u; kernExeIdx < numKernExecutions; ++kernExeIdx)
    {
        ComputeOctTreeNodeType<<<numBlocksPerGrid, numThreadsPerBlock>>>(voxelColors, 
                                                                         pOctTreeNodesDevPtr,
                                                                         pOctTreeNodesConstColorPtr,
                                                                         voxOffset,
                                                                         glm::ivec3(offsetToBrickBorder.x,
                                                                                    offsetToBrickBorder.y,
                                                                                    offsetToBrickBorder.z),
                                                                         //brickSampleOffset,
                                                                         glm::ivec3(mipMapChunkDim.x,
                                                                                    mipMapChunkDim.y,
                                                                                    mipMapChunkDim.z),
                                                                         fullVoxDim,
                                                                         octTreeNodesDim,
                                                                         _brickDim,
                                                                         kernXOffset);
        s_cudaStatus = hipDeviceSynchronize();
        if(s_cudaStatus != hipSuccess)
        {
            _error << "computeOctTreeNodes() error: " << s_cudaStatus << "." << std::endl;
            return;
        }

        kernXOffset += (numBlocksPerGrid.x * numThreadsPerBlock.x);
    }

    const VoxelColorMipMap& voxelColorMM = voxelColorMipMaps[voxelColorMipMaps.size() - depth - 1u];
    const VoxelNormalMipMap& voxelNormalMM = voxelNormalMipMaps[voxelNormalMipMaps.size() - depth - 1u];
    VoxelBrickWriter& brickWriter = voxelBrickWriters[depth];
    
    size_t rowReadSize = nodeXEnd - nodeXStart + 1u;  
    
    int nodeBaseX = (nodeXStart * _brickDim.x) - xOffset + offsetToBrickBorder.x;
    int nodeBaseY = (nodeYStart * _brickDim.y) - yOffset + offsetToBrickBorder.y;
    int nodeBaseZ = (nodeZStart * _brickDim.z) - zOffset + offsetToBrickBorder.z;

    //std::cout << std::endl << "Writing bricks";
    //TODO if nothing voxelized then replace this function with just a loop like the one
    //below that just sets all nodes to constant unless it is already set to non-const
    //int loopCount = 0;
    int mipMapZOffset = nodeBaseZ;
    for(size_t nodeZ = nodeZStart; 
        nodeZ <= nodeZEnd; 
        ++nodeZ,
        mipMapZOffset += _brickDim.z)
    {
        //if(loopCount++ % 3 == 0)
        //    std::cout << ".";
        int mipMapYOffset = nodeBaseY;
        for(size_t nodeY = nodeYStart;
            nodeY <= nodeYEnd;
            ++nodeY,
            mipMapYOffset += _brickDim.y)
        {
            size_t rowReadIndex = (nodeZ * octTreeNodesDim.y * octTreeNodesDim.x)
                                 + (nodeY * octTreeNodesDim.x) 
                                 + nodeXStart;
            size_t rowWriteIndex = writeIndex + rowReadIndex;
            s_cudaStatus = hipMemcpy(&octTreeNodes[rowWriteIndex], 
                                      &pOctTreeNodesDevPtr[rowReadIndex], 
                                      sizeof(Voxelizer::OctTreeNode) * rowReadSize,
                                      hipMemcpyDeviceToHost);
            if(s_cudaStatus != hipSuccess)
            {
                _error << "computeOctTreeNodes() error: " << s_cudaStatus << "." << std::endl;
                return;
            }

            s_cudaStatus = hipMemcpy(&octTreeConstColors[rowWriteIndex], 
                                      &pOctTreeNodesConstColorPtr[rowReadIndex], 
                                      sizeof(cuda::VoxColor) * rowReadSize,
                                      hipMemcpyDeviceToHost);
            if(s_cudaStatus != hipSuccess)
            {
                _error << "computeOctTreeNodes() error: " << s_cudaStatus << "." << std::endl;
                return;
            }

            int mipMapXOffset = nodeBaseX;
            for(size_t nodeX = nodeXStart; 
                nodeX <= nodeXEnd; 
                ++nodeX, 
                mipMapXOffset += _brickDim.x)
            {
                size_t curNodeIndex = rowWriteIndex + (nodeX - nodeXStart);
                    
                //compute x,y,z of first element of this node's brick
                if(mipMapXOffset + _brickDim.x >= mipMapChunkDim.x || mipMapXOffset <= 0 ||
                   mipMapYOffset + _brickDim.y >= mipMapChunkDim.y || mipMapYOffset <= 0 ||
                   mipMapZOffset + _brickDim.z >= mipMapChunkDim.z || mipMapZOffset <= 0)
                {
                    unsigned int brickX = 1u;
                    size_t mmXOffset = mipMapXOffset;
                    size_t brickSizeX = glm::min(_brickDim.x, mipMapChunkDim.x - mipMapXOffset);
                    if(mipMapXOffset < 0)
                    {
                        brickX = 0u - mipMapXOffset + 1u;
                        mmXOffset = 0u;
                        brickSizeX = _brickDim.x - brickX + 1u;
                    }

                    unsigned int brickY = 1u;
                    size_t mmYOffset = mipMapYOffset;
                    size_t brickSizeY = glm::min(_brickDim.y, mipMapChunkDim.y - mipMapYOffset);
                    if(mipMapYOffset < 0)
                    {
                        brickY = 0u - mipMapYOffset + 1u;
                        mmYOffset = 0u;
                        brickSizeY = _brickDim.y - brickY + 1u;
                    }
                    
                    unsigned int brickZ = 1u;
                    size_t mmZOffset = mipMapZOffset;
                    size_t brickSizeZ = glm::min(_brickDim.z, mipMapChunkDim.z - mipMapZOffset);
                    if(mipMapZOffset < 0)
                    {
                        brickZ = 0u - mipMapZOffset + 1u;
                        mmZOffset = 0u;
                        brickSizeZ = _brickDim.z - brickZ + 1u;
                    }

                    brickWriter.storePartialBrick(nodeX, nodeY, nodeZ,
                                                  brickX, brickY, brickZ,
                                                  mmXOffset, 
                                                  mmYOffset, 
                                                  mmZOffset,
                                                  brickSizeX,
                                                  brickSizeY,
                                                  brickSizeZ,
                                                  _brickDim.x, _brickDim.y, _brickDim.z,
                                                  voxelColorMM.colors, voxelNormalMM.normals,
                                                  voxelColorMM.dim.x, voxelColorMM.dim.y, voxelColorMM.dim.z);
                }
                else
                {
#ifdef __DEBUG__
                    if(!ValidateNode(mipMapXOffset, mipMapYOffset, mipMapZOffset,
                                     voxelColorMM.dim.x, voxelColorMM.dim.y, voxelColorMM.dim.z,
                                     _brickDim,
                                     voxelColorMM.colors,
                                     octTreeNodes[curNodeIndex]))
                    {
                        _error << "Failed to validate node." << std::endl;
                    }
#endif
                    if(octTreeNodes[curNodeIndex] == 1u)
                    {
                        if(!brickWriter.writeBrick(nodeX, nodeY, nodeZ,
                                                static_cast<size_t>(mipMapXOffset), 
                                                static_cast<size_t>(mipMapYOffset), 
                                                static_cast<size_t>(mipMapZOffset),
                                                voxelColorMM.dim.x, voxelColorMM.dim.y, voxelColorMM.dim.z,
                                                _brickDim.x, _brickDim.y, _brickDim.z,
                                                voxelColorMM.colors, voxelNormalMM.normals))
                        {
                            _error << "Failed to write brick data." << std::endl;
                        }

                    }
                }
            }
        }
    }

    //std::cout << std::endl;

    if(depth == 0)
        return;
    --depth;

    glm::uvec3 nextMipMapDim(mipMapChunkDim.x >> 1u,
                             mipMapChunkDim.y >> 1u,
                             mipMapChunkDim.z >> 1u);

    glm::uvec3 nextOffsetToBrickBorder(glm::max(offsetToBrickBorder.x >> 1u, 1u),//there is at least a one voxel border
                                       glm::max(offsetToBrickBorder.y >> 1u, 1u),
                                       glm::max(offsetToBrickBorder.z >> 1u, 1u));

    size_t nextNumNodes = static_cast<size_t>(glm::pow(8.0f, static_cast<float>(depth)));
    size_t nextXOffset = xOffset >> 1;
    size_t nextYOffset = yOffset >> 1;
    size_t nextZOffset = zOffset >> 1;
    computeOctTreeNodes(nextXOffset, nextYOffset, nextZOffset,
                        voxelDeviceMipMaps,
                        voxelColorMipMaps,
                        voxelNormalMipMaps,
                        voxelBrickWriters,
                        nextMipMapDim,
                        nextOffsetToBrickBorder,
                        octTreeDevBuffers,
                        octTreeNodes,
                        octTreeConstColors,
                        writeIndex - nextNumNodes,
                        depth);
}

bool Voxelizer::allocateDeviceMipMaps(const glm::uvec3& voxChunkDim,
                                     hipPitchedPtr& voxelColorsDevPtr,
                                     hipPitchedPtr& voxelNormalsDevPtr,
                                     VoxelDeviceMipMaps& voxelDeviceMipMaps)
{
    hipExtent voxExt = make_hipExtent(voxChunkDim.x * sizeof(cuda::VoxColor),
                                        voxChunkDim.y,
                                        voxChunkDim.z);
    
    //allocate gpu memory to hold voxel colors
    s_cudaStatus = hipMalloc3D(&voxelColorsDevPtr, voxExt);
    if (s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceMipMaps() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += (voxExt.width * voxExt.height * voxExt.depth);

    s_cudaStatus = hipMemset(voxelColorsDevPtr.ptr, 0, 
                              voxelColorsDevPtr.pitch * voxExt.height * voxExt.depth);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceMipMaps() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    //allocate gpu memory to hold voxel normals
    voxExt.width = voxChunkDim.x * sizeof(cuda::VoxNorm);
    s_cudaStatus = hipMalloc3D(&voxelNormalsDevPtr, voxExt);
    if (s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceMipMaps() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += (voxExt.width * voxExt.height * voxExt.depth);

    s_cudaStatus = hipMemset(voxelNormalsDevPtr.ptr, 0, 
                              voxelNormalsDevPtr.pitch * voxExt.height * voxExt.depth);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceMipMaps() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    
    //mip maps stored on device
    voxelDeviceMipMaps.push_back(VoxelDataPtrs(voxelColorsDevPtr, voxelNormalsDevPtr));

    if(!allocateDeviceMipMaps(voxChunkDim.x, 
                              voxChunkDim.y, 
                              voxChunkDim.z, 
                              voxelDeviceMipMaps))
    {
        return false;
    }

    return true;
}

bool Voxelizer::allocateDeviceVoxelizationMemory(const glm::uvec3& voxChunkDim,
                                                hipPitchedPtr& voxelTriIndicesDevPtr,
                                                hipPitchedPtr& voxelTriCountsDevPtr)
{
    hipExtent voxExt = make_hipExtent(voxChunkDim.x * sizeof(glm::lowp_ivec4),
                                        voxChunkDim.y,
                                        voxChunkDim.z);
    
    //allocate gpu memory to hold voxel colors
    s_cudaStatus = hipMalloc3D(&voxelTriIndicesDevPtr, voxExt);
    if (s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceVoxelizationMemory() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += (voxelTriIndicesDevPtr.pitch * voxExt.height * voxExt.depth);

    s_cudaStatus = hipMemset(voxelTriIndicesDevPtr.ptr, -1, 
                              voxelTriIndicesDevPtr.pitch * voxExt.height * voxExt.depth);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceVoxelizationMemory() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    //allocate gpu memory to hold voxel normals
    /*voxExt.width = voxChunkDim.x * sizeof(cuda::VoxNorm);
    s_cudaStatus = hipMalloc3D(&voxelNormalsDevPtr, voxExt);
    if (s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceVoxelizationMemory() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += (voxExt.width * voxExt.height * voxExt.depth);

    s_cudaStatus = hipMemset(voxelNormalsDevPtr.ptr, 0, 
                              voxelNormalsDevPtr.pitch * voxExt.height * voxExt.depth);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceVoxelizationMemory() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }*/
    
    //allocate gpu memory to hold voxel tri counts
    voxExt.width = voxChunkDim.x * sizeof(glm::uint);
    s_cudaStatus = hipMalloc3D(&voxelTriCountsDevPtr, voxExt);
    if (s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceVoxelizationMemory() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }
    s_totalAllocatedDeviceMemory += (voxelTriCountsDevPtr.pitch * voxExt.height * voxExt.depth);

    s_cudaStatus = hipMemset(voxelTriCountsDevPtr.ptr, 0,
                              voxelTriCountsDevPtr.pitch * voxExt.height * voxExt.depth);
    if(s_cudaStatus != hipSuccess)
    {
        _error << "allocateDeviceVoxelizationMemory() error: " << s_cudaStatus << "." << std::endl;
        hipFree(voxelTriCountsDevPtr.ptr);
        return false;
    }

    return true;
}

//static void PrintVoxelColors(const VoxelColors& voxelColors, const glm::uvec3& voxDim)
//{
//    for(size_t z = 0; z < voxDim.z; ++z)
//    {
//        for(size_t y = 0; y < voxDim.y; ++y)
//        {
//            for(size_t x = 0; x < voxDim.x; ++x)
//            {
//                const cuda::VoxColor& color = voxelColors[(z * voxDim.y * voxDim.x) + (y * voxDim.x) + x];
//                if(color.w > 0.0f)
//                {
//                    std::cout << "Color at [" 
//                              << x << ", " << y << "," << z 
//                              << "]="
//                              << "[" 
//                              << color.x << ", " << color.y << ", " << color.z << ", " << color.w 
//                              << "]" 
//                              << std::endl;
//                }
//            }
//        }
//    }
//}

static inline bool VEC4_EQUAL(const glm::vec4& v1, const glm::vec4& v2)
{
    static float epsilon = 0.001f;
    glm::vec4 diff = v1 - v2;

    return (diff.x > -epsilon && diff.x < epsilon) &&
           (diff.y > -epsilon && diff.y < epsilon) &&
           (diff.z > -epsilon && diff.z < epsilon) &&
           (diff.w > -epsilon && diff.w < epsilon);
}

static inline bool VEC4_EQUAL(const uchar4& v1, const uchar4& v2)
{
    return v1.x == v2.x && v1.y == v2.y && v1.z == v2.z && v1.w == v2.w;
}

static inline bool VEC3_EQUAL(const glm::vec3& v1, const glm::vec3& v2)
{
    static float epsilon = 0.001f;
    glm::vec3 diff = v1 - v2;

    return (diff.x > -epsilon && diff.x < epsilon) &&
           (diff.y > -epsilon && diff.y < epsilon) &&
           (diff.z > -epsilon && diff.z < epsilon);
}

static inline bool VEC3_EQUAL(const uchar3& v1, const uchar3& v2)
{
    return v1.x == v2.x && v1.y == v2.y && v1.z == v2.z;
}

float VectorLenSq(const glm::vec3& vec)
{
     return vec.x * vec.x + vec.y * vec.y + vec.z * vec.z;
}

//static bool ValidateVoxelMipMaps(const VoxelColorMipMaps& voxelColorMipMaps,
//                                 const VoxelNormalMipMaps& voxelNormalMipMaps,
//                                 const glm::uvec3& voxDim,
//                                 size_t numMipMapLevels,
//                                 size_t curDepth=0)
//{
//    const VoxelColors& curLevelColors = voxelColorMipMaps.at(curDepth).colors;
//    const VoxelNormals& curLevelNormals = voxelNormalMipMaps.at(curDepth).normals;
//
//    const VoxelColors& nxtLevelColors = voxelColorMipMaps.at(curDepth+1).colors;
//    const VoxelNormals& nxtLevelNormals = voxelNormalMipMaps.at(curDepth+1).normals;
//
//    glm::uvec3 mmDim(voxDim);
//    mmDim.x >>= 1;
//    mmDim.y >>= 1;
//    mmDim.z >>= 1;
//
//    glm::uint xScale = voxDim.x / mmDim.x;
//    glm::uint yScale = voxDim.y / mmDim.y;
//    glm::uint zScale = voxDim.z / mmDim.z;
//
//    for(glm::uint z = 0; z < mmDim.z; ++z)
//    {
//        glm::uint baseZ = z * zScale;
//        glm::uint baseZPlus1 = baseZ + 1;
//        for(glm::uint y = 0; y < mmDim.y; ++y)
//        {
//            glm::uint baseY = y * yScale;
//            glm::uint baseYPlus1 = baseY + 1;
//            for(glm::uint x = 0; x < mmDim.x; ++x)
//            {
//                glm::uint baseX = x * xScale;
//                glm::uint baseXPlus1 = baseX + 1;
//    
//                const cuda::VoxColor& color = nxtLevelColors[(z * mmDim.y * mmDim.x) + (y * mmDim.x) + x];
//                const cuda::VoxNorm& normal = nxtLevelNormals[(z * mmDim.y * mmDim.x) + (y * mmDim.x) + x];
//
//                cuda::VoxColor colorBox[8] = {
//                    curLevelColors[(baseZ * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseX],//x, y, z
//                    curLevelColors[(baseZ * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseXPlus1],//x+1, y, z
//                    curLevelColors[(baseZ * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseX],//x, y+1, z
//                    curLevelColors[(baseZ * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseXPlus1],//x+1, y+1, z
//                    curLevelColors[(baseZPlus1 * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseX],//x, y, z+1
//                    curLevelColors[(baseZPlus1 * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseXPlus1],//x+1, y, z+1 
//                    curLevelColors[(baseZPlus1 * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseX],//x, y+1, z+1
//                    curLevelColors[(baseZPlus1 * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseXPlus1]//x+1, y+1, z+1
//                };
//
//                float alphaSum = colorBox[0].w;
//                alphaSum += colorBox[1].w;
//                alphaSum += colorBox[2].w;
//                alphaSum += colorBox[3].w;
//                alphaSum += colorBox[4].w;
//                alphaSum += colorBox[5].w;
//                alphaSum += colorBox[6].w;
//                alphaSum += colorBox[7].w;
//
//                float alphaWeights[8] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
//                if(alphaSum > 0.0f)
//                {
//                    alphaWeights[0] = colorBox[0].w / alphaSum;
//                    alphaWeights[1] = colorBox[1].w / alphaSum;
//                    alphaWeights[2] = colorBox[2].w / alphaSum;
//                    alphaWeights[3] = colorBox[3].w / alphaSum;
//                    alphaWeights[4] = colorBox[4].w / alphaSum;
//                    alphaWeights[5] = colorBox[5].w / alphaSum;
//                    alphaWeights[6] = colorBox[6].w / alphaSum;
//                    alphaWeights[7] = colorBox[7].w / alphaSum;
//                }
//
//                cuda::VoxColor avgColor;
//                cuda::VoxColor curColor4;
//                glm::vec3 curColor3;
//                curColor4 = colorBox[0];
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[0];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                curColor4 = colorBox[1]; 
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[1];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                curColor4 = colorBox[2];
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[2];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                curColor4 = colorBox[3];
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[3];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                curColor4 = colorBox[4];
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[4];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                curColor4 = colorBox[5];
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[5];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                curColor4 = colorBox[6];
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[6];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                curColor4 = colorBox[7];
//                curColor3 = glm::vec3(curColor4.x, curColor4.y, curColor4.z);
//                curColor3 *= alphaWeights[7];//glm::pow(curColor4.w, 2.0f);
//                avgColor += cuda::VoxColor(curColor3, curColor4.w);
//
//                float oneOverEight = (1.0f / 8.0f);
//                //avgColor.x *= oneOverEight;
//                //avgColor.y *= oneOverEight;
//                //avgColor.z *= oneOverEight;
//                avgColor.w *= oneOverEight;//glm::min(avgColor.w, 1.0f);
//                avgColor.w = glm::pow(avgColor.w, 1.0f - avgColor.w);
//                avgColor.x *= avgColor.w;
//                avgColor.y *= avgColor.w;
//                avgColor.z *= avgColor.w;
//
//                if(VEC4_EQUAL(avgColor, color) == false)
//                    return false;
//
//                cuda::VoxNorm avgNormal;
//                avgNormal  = curLevelNormals[(baseZ * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseX] 
//                            * alphaWeights[0];//colorBox[0].w;//x, y, z
//                avgNormal += curLevelNormals[(baseZ * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseXPlus1]
//                            * alphaWeights[1];//colorBox[1].w;//x+1, y, z 
//                avgNormal += curLevelNormals[(baseZ * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseX]
//                            * alphaWeights[2];//colorBox[2].w;//x, y+1, z
//                avgNormal += curLevelNormals[(baseZ * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseXPlus1]
//                            * alphaWeights[3];//colorBox[3].w;//x+1, y+1, z
//
//                avgNormal += curLevelNormals[(baseZPlus1 * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseX]
//                            * alphaWeights[4];//colorBox[4].w;//x, y, z+1
//                avgNormal += curLevelNormals[(baseZPlus1 * voxDim.y * voxDim.x) + (baseY * voxDim.x) + baseXPlus1]
//                            * alphaWeights[5];//colorBox[5].w;//x+1, y, z+1 
//                avgNormal += curLevelNormals[(baseZPlus1 * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseX]
//                            * alphaWeights[6];//colorBox[6].w;//x, y+1, z+1
//                avgNormal += curLevelNormals[(baseZPlus1 * voxDim.y * voxDim.x) + (baseYPlus1 * voxDim.x) + baseXPlus1]
//                            * alphaWeights[7];//colorBox[7].w;//x+1, y+1, z+1
//
//                //avgNormal *= oneOverEight;
//                float len = glm::length(avgNormal);
//                if(len > 0.0f)
//                    avgNormal /= len;
//                else
//                    avgNormal = cuda::VoxNorm(0);
//
//                if(VEC3_EQUAL(avgNormal, normal) == false)
//                    return false;
//            }
//        }
//    }
//
//    ++curDepth;
//    if(curDepth+1 == numMipMapLevels)
//    {
//        return true;
//    }
//
//    return ValidateVoxelMipMaps(voxelColorMipMaps, voxelNormalMipMaps, mmDim, numMipMapLevels, curDepth);
//}

//static void CopyRootBrickChunkToRootBrick(glm::uint xOffset, glm::uint yOffset, glm::uint zOffset,
//                                     const VoxelColorMipMap& chunkColors,
//                                     const VoxelNormalMipMap& chunkNormals,
//                                     VoxelColorMipMap& rootBrickColors,
//                                     VoxelNormalMipMap& rootBrickNormals)
//{
//    
//    for(glm::uint zRead = 0u; zRead < chunkColors.dim.z; ++zRead)
//    {
//        for(glm::uint yRead = 0u; yRead < chunkColors.dim.y; ++yRead)
//        {
//            glm::uint readIndex = (zRead * chunkColors.dim.y * chunkColors.dim.x)
//                                  + (yRead * chunkColors.dim.x);
//
//            glm::uint writeIndex = ((zOffset + zRead) * rootBrickColors.dim.y * rootBrickColors.dim.x)
//                           + ((yOffset + yRead) * rootBrickColors.dim.x)
//                           + xOffset;
//
//            memcpy(&rootBrickColors.colors[writeIndex],
//                   &chunkColors.colors[readIndex],
//                   sizeof(cuda::VoxColor) * chunkColors.dim.x);
//            
//            memcpy(&rootBrickNormals.normals[writeIndex],
//                   &chunkNormals.normals[readIndex],
//                   sizeof(cuda::VoxNorm) * chunkNormals.dim.x);
//        }
//    }
//}

bool Voxelizer::initVoxelBrickWriters(const std::string& outputDir, 
                                      bool outputBinary,
                                      bool outputCompressed,
                                      VoxelBrickWriters& voxelBrickWriters)
{
    std::string voxelMipMapPrefix = "voxels_";
    for(size_t i = 0; i < _voxelColorMipMaps.size(); ++i)
    {
        std::stringstream voxelOutputFile;
        voxelOutputFile << outputDir << "/" << voxelMipMapPrefix << i;

        if(i == 0 && 
           outputBinary && 
           voxelBrickWriters.at(i).createOpenGLContext() == false)
        {
            _error << "Failed to create compression context for compressed output." << std::endl;
            return false;
        }

        std::stringstream outputPartialBricksDir;
        outputPartialBricksDir << outputDir << "/partial_bricks_" << i;
        if(!voxelBrickWriters.at(i).startBricksFile(outputPartialBricksDir.str(), voxelOutputFile.str(), outputBinary, outputCompressed))
        {
            _error << "Failed to open " << voxelOutputFile.str() << " for output." << std::endl;
            return false;
        }
    }

    return true;
}

bool WriteCompletedStoredBricks(VoxelBrickWriters& voxelBrickWriters,
                       const Voxelizer::OctTreeNodes& octTreeNodes,
                       size_t octTreeNodeStartIndex=0,
                       size_t depth=0)
{
    size_t numNodes = static_cast<size_t>(glm::pow(8.0f, static_cast<float>(depth)));
    size_t cubeRoot = static_cast<size_t>(glm::pow(static_cast<float>(numNodes), 1.0f / 3.0f));

    size_t index = octTreeNodeStartIndex;
    const glm::uint* pOctTreeNodes = &octTreeNodes[index];
    
    if(!voxelBrickWriters.at(depth).writeCompletedStoredBricks(pOctTreeNodes, cubeRoot, cubeRoot))
        return false;

    if(depth == voxelBrickWriters.size() - 1)
        return true;

    return WriteCompletedStoredBricks(voxelBrickWriters,
                             octTreeNodes,
                             octTreeNodeStartIndex + numNodes,
                             depth+1);
}

static void SvgCircle(std::ofstream& svg,
                      float radius,
                      size_t x, size_t y, size_t triIndex,
                      size_t voxX, size_t voxY, size_t voxZ)
{
    static std::string colors[] = {
        "red", "green", "blue", 
        "darkred", "darkgreen", "darkblue",
        "gold", "gray", "deeppink",
        "darkslategray", "blueviolet", "brown", 
        "crimson", "hotpink", "yellow",
        "tomato", "seagreen", "purple"
    };
    size_t colorIndex = triIndex % 18;
    const std::string& color = colors[colorIndex];
    svg << "<circle stroke-width=\"1\" "
        << "r=\"" << radius << "\" "
        << "cx=\"" << x << "\" "
        << "cy=\"" << y << "\" "
        << "fill=\"" << color << "\" "
        << "id=\"" << triIndex << "\" "
        << "onclick=\"alert('" << voxX << ", " << voxY << ", " << voxZ << "')\" "
        << "/>" << std::endl;
}

static void SvgCircle(std::ofstream& svg,
                      float radius,
                      size_t x, size_t y,
                      const cuda::VoxColor& color,
                      size_t voxX, size_t voxY, size_t voxZ)
{
    svg << "<circle stroke-width=\"1\" "
        << "r=\"" << radius << "\" "
        << "cx=\"" << x << "\" "
        << "cy=\"" << y << "\" "
        << "fill=\"rgb("
        << static_cast<int>(sizeof(color) > 4 ? color.x * 255.0 : color.x) << ","
        << static_cast<int>(sizeof(color) > 4 ? color.y * 255.0 : color.y) << ","
        << static_cast<int>(sizeof(color) > 4 ? color.z * 255.0 : color.z) << "\" "
        << "opacity=\""
        << color.w
        << "\" "
        << "onclick=\"alert('" << voxX << ", " << voxY << ", " << voxZ << "')\" "
        << "/>" << std::endl;
}

static void SvgTriangle(std::ofstream& svg,
                        float x0, float y0,
                        float x1, float y1,
                        float x2, float y2,
                        size_t triIndex,
                        const glm::vec3& v0,
                        const glm::vec3& v1,
                        const glm::vec3& v2)
{
    svg << "<path stroke-width=\"0.25\" "
        << "d=\"M" 
        << x0 << " " 
        << y0 << "L" 
        << x1 << " " 
        << y1 << "L" 
        << x2 << " " 
        << y2 << " Z\" "
        << "stroke=\"black\" "
        << "id=\"" << triIndex << "\" "
        << "opacity=\"0.25\" fill=\"black\" "
        << "onclick=\"alert('v0=[" 
        << v0.x << ", " << v0.y << ", " << v0.z << "] v1=["
        << v1.x << ", " << v1.y << ", " << v1.z << "] v2=["
        << v2.x << ", " << v2.y << ", " << v2.z << "]')\""
        << "/>" << std::endl;
}

static void PrintSvgHeader(std::ofstream& svg)
{
    svg << "<svg" << std::endl;
    svg << "    xmlns:svg='http://www.w3.org/2000/svg'" << std::endl;
    svg << "    xmlns='http://www.w3.org/2000/svg'" << std::endl;
    svg << "    version='1.1'" << std::endl;
    svg << "    width='800'" << std::endl;
    svg << "    height='600'" << std::endl;
    svg << "    onload='init(evt)'>" << std::endl;
    svg << "   " << std::endl;
    svg << "  <style>" << std::endl;
    svg << "  		.territory:hover{" << std::endl;
    svg << "			fill:           #22aa44;" << std::endl;
    svg << "		}" << std::endl;
    svg << "  		.compass{" << std::endl;
    svg << "  			fill:			#fff;" << std::endl;
    svg << "  			stroke:			#000;" << std::endl;
    svg << "  			stroke-width:	1.5;" << std::endl;
    svg << "  		}" << std::endl;
    svg << "   		.button{" << std::endl;
    svg << "		    fill:           	#225EA8;" << std::endl;
    svg << "			stroke:   			#0C2C84;" << std::endl;
    svg << "			stroke-miterlimit:	6;" << std::endl;
    svg << "			stroke-linecap:		round;" << std::endl;
    svg << "		}" << std::endl;
    svg << "		.button:hover{" << std::endl;
    svg << "			stroke-width:   	2;" << std::endl;
    svg << "		}" << std::endl;
    svg << "		.plus-minus{" << std::endl;
    svg << "			fill:	#fff;" << std::endl;
    svg << "			pointer-events: none;" << std::endl;
    svg << "		}" << std::endl;
    svg << "  </style>" << std::endl;
    svg << "  " << std::endl;
    svg << "  <script type='text/ecmascript'>" << std::endl;
    svg << "    <![CDATA[" << std::endl;
    svg << "		var transMatrix = [2,0,0,2,0,0];" << std::endl;
    svg << "        " << std::endl;
    svg << "        function init(evt)" << std::endl;
    svg << "        {" << std::endl;
    svg << "            if ( window.svgDocument == null )" << std::endl;
    svg << "            {" << std::endl;
    svg << "                svgDoc = evt.target.ownerDocument;" << std::endl;
    svg << "" << std::endl;
    svg << "            }" << std::endl;
    svg << "            mapMatrix = svgDoc.getElementById('map-matrix');" << std::endl;
    svg << "            width  = evt.target.getAttributeNS(null, 'width');" << std::endl;
    svg << "            height = evt.target.getAttributeNS(null, 'height');" << std::endl;
    svg << "        }" << std::endl;
    svg << "        " << std::endl;
    svg << "        function pan(dx, dy)" << std::endl;
    svg << "        {" << std::endl;
    svg << "        	" << std::endl;
    svg << "			transMatrix[4] += dx;" << std::endl;
    svg << "			transMatrix[5] += dy;" << std::endl;
    svg << "            " << std::endl;
    svg << "			newMatrix = 'matrix(' +  transMatrix.join(' ') + ')';" << std::endl;
    svg << "			mapMatrix.setAttributeNS(null, 'transform', newMatrix);" << std::endl;
    svg << "        }" << std::endl;
    svg << "        " << std::endl;
    svg << "		function zoom(scale)" << std::endl;
    svg << "		{" << std::endl;
    svg << "			for (var i=0; i<transMatrix.length; i++)" << std::endl;
    svg << "			{" << std::endl;
    svg << "				transMatrix[i] *= scale;" << std::endl;
    svg << "			}" << std::endl;
    svg << "			transMatrix[4] += (1-scale)*width/2;" << std::endl;
    svg << "			transMatrix[5] += (1-scale)*height/2;" << std::endl;
    svg << "		        " << std::endl;
    svg << "			newMatrix = 'matrix(' +  transMatrix.join(' ') + ')';" << std::endl;
    svg << "			mapMatrix.setAttributeNS(null, 'transform', newMatrix);" << std::endl;
    svg << "        }" << std::endl;
    svg << "    ]]>" << std::endl;
    svg << "  </script>" << std::endl;
    svg << "  <g id='map-matrix' transform='matrix(2 0 0 2 0 0)'>" << std::endl;
}

static void PrintSvgFooter(std::ofstream& svg)
{
    svg << "  </g>" << std::endl; 
    svg << "  <circle cx='50' cy='50' r='42' fill='white' opacity='0.75'/>" << std::endl;
    svg << "  <path class='button' onclick='pan( 0, 50)' d='M50 10 l12   20 a40, 70 0 0,0 -24,  0z' />" << std::endl;
    svg << "  <path class='button' onclick='pan( 50, 0)' d='M10 50 l20  -12 a70, 40 0 0,0   0, 24z' />" << std::endl;
    svg << "  <path class='button' onclick='pan( 0,-50)' d='M50 90 l12  -20 a40, 70 0 0,1 -24,  0z' />" << std::endl;
    svg << "  <path class='button' onclick='pan(-50, 0)' d='M90 50 l-20 -12 a70, 40 0 0,1   0, 24z' />" << std::endl;
  
    svg << "  <circle class='compass' cx='50' cy='50' r='20'/>" << std::endl;
    svg << "  <circle class='button'  cx='50' cy='41' r='8' onclick='zoom(0.8)'/>" << std::endl;
    svg << "  <circle class='button'  cx='50' cy='59' r='8' onclick='zoom(1.25)'/>" << std::endl;

    svg << "  <rect class='plus-minus' x='46' y='39.5' width='8' height='3'/>" << std::endl;
    svg << "  <rect class='plus-minus' x='46' y='57.5' width='8' height='3'/>" << std::endl;
    svg << "  <rect class='plus-minus' x='48.5' y='55' width='3' height='8'/>" << std::endl;
        
  svg << "</svg>" << std::endl;
}

static void ExportMipMapToSvg(const VoxelColors& voxelColors,
                              const glm::uvec3& voxDim,
                              const glm::vec3& voxOrigin,
                              const glm::vec3& voxScale,
                              const std::string& outputFileName)
{
    std::ofstream svgX, svgY, svgZ;
    svgX.open(outputFileName + ".x.svg");
    svgY.open(outputFileName + ".y.svg");
    svgZ.open(outputFileName + ".z.svg");
    
    PrintSvgHeader(svgX);
    PrintSvgHeader(svgY);
    PrintSvgHeader(svgZ);
    
    for(size_t z = 0; z < voxDim.z; ++z)
    {
        for(size_t y = 0; y < voxDim.y; ++y)
        {
            for(size_t x = 0; x < voxDim.x; ++x)
            {
                const cuda::VoxColor& color = voxelColors.at((z * voxDim.x * voxDim.y) + (y * voxDim.x) + x);
                float radius = 0.25f;
                if(color.w > 0.0f)
                {
                    SvgCircle(svgX, radius, z, y, color, x, y, z);
                    SvgCircle(svgY, radius, x, z, color, x, y, z);
                    SvgCircle(svgZ, radius, x, y, color, x, y, z);
                }
            }
        }
    }
    
    svgX << "<rect "
         << "x=\"0\" "
         << "y=\"0\" "
         << "width=\"" << voxDim.z << "\" "
         << "height=\"" << voxDim.y << "\" "
         << "stroke-width=\"1\" "
         << "fill=\"none\" "
         << "stroke=\"black\" "
         << "opacity=\"0.5\" />"
         << std::endl;
    svgY << "<rect "
         << "x=\"0\" "
         << "y=\"0\" "
         << "width=\"" << voxDim.x << "\" "
         << "height=\"" << voxDim.z << "\" "
         << "stroke-width=\"1\" "
         << "fill=\"none\" "
         << "stroke=\"black\" "
         << "opacity=\"0.5\" />"
         << std::endl;
    svgZ << "<rect "
         << "x=\"0\" "
         << "y=\"0\" "
         << "width=\"" << voxDim.x << "\" "
         << "height=\"" << voxDim.y << "\" "
         << "stroke-width=\"1\" "
         << "fill=\"none\" "
         << "stroke=\"black\" "
         << "opacity=\"0.5\" />"
         << std::endl;

    PrintSvgFooter(svgX);
    PrintSvgFooter(svgY);
    PrintSvgFooter(svgZ);
}

static void ExportToSvg(const glm::uvec3& voxDim, 
                        size_t pitch,
                        const glm::lowp_ivec4* pTriIndicesBuf,
                        const std::vector<glm::vec3>* pTriVerts,
                        size_t triGroupOffset,
                        const glm::vec3& voxOrigin,
                        const glm::vec3& voxScale,
                        const std::string& outputFileName)
{   
    std::ofstream svgX, svgY, svgZ;
    svgX.open(outputFileName + ".x.svg");
    svgY.open(outputFileName + ".y.svg");
    svgZ.open(outputFileName + ".z.svg");
    
    PrintSvgHeader(svgX);
    PrintSvgHeader(svgY);
    PrintSvgHeader(svgZ);

    for(size_t vertIndex = 0;
        vertIndex < pTriVerts->size();
        vertIndex += 3)
    {
        size_t triIndex = static_cast<size_t>(static_cast<float>(vertIndex) / 3.0f);
        glm::vec3 v00 = pTriVerts->at(vertIndex);
        glm::vec3 v11 = pTriVerts->at(vertIndex+1);
        glm::vec3 v22 = pTriVerts->at(vertIndex+2);
        glm::vec3 v0 = ((v00 - voxOrigin) * voxScale.x);
        glm::vec3 v1 = ((v11 - voxOrigin) * voxScale.y);
        glm::vec3 v2 = ((v22 - voxOrigin) * voxScale.z);
        /*if((v0.x >= 0.0f && static_cast<unsigned int>(v0.x) < voxDim.x &&
            v0.y >= 0.0f && static_cast<unsigned int>(v0.y) < voxDim.y &&
            v0.z >= 0.0f && static_cast<unsigned int>(v0.z) < voxDim.z) ||
           (v1.x >= 0.0f && static_cast<unsigned int>(v1.x) < voxDim.x &&
            v1.y >= 0.0f && static_cast<unsigned int>(v1.y) < voxDim.y &&
            v1.z >= 0.0f && static_cast<unsigned int>(v1.z) < voxDim.z) ||
           (v2.x >= 0.0f && static_cast<unsigned int>(v2.x) < voxDim.x &&
            v2.y >= 0.0f && static_cast<unsigned int>(v2.y) < voxDim.y &&
            v2.z >= 0.0f && static_cast<unsigned int>(v2.z) < voxDim.z))*/
        {
            SvgTriangle(svgX, v0.z, v0.y, v1.z, v1.y, v2.z, v2.y, triIndex, 
                            v00, v11, v22);
            SvgTriangle(svgY, v0.x, v0.z, v1.x, v1.z, v2.x, v2.z, triIndex,
                            v00, v11, v22);
            SvgTriangle(svgZ, v0.x, v0.y, v1.x, v1.y, v2.x, v2.y, triIndex,
                            v00, v11, v22);
        }
    }
    
    std::set<size_t> triangles;
    char* pReadPtrTriIndices = (char*)pTriIndicesBuf;
    size_t triIndicesSlicePitch = pitch * voxDim.y;
    for(size_t z = 0; z < voxDim.z; ++z)
    {
        char* pSliceReadTriIndices = pReadPtrTriIndices + z * triIndicesSlicePitch;
        for(size_t y = 0; y < voxDim.y; ++y)
        {
            glm::lowp_ivec4* pVoxelReadTriIndices = 
                (glm::lowp_ivec4*)(pSliceReadTriIndices + y * pitch);
            for(size_t x = 0; x < voxDim.x; ++x)
            {
                const glm::lowp_ivec4& indices = pVoxelReadTriIndices[x];
                for(size_t i = 0; i < 4 && indices[i] != -1; ++i)
                {
                    size_t triIndex = indices[i] - triGroupOffset;
                    float radius = (1.0f / static_cast<float>(i+1)) * 0.5f;
                    SvgCircle(svgX, radius, z, y, triIndex, x, y, z);
                    SvgCircle(svgY, radius, x, z, triIndex, x, y, z);
                    SvgCircle(svgZ, radius, x, y, triIndex, x, y, z);
                    triangles.insert(triIndex);
                }
            }
        }
    }
    
    svgX << "<rect "
         << "x=\"0\" "
         << "y=\"0\" "
         << "width=\"255\" "
         << "height=\"255\" "
         << "stroke-width=\"1\" "
         << "fill=\"none\" "
         << "stroke=\"black\" "
         << "opacity=\"0.5\" />"
         << std::endl;
    svgY << "<rect "
         << "x=\"0\" "
         << "y=\"0\" "
         << "width=\"255\" "
         << "height=\"255\" "
         << "stroke-width=\"1\" "
         << "fill=\"none\" "
         << "stroke=\"black\" "
         << "opacity=\"0.5\" />"
         << std::endl;
    svgZ << "<rect "
         << "x=\"0\" "
         << "y=\"0\" "
         << "width=\"255\" "
         << "height=\"255\" "
         << "stroke-width=\"1\" "
         << "fill=\"none\" "
         << "stroke=\"black\" "
         << "opacity=\"0.5\" />"
         << std::endl;

    PrintSvgFooter(svgX);
    PrintSvgFooter(svgY);
    PrintSvgFooter(svgZ);
}

static void ExportToScalarFileHeader(std::ofstream& voxFile,
                                     size_t voxDimX, size_t voxDimY, size_t voxDimZ,
                                     const std::string& headerFileName)
{
    voxFile.open(headerFileName, std::fstream::out);
    if(!voxFile.is_open())
    {
        return;
    }

    voxFile << "VOXEL_HEADER" << std::endl;
    voxFile << "POSITION " 
            << -1.0f * static_cast<float>(voxDimX >> 1) << " "
            << -1.0f * static_cast<float>(voxDimY >> 1) << " "
            << -1.0f * static_cast<float>(voxDimZ >> 1) << std::endl;
    voxFile << "ORIENTATION 0 0 1 0" << std::endl;
    voxFile << "SCALE 1" << std::endl;
    voxFile << "DIMENSIONS "
            << voxDimX << " "
            << voxDimY << " "
            << voxDimZ << std::endl;
    voxFile << "TYPE SCALARS" << std::endl;
    voxFile << "FORMAT GL_R8" << std::endl;
    voxFile << "VOXEL_HEADER_END" << std::endl;
    voxFile << "VOXEL_SUB_IMAGE_FILES" << std::endl;
}

static void ExportToScalarFile(
                        std::ofstream& voxScalarHeaderFile,
                        const glm::uvec3& voxDim, 
                        size_t pitch,
                        size_t xStart, size_t yStart, size_t zStart,
                        size_t xEnd, size_t yEnd, size_t zEnd,
                        const glm::uint* pTriCountsBuf,
                        const std::string& outputFileName,
                        const glm::uvec3& subRangeStart,
                        const glm::uvec3& subRangeEnd)
{
    voxScalarHeaderFile << "VOXEL_SUB_IMAGE_FILE " 
                << subRangeStart.x << " " << subRangeStart.y << " " << subRangeStart.z << " "
                << subRangeEnd.x << " " << subRangeEnd.y << " " << subRangeEnd.z << " "
                << outputFileName
                << std::endl;

    std::ofstream voxOut;
    voxOut.open(outputFileName, std::ios_base::out | std::ios_base::binary);
    if(voxOut.is_open() == false)
        return;

    size_t xSize = xEnd - xStart;
    size_t ySize = yEnd - yStart;
    size_t zSize = zEnd - zStart;
    
    std::vector<unsigned char> voxelScalars(xSize * ySize * zSize);

    char* pReadPtrTriCounts = (char*)pTriCountsBuf;
    size_t triCountsSlicePitch = pitch * voxDim.y;
    size_t zWrite = 0;
    for(size_t z = zStart; z < zEnd; ++z, ++zWrite)
    {
        char* pSliceReadTriCounts = pReadPtrTriCounts + z * triCountsSlicePitch;
        size_t yWrite = 0;
        for(size_t y = yStart; y < yEnd; ++y, ++yWrite)
        {
            glm::uint* pVoxelReadTriCounts = 
                (glm::uint*)(pSliceReadTriCounts + y * pitch);
            size_t writeIndex = (zWrite * ySize * xSize) + (yWrite * xSize);
            unsigned char* pVoxelScalars = &voxelScalars[writeIndex];
            size_t xWrite = 0;
            for(size_t x = xStart; x < xEnd; ++x, ++xWrite)
            {
                const glm::uint& triCount = pVoxelReadTriCounts[x];
                if(triCount > 0)
                    pVoxelScalars[xWrite] = 255;
                else
                    pVoxelScalars[xWrite] = 0;
            }
        }
    }
    int formatIsUByte = 1;
    voxOut.write((const char*)&formatIsUByte, sizeof(formatIsUByte));
    unsigned int dataSize = sizeof(unsigned char)
                      * xSize * ySize * zSize;
    voxOut.write((const char*)&dataSize, sizeof(dataSize));
    
    voxOut.write((const char*)voxelScalars.data(), dataSize);
}

struct VoxelChunkID
{
    size_t xOffset; 
    size_t yOffset; 
    size_t zOffset;
    VoxelChunkID(size_t x, size_t y, size_t z) : 
        xOffset(x), yOffset(y), zOffset(z) {}
    bool operator<(const VoxelChunkID& rhs) const
    {
        return zOffset < rhs.zOffset || 
                (zOffset == rhs.zOffset && yOffset < rhs.yOffset) || 
                (zOffset == rhs.zOffset && yOffset == rhs.yOffset 
                    && xOffset < rhs.xOffset);
    }
};

typedef std::set<VoxelChunkID> QueuedVoxelChunks;

typedef std::list<VoxelChunkID> VoxelChunkQueue;

int Voxelizer::generateVoxelsAndOctTree(const std::string& outputDir, 
                                        bool outputBinary,
                                        bool outputCompressed)
{
    glm::uvec3 voxChunkDim = _voxChunkDim;
    //these two vectors will hold the chunks of color and normal texture after reading back from gpu
    VoxelColorMipMaps& voxelColorMipMaps = _voxelColorMipMaps;
    VoxelNormalMipMaps& voxelNormalMipMaps = _voxelNormalMipMaps;

    //allocate memory for voxel mip maps
    hipPitchedPtr& voxelTriIndicesDevPtr = _voxelTriIndicesDevPtr;
    hipPitchedPtr& voxelTriCountsDevPtr = _voxelTriCountsDevPtr;

    VoxelBrickWriters voxelBrickWriters(_voxelColorMipMaps.size());
    if(!initVoxelBrickWriters(outputDir, outputBinary, outputCompressed, voxelBrickWriters))
    {
        return false;
    }

    hipPitchedPtr& voxelColorsDevMipMapPtr = _voxelColorsDevMipMapPtr;
    hipPitchedPtr& voxelNormalsDevMipMapPtr = _voxelNormalsDevMipMapPtr;

    OctTreeDeviceBuffers& octTreeDeviceBuffers = _octTreeDeviceBuffers;
    if(!initOctTreeDeviceBuffers(_octTreeDeviceBuffers,
                                 voxelColorMipMaps.size()-1u))
    {
        return false;
    }

    OctTreeNodes& octTreeNodes = _octTreeNodes;
    octTreeNodes.assign(octTreeNodes.size(), 0u);
    OctTreeConstColors& octTreeConstColors = _octTreeConstColors;
    cuda::VoxColor zeros;
    zeros.x = zeros.y = zeros.z = zeros.w = 0;
    octTreeConstColors.assign(octTreeConstColors.size(), zeros);
     
    dim3 numBlocksPerGrid(1u, 1u, 1u);
    dim3 numThreadsPerBlock(voxChunkDim.x, voxChunkDim.y, voxChunkDim.z);
    if(numThreadsPerBlock.x > static_cast<glm::uint>(_maxBlockDim.x) || 
       numThreadsPerBlock.y > static_cast<glm::uint>(_maxBlockDim.y) || 
       numThreadsPerBlock.z > static_cast<glm::uint>(_maxBlockDim.z))
    {
        numThreadsPerBlock.x = glm::min(numThreadsPerBlock.x, static_cast<glm::uint>(_maxBlockDim.x));
        numThreadsPerBlock.y = glm::min(numThreadsPerBlock.y, static_cast<glm::uint>(_maxBlockDim.y));
        numThreadsPerBlock.z = glm::min(numThreadsPerBlock.z, static_cast<glm::uint>(_maxBlockDim.z));

        numBlocksPerGrid.x = 
            static_cast<size_t>(glm::ceil((static_cast<float>(voxChunkDim.x) / static_cast<float>(numThreadsPerBlock.x))));
        numBlocksPerGrid.y = 
            static_cast<size_t>(glm::ceil((static_cast<float>(voxChunkDim.y) / static_cast<float>(numThreadsPerBlock.y))));
        numBlocksPerGrid.z = 
            static_cast<size_t>(glm::ceil((static_cast<float>(voxChunkDim.z) / static_cast<float>(numThreadsPerBlock.z))));
    }

    dim3 threadsPerBlock(numThreadsPerBlock.x,
                         numThreadsPerBlock.y,
                         numThreadsPerBlock.z);

    dim3 computeCNBlocksPerGrid(1u, 1u, 1u);
    dim3 computeCNThreadsPerBlock(voxChunkDim.x, voxChunkDim.y, voxChunkDim.z);
    if(voxChunkDim.x > static_cast<glm::uint>(_maxBlockDim.x) || 
       voxChunkDim.y > static_cast<glm::uint>(_maxBlockDim.y) || 
       voxChunkDim.z > static_cast<glm::uint>(_maxBlockDim.z))
    {
        computeCNThreadsPerBlock.x = glm::min(voxChunkDim.x, static_cast<glm::uint>(_maxBlockDim.x));
        computeCNThreadsPerBlock.y = glm::min(voxChunkDim.y, static_cast<glm::uint>(_maxBlockDim.y));
        computeCNThreadsPerBlock.z = glm::min(voxChunkDim.z, static_cast<glm::uint>(_maxBlockDim.z));

        computeCNBlocksPerGrid.x = 
            static_cast<size_t>(glm::ceil((static_cast<float>(voxChunkDim.x) / static_cast<float>(computeCNThreadsPerBlock.x))));
        computeCNBlocksPerGrid.y = 
            static_cast<size_t>(glm::ceil((static_cast<float>(voxChunkDim.y) / static_cast<float>(computeCNThreadsPerBlock.y))));
        computeCNBlocksPerGrid.z = 
            static_cast<size_t>(glm::ceil((static_cast<float>(voxChunkDim.z) / static_cast<float>(computeCNThreadsPerBlock.z))));
    }

#ifdef __DEBUG__
    std::vector<std::string> debugVoxelColorFiles;
    bool debugFormatIsUByte = true;
    std::vector<glm::uvec3> debugVoxelColorStartRanges;
    std::vector<glm::uvec3> debugVoxelColorEndRanges;
    std::ofstream voxScalarFile;
    if(s_debug_output || s_debug_scalars)
    {
        std::stringstream scalarFileHeader;
        scalarFileHeader << outputDir << "/VoxelizerScalars.voxt";
        ExportToScalarFileHeader(voxScalarFile,
                                 _voxDim.x,
                                 _voxDim.y,
                                 _voxDim.z,
                                 scalarFileHeader.str());
    }
#endif

    bool atLeastOneTriGroupOverlapsVoxChunk = false;
    //use this for texture map
    unsigned char* pCurImage = nullptr;
    //iterate through entire highest res voxel mip map
    QueuedVoxelChunks queuedVoxelChunks;
    VoxelChunkQueue voxelChunkQueue;

    voxelChunkQueue.push_back(VoxelChunkID(0u, 0u, 0u));
    queuedVoxelChunks.insert(voxelChunkQueue.front());

    while(voxelChunkQueue.size() > 0)
    {
        VoxelChunkID voxelChunkID = voxelChunkQueue.front();
        voxelChunkQueue.pop_front();

        glm::uint x, y, z;
        x = voxelChunkID.xOffset;
        y = voxelChunkID.yOffset;
        z = voxelChunkID.zOffset;
        for(glm::uint nx = x; nx <= x + voxChunkDim.x; nx += voxChunkDim.x)
        {
            for(glm::uint ny = y; ny <= y + voxChunkDim.y; ny += voxChunkDim.y)
            {
                for(glm::uint nz = z; nz <= z + voxChunkDim.z; nz += voxChunkDim.z)
                {
                    VoxelChunkID neighborVoxelChunk(nx, ny, nz);
                    if(neighborVoxelChunk.xOffset < _voxDim.x
                        && neighborVoxelChunk.yOffset < _voxDim.y
                        && neighborVoxelChunk.zOffset < _voxDim.z
                        && queuedVoxelChunks.insert(neighborVoxelChunk).second == true)
                    {
                        voxelChunkQueue.push_back(neighborVoxelChunk);
                    }
                }
            }
        }
    
        glm::uvec3 minVoxChunk(x, y, z);
        glm::uvec3 maxVoxChunk(minVoxChunk.x + voxChunkDim.x - 1u,
                                minVoxChunk.y + voxChunkDim.y - 1u,
                                minVoxChunk.z + voxChunkDim.z - 1u);

        std::cout << "Voxelizing " 
                    << minVoxChunk.x << " " 
                    << minVoxChunk.y << " " 
                    << minVoxChunk.z << std::endl;

        glm::vec3 voxOffsetOrigin(_offsetP.x + (x * _deltaP.x),
                                    _offsetP.y + (y * _deltaP.y),
                                    _offsetP.z + (z * _deltaP.z));
                
        hipMemset(voxelTriCountsDevPtr.ptr, 0,
                    voxelTriCountsDevPtr.pitch * voxChunkDim.y * voxChunkDim.z);

        hipMemset(voxelColorsDevMipMapPtr.ptr, 0, 
                    voxelColorsDevMipMapPtr.pitch * voxChunkDim.y * voxChunkDim.z);

        hipMemset(voxelNormalsDevMipMapPtr.ptr, 0,
                    voxelNormalsDevMipMapPtr.pitch * voxChunkDim.y * voxChunkDim.z);

        size_t triGroupOffset = 0;
        size_t uvOffset = 0;
        size_t normalsOffset = 0;
        for(TriangleGroups::iterator triGrp = _triangleGroups.begin();
            triGrp != _triangleGroups.end();
            ++triGrp)
        {
            bool trisOverlapVoxChunk = true;
            if((triGrp->voxelBBox.minVox.x > maxVoxChunk.x ||
                triGrp->voxelBBox.maxVox.x < minVoxChunk.x))
                trisOverlapVoxChunk = false;
            if((triGrp->voxelBBox.minVox.y > maxVoxChunk.y ||
                triGrp->voxelBBox.maxVox.y < minVoxChunk.y))
                trisOverlapVoxChunk = false;
            if((triGrp->voxelBBox.minVox.z > maxVoxChunk.z ||
                triGrp->voxelBBox.maxVox.z < minVoxChunk.z))
                trisOverlapVoxChunk = false;

            if(trisOverlapVoxChunk)
                atLeastOneTriGroupOverlapsVoxChunk = true;

            size_t numTris = triGrp->pVerts->size() / 3;

            dim3 numBlocks(numBlocksPerGrid.x,
                            numBlocksPerGrid.y,
                            static_cast<glm::uint>(numTris));

            size_t maxTrisPerVoxelization = 512u;
            size_t voxelizationBlockZDim = numTris;
            if(numTris > maxTrisPerVoxelization)
            {
                voxelizationBlockZDim = maxTrisPerVoxelization;
            }

            bool perVtxNormals = triGrp->pVertNormals != nullptr;
            for(size_t triOffset = triGroupOffset;
                trisOverlapVoxChunk && triOffset < (triGroupOffset + numTris);
                triOffset += maxTrisPerVoxelization)
            {
                if((triOffset-triGroupOffset) + voxelizationBlockZDim > numTris)
                    numBlocks.z = static_cast<glm::uint>(numTris - (triOffset-triGroupOffset));
                else
                    numBlocks.z = static_cast<glm::uint>(voxelizationBlockZDim);

                glm::uint zStart = glm::max(triGrp->voxelBBox.minVox.z, minVoxChunk.z);
                glm::uint zEnd = glm::min(triGrp->voxelBBox.maxVox.z, maxVoxChunk.z);
                for(glm::uint threadZ = zStart;
                    threadZ <= zEnd;
                    threadZ += threadsPerBlock.z)
                {
                    dim3 curThreadsPerBlock = threadsPerBlock;
                    if(threadZ + curThreadsPerBlock.z > zEnd)
                        curThreadsPerBlock.z = zEnd - threadZ + 1;
                    ComputeVoxelization<<<numBlocks, curThreadsPerBlock>>>(_pgVerts, triOffset, 
                                                                        _pgEdges, 
                                                                        _pgFaceNormals,
                                                                        _pgBounds, 
                                                                        _offsetP, _deltaP, 
                                                                        minVoxChunk, maxVoxChunk,
                                                                        threadZ,
                                                                        voxelTriCountsDevPtr,
                                                                        voxelTriIndicesDevPtr);
                    s_cudaStatus = hipDeviceSynchronize();
                    if(s_cudaStatus != hipSuccess)
                    {
                        _error << "generateVoxelsAndOctTree() error: " << s_cudaStatus << "." << std::endl;
                        return 0;
                    }
                }
            }
#ifdef __DEBUG__
            if(trisOverlapVoxChunk && 
                (s_debug_output || s_debug_svg == true))
            {
                if(s_debug_svg)
                {
                    std::cout << "Debugging svg, ";
                            
                    std::stringstream dbgFileName;
                    dbgFileName << outputDir << "/" << "VoxelizerDebug-" 
                                << minVoxChunk.x << "-"
                                << minVoxChunk.y << "-"
                                << minVoxChunk.z << "-"
                                << triGroupOffset;

                    size_t bufSize = voxelTriIndicesDevPtr.pitch 
                                    * voxChunkDim.y
                                    * voxChunkDim.z;
                    glm::lowp_ivec4* pTriIndicesBuf = (glm::lowp_ivec4*)malloc(bufSize);
                    if(pTriIndicesBuf != NULL)
                    {
                        hipMemcpy(pTriIndicesBuf, voxelTriIndicesDevPtr.ptr, bufSize, hipMemcpyDeviceToHost);
                     
                        ExportToSvg(voxChunkDim, voxelTriIndicesDevPtr.pitch,
                                    pTriIndicesBuf, 
                                    triGrp->pVerts, triGroupOffset, 
                                    voxOffsetOrigin,
                                    glm::vec3(1.0f / _deltaP.x,
                                                1.0f / _deltaP.y,
                                                1.0f / _deltaP.z),
                                    dbgFileName.str());

                        free(pTriIndicesBuf);
                    }
                    else
                    {
                        std::cout << "Failed to allocate svg debug memory for " 
                                    << dbgFileName.str() << "."
                                    << std::endl;
                    }
                }
                        
                if(s_debug_output == true)
                {
                    s_debug_voxels = true;
                    s_debug_scalars = true;
                }
            }
#endif

            if(triGrp->pImageData != nullptr && triGrp->pUVs != nullptr)
            { 
                ImageMap::iterator findIt = _imageMap.find(triGrp->pImageData);
                if(findIt == _imageMap.end())
                {
                    findIt = _imageMap.insert(std::make_pair(triGrp->pImageData,
                                                                nullptr)).first;
                }

                if(findIt->second == nullptr && trisOverlapVoxChunk)
                {
                    s_cudaStatus = hipMallocArray(&findIt->second, 
                                                    &triGrp->imageDesc, 
                                                    triGrp->imageWidth,
                                                    triGrp->imageHeight);
                    if(s_cudaStatus != hipSuccess)
                    {
                        _error << "generateVoxelsAndOctTree() failed to allocate image array, error: " << s_cudaStatus << "." << std::endl;
                        return 0;
                    }

                    s_totalAllocatedDeviceMemory += (sizeof(unsigned char) * 4 * triGrp->imageWidth * triGrp->imageHeight);

                    s_cudaStatus = hipMemcpyToArray(findIt->second,
                                                        0u, 0u, 
                                                        triGrp->pImageData,
                                                        sizeof(unsigned char) * 4
                                                        * triGrp->imageWidth 
                                                        * triGrp->imageHeight,
                                                        hipMemcpyHostToDevice);
                    if(s_cudaStatus != hipSuccess)
                    {
                        _error << "generateVoxelsAndOctTree() failed to memcpy image data to image array, error: " << s_cudaStatus << "." << std::endl;
                        return 0;
                    }
                }
                        
                if(pCurImage != triGrp->pImageData && trisOverlapVoxChunk)
                {
                    pCurImage = triGrp->pImageData;
                    if(!BindTextureToArray(findIt->second,
                                            triGrp->imageDesc,
                                            triGrp->texAddressMode0,
                                            triGrp->texAddressMode1))
                    {
                        _error << "generateVoxelsAndOctTree() failed to bind texture to array, error: " << s_cudaStatus << "." << std::endl;
                        return 0;
                    }
                }
                        
                if(trisOverlapVoxChunk)
                {
                    ComputeColorsAndNormals<<<computeCNBlocksPerGrid, computeCNThreadsPerBlock>>>(triGroupOffset,
                                                                                        _pgVerts, 
                                                                                        perVtxNormals ? 
                                                                                            &_pgVtxNormals[normalsOffset] : nullptr,
                                                                                        _pgFaceNormals,
                                                                                        &_pgUVs[uvOffset],
                                                                                        triGrp->isTerrain,
                                                                                        voxOffsetOrigin,
                                                                                        _deltaP,
                                                                                        voxelTriCountsDevPtr,
                                                                                        voxelTriIndicesDevPtr,
                                                                                        voxelColorsDevMipMapPtr,
                                                                                        voxelNormalsDevMipMapPtr,
                                                                                        voxChunkDim);
                    //uvOffset += triGrp->pUVs->size();
                }
            }
            else if(trisOverlapVoxChunk)
            {
                ComputeColorsAsNormals<<<computeCNBlocksPerGrid, computeCNThreadsPerBlock>>>(triGroupOffset,
                                                                                        _pgVerts, 
                                                                                        perVtxNormals ? 
                                                                                            &_pgVtxNormals[normalsOffset] : nullptr,
                                                                                        _pgFaceNormals,
                                                                                        voxOffsetOrigin,
                                                                                        _deltaP,
                                                                                        voxelTriCountsDevPtr,
                                                                                        voxelTriIndicesDevPtr,
                                                                                        voxelColorsDevMipMapPtr,
                                                                                        voxelNormalsDevMipMapPtr,
                                                                                        voxChunkDim);
            }
                    
            if(trisOverlapVoxChunk)
            {
                s_cudaStatus = hipDeviceSynchronize();
                if(s_cudaStatus != hipSuccess)
                {
                    _error << "generateVoxelsAndOctTree() error: " << s_cudaStatus << "." << std::endl;
                    return 0;
                }
            }

            if(perVtxNormals)
                normalsOffset += triGrp->pVertNormals->size();

            if(triGrp->pUVs != nullptr)
                uvOffset += triGrp->pUVs->size();

            triGroupOffset += numTris;
        }
                
        //std::cout << "Computing mipmaps, ";
        // TODO if no data was voxelized then we can replace this with memset to zero
        if(!computeMipMaps(voxChunkDim.x,//host memory chunk dimensions 
                            voxChunkDim.y, 
                            voxChunkDim.z, 
                            voxChunkDim.x>>1,
                            voxChunkDim.y>>1,
                            voxChunkDim.z>>1,
                            _voxelDeviceMipMaps,
                            voxelColorMipMaps,
                            voxelNormalMipMaps))
        {
            return 0;
        }

        VoxelColors& voxelColors = voxelColorMipMaps.front().colors;
        VoxelNormals& voxelNormals = voxelNormalMipMaps.front().normals;

        //update headers, update compute oct-tree, write data in chunks
        copyDeviceChunkToHostMipMapChunk(voxelColors,
                                            voxelNormals,
                                            voxChunkDim.x,//host memory chunk dimensions 
                                            voxChunkDim.y, 
                                            voxChunkDim.z,
                                            voxelColorsDevMipMapPtr,
                                            voxelNormalsDevMipMapPtr);//device memory dimensions
#ifdef __DEBUG__
        if(s_debug_mipmaps)
        {
            std::cout << "Debugging mipmaps, ";
            
            for(size_t depth = 1u; depth < voxelColorMipMaps.size(); ++depth)
            {
                const VoxelColors& voxelMMColors = voxelColorMipMaps[depth].colors;
                VoxelNormals& voxelMMNormals = voxelNormalMipMaps[depth].normals;
                const glm::uvec3& dim = voxelColorMipMaps[depth].dim;
        
                std::stringstream dbgFileName;
                dbgFileName << outputDir << "/" << "VoxelizerMipMapDebug-"
                            << x << "-"
                            << y << "-"
                            << z << "-"
                            << dim.x << "-"
                            << dim.y << "-"
                            << dim.z << "-"
                            << depth
                            << ".voxb";

                glm::uvec3 rangeStart(0);
                glm::uvec3 rangeEnd(dim);
                std::vector<std::string> debugVoxelMMColorFiles;
                std::vector<glm::uvec3> debugVoxelMMColorStartRanges;
                std::vector<glm::uvec3> debugVoxelMMColorEndRanges;
                //ExportMipMapToSvg(voxelColors,
                //                  dim,
                //                  voxOffsetOrigin,
                //                  glm::vec3(1.0f / _deltaP.x,
                //                            1.0f / _deltaP.y,
                //                            1.0f / _deltaP.z),
                //                  dbgFileName.str());
                debugVoxelMMColorFiles.push_back(dbgFileName.str());
                debugVoxelMMColorStartRanges.push_back(rangeStart);
                debugVoxelMMColorEndRanges.push_back(rangeEnd);
                VoxelBrickWriter::ExportToFile(voxelMMColors, voxelNormals,
                                              false,
                                              dim.x,//host memory chunk dimensions 
                                              dim.y, 
                                              dim.z,
                                              0, 0, 0,
                                              dim.x, dim.y, dim.z,
                                              dbgFileName.str());
                std::stringstream debugFileHeader;
                debugFileHeader << dbgFileName.str() << ".voxt";
                VoxelBrickWriter::ExportToFileHeader(dim.x,
                                                     dim.y,
                                                     dim.z,
                                                     debugFileHeader.str(),
                                                     false,
                                                     debugVoxelMMColorFiles,
                                                     debugVoxelMMColorStartRanges,
                                                     debugVoxelMMColorEndRanges);
            }
        }
#endif
#ifdef __DEBUG__
        if(s_debug_voxels == true || s_debug_scalars == true)
        {
            glm::uvec3 voxChunkStart(_extraVoxChunk);
            glm::uvec3 voxChunkEnd(maxVoxChunk + glm::uvec3(1));
            glm::uvec3 rangeStart(0);
            if(minVoxChunk.x != 0)
            {
                voxChunkStart.x = 0;
                rangeStart.x = minVoxChunk.x - _extraVoxChunk.x;
                if(maxVoxChunk.x > _voxDim.x)
                    voxChunkEnd.x = _voxDim.x - rangeStart.x;
                else
                    voxChunkEnd.x = maxVoxChunk.x - minVoxChunk.x + 1u;
            }

            if(minVoxChunk.y != 0)
            {
                voxChunkStart.y = 0;
                rangeStart.y = minVoxChunk.y - _extraVoxChunk.y;
                if(maxVoxChunk.y > _voxDim.y)
                    voxChunkEnd.y = _voxDim.y - rangeStart.y;
                else
                    voxChunkEnd.y = maxVoxChunk.y - minVoxChunk.y + 1u;
            }

            if(minVoxChunk.z != 0)
            {
                voxChunkStart.z = 0;
                rangeStart.z = minVoxChunk.z - _extraVoxChunk.z;
                if(maxVoxChunk.z > _voxDim.z)
                    voxChunkEnd.z = _voxDim.z - rangeStart.z;
                else
                    voxChunkEnd.z = maxVoxChunk.z - minVoxChunk.z + 1u;
            }
            
            glm::uvec3 rangeEnd(rangeStart + (voxChunkEnd - voxChunkStart));
            if(s_debug_voxels)
            {
                std::cout << "Debugging voxels, ";
                                        
                std::stringstream dbgFileName;
                dbgFileName << outputDir << "/" << "VoxelizerDebug-" 
                            << rangeStart.x << "-"
                            << rangeStart.y << "-"
                            << rangeStart.z << ".voxb";
                debugVoxelColorFiles.push_back(dbgFileName.str());
                debugVoxelColorStartRanges.push_back(rangeStart);
                debugVoxelColorEndRanges.push_back(rangeEnd);
                VoxelBrickWriter::ExportToFile(voxelColors, voxelNormals,
                                                debugFormatIsUByte,
                                                voxChunkDim.x,//host memory chunk dimensions 
                                                voxChunkDim.y, 
                                                voxChunkDim.z,
                                                voxChunkStart.x, voxChunkStart.y, voxChunkStart.z,
                                                voxChunkEnd.x, voxChunkEnd.y, voxChunkEnd.z,
                                                dbgFileName.str());
                s_debug_voxels = false;
            }

            if(s_debug_scalars)
            {
                std::cout << "Debugging scalars, ";

                std::stringstream dbgFileName;
                dbgFileName << outputDir << "/" << "VoxelizerScalars-" 
                            << rangeStart.x << "-"
                            << rangeStart.y << "-"
                            << rangeStart.z << ".voxs";

                size_t bufSize = voxelTriCountsDevPtr.pitch 
                                * voxChunkDim.y
                                * voxChunkDim.z;
                glm::uint* pTriCountsBuf = (glm::uint*)malloc(bufSize);

                if(pTriCountsBuf != NULL)
                {
                    hipMemcpy(pTriCountsBuf, voxelTriCountsDevPtr.ptr, bufSize, hipMemcpyDeviceToHost);

                    ExportToScalarFile(voxScalarFile,
                                        voxChunkDim,
                                        voxelTriCountsDevPtr.pitch,
                                        voxChunkStart.x, voxChunkStart.y, voxChunkStart.z,
                                        voxChunkEnd.x, voxChunkEnd.y, voxChunkEnd.z,
                                        pTriCountsBuf,
                                        dbgFileName.str(),
                                        rangeStart,
                                        rangeEnd);

                    free(pTriCountsBuf);
                }
                else
                {
                    std::cout << "Failed to allocate debug scalars memory for " 
                                << dbgFileName.str() << "."
                                << std::endl;
                }

                s_debug_voxels = false;
                s_debug_scalars = false;
            }
        }
#endif
        //copy root brick chunk into full res root brick
        //CopyRootBrickChunkToRootBrick(rootBrickXOffset, 
        //                              rootBrickYOffset, 
        //                              rootBrickZOffset,
        //                              voxelColorMipMaps.back(),
        //                              voxelNormalMipMaps.back(),
        //                              rootBrickColors,
        //                              rootBrickNormals);
#ifdef __DEBUG_MIPMAPS__
        //PrintVoxelColors(voxelColorMipMaps.front(), _voxDim);
        if(!ValidateVoxelMipMaps(voxelColorMipMaps, voxelNormalMipMaps, voxChunkDim, _numMipMapLevels))
        {
            _error << "Failed to validate mip maps." << std::endl;
            return 0;
        }
#endif
        //std::cout << "Compute octree.";
                
        computeOctTreeNodes(x, y, z,
                            _voxelDeviceMipMaps,
                            voxelColorMipMaps,
                            voxelNormalMipMaps,
                            voxelBrickWriters,
                            voxChunkDim,
                            _extraVoxChunk,
                            octTreeDeviceBuffers,
                            octTreeNodes,
                            octTreeConstColors,
                            _octTreeNodesWriteIndex,
                            _numMipMapLevels-1);
                
        //std::cout << "Write stored bricks, ";

        WriteCompletedStoredBricks(voxelBrickWriters, octTreeNodes);

        std::cout << "Done." << std::endl;
    }
#ifdef __DEBUG__
    if(debugVoxelColorFiles.size() > 0)
    {
        std::stringstream debugFileHeader;
        debugFileHeader << outputDir << "/VoxelizerDebug.voxt";
        VoxelBrickWriter::ExportToFileHeader(_voxDim.x,
                                             _voxDim.y,
                                             _voxDim.z,
                                             debugFileHeader.str(),
                                             debugFormatIsUByte,
                                             debugVoxelColorFiles,
                                             debugVoxelColorStartRanges,
                                             debugVoxelColorEndRanges);
    }
#endif

    if(atLeastOneTriGroupOverlapsVoxChunk)
    {
        std::cout << "Collapsed constant branches, ";

        CollapseConstantBranches(octTreeNodes);

        std::cout << "Write octree, ";

        bool retVal = writeOctTree(outputDir, outputBinary, outputCompressed, voxelBrickWriters);

        std::cout << "Done." << std::endl;

        return 1;
    }

    return -1;
}

static bool WriteOctTreeNode(std::ofstream& treeFile,
                             std::stringstream& error,
                             VoxelBrickWriters& brickWriters,
                             const Voxelizer::OctTreeNodes& octTreeNodes,
                             const Voxelizer::OctTreeConstColors& octTreeConstColors,
                             size_t xOffset,//offset in mip map
                             size_t yOffset,
                             size_t zOffset,
                             size_t xSize,//size of mip map
                             size_t ySize,
                             size_t zSize,
                             const glm::uvec3& brickDim,
                             size_t octTreeDepth,
                             size_t curX=0,
                             size_t curY=0,
                             size_t curZ=0,
                             size_t curXSize=1,
                             size_t curYSize=1,
                             size_t curZSize=1,
                             size_t curStart=0,
                             size_t childX=0,
                             size_t childY=0,
                             size_t childZ=0,
                             size_t childXSize=2,
                             size_t childYSize=2,
                             size_t childZSize=2,
                             size_t nextStartIndex=1)
{
    //size_t maxOctTreeDepth = voxelColorMipMaps.size()-1;
    
    //const VoxelColors& voxelColors = voxelColorMipMaps.at(voxelColorMipMaps.size() - 1 - octTreeDepth).colors;
    //const VoxelNormals& voxelNormals = voxelNormalMipMaps.at(voxelNormalMipMaps.size() - 1 - octTreeDepth).normals;

    size_t childStartIndex = nextStartIndex + 
                             ((childZ * childYSize * childXSize) + (childY * childXSize) + childX);
    size_t curIndex = curStart + 
                      ((curZ * curYSize * curXSize) + (curY * curXSize) + curX);

    const Voxelizer::OctTreeNode& curOctTreeNode = octTreeNodes[curIndex];
    
    for(size_t i = 0; i <= octTreeDepth; ++i)
        treeFile << "    ";//indent

    if(curOctTreeNode == 1u)//non-const
    {
        VoxelBrickWriter& brickWriter = brickWriters.at(octTreeDepth);
        treeFile << "<Node "
                 << "MipMapX=\"" << xOffset << "\" "
                 << "MipMapY=\"" << yOffset << "\" "
                 << "MipMapZ=\"" << zOffset << "\" "
                 << "Type=\"NON-CONST\" "
                 << "Brick=\"" << brickWriter.getBrickIndex(curX, curY, curZ) << "\" "
                 << "Depth=\"" << octTreeDepth << "\"";
        
        /*bool isHighestResBrick = octTreeDepth == maxOctTreeDepth;

        if(!brickWriter.writeBrick(xOffset, yOffset, zOffset,
                               xSize, ySize, zSize,
                               brickDim,
                               isHighestResBrick,
                               voxelColors, voxelNormals))
        {
            error << "Failed to write brick " 
                  << (brickWriter.getBrickCount()-1) 
                  << " to file: " 
                  << brickWriter.getOutputFileName() << "." << std::endl;
            return false;
        }*/
    }
    else//const node
    {
        //size_t firstVoxelInBrick = (zOffset * ySize * xSize) + (yOffset * xSize) + xOffset;
        //node type constant-color
        treeFile << "<Node "
                 << "MipMapX=\"" << xOffset << "\" "
                 << "MipMapY=\"" << yOffset << "\" "
                 << "MipMapZ=\"" << zOffset << "\" "
                 << "Type=\""
                 << (curOctTreeNode == 0 ? "CONST" : "LEAF-CONST")//LEAF-CONST means that I am const and all my children are too
                 << "\" "
                 << "ColorR=\"" 
                 << (sizeof(cuda::VoxColor) == 4 ?
                        static_cast<float>(octTreeConstColors.at(curIndex).x) / 255.0 :
                        octTreeConstColors.at(curIndex).x)
                 << "\" "
                 << "ColorG=\"" 
                 << (sizeof(cuda::VoxColor) == 4 ?
                        static_cast<float>(octTreeConstColors.at(curIndex).y) / 255.0 :
                        octTreeConstColors.at(curIndex).y)
                 << "\" "
                 << "ColorB=\""
                 << (sizeof(cuda::VoxColor) == 4 ?
                        static_cast<float>(octTreeConstColors.at(curIndex).z) / 255.0 :
                        octTreeConstColors.at(curIndex).z)
                 << "\" "
                 << "ColorA=\""
                 << (sizeof(cuda::VoxColor) == 4 ?
                        static_cast<float>(octTreeConstColors.at(curIndex).w) / 255.0 :
                        octTreeConstColors.at(curIndex).w)
                 << "\" "
                 << "Depth=\"" 
                 << octTreeDepth << "\"";
    }

    if(childStartIndex < octTreeNodes.size() //if we are not at leaf node
#ifndef __DEBUG__  
        && curOctTreeNode != 2//if current node is 2 then all our children are constant so we can skip them
#endif
      )
    {
        //end <Node> tag
        treeFile << ">"
                 << std::endl;

        size_t childMipMapXSize = xSize << 1;//child mip map size
        size_t childMipMapYSize = ySize << 1; 
        size_t childMipMapZSize = zSize << 1;

        size_t gChildXSize = childXSize << 1;
        size_t gChildYSize = childYSize << 1;
        size_t gChildZSize = childZSize << 1;
        size_t gChildStartIndex = nextStartIndex + (childXSize * childYSize * childZSize);

        size_t endX = childX + 2;
        size_t endY = childY + 2;
        size_t endZ = childZ + 2;
        for(size_t zChild = childZ, childZOffset = (zOffset << 1); 
            zChild < endZ; 
            ++zChild, childZOffset += brickDim.z)
        {
            size_t gChildZ = zChild << 1;
            for(size_t yChild = childY, childYOffset = (yOffset << 1);
                yChild < endY; 
                ++yChild, childYOffset += brickDim.y)
            {
                size_t gChildY = yChild << 1;
                for(size_t xChild = childX, childXOffset = (xOffset << 1);
                    xChild < endX;
                    ++xChild, childXOffset += brickDim.x)
                {
                    size_t gChildX = xChild << 1;
#ifdef __DEBUG__
                    if(curOctTreeNode == 2)
                    {
                        size_t childIndex = nextStartIndex + 
                                ((zChild * childYSize * childXSize) + (yChild * childXSize) + xChild);
                        //validate that all my children are constant
                        const Voxelizer::OctTreeNode& childOctTreeNode = octTreeNodes[childIndex];
                        if(childOctTreeNode == 1)
                        {
                            error << "Collapsed constant node is not valid." << std::endl;
                            return false;
                        }
                    }
#endif
                    if(!WriteOctTreeNode(treeFile, 
                                         error, 
                                         brickWriters, 
                                         //voxelColorMipMaps, 
                                         //voxelNormalMipMaps, 
                                         octTreeNodes,
                                         octTreeConstColors,
                                         childXOffset, childYOffset, childZOffset,
                                         childMipMapXSize, childMipMapYSize, childMipMapZSize,
                                         brickDim,
                                         octTreeDepth + 1, 
                                         xChild, yChild, zChild, 
                                         childXSize, childYSize, childZSize, 
                                         nextStartIndex, 
                                         gChildX, gChildY, gChildZ,
                                         gChildXSize, gChildYSize, gChildZSize,
                                         gChildStartIndex))
                    {
                        return false;
                    }
                }
            }
        }

        for(size_t i = 0; i <= octTreeDepth; ++i)
            treeFile << "    ";//indent
        treeFile << "</Node>" << std::endl;
    }
    else
    {
        //end <Node> tag
        treeFile << " />"
                 << std::endl;
    }

    return true;
}

bool Voxelizer::writeOctTree(const std::string& outputDir, 
                             bool outputBinary,
                             bool outputCompressed,
                             VoxelBrickWriters& voxelBrickWriters)

{
    std::string outFileName = outputDir + "/tree.gvx";
    std::ofstream treeFile(outFileName, std::fstream::out);
    if(!treeFile.is_open())
    {
        _error << "Failed to open " << outFileName << " for output." << std::endl;
        return false;
    }

    treeFile << "<GigaVoxelsOctTree "
             << "MaxDepth=\"" << _voxelColorMipMaps.size() << "\" "
             << "X=\"" << _p.x << "\" "
             << "Y=\"" << _p.y << "\" "
             << "Z=\"" << _p.z << "\" "
             << "DeltaX=\"" << _deltaP.x << "\" "
             << "DeltaY=\"" << _deltaP.y << "\" "
             << "DeltaZ=\"" << _deltaP.z << "\" "
             << "VolumeXSize=\"" << _voxDim.x << "\" "
             << "VolumeYSize=\"" << _voxDim.y << "\" "
             << "VolumeZSize=\"" << _voxDim.z << "\" "
             << "BrickXSize=\"" << _brickDim.x << "\" "
             << "BrickYSize=\"" << _brickDim.y << "\" "
             << "BrickZSize=\"" << _brickDim.z << "\" "
             << "Binary=\"" << (outputBinary ? "YES" : "NO") << "\" "
             << "Compressed=\"" << (outputCompressed ? "YES" : "NO") << "\" >"
             << std::endl;

    size_t xOffset = 0;
    size_t yOffset = 0;
    size_t zOffset = 0;
    size_t xSize = _brickDim.x;
    size_t ySize = _brickDim.y;
    size_t zSize = _brickDim.z;
    size_t octTreeDepth = 0;

    bool retVal = WriteOctTreeNode(treeFile, _error, 
                                   voxelBrickWriters, 
                                   _octTreeNodes,
                                   _octTreeConstColors,
                                   xOffset, yOffset, zOffset,
                                   xSize, ySize, zSize,
                                   _brickDim, octTreeDepth);

    treeFile << "</GigaVoxelsOctTree>"
             << std::endl;

    return retVal;
}

void Voxelizer::freeVoxelDeviceMipMaps()
{
    for(VoxelDeviceMipMaps::iterator itr = _voxelDeviceMipMaps.begin();
        itr != _voxelDeviceMipMaps.end();
        ++itr)
    {
        hipFree(itr->first.ptr);
        hipFree(itr->second.ptr);
    }

    _voxelDeviceMipMaps.clear();
}

void Voxelizer::freeVoxelMipMapsAndOctTree()
{
    VoxelColorMipMaps empty1;
    _voxelColorMipMaps.swap(empty1);

    VoxelNormalMipMaps empty2;
    _voxelNormalMipMaps.swap(empty2);

    OctTreeNodes empty3;
    _octTreeNodes.swap(empty3);

    OctTreeConstColors empty4;
    _octTreeConstColors.swap(empty4);
}

bool Voxelizer::copyDeviceChunkToHostMipMapChunk(VoxelColors& voxelColors,
                                                 VoxelNormals& voxelNormals,
                                                 size_t voxDimX, size_t voxDimY, size_t voxDimZ,
                                                 const hipPitchedPtr& voxelColorsDevPtr,
                                                 const hipPitchedPtr& voxelNormalsDevPtr)
{
    cuda::VoxColor* pVoxelChunkColors = NULL;
    size_t sizeOfColorsRow = sizeof(cuda::VoxColor) * voxDimX;
    if(sizeOfColorsRow == voxelColorsDevPtr.pitch)
    {
        pVoxelChunkColors = &voxelColors[0];
    }
    else
    {
        pVoxelChunkColors = (cuda::VoxColor*)malloc(voxDimZ *  voxelColorsDevPtr.ysize * voxelColorsDevPtr.pitch);
    }

    s_cudaStatus = hipMemcpy(pVoxelChunkColors,
                              voxelColorsDevPtr.ptr, 
                              voxDimZ *  voxelColorsDevPtr.ysize * voxelColorsDevPtr.pitch, 
                              hipMemcpyDeviceToHost);
    if (s_cudaStatus != hipSuccess)
    {
        _error << "generateVoxelsAndOctTree() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }

    cuda::VoxNorm* pVoxelChunkNormals = NULL;
    size_t sizeOfNormalsRow = sizeof(cuda::VoxNorm) * voxDimX;
    if(sizeOfNormalsRow == voxelNormalsDevPtr.pitch)
    {
        pVoxelChunkNormals = &voxelNormals[0];
    }
    else
    {
        pVoxelChunkNormals = (cuda::VoxNorm*)malloc(voxDimZ * voxelNormalsDevPtr.ysize * voxelNormalsDevPtr.pitch);
    }
                        
    s_cudaStatus = hipMemcpy(pVoxelChunkNormals,
                              voxelNormalsDevPtr.ptr,  
                              voxDimZ * voxelNormalsDevPtr.ysize * voxelNormalsDevPtr.pitch, 
                              hipMemcpyDeviceToHost);
    if (s_cudaStatus != hipSuccess)
    {
        _error << "generateVoxelsAndOctTree() error: " << s_cudaStatus << "." << std::endl;
        return false;
    }

    if(sizeOfColorsRow == voxelColorsDevPtr.pitch && sizeOfNormalsRow == voxelNormalsDevPtr.pitch)
        return true;

    char* colorReadPtr = (char*)pVoxelChunkColors;
    size_t colorPitch = voxelColorsDevPtr.pitch;
    size_t colorSlicePitch = colorPitch * voxelColorsDevPtr.ysize;

    char* normalsReadPtr = (char*)pVoxelChunkNormals;
    size_t normalsPitch = voxelNormalsDevPtr.pitch;
    size_t normalsSlicePitch = normalsPitch * voxelNormalsDevPtr.ysize;
    
    for(size_t z = 0; z < voxDimZ; ++z)
    {
        char* colorSlice = colorReadPtr + z * colorSlicePitch;
        char* normalsSlice = normalsReadPtr + z * normalsSlicePitch;
        for(size_t y = 0; y < voxDimY; ++y)
        {
            cuda::VoxColor* readColorRowPtr = (cuda::VoxColor*)(colorSlice + (y * colorPitch));
            cuda::VoxNorm* readNormalsRowPtr = (cuda::VoxNorm*)(normalsSlice + (y * normalsPitch));

            size_t voxIndex = (z * voxDimX * voxDimY) + (y * voxDimX);
    
            if(sizeOfColorsRow != voxelColorsDevPtr.pitch)
                memcpy(&voxelColors[voxIndex], readColorRowPtr, sizeOfColorsRow);
            if(sizeOfNormalsRow != voxelNormalsDevPtr.pitch)
                memcpy(&voxelNormals[voxIndex], readNormalsRowPtr, sizeOfNormalsRow);
        }
    }

    if(sizeOfColorsRow != voxelColorsDevPtr.pitch)
        free(pVoxelChunkColors);
    if(sizeOfNormalsRow != voxelNormalsDevPtr.pitch)
        free(pVoxelChunkNormals);

    return true;
}

const std::string& Voxelizer::getErrorMessage()
{
    _error << " Cuda Error Code: " 
           << hipPeekAtLastError() 
           << " '" 
           << hipGetErrorString(hipGetLastError()) 
           << "'" 
           << std::endl;
    static std::string errorMsg;
    errorMsg = _error.str();
    return errorMsg;
}

//copy voxel colors to cpu buffer
                //this is only a portion of the full 3D
                //s_cudaStatus = hipMemcpy(&voxelChunkTriCounts.front(), 
                //                            voxelTriCountsDevPtr.ptr,  
                //                            voxExt.depth * voxExt.height * voxelTriCountsDevPtr.pitch, 
                //                            hipMemcpyDeviceToHost);
                //if (s_cudaStatus != hipSuccess)
                //{
                //    _error << "generateVoxelsAndOctTree() error: " << s_cudaStatus << "." << std::endl;
                //    return false;
                //}
                //debugging code
                //size_t voxIndex = (z * _voxDim.x * _voxDim.y) + (y * _voxDim.x) + x;

                //char* triCountsReadPtr = (char*)&voxelChunkTriCounts.front();
                //size_t triCountsPitch = voxelTriCountsDevPtr.pitch;
                //size_t triCountsSlicePitch = triCountsPitch * voxExt.height;
                //for(size_t zIndex = 0; zIndex < zSize; ++zIndex)
                //{
                //    char* triCountsSlice = triCountsReadPtr + zIndex * triCountsSlicePitch;
                //    for(size_t yIndex = 0; yIndex < ySize; ++yIndex)
                //    {
                //        glm::uint* readTriCountsRowPtr = (glm::uint*)(triCountsSlice + yIndex * triCountsPitch);
                //        for(size_t voxIndexItr = voxIndex; voxIndexItr < voxIndex + xSize; ++voxIndexItr)
                //        {
                //            if(readTriCountsRowPtr[voxIndexItr-voxIndex] > 0)
                //            {
                //                std::cout << "[" << (voxIndexItr - voxIndex) << "][" << yIndex << "][" << zIndex << "]=(" 
                //                        << voxelColors[voxIndexItr].x << ", "
                //                        << voxelColors[voxIndexItr].y << ", "
                //                        << voxelColors[voxIndexItr].z << ", "
                //                        << voxelColors[voxIndexItr].w << ") ("
                //                        << voxelNormals[voxIndexItr].x << ", "
                //                        << voxelNormals[voxIndexItr].y << ", "
                //                        << voxelNormals[voxIndexItr].z << ") " 
                //                        << readTriCountsRowPtr[voxIndexItr-voxIndex]
                //                        << std::endl;
                //            }
                //            else if(voxelColors[voxIndexItr].w > 0.0f)
                //                std::cout << "Huh?";
                //        }
                //        voxIndex += _voxDim.x;//advance to next row
                //    }
                //}

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <cmath>
#include <stdio.h>
#include <iostream>
#include <string>
#include <vector>

#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
#include <glm/gtx/norm.hpp>

#include "Voxelizer.h"

__device__ __host__ bool TrianglePlaneOverlapsVoxel(const glm::vec3& v0, 
                                const glm::vec3& normal,
                                const glm::vec3& p, 
                                const glm::vec3& deltaP,
                                const glm::vec3& uvw)//, 
                                //bool debug)
{
    glm::vec3 voxOffset = (uvw * deltaP);
    glm::vec3 p0 = p + voxOffset;//lower left
    glm::vec3 cornerDelta[] = {
        deltaP, //back upper right
        glm::vec3(deltaP.x, deltaP.y, 0.0f), //back lower right
        glm::vec3(deltaP.x, 0.0f, 0.0f), //front lower right
        glm::vec3(0.0f, 0.0f, deltaP.z), //front upper left
        glm::vec3(0.0f, deltaP.y, deltaP.z), //back upper left
        glm::vec3(deltaP.x, 0.0f, deltaP.z), //front upper right
        glm::vec3(0.0f, deltaP.y, 0.0f), //back lower left
    };
    float D = glm::dot(-normal, v0);
    float dist1 = glm::dot(normal, p0) + D;
    for(int i = 0; i < 7; ++i)
    {
        glm::vec3 p1 = p0 + cornerDelta[i];
        float dist2 = glm::dot(normal, p1) + D;

        /*if(debug)
        {
            printf("p0=[%f %f %f] p1=[%f %f %f] D=%f dist1=%f dist2=%f\n",
                   p0.x, p0.y, p0.z,
                   p1.x, p1.y, p1.z,
                   D, dist1, dist2);
        }*/
        if(dist1 * dist2 <= 0.0f)
            return true;
    }
    return false;
}

static const int X_AXIS = 0;
static const int Y_AXIS = 1;
static const int Z_AXIS = 2;

//bool TriangleOverlapsVoxelOnAxis(const glm::vec3* pVerts,
//                                 const glm::vec3* pEdges,
//                                 const glm::vec3* pEdgeNormals,
//                                 const glm::vec3& normal,
//                                 const glm::vec3& voxMin,
//                                 const glm::vec3& voxDelta,
//                                 int axisIndex1, int axisIndex2)
//{
//    static float epsilon = 0.0001f;
//    glm::vec2 voxMinProj = glm::vec2(voxMin[axisIndex1], voxMin[axisIndex2]);
//    //for each edge
//    for(int i = 0; i < 3; ++i)
//    {
//        const glm::vec2& edge = glm::vec2(pEdges[i][axisIndex1], pEdges[i][axisIndex2]);
//        glm::vec2 edgeNorm = glm::vec2(pEdgeNormals[i][axisIndex1], pEdgeNormals[i][axisIndex2]);
//        
//        if(glm::abs(edgeNorm.x) + glm::abs(edgeNorm.y) < epsilon)//skip degenerate normals
//            continue;
//        //compute critical point
//        glm::vec2 critPtDelta = glm::vec2((edgeNorm.x < 0.0f ? 0.0f : voxDelta.x),
//                                          (edgeNorm.y < 0.0f ? 0.0f : voxDelta.y));
//
//        glm::vec2 critPt = voxMinProj + critPtDelta;
//        
//        //compute edge function test result
//        float edgeFuncResult = ((critPt.x - pVerts[i][axisIndex1]) * edge.y)
//                                - ((critPt.y - pVerts[i][axisIndex2]) * edge.x);
//        if(edgeFuncResult > 0)
//            return false;
//    }
//
//    return true;
//}

__device__ __host__ bool TriangleOverlapsVoxelOnAxis(const glm::vec3* pVerts,
                                 const glm::vec3* pEdges,
                                 const glm::vec3& normal,
                                 const glm::vec3& p,
                                 const glm::vec3& deltaP,
                                 const glm::vec3& uvw,
                                 int axisIndex1, int axisIndex2, int otherAxisIndex)
{
    float projScalar = normal[otherAxisIndex] >= 0.0f ? 1.0f : -1.0f;
    glm::vec2 edgeNormsProjToPlane[3] = {
        (glm::vec2(-pEdges[0][axisIndex2], pEdges[0][axisIndex1]) * projScalar),
        (glm::vec2(-pEdges[1][axisIndex2], pEdges[1][axisIndex1]) * projScalar),
        (glm::vec2(-pEdges[2][axisIndex2], pEdges[2][axisIndex1]) * projScalar)
    };

    glm::vec2 voxProjToPlane[3] = {
        glm::vec2(edgeNormsProjToPlane[0].x * deltaP[axisIndex1], edgeNormsProjToPlane[0].y * deltaP[axisIndex2]),
        glm::vec2(edgeNormsProjToPlane[1].x * deltaP[axisIndex1], edgeNormsProjToPlane[1].y * deltaP[axisIndex2]),
        glm::vec2(edgeNormsProjToPlane[2].x * deltaP[axisIndex1], edgeNormsProjToPlane[2].y * deltaP[axisIndex2])
    };

    float deltaProjToPlane[3] = {
        (glm::dot(-edgeNormsProjToPlane[0], glm::vec2(pVerts[0][axisIndex1], pVerts[0][axisIndex2])))  
            + (glm::dot(edgeNormsProjToPlane[0], glm::vec2(p[axisIndex1], p[axisIndex2]))) 
            + (glm::max(0.0f, deltaP[axisIndex1] * edgeNormsProjToPlane[0].x))
            + (glm::max(0.0f, deltaP[axisIndex2] * edgeNormsProjToPlane[0].y)),
        (glm::dot(-edgeNormsProjToPlane[1], glm::vec2(pVerts[1][axisIndex1], pVerts[1][axisIndex2])))  
            + (glm::dot(edgeNormsProjToPlane[1], glm::vec2(p[axisIndex1], p[axisIndex2]))) 
            + (glm::max(0.0f, deltaP[axisIndex1] * edgeNormsProjToPlane[1].x))
            + (glm::max(0.0f, deltaP[axisIndex2] * edgeNormsProjToPlane[1].y)),
        (glm::dot(-edgeNormsProjToPlane[2], glm::vec2(pVerts[2][axisIndex1], pVerts[2][axisIndex2])))  
            + (glm::dot(edgeNormsProjToPlane[2], glm::vec2(p[axisIndex1], p[axisIndex2]))) 
            + (glm::max(0.0f, deltaP[axisIndex1] * edgeNormsProjToPlane[2].x))
            + (glm::max(0.0f, deltaP[axisIndex2] * edgeNormsProjToPlane[2].y))
    };

    float overlapTest0 = deltaProjToPlane[0] 
                            + (uvw[axisIndex2] * voxProjToPlane[0].y) 
                            + (uvw[axisIndex1] * voxProjToPlane[0].x);

                            
    float overlapTest1 = deltaProjToPlane[1] 
                            + (uvw[axisIndex2] * voxProjToPlane[1].y) 
                            + (uvw[axisIndex1] * voxProjToPlane[1].x);

                            
    float overlapTest2 = deltaProjToPlane[2] 
                            + (uvw[axisIndex2] * voxProjToPlane[2].y) 
                            + (uvw[axisIndex1] * voxProjToPlane[2].x);

    return (overlapTest0 >= 0.0f && overlapTest1 >= 0.0f && overlapTest2 >= 0.0f);
}

__device__ __host__ bool TriangleOverlapsVoxel(const glm::vec3* pVerts,
                           const glm::vec3* pEdges,
                           //const glm::vec3* pEdgeNormals,
                           const glm::vec3& normal,
                           const glm::vec3& p,
                           const glm::vec3& deltaP,
                           const glm::vec3& uvw)
{
    ////xy plane
    //if(!TriangleOverlapsVoxelOnAxis(pVerts, pEdges, pEdgeNormals, normal, p, deltaP, X_AXIS, Y_AXIS))
    //    return false;
    ////zx plane
    //if(!TriangleOverlapsVoxelOnAxis(pVerts, pEdges, pEdgeNormals, normal, p, deltaP, Z_AXIS, X_AXIS))
    //    return false;
    ////yz plane
    //return TriangleOverlapsVoxelOnAxis(pVerts, pEdges, pEdgeNormals, normal, p, deltaP, Y_AXIS, Z_AXIS);
    //xy plane
    if(!TriangleOverlapsVoxelOnAxis(pVerts, pEdges, normal, p, deltaP, uvw, X_AXIS, Y_AXIS, Z_AXIS))
        return false;
    //zx plane
    if(!TriangleOverlapsVoxelOnAxis(pVerts, pEdges, normal, p, deltaP, uvw, Z_AXIS, X_AXIS, Y_AXIS))
        return false;
    //yz plane
    return TriangleOverlapsVoxelOnAxis(pVerts, pEdges, normal, p, deltaP, uvw, Y_AXIS, Z_AXIS, X_AXIS);
}

void ComputeTriangleVoxelBounds(const glm::vec3* pVerts, 
                        const glm::vec3& p, 
                        const glm::vec3& deltaP, 
                        const glm::uvec3& voxDim, 
                        glm::uvec3& minVox, 
                        glm::uvec3& maxVox)
{
    const glm::ivec3 maxVoxIndex = glm::ivec3(voxDim) - glm::ivec3(1);
    const glm::ivec3 minVoxIndex = glm::ivec3(0);
    minVox = glm::uvec3(maxVoxIndex);
    maxVox = glm::uvec3(minVoxIndex);

    for(int i = 0; i < 3; ++i)
    {
        const glm::vec3& vert = pVerts[i];
        glm::vec3 diff = vert - p, p, maxP;

        glm::uvec3 uvw = glm::uvec3(glm::clamp(glm::ivec3(diff / deltaP),
                                    minVoxIndex, maxVoxIndex));
        if(uvw.x < minVox.x)
            minVox.x = uvw.x;
        else if(uvw.x > maxVox.x)
            maxVox.x = uvw.x;

        if(uvw.y < minVox.y)
            minVox.y = uvw.y;
        else if(uvw.y > maxVox.y)
            maxVox.y = uvw.y;

        if(uvw.z < minVox.z)
            minVox.z = uvw.z;
        else if(uvw.z > maxVox.z)
            maxVox.z = uvw.z;
    }
}

__global__ void ComputeFaceNormals(const glm::vec3* pVerts,
                                   glm::vec3* pNormals,
                                   size_t numVerts)
{
    glm::uint triIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    glm::uint vertIndex = triIndex * 3u;
    if((vertIndex+2) >= numVerts)
    {
        //printf("CEFNB: vertIndex=%d numVerts=%d\n", vertIndex, numVerts);
        return;
    }    

    glm::vec3 v0 = pVerts[vertIndex];
    glm::vec3 v1 = pVerts[vertIndex+1];
    glm::vec3 v2 = pVerts[vertIndex+2];

    glm::vec3 p0p1 = v1 - v0;
    glm::vec3 p0p2 = v2 - v0;
    pNormals[triIndex] = glm::normalize(glm::cross(p0p1, p0p2));
}

__global__ void ComputeEdgesFaceNormalsAndBounds(const glm::vec3* pVerts,
                                                 size_t numVerts, 
                                                 glm::vec3 p,
                                                 glm::vec3 deltaP,
                                                 glm::uvec3 voxDim,
                                                 glm::vec3* pEdges, 
                                                 glm::vec3* pNormals, 
                                                 cuda::VoxelBBox* pBounds)
{
    glm::uint triIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
    
    glm::uint vertIndex = triIndex * 3u;
    if((vertIndex+2) >= numVerts)
    {
        //printf("CEFNB: vertIndex=%d numVerts=%d\n", vertIndex, numVerts);
        return;
    }    

    glm::vec3 v0 = pVerts[vertIndex];
    glm::vec3 v1 = pVerts[vertIndex+1];
    glm::vec3 v2 = pVerts[vertIndex+2];

    glm::vec3 edges[3] = {
        glm::vec3(v1 - v0),
        glm::vec3(v2 - v1),
        glm::vec3(v0 - v2)
    };

    pEdges[vertIndex] = edges[0];
    pEdges[vertIndex+1] = edges[1];
    pEdges[vertIndex+2] = edges[2];

    const glm::vec3& p0p1 = edges[0];
    glm::vec3 p0p2 = v2 - v0;
    pNormals[triIndex] = glm::normalize(glm::cross(p0p1, p0p2));

    //printf("%d=(%f %f %f)\n", 
    //       triIndex,
    //       pNormals[triIndex].x,
    //       pNormals[triIndex].y,
    //       pNormals[triIndex].z);

    const glm::ivec3 maxVoxIndex = glm::ivec3(voxDim) - glm::ivec3(1);
    const glm::ivec3 minVoxIndex = glm::ivec3(0);
    cuda::VoxelBBox bbox;
    bbox.minVox = glm::uvec3(maxVoxIndex);
    bbox.maxVox = glm::uvec3(minVoxIndex);

    {
        glm::vec3 diff = (v0 - p) / deltaP;

        glm::uvec3 uvw = glm::uvec3(glm::clamp(glm::ivec3(diff),
                                                minVoxIndex, maxVoxIndex));

        bbox.minVox = uvw;
        bbox.maxVox = uvw;
    }

    {
        glm::vec3 diff = (v1 - p) / deltaP;
        //printf("diff1=(%f %f %f)\n",
        //       diff.x,
        //       diff.y,
        //       diff.z);

        glm::uvec3 uvw = glm::uvec3(glm::clamp(glm::ivec3(diff),
                                                minVoxIndex, maxVoxIndex));
        if(uvw.x < bbox.minVox.x)
            bbox.minVox.x = uvw.x;
        else if(uvw.x > bbox.maxVox.x)
            bbox.maxVox.x = uvw.x;

        if(uvw.y < bbox.minVox.y)
            bbox.minVox.y = uvw.y;
        else if(uvw.y > bbox.maxVox.y)
            bbox.maxVox.y = uvw.y;

        if(uvw.z < bbox.minVox.z)
            bbox.minVox.z = uvw.z;
        else if(uvw.z > bbox.maxVox.z)
            bbox.maxVox.z = uvw.z;
    }

    {
        glm::vec3 diff = (v2 - p) / deltaP;
        //printf("diff2=(%f %f %f)\n",
        //       diff.x,
        //       diff.y,
        //       diff.z);

        glm::uvec3 uvw = glm::uvec3(glm::clamp(glm::ivec3(diff),
                                                minVoxIndex, maxVoxIndex));
        if(uvw.x < bbox.minVox.x)
            bbox.minVox.x = uvw.x;
        else if(uvw.x > bbox.maxVox.x)
            bbox.maxVox.x = uvw.x;

        if(uvw.y < bbox.minVox.y)
            bbox.minVox.y = uvw.y;
        else if(uvw.y > bbox.maxVox.y)
            bbox.maxVox.y = uvw.y;

        if(uvw.z < bbox.minVox.z)
            bbox.minVox.z = uvw.z;
        else if(uvw.z > bbox.maxVox.z)
            bbox.maxVox.z = uvw.z;
    }

    //printf("bounds[%d](%d %d %d) - (%d %d %d)\n", 
    //       triIndex, 
    //       bbox.minVox.x,
    //       bbox.minVox.y,
    //       bbox.minVox.z,
    //       bbox.maxVox.x,
    //       bbox.maxVox.y,
    //       bbox.maxVox.z);
    pBounds[triIndex] = bbox;
}

__global__ void ComputeVoxelization(const glm::vec3* pVerts,
                                    size_t triOffset, 
                                    const glm::vec3* pEdges, 
                                    const glm::vec3* pNormals, 
                                    const cuda::VoxelBBox* pBounds,
                                    glm::vec3 p,
                                    glm::vec3 deltaP,
                                    glm::uvec3 minVoxChunk,
                                    glm::uvec3 maxVoxChunk,
                                    glm::uint zOffset,
                                    hipPitchedPtr pdevVoxelTriCounts,
                                    hipPitchedPtr pdevVoxelTriIndices)
{
    //glm::uint triIndex = (blockDim.x * blockIdx.x) + threadIdx.x;
    glm::uint triIndex = blockIdx.z + triOffset;
    
    glm::uint vertIndex = triIndex * 3u;
    
    //if(triIndex == 5000 && threadIdx.x == 0u && threadIdx.y == 0u)
    //    printf("CEFNB: triIndex = %d blockIdx=(%d %d %d) blockDim=(%d %d %d)\n", triIndex, blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z);
    
    glm::uint xIndex = minVoxChunk.x + ((blockIdx.x * blockDim.x) + threadIdx.x);
    glm::uint yIndex = minVoxChunk.y + ((blockIdx.y * blockDim.y) + threadIdx.y);
    glm::uint zIndex = zOffset + threadIdx.z;

    const cuda::VoxelBBox& bounds = pBounds[triIndex];
    glm::uvec3 minVox = glm::max(bounds.minVox, minVoxChunk);
    glm::uvec3 maxVox = glm::min(bounds.maxVox, maxVoxChunk);

    //check if current chunk is outside bounds of current triangle
    if(minVox.x > maxVox.x)
        return;

    if(minVox.y > maxVox.y)
        return;

    if(minVox.z > maxVox.z)
        return;

    //check if my voxel is outside bounds of triangle
    if(xIndex < minVox.x || xIndex > maxVox.x)
        return;

    if(yIndex < minVox.y || yIndex > maxVox.y)
        return;

    if(zIndex < minVox.z || zIndex > maxVox.z)
        return;

    glm::vec3 uvw(0.0f, 0.0f, 0.0f);

    const glm::vec3* pTriVerts = &pVerts[vertIndex];
    const glm::vec3* pTriEdges = &pEdges[vertIndex];
    const glm::vec3& normal = pNormals[triIndex];
    
    //if(threadIdx.z < 3)
    //{
    //    printf("%d %d - %d minVox=[%d %d %d] maxVox=[%d %d %d]\n",
    //           threadIdx.z,
    //           threadZStart,
    //           threadZEnd,
    //           minVox.x,
    //           minVox.y,
    //           minVox.z,
    //           maxVox.x,
    //           maxVox.y,
    //           maxVox.z);
    //}

    //if(zIndex == 141)
    //    printf("Checking tri=%d at voxel=%d %d %d\n", triIndex, xIndex, yIndex, zIndex);

    char* devPtrTriCounts = (char*)pdevVoxelTriCounts.ptr;
    size_t triCountsPitch = pdevVoxelTriCounts.pitch;
    size_t triCountsSlicePitch = triCountsPitch * pdevVoxelTriCounts.ysize;

    char* devPtrTriIndices = (char*)pdevVoxelTriIndices.ptr;
    size_t triIndicesPitch = pdevVoxelTriIndices.pitch;
    size_t triIndicesSlicePitch = triIndicesPitch * pdevVoxelTriIndices.ysize;

    //char* devPtrNormals = (char*)pdevVoxelNormals.ptr;
    //size_t normalsPitch = pdevVoxelNormals.pitch;
    //size_t normalsSlicePitch = normalsPitch * pdevVoxelNormals.ysize;

    //for(glm::uint w = minVox.z; w <= maxVox.z; ++w)
    glm::uint w = zIndex;
    {
        uvw.z = (float)w;
        glm::uint z = (w - minVoxChunk.z);

        char* sliceTriCounts = devPtrTriCounts + z * triCountsSlicePitch;
        char* sliceTriIndices = devPtrTriIndices + z * triIndicesSlicePitch;
        //char* sliceNormals = devPtrNormals + z * normalsSlicePitch;

        //printf("uvw.z=%f\n", uvw.z);

        //for(glm::uint v = minVox.y; v <= maxVox.y; ++v)
        {
            glm::uint v = yIndex;
            uvw.y = (float)v;
            glm::uint y = (v - minVoxChunk.y);
            glm::uint* voxelCounts = (glm::uint*)(sliceTriCounts + y * triCountsPitch);
            glm::lowp_ivec4* voxelTriIndices = (glm::lowp_ivec4*)(sliceTriIndices + y * triIndicesPitch);
            //glm::vec3* voxelNormals = (glm::vec3*)(sliceNormals + y * normalsPitch);
            //for(glm::uint u = minVox.x; u <= maxVox.x; ++u)
            {
                glm::uint u = xIndex;
                glm::uint x = (u - minVoxChunk.x);
                uvw.x = (float)u;

                /*bool debug = (triIndex == 8 &&
                       (x == 179 || x == 180) &&
                       y == 158 && 
                       (z >= 221 && z <= 225));*/
                /*if(triIndex == 8 &&
                       (x == 179 || x == 180) &&
                       y == 158 && 
                       (z >= 221 && z <= 225))
                {
                    printf("[%d %d %d] tri=%d v0=[%f %f %f] normal=[%f %f %f] uvw=[%f %f %f]\n",
                            x, y, z, triIndex,
                            pTriVerts[0].x, pTriVerts[0].y, pTriVerts[0].z,
                            normal.x, normal.y, normal.z,
                            uvw.x, uvw.y, uvw.z);
                }*/

                //voxelColors[x] = cuda::VoxColor(uvw, 1.0f);
                //voxelNormals[x] = glm::vec3(uvw);
                if(TrianglePlaneOverlapsVoxel(pTriVerts[0], normal, p, deltaP, uvw))//, debug))
                {
                    /*if(triIndex == 8 &&
                       (x == 179 || x == 180) &&
                       y == 158 && 
                       (z >= 221 && z <= 225))
                    {
                        printf("[%d %d %d] tri=%d passed plane overlap check\n",
                               x, y, z, triIndex);
                    }*/
                    if(!TriangleOverlapsVoxel(pTriVerts, 
                                              pTriEdges, 
                                              normal, 
                                              p, deltaP, uvw))
                    {
                        //continue;
                        return;
                    }
                }
                else
                {
                    //continue;
                    return;
                }

                //printf("[%d %d %d] tri=%d found overlap\n",
                //       x, y, z, triIndex);
                
                atomicAdd(&voxelCounts[x], 1u);
                
                if(voxelTriIndices[x][0] == -1)
                    voxelTriIndices[x][0] = triIndex;
                else if(voxelTriIndices[x][1] == -1)
                    voxelTriIndices[x][1] = triIndex;
                else if(voxelTriIndices[x][2] == -1)
                    voxelTriIndices[x][2] = triIndex;
                else
                    voxelTriIndices[x][3] = triIndex;

                //TODO get color from texture and mix rather than just set
                /*cuda::VoxColor color = cuda::VoxColor(std::abs(normal.x), std::abs(normal.y), std::abs(normal.z), 1.0f);
                atomicAdd(&voxelColors[x].x, color.x);
                atomicAdd(&voxelColors[x].y, color.y);
                atomicAdd(&voxelColors[x].z, color.z);
                voxelColors[x].w = color.w;
                */
                //atomicAdd(&voxelNormals[x].x, normal.x),
                //atomicAdd(&voxelNormals[x].y, normal.y),
                //atomicAdd(&voxelNormals[x].z, normal.z);
                //in next step we'll average and normalize
            }
        }
    }
}

//__global__ void ComputeVoxelizationAverages(hipPitchedPtr pdevVoxelTriCounts,
//                                            hipPitchedPtr pdevVoxelColors,
//                                            glm::uvec3 voxDim)
//{
//    glm::uint xIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
//    if(xIndex >= voxDim.x)
//        return;
//    
//    glm::uint yIndex = (blockIdx.y * blockDim.y) + threadIdx.y;
//    if(yIndex >= voxDim.y)
//        return;
//
//    glm::uint zIndex = (blockIdx.z * blockDim.z) + threadIdx.z;
//    if(zIndex >= voxDim.z)
//        return;
//
//    char* devPtrTriCounts = (char*)pdevVoxelTriCounts.ptr;
//    size_t triCountsPitch = pdevVoxelTriCounts.pitch;
//    size_t triCountsSlicePitch = triCountsPitch * pdevVoxelTriCounts.ysize;
//
//    char* sliceTriCounts = devPtrTriCounts + zIndex * triCountsSlicePitch;
//
//    glm::uint* voxelCounts = (glm::uint*)(sliceTriCounts + yIndex * triCountsPitch);
//
//    glm::uint triCount = voxelCounts[xIndex];
//    if(triCount <= 1u)
//        return;
//
//    char* devPtrColors = (char*)pdevVoxelColors.ptr;
//    size_t colorsPitch = pdevVoxelColors.pitch;
//    size_t colorsSlicePitch = colorsPitch * pdevVoxelColors.ysize;
//
//    //char* devPtrNormals = (char*)pdevVoxelNormals.ptr;
//    //size_t normalsPitch = pdevVoxelNormals.pitch;
//    //size_t normalsSlicePitch = normalsPitch * pdevVoxelNormals.ysize;
//
//    char* sliceColors = devPtrColors + zIndex * colorsSlicePitch;
//    //char* sliceNormals = devPtrNormals + zIndex * normalsSlicePitch;
//
//    cuda::VoxColor* voxelColors = (cuda::VoxColor*)(sliceColors + yIndex * colorsPitch);
//    //glm::vec3* voxelNormals = (glm::vec3*)(sliceNormals + yIndex * normalsPitch);
//
//    float oneOverTriCount = 1.0f / static_cast<float>(triCount);
//
//    voxelColors[xIndex].x *= oneOverTriCount;
//    voxelColors[xIndex].y *= oneOverTriCount;
//    voxelColors[xIndex].z *= oneOverTriCount;
//    //leave the alpha component alone
//    //glm::vec3 normal = voxelNormals[xIndex];
//
//    //normal *= oneOverTriCount;
//    //voxelNormals[xIndex] = glm::normalize(normal);
//}

texture<uchar4, hipTextureType2D, hipReadModeNormalizedFloat> colorTex;

bool BindTextureToArray(hipArray* pgImageArray,
                        hipChannelFormatDesc imageDesc,
                        hipTextureAddressMode addressMode0,
                        hipTextureAddressMode addressMode1)
{
    colorTex.addressMode[0] = addressMode0;
    colorTex.addressMode[1] = addressMode1;
    colorTex.filterMode = hipFilterModeLinear;
    colorTex.normalized = true;    // access with normalized texture coordinates

    // Bind the array to the texture
    return hipBindTextureToArray(&colorTex, pgImageArray, &imageDesc) == hipSuccess;
}

__device__ glm::vec2 ComputeVoxelBarycentricCoords(glm::vec2 P,
                                           glm::vec2 A,
                                           glm::vec2 B,
                                           glm::vec2 C)
{
    // Compute vectors        
    glm::vec2 v0 = C - A;
    glm::vec2 v1 = B - A;
    glm::vec2 v2 = P - A;

    // Compute dot products
    float dot00 = glm::dot(v0, v0);
    float dot01 = glm::dot(v0, v1);
    float dot02 = glm::dot(v0, v2);
    float dot11 = glm::dot(v1, v1);
    float dot12 = glm::dot(v1, v2);

    // Compute barycentric coordinates
    float invDenom = 1.0f / (dot00 * dot11 - dot01 * dot01);
    glm::vec2 uv(glm::clamp((dot11 * dot02 - dot01 * dot12) * invDenom,
                            0.0f, 1.0f),
                 glm::clamp((dot00 * dot12 - dot01 * dot02) * invDenom,
                            0.0f, 1.0f));

    return uv;
}

__device__ glm::vec2 ComputeVoxelTexCoords(glm::vec2 uv,
                                glm::vec2 uvA, 
                                glm::vec2 uvB, 
                                glm::vec2 uvC)//,
                                //bool debug=false)
{

    glm::vec2 V = (uvB - uvA) * uv.y;
    glm::vec2 U = (uvC - uvA) * uv.x;
    glm::vec2 interpUV = uvA + U + V;
    /*if(debug)
    {
        printf("u=%f v=%f U=[%f %f] V=[%f %f]\n", uv.x, uv.y, U.x, U.y, V.x, V.y);
    }*/
    return interpUV;
}

__device__ glm::vec3 ComputeVoxelNormal(glm::vec2 uv,
                             glm::vec3 nA, 
                             glm::vec3 nB, 
                             glm::vec3 nC)//,
                             //bool debug=false)
{
    glm::vec3 V = (nB - nA) * uv.y;
    glm::vec3 U = (nC - nA) * uv.x;
    glm::vec3 interpNorm = nA + U + V;
    /*if(debug)
    {
        printf("u=%f v=%f U=[%f %f %f] V=[%f %f %f]\n", 
               uv.x, uv.y, U.x, U.y, U.z, V.x, V.y, V.z);
    }*/
    return interpNorm;
}

__device__ void VoxColorToVec4(glm::vec4* pVec4, glm::vec4 color)
{
    *pVec4 = color;
}

__device__ void VoxColorToVec4(glm::vec4* pVec4, uchar4 color)
{
    pVec4->x = static_cast<float>(color.x) / 255.0f;
    pVec4->y = static_cast<float>(color.y) / 255.0f;
    pVec4->z = static_cast<float>(color.z) / 255.0f;
    pVec4->w = static_cast<float>(color.w) / 255.0f;
}

__device__ void Vec4ToVoxColor(glm::vec4* pVoxColor, glm::vec4 vec4Color)
{
    *pVoxColor = vec4Color;
}

__device__ void Vec4ToVoxColor(uchar4* pVoxColor, glm::vec4 vec4Color)
{
    pVoxColor->x = static_cast<unsigned char>(vec4Color.x * 255.0f);
    pVoxColor->y = static_cast<unsigned char>(vec4Color.y * 255.0f);
    pVoxColor->z = static_cast<unsigned char>(vec4Color.z * 255.0f);
    pVoxColor->w = static_cast<unsigned char>(vec4Color.w * 255.0f);
}

__global__ void ComputeColorsAndNormals(size_t triGrpOffset,
                                        const glm::vec3* pVerts,
                                        const glm::vec3* pVtxNormals,
                                        const glm::vec3* pFaceNormals,
                                        const glm::vec2* pUVs,
                                        bool isTerrain,
                                        glm::vec3 voxOrigin,
                                        glm::vec3 voxSizeMeters,
                                        hipPitchedPtr voxelTriCountsDevPtr,
                                        hipPitchedPtr voxelTriIndicesDevPtr,
                                        hipPitchedPtr voxelColorsDevMipMapPtr,
                                        hipPitchedPtr voxelNormalsDevMipMapPtr,
                                        glm::uvec3 voxWriteDim)
{
    glm::uint xIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(xIndex >= voxWriteDim.x)
        return;
    
    glm::uint yIndex = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(yIndex >= voxWriteDim.y)
        return;

    glm::uint zIndex = (blockIdx.z * blockDim.z) + threadIdx.z;
    if(zIndex >= voxWriteDim.z)
        return;

    glm::vec3 voxCenter = voxOrigin
                        + glm::vec3(xIndex * voxSizeMeters.x,
                                    yIndex * voxSizeMeters.y,
                                    zIndex * voxSizeMeters.z)
                        + (voxSizeMeters * 0.5f);

    glm::lowp_ivec4* voxelReadTriIndices = NULL;
    glm::uint* voxelReadTriCounts = NULL;
    {
        char* devReadPtrTriIndices = (char*)voxelTriIndicesDevPtr.ptr;
        size_t triIndicesPitch = voxelTriIndicesDevPtr.pitch;
        size_t triIndicesSlicePitch = triIndicesPitch * voxelTriIndicesDevPtr.ysize;

        char* devReadPtrTriCounts = (char*)voxelTriCountsDevPtr.ptr;
        size_t normalsPitch = voxelTriCountsDevPtr.pitch;
        size_t normalsSlicePitch = normalsPitch * voxelTriCountsDevPtr.ysize;

        char* sliceReadTriIndices = devReadPtrTriIndices + zIndex * triIndicesSlicePitch;
        char* sliceReadTriCounts = devReadPtrTriCounts + zIndex * normalsSlicePitch;

        voxelReadTriIndices = (glm::lowp_ivec4*)(sliceReadTriIndices + yIndex * triIndicesPitch);
        voxelReadTriCounts = (glm::uint*)(sliceReadTriCounts + yIndex * normalsPitch);
    }

    cuda::VoxColor* voxelWriteColors = NULL;
    cuda::VoxNorm* voxelWriteNormals = NULL;

    {
        char* devWritePtrColors = (char*)voxelColorsDevMipMapPtr.ptr;
        size_t colorsPitch = voxelColorsDevMipMapPtr.pitch;
        size_t colorsSlicePitch = colorsPitch * voxelColorsDevMipMapPtr.ysize;

        char* devWritePtrNormals = (char*)voxelNormalsDevMipMapPtr.ptr;
        size_t normalsPitch = voxelNormalsDevMipMapPtr.pitch;
        size_t normalsSlicePitch = normalsPitch * voxelNormalsDevMipMapPtr.ysize;

        char* sliceWriteColors = devWritePtrColors + zIndex * colorsSlicePitch;
        char* sliceWriteNormals = devWritePtrNormals + zIndex * normalsSlicePitch;

        voxelWriteColors = (cuda::VoxColor*)(sliceWriteColors + yIndex * colorsPitch);
        voxelWriteNormals = (cuda::VoxNorm*)(sliceWriteNormals + yIndex * normalsPitch);
    }

    glm::vec4 color;
    VoxColorToVec4(&color, voxelWriteColors[xIndex]);

    cuda::VoxNorm normal = voxelWriteNormals[xIndex];
    glm::lowp_ivec4& triIndices = voxelReadTriIndices[xIndex];
    
    //if first item in triIndices is negative
    //then no new triangle intersections were found
    bool newTriIsects = (triIndices[0] >= 0);
    if(newTriIsects)
    {
        glm::uint triCount = glm::min(voxelReadTriCounts[xIndex], 4u);
        //reset this to one because after this execution the
        //stored color is worth one in the average
        voxelReadTriCounts[xIndex] = 1u;
        for(glm::uint i = 0u; i < 4u && triIndices[i] != -1; ++i)
        {
            short int triIndex = triIndices[i];
            triIndices[i] = -1;//reset this to -1 for next loop
            short int vertBaseIndex = triIndex * 3;
            const glm::vec3& v1 = pVerts[vertBaseIndex];
            const glm::vec3& v2 = pVerts[vertBaseIndex+1];
            const glm::vec3& v3 = pVerts[vertBaseIndex+2];

            short int triGrpTriIndex = (triIndex - triGrpOffset);
            short int triGrpBaseVtxIndex = triGrpTriIndex * 3;
            const glm::vec2& uv1 = pUVs[triGrpBaseVtxIndex];
            const glm::vec2& uv2 = pUVs[triGrpBaseVtxIndex+1];
            const glm::vec2& uv3 = pUVs[triGrpBaseVtxIndex+2];
            glm::vec2 bcCoords;
            //find dominant axis
            const glm::vec3& faceNormal = pFaceNormals[triIndex];
            float absFaceNormalX = std::abs(faceNormal.x);
            float absFaceNormalY = std::abs(faceNormal.y);
            float absFaceNormalZ = std::abs(faceNormal.z);
            //glm::vec3 absFaceNormal = glm::abs(faceNormal);
            if(absFaceNormalX > absFaceNormalY && absFaceNormalX > absFaceNormalZ)
            {
               bcCoords = ComputeVoxelBarycentricCoords(glm::vec2(voxCenter.z, voxCenter.y),
                                                 glm::vec2(v1.z, v1.y),
                                                 glm::vec2(v2.z, v2.y),
                                                 glm::vec2(v3.z, v3.y));
            }
            else if(absFaceNormalY > absFaceNormalZ)
            { 
               bcCoords = ComputeVoxelBarycentricCoords(glm::vec2(voxCenter.x, voxCenter.z),
                                                 glm::vec2(v1.x, v1.z),
                                                 glm::vec2(v2.x, v2.z),
                                                 glm::vec2(v3.x, v3.z));
            }
            else
            {
                bcCoords = ComputeVoxelBarycentricCoords(glm::vec2(voxCenter.x, voxCenter.y),
                                                         glm::vec2(v1.x, v1.y),
                                                         glm::vec2(v2.x, v2.y),
                                                         glm::vec2(v3.x, v3.y));
            }

            /*bool debug = false;
            if(xIndex == 25 && yIndex == 25)
                debug = true;*/
            if(pVtxNormals != nullptr)
            {
                const glm::vec3& n1 = pVtxNormals[triGrpBaseVtxIndex];
                const glm::vec3& n2 = pVtxNormals[triGrpBaseVtxIndex+1];
                const glm::vec3& n3 = pVtxNormals[triGrpBaseVtxIndex+2];
                normal += ComputeVoxelNormal(bcCoords,
                                            n1, n2, n3);//,
                                            //debug);
            }
            else
                normal += faceNormal;

            glm::vec2 texCoords = ComputeVoxelTexCoords(bcCoords,
                                                        uv1, uv2, uv3);//,
                                                        //debug);
            
            float4 texel = tex2D(colorTex, 
                                 static_cast<float>(texCoords.x), 
                                 static_cast<float>(texCoords.y));
            //don't apply completely translucent texels to the voxel
            if(texel.w <= 0.005)
                --triCount;
            else
            {
                texel.x *= texel.w;
                texel.y *= texel.w;
                texel.z *= texel.w;
                color.x += texel.x;
                color.y += texel.y;
                color.z += texel.z;
                color.w += texel.w;
            }
            //if((xIndex == 25 && yIndex == 25) || triCount > 2)
            //{
            //    printf("normal=[%f %f %f]\n", normal.x, normal.y, normal.z);
            //    printf("voxCenter=[%f %f %f]\n", voxCenter.x, voxCenter.y, voxCenter.z);
            //    printf("triVerts=[%f %f %f][%f %f %f][%f %f %f]\n", 
            //            v1.x, v1.y, v1.z,
            //            v2.x, v2.y, v2.z,
            //            v3.x, v3.y, v3.z);
            //    printf("uvs=[%f %f][%f %f][%f %f]\n", 
            //            uv1.x, uv1.y,
            //            uv2.x, uv2.y,
            //            uv3.x, uv3.y);
            //    printf("voxel=[%d %d %d] triIndices[%d]=[%d] texCoords=[%f %f] = texel=[%f %f %f %f] ... color=[%f %f %f %f] / triCount=%d\n", 
            //        xIndex, yIndex, zIndex, i,
            //        triIndex,
            //        texCoords.x, texCoords.y,
            //        texel.x, texel.y, texel.z, texel.w,
            //        color.x, color.y, color.z, color.w,
            //        triCount);
            //}
        }

        if(triCount > 0u)
        {
            color.x /= triCount;
            color.y /= triCount;
            color.z /= triCount;
            color.w = glm::min(color.w, 1.0f);//additive alpha blending
            normal.x /= triCount;
            normal.y /= triCount;
            normal.z /= triCount;
            normal = glm::normalize(normal);
        }
    }
    
    Vec4ToVoxColor(&voxelWriteColors[xIndex], color);
    voxelWriteNormals[xIndex] = normal;
    if(newTriIsects && isTerrain)
    {
        //fill in voxels under ground
        for(glm::uint groundZ = 1; groundZ < 20u; ++groundZ)
        {
            if(zIndex == 0u)
                break;
            --zIndex;

            char* devWritePtrColors = (char*)voxelColorsDevMipMapPtr.ptr;
            size_t colorsPitch = voxelColorsDevMipMapPtr.pitch;
            size_t colorsSlicePitch = colorsPitch * voxelColorsDevMipMapPtr.ysize;

            char* devWritePtrNormals = (char*)voxelNormalsDevMipMapPtr.ptr;
            size_t normalsPitch = voxelNormalsDevMipMapPtr.pitch;
            size_t normalsSlicePitch = normalsPitch * voxelNormalsDevMipMapPtr.ysize;

            char* sliceWriteColors = devWritePtrColors + zIndex * colorsSlicePitch;
            char* sliceWriteNormals = devWritePtrNormals + zIndex * normalsSlicePitch;

            voxelWriteColors = (cuda::VoxColor*)(sliceWriteColors + yIndex * colorsPitch);
            voxelWriteNormals = (cuda::VoxNorm*)(sliceWriteNormals + yIndex * normalsPitch);
            //if((xIndex == 25 && yIndex == 25))
            //{
            //    printf("Filling ground %d\n", zIndex);
            //}
            Vec4ToVoxColor(&voxelWriteColors[xIndex], color);
            voxelWriteNormals[xIndex] = normal;
        }
    }
}

__global__ void ComputeColorsAsNormals(size_t triGrpOffset,
                                       const glm::vec3* pVerts,
                                       const glm::vec3* pVtxNormals,
                                       const glm::vec3* pFaceNormals,
                                       glm::vec3 voxOrigin,
                                       glm::vec3 voxSizeMeters,
                                       hipPitchedPtr voxelTriCountsDevPtr,
                                       hipPitchedPtr voxelTriIndicesDevPtr,
                                       hipPitchedPtr voxelColorsDevMipMapPtr,
                                       hipPitchedPtr voxelNormalsDevMipMapPtr,
                                       glm::uvec3 voxWriteDim)
{
    glm::uint xIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(xIndex >= voxWriteDim.x)
        return;
    
    glm::uint yIndex = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(yIndex >= voxWriteDim.y)
        return;

    glm::uint zIndex = (blockIdx.z * blockDim.z) + threadIdx.z;
    if(zIndex >= voxWriteDim.z)
        return;

    glm::vec3 voxCenter = voxOrigin
                        + glm::vec3(xIndex * voxSizeMeters.x,
                                    yIndex * voxSizeMeters.y,
                                    zIndex * voxSizeMeters.z)
                        + (voxSizeMeters * 0.5f);

    glm::lowp_ivec4* voxelReadTriIndices = NULL;
    glm::uint* voxelReadTriCounts = NULL;
    {
        char* devReadPtrTriIndices = (char*)voxelTriIndicesDevPtr.ptr;
        size_t triIndicesPitch = voxelTriIndicesDevPtr.pitch;
        size_t triIndicesSlicePitch = triIndicesPitch * voxelTriIndicesDevPtr.ysize;

        char* devReadPtrTriCounts = (char*)voxelTriCountsDevPtr.ptr;
        size_t normalsPitch = voxelTriCountsDevPtr.pitch;
        size_t normalsSlicePitch = normalsPitch * voxelTriCountsDevPtr.ysize;

        char* sliceReadTriIndices = devReadPtrTriIndices + zIndex * triIndicesSlicePitch;
        char* sliceReadTriCounts = devReadPtrTriCounts + zIndex * normalsSlicePitch;

        voxelReadTriIndices = (glm::lowp_ivec4*)(sliceReadTriIndices + yIndex * triIndicesPitch);
        voxelReadTriCounts = (glm::uint*)(sliceReadTriCounts + yIndex * normalsPitch);
    }

    cuda::VoxColor* voxelWriteColors = NULL;
    cuda::VoxNorm* voxelWriteNormals = NULL;

    {
        char* devWritePtrColors = (char*)voxelColorsDevMipMapPtr.ptr;
        size_t colorsPitch = voxelColorsDevMipMapPtr.pitch;
        size_t colorsSlicePitch = colorsPitch * voxelColorsDevMipMapPtr.ysize;

        char* devWritePtrNormals = (char*)voxelNormalsDevMipMapPtr.ptr;
        size_t normalsPitch = voxelNormalsDevMipMapPtr.pitch;
        size_t normalsSlicePitch = normalsPitch * voxelNormalsDevMipMapPtr.ysize;

        char* sliceWriteColors = devWritePtrColors + zIndex * colorsSlicePitch;
        char* sliceWriteNormals = devWritePtrNormals + zIndex * normalsSlicePitch;

        voxelWriteColors = (cuda::VoxColor*)(sliceWriteColors + yIndex * colorsPitch);
        voxelWriteNormals = (cuda::VoxNorm*)(sliceWriteNormals + yIndex * normalsPitch);
    }

    glm::vec4 color;
    VoxColorToVec4(&color, voxelWriteColors[xIndex]);

    cuda::VoxNorm normal = voxelWriteNormals[xIndex];
    glm::lowp_ivec4& triIndices = voxelReadTriIndices[xIndex];
    
    //if first item in triIndices is negative
    //then no new triangle intersections were found
    if(triIndices[0] >= 0)
    {
        glm::uint triCount = voxelReadTriCounts[xIndex];
        //reset this to one because after this execution the
        //stored color is worth one in the average
        voxelReadTriCounts[xIndex] = 1u;
        for(glm::uint i = 0u; i < 4u && triIndices[i] != -1; ++i)
        {
            short int triIndex = triIndices[i];
            triIndices[i] = -1;//reset this to -1 for next loop
            short int vertBaseIndex = triIndex * 3;
            const glm::vec3& v1 = pVerts[vertBaseIndex];
            const glm::vec3& v2 = pVerts[vertBaseIndex+1];
            const glm::vec3& v3 = pVerts[vertBaseIndex+2];
            
            const glm::vec3& faceNormal = pFaceNormals[triIndex];
            if(pVtxNormals != nullptr)
            {
                short int triGrpTriIndex = (triIndex - triGrpOffset);
                short int triGrpBaseVtxIndex = triGrpTriIndex * 3;
                const glm::vec3& n1 = pVtxNormals[triGrpBaseVtxIndex];
                const glm::vec3& n2 = pVtxNormals[triGrpBaseVtxIndex+1];
                const glm::vec3& n3 = pVtxNormals[triGrpBaseVtxIndex+2];
                //this will be used to determine which plane
                //of coordinate system to compute barycentric
                //coordinates
                glm::vec2 bcCoords;
                //find dominant axis
                if(faceNormal.x > faceNormal.y && faceNormal.x > faceNormal.z)
                {
                   bcCoords = ComputeVoxelBarycentricCoords(glm::vec2(voxCenter.z, voxCenter.y),
                                                     glm::vec2(v1.z, v1.y),
                                                     glm::vec2(v2.z, v2.y),
                                                     glm::vec2(v3.z, v3.y));
                }
                else if(faceNormal.y > faceNormal.z)
                { 
                   bcCoords = ComputeVoxelBarycentricCoords(glm::vec2(voxCenter.x, voxCenter.z),
                                                     glm::vec2(v1.x, v1.z),
                                                     glm::vec2(v2.x, v2.z),
                                                     glm::vec2(v3.x, v3.z));
                }
                else
                {
                    bcCoords = ComputeVoxelBarycentricCoords(glm::vec2(voxCenter.x, voxCenter.y),
                                                             glm::vec2(v1.x, v1.y),
                                                             glm::vec2(v2.x, v2.y),
                                                             glm::vec2(v3.x, v3.y));
                }
                color += glm::vec4(ComputeVoxelNormal(bcCoords, n1, n2, n3), 1.0);
            }
            else
            {
                color += glm::vec4(faceNormal, 1.0f);
            }

            //if((xIndex > 25 && xIndex < 35) || triCount > 2)
            //{
            //    printf("voxCenter=[%f %f %f]\n", voxCenter.x, voxCenter.y, voxCenter.z);
            //    printf("triVerts=[%f %f %f][%f %f %f][%f %f %f]\n", 
            //            v1.x, v1.y, v1.z,
            //            v2.x, v2.y, v2.z,
            //            v3.x, v3.y, v3.z);
            //    printf("voxel=[%d %d %d] triIndices[%d]=[%d] color=[%f %f %f %f] / triCount=%d\n", 
            //        xIndex, yIndex, zIndex, i,
            //        triIndex,
            //        color.x, color.y, color.z, color.w,
            //        triCount);
            //}
        }
        
        color.x /= triCount;
        color.y /= triCount;
        color.z /= triCount;
        color.w = glm::min(color.w, 1.0f);//additive alpha blending
        
        normal = glm::normalize(glm::vec3(color));

        if(color.x < 0.0f)
            color.x *= -1.0f;
        if(color.y < 0.0f)
            color.y *= -1.0f;
        if(color.z < 0.0f)
            color.z *= -1.0f;
    }
    
    Vec4ToVoxColor(&voxelWriteColors[xIndex], color);
    voxelWriteNormals[xIndex] = normal;
}

__device__ float VectorLenSq(glm::vec3 vec)
{
     return vec.x * vec.x + vec.y * vec.y + vec.z * vec.z;
}

__global__ void ComputeVoxelMipMap64(
                        glm::uint voxInputDimX, glm::uint voxInputDimY, glm::uint voxInputDimZ,//input size
                        hipPitchedPtr inputVoxelColors,//input colors
                        hipPitchedPtr inputVoxelNormals,//input normals
                        glm::uint voxDimX, glm::uint voxDimY, glm::uint voxDimZ,//output size
                        hipPitchedPtr mipMapColors,//output colors
                        hipPitchedPtr mipMapNormals)//output normals
{
    glm::uint xIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(xIndex >= voxDimX)
        return;
    
    glm::uint yIndex = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(yIndex >= voxDimY)
        return;

    glm::uint zIndex = (blockIdx.z * blockDim.z) + threadIdx.z;
    if(zIndex >= voxDimZ)
        return;

    glm::uint mmBaseX = xIndex * 4;
    glm::uint mmXPlus1 = glm::min(mmBaseX + 1, voxInputDimX-1);
    glm::uint mmXPlus2 = glm::min(mmBaseX + 2, voxInputDimX-1);
    glm::uint mmXPlus3 = glm::min(mmBaseX + 3, voxInputDimX-1);
    glm::uint mmBaseY = yIndex * 4;
    glm::uint mmYPlus1 = glm::min(mmBaseY + 1, voxInputDimY-1);
    glm::uint mmYPlus2 = glm::min(mmBaseY + 2, voxInputDimY-1);
    glm::uint mmYPlus3 = glm::min(mmBaseY + 3, voxInputDimY-1);
    glm::uint mmBaseZ = zIndex * 4;
    glm::uint mmZPlus1 = glm::min(mmBaseZ + 1, voxInputDimZ-1);
    glm::uint mmZPlus2 = glm::min(mmBaseZ + 2, voxInputDimZ-1);
    glm::uint mmZPlus3 = glm::min(mmBaseZ + 3, voxInputDimZ-1);

    char* pVoxels = (char*)inputVoxelColors.ptr;
    size_t pitch = inputVoxelColors.pitch;
    size_t slicePitch = inputVoxelColors.ysize * pitch;

    glm::uint mmBaseZOffset = mmBaseZ * slicePitch;
    glm::uint mmZPlus1Offset = mmZPlus1 * slicePitch;
    glm::uint mmZPlus2Offset = mmZPlus2 * slicePitch;
    glm::uint mmZPlus3Offset = mmZPlus3 * slicePitch;

    glm::uint mmBaseYOffset = mmBaseY * pitch;
    glm::uint mmYPlus1Offset = mmYPlus1 * pitch;
    glm::uint mmYPlus2Offset = mmYPlus2 * pitch;
    glm::uint mmYPlus3Offset = mmYPlus3 * pitch;

    cuda::VoxColor colorBox[64] = {
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmBaseX],//x, y, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus1],//x+1, y, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus2],//x+2, y, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus3],//x+3, y, z

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmBaseX],//x, y+1, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmBaseX],//x, y+2, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmBaseX],//x, y+3, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmXPlus3],//x+3, y+3, z

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmBaseX],//x, y, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus2],//x+2, y, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus3],//x+3, y, z+1

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z+1

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmBaseX],//x, y+2, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z+1

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmBaseX],//x, y+3, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmXPlus3],//x+3, y+3, z+1

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmBaseX],//x, y, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmXPlus2],//x+2, y, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmXPlus3],//x+3, y, z+2

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z+2

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmBaseX],//x, y+2, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z+2

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmBaseX],//x, y+3, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z+2
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmXPlus3],//x+3, y+3, z+2

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmBaseX],//x, y, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmXPlus2],//x+2, y, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmXPlus3],//x+3, y, z+3

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z+3

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmBaseX],//x, y+2, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z+3

        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmBaseX],//x, y+3, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z+3
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmXPlus3]//x+3, y+3, z+3
    };

    float alphaSum = colorBox[0].w;
    alphaSum += colorBox[1].w;
    alphaSum += colorBox[2].w;
    alphaSum += colorBox[3].w;
    alphaSum += colorBox[4].w;
    alphaSum += colorBox[5].w;
    alphaSum += colorBox[6].w;
    alphaSum += colorBox[7].w;
    alphaSum += colorBox[8].w;
    alphaSum += colorBox[9].w;
    alphaSum += colorBox[10].w;
    alphaSum += colorBox[11].w;
    alphaSum += colorBox[12].w;
    alphaSum += colorBox[13].w;
    alphaSum += colorBox[14].w;
    alphaSum += colorBox[15].w;
    alphaSum += colorBox[16].w;
    alphaSum += colorBox[17].w;
    alphaSum += colorBox[18].w;
    alphaSum += colorBox[19].w;
    alphaSum += colorBox[20].w;
    alphaSum += colorBox[21].w;
    alphaSum += colorBox[22].w;
    alphaSum += colorBox[23].w;
    alphaSum += colorBox[24].w;
    alphaSum += colorBox[25].w;
    alphaSum += colorBox[26].w;
    alphaSum += colorBox[27].w;
    alphaSum += colorBox[28].w;
    alphaSum += colorBox[29].w;
    alphaSum += colorBox[30].w;
    alphaSum += colorBox[31].w;
    alphaSum += colorBox[32].w;
    alphaSum += colorBox[33].w;
    alphaSum += colorBox[34].w;
    alphaSum += colorBox[35].w;
    alphaSum += colorBox[36].w;
    alphaSum += colorBox[37].w;
    alphaSum += colorBox[38].w;
    alphaSum += colorBox[39].w;
    alphaSum += colorBox[40].w;
    alphaSum += colorBox[41].w;
    alphaSum += colorBox[42].w;
    alphaSum += colorBox[43].w;
    alphaSum += colorBox[44].w;
    alphaSum += colorBox[45].w;
    alphaSum += colorBox[46].w;
    alphaSum += colorBox[47].w;
    alphaSum += colorBox[48].w;
    alphaSum += colorBox[49].w;
    alphaSum += colorBox[50].w;
    alphaSum += colorBox[51].w;
    alphaSum += colorBox[52].w;
    alphaSum += colorBox[53].w;
    alphaSum += colorBox[54].w;
    alphaSum += colorBox[55].w;
    alphaSum += colorBox[56].w;
    alphaSum += colorBox[57].w;
    alphaSum += colorBox[58].w;
    alphaSum += colorBox[59].w;
    alphaSum += colorBox[60].w;
    alphaSum += colorBox[61].w;
    alphaSum += colorBox[62].w;
    alphaSum += colorBox[63].w;
    
    float alphaWeights[64] = { 
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
        0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f
    };

    if(alphaSum > 0.0f)
    {
        alphaWeights[0] = colorBox[0].w / alphaSum;
        alphaWeights[1] = colorBox[1].w / alphaSum;
        alphaWeights[2] = colorBox[2].w / alphaSum;
        alphaWeights[3] = colorBox[3].w / alphaSum;
        alphaWeights[4] = colorBox[4].w / alphaSum;
        alphaWeights[5] = colorBox[5].w / alphaSum;
        alphaWeights[6] = colorBox[6].w / alphaSum;
        alphaWeights[7] = colorBox[7].w / alphaSum;
        alphaWeights[8] = colorBox[8].w / alphaSum;
        alphaWeights[9] = colorBox[9].w / alphaSum;

        alphaWeights[10] = colorBox[10].w / alphaSum;
        alphaWeights[11] = colorBox[11].w / alphaSum;
        alphaWeights[12] = colorBox[12].w / alphaSum;
        alphaWeights[13] = colorBox[13].w / alphaSum;
        alphaWeights[14] = colorBox[14].w / alphaSum;
        alphaWeights[15] = colorBox[15].w / alphaSum;
        alphaWeights[16] = colorBox[16].w / alphaSum;
        alphaWeights[17] = colorBox[17].w / alphaSum;
        alphaWeights[18] = colorBox[18].w / alphaSum;
        alphaWeights[19] = colorBox[19].w / alphaSum;
        
        alphaWeights[20] = colorBox[20].w / alphaSum;
        alphaWeights[21] = colorBox[21].w / alphaSum;
        alphaWeights[22] = colorBox[22].w / alphaSum;
        alphaWeights[23] = colorBox[23].w / alphaSum;
        alphaWeights[24] = colorBox[24].w / alphaSum;
        alphaWeights[25] = colorBox[25].w / alphaSum;
        alphaWeights[26] = colorBox[26].w / alphaSum;
        alphaWeights[27] = colorBox[27].w / alphaSum;
        alphaWeights[28] = colorBox[28].w / alphaSum;
        alphaWeights[29] = colorBox[29].w / alphaSum;
        
        alphaWeights[30] = colorBox[30].w / alphaSum;
        alphaWeights[31] = colorBox[31].w / alphaSum;
        alphaWeights[32] = colorBox[32].w / alphaSum;
        alphaWeights[33] = colorBox[33].w / alphaSum;
        alphaWeights[34] = colorBox[34].w / alphaSum;
        alphaWeights[35] = colorBox[35].w / alphaSum;
        alphaWeights[36] = colorBox[36].w / alphaSum;
        alphaWeights[37] = colorBox[37].w / alphaSum;
        alphaWeights[38] = colorBox[38].w / alphaSum;
        alphaWeights[39] = colorBox[39].w / alphaSum;
        
        alphaWeights[40] = colorBox[40].w / alphaSum;
        alphaWeights[41] = colorBox[41].w / alphaSum;
        alphaWeights[42] = colorBox[42].w / alphaSum;
        alphaWeights[43] = colorBox[43].w / alphaSum;
        alphaWeights[44] = colorBox[44].w / alphaSum;
        alphaWeights[45] = colorBox[45].w / alphaSum;
        alphaWeights[46] = colorBox[46].w / alphaSum;
        alphaWeights[47] = colorBox[47].w / alphaSum;
        alphaWeights[48] = colorBox[48].w / alphaSum;
        alphaWeights[49] = colorBox[49].w / alphaSum;
        
        alphaWeights[50] = colorBox[50].w / alphaSum;
        alphaWeights[51] = colorBox[51].w / alphaSum;
        alphaWeights[52] = colorBox[52].w / alphaSum;
        alphaWeights[53] = colorBox[53].w / alphaSum;
        alphaWeights[54] = colorBox[54].w / alphaSum;
        alphaWeights[55] = colorBox[55].w / alphaSum;
        alphaWeights[56] = colorBox[56].w / alphaSum;
        alphaWeights[57] = colorBox[57].w / alphaSum;
        alphaWeights[58] = colorBox[58].w / alphaSum;
        alphaWeights[59] = colorBox[59].w / alphaSum;
        
        alphaWeights[60] = colorBox[60].w / alphaSum;
        alphaWeights[61] = colorBox[61].w / alphaSum;
        alphaWeights[62] = colorBox[62].w / alphaSum;
        alphaWeights[63] = colorBox[63].w / alphaSum;
    }

    pVoxels = (char*)inputVoxelNormals.ptr;
    pitch = inputVoxelNormals.pitch;
    slicePitch = inputVoxelNormals.ysize * pitch;

    mmBaseZOffset = mmBaseZ * slicePitch;
    mmZPlus1Offset = mmZPlus1 * slicePitch;
    mmZPlus2Offset = mmZPlus2 * slicePitch;
    mmZPlus3Offset = mmZPlus3 * slicePitch;

    mmBaseYOffset = mmBaseY * pitch;
    mmYPlus1Offset = mmYPlus1 * pitch;
    mmYPlus2Offset = mmYPlus2 * pitch;
    mmYPlus3Offset = mmYPlus3 * pitch;

    cuda::VoxNorm normals[64] = 
    {
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmBaseX],//x, y, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus1],//x+1, y, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus2],//x+2, y, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus3],//x+3, y, z

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmBaseX],//x, y+1, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmBaseX],//x, y+2, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmBaseX],//x, y+3, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmBaseZOffset + mmYPlus3Offset)[mmXPlus3],//x+3, y+3, z

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmBaseX],//x, y, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus2],//x+2, y, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus3],//x+3, y, z+1

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z+1

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmBaseX],//x, y+2, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z+1

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmBaseX],//x, y+3, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z+1
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus1Offset + mmYPlus3Offset)[mmXPlus3],//x+3, y+3, z+1

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmBaseX],//x, y, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmXPlus2],//x+2, y, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmBaseYOffset)[mmXPlus3],//x+3, y, z+2

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z+2

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmBaseX],//x, y+2, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z+2

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmBaseX],//x, y+3, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z+2
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus2Offset + mmYPlus3Offset)[mmXPlus3],//x+3, y+3, z+2

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmBaseX],//x, y, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmXPlus2],//x+2, y, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmBaseYOffset)[mmXPlus3],//x+3, y, z+3

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmXPlus2],//x+2, y+1, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus1Offset)[mmXPlus3],//x+3, y+1, z+3

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmBaseX],//x, y+2, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmXPlus1],//x+1, y+2, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmXPlus2],//x+2, y+2, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus2Offset)[mmXPlus3],//x+3, y+2, z+3

        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmBaseX],//x, y+3, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmXPlus1],//x+1, y+3, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmXPlus2],//x+2, y+3, z+3
        reinterpret_cast<cuda::VoxNorm*>(pVoxels + mmZPlus3Offset + mmYPlus3Offset)[mmXPlus3]//x+3, y+3, z+3
    };

    glm::vec4 avgColor;
    VoxColorToVec4(&avgColor, colorBox[0]);
    avgColor.x *= alphaWeights[0];
    avgColor.y *= alphaWeights[0];
    avgColor.z *= alphaWeights[0];

    glm::vec4 curColor;

    cuda::VoxNorm avgNormal = normals[0];
    for(int i = 1; i < 64; ++i)
    {
        VoxColorToVec4(&curColor, colorBox[i]);
        curColor.x *= alphaWeights[i];
        curColor.y *= alphaWeights[i];
        curColor.z *= alphaWeights[i];

        avgColor += curColor;

        avgNormal += normals[i];
    }

    const float oneOverSixtyFour = 1.0f / 64.0f;

    avgColor.w *= oneOverSixtyFour;
    avgColor.w = glm::pow(avgColor.w, 1.0f - avgColor.w);
    avgColor.x *= avgColor.w;
    avgColor.y *= avgColor.w;
    avgColor.z *= avgColor.w;

    avgNormal *= oneOverSixtyFour;
    float len2 = glm::length2(avgNormal);
    if(len2 > 0.0f)
        avgNormal /= (glm::sqrt(len2));//normalize
    else
        avgNormal = glm::vec3(0.0f);

    pVoxels = (char*)mipMapColors.ptr;
    pitch = mipMapColors.pitch;
    slicePitch = mipMapColors.ysize * pitch;
    //if(xIndex >= 110 && xIndex <= 112 && yIndex == 7 && zIndex == 8)
    //    printf("[%d %d %d]=[%f %f %f %f]\n", xIndex-7, yIndex-7, zIndex-7, avgColor.x, avgColor.y, avgColor.z, avgColor.w);
    cuda::VoxColor voxColor;
    Vec4ToVoxColor(&voxColor, avgColor);
    reinterpret_cast<cuda::VoxColor*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = voxColor;
    //reinterpret_cast<cuda::VoxColor*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = cuda::VoxColor(xIndex, yIndex, zIndex, 0.0f);

    pVoxels = (char*)mipMapNormals.ptr;
    pitch = mipMapNormals.pitch;
    slicePitch = mipMapNormals.ysize * pitch;

    reinterpret_cast<glm::vec3*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = avgNormal;
    //reinterpret_cast<glm::vec3*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = glm::vec3(xIndex, yIndex, zIndex);
}

__global__ void ComputeVoxelMipMap8(
                        glm::uint voxInputDimX, glm::uint voxInputDimY, glm::uint voxInputDimZ,//input size
                        hipPitchedPtr inputVoxelColors,//input colors
                        hipPitchedPtr inputVoxelNormals,//input normals
                        glm::uint voxDimX, glm::uint voxDimY, glm::uint voxDimZ,//output size
                        hipPitchedPtr mipMapColors,//output colors
                        hipPitchedPtr mipMapNormals)//output normals
{
    glm::uint xIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(xIndex >= voxDimX)
        return;
    
    glm::uint yIndex = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(yIndex >= voxDimY)
        return;

    glm::uint zIndex = (blockIdx.z * blockDim.z) + threadIdx.z;
    if(zIndex >= voxDimZ)
        return;

    glm::uint mmBaseX = xIndex * 2;
    glm::uint mmXPlus1 = glm::min(mmBaseX + 1, voxInputDimX-1);
    glm::uint mmBaseY = yIndex * 2;
    glm::uint mmYPlus1 = glm::min(mmBaseY + 1, voxInputDimY-1);
    glm::uint mmBaseZ = zIndex * 2;
    glm::uint mmZPlus1 = glm::min(mmBaseZ + 1, voxInputDimZ-1);

    char* pVoxels = (char*)inputVoxelColors.ptr;
    size_t pitch = inputVoxelColors.pitch;
    size_t slicePitch = inputVoxelColors.ysize * pitch;

    glm::uint mmBaseZOffset = mmBaseZ * slicePitch;
    glm::uint mmZPlus1Offset = mmZPlus1 * slicePitch;

    glm::uint mmBaseYOffset = mmBaseY * pitch;
    glm::uint mmYPlus1Offset = mmYPlus1 * pitch;

    cuda::VoxColor colorBox[8] = {
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmBaseX],//x, y, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus1],//x+1, y, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmBaseX],//x, y+1, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmBaseX],//x, y, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+1
        reinterpret_cast<cuda::VoxColor*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus1]//x+1, y+1, z+1
    };

    /*float alphaSum = colorBox[0].w;
    alphaSum += colorBox[1].w;
    alphaSum += colorBox[2].w;
    alphaSum += colorBox[3].w;
    alphaSum += colorBox[4].w;
    alphaSum += colorBox[5].w;
    alphaSum += colorBox[6].w;
    alphaSum += colorBox[7].w;
    float alphaWeights[8] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
    if(alphaSum > 0.0f)
    {
        alphaWeights[0] = colorBox[0].w / alphaSum;
        alphaWeights[1] = colorBox[1].w / alphaSum;
        alphaWeights[2] = colorBox[2].w / alphaSum;
        alphaWeights[3] = colorBox[3].w / alphaSum;
        alphaWeights[4] = colorBox[4].w / alphaSum;
        alphaWeights[5] = colorBox[5].w / alphaSum;
        alphaWeights[6] = colorBox[6].w / alphaSum;
        alphaWeights[7] = colorBox[7].w / alphaSum;
    }*/

    pVoxels = (char*)inputVoxelNormals.ptr;
    pitch = inputVoxelNormals.pitch;
    slicePitch = inputVoxelNormals.ysize * pitch;

    mmBaseZOffset = mmBaseZ * slicePitch;
    mmZPlus1Offset = mmZPlus1 * slicePitch;

    mmBaseYOffset = mmBaseY * pitch;
    mmYPlus1Offset = mmYPlus1 * pitch;

    glm::vec3 normals[8] = 
    {
        reinterpret_cast<glm::vec3*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmBaseX],//x, y, z
        reinterpret_cast<glm::vec3*>(pVoxels + mmBaseZOffset + mmBaseYOffset)[mmXPlus1],//x+1, y, z
        reinterpret_cast<glm::vec3*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmBaseX],//x, y+1, z
        reinterpret_cast<glm::vec3*>(pVoxels + mmBaseZOffset + mmYPlus1Offset)[mmXPlus1],//x+1, y+1, z
        reinterpret_cast<glm::vec3*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmBaseX],//x, y, z+1
        reinterpret_cast<glm::vec3*>(pVoxels + mmZPlus1Offset + mmBaseYOffset)[mmXPlus1],//x+1, y, z+1
        reinterpret_cast<glm::vec3*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmBaseX],//x, y+1, z+1
        reinterpret_cast<glm::vec3*>(pVoxels + mmZPlus1Offset + mmYPlus1Offset)[mmXPlus1]//x+1, y+1, z+1
    };

    glm::vec4 avgColor;
    VoxColorToVec4(&avgColor, colorBox[0]);
    /*avgColor.x *= alphaWeights[0];
    avgColor.y *= alphaWeights[0];
    avgColor.z *= alphaWeights[0];*/

    glm::vec4 curColor;

    cuda::VoxNorm avgNormal = normals[0];
    for(int i = 1; i < 8; ++i)
    {
        VoxColorToVec4(&curColor, colorBox[i]);
        /*curColor.x *= alphaWeights[i];
        curColor.y *= alphaWeights[i];
        curColor.z *= alphaWeights[i];*/

        avgColor += curColor;

        avgNormal += normals[i];
    }

    const float oneOverEight = 1.0f / 8.0f;
    avgColor.w *= oneOverEight;
    //avgColor.w = glm::pow(avgColor.w, 1.0f - avgColor.w);
    avgColor.x *= oneOverEight;//avgColor.w;
    avgColor.y *= oneOverEight;//avgColor.w;
    avgColor.z *= oneOverEight;//avgColor.w;

    avgNormal *= oneOverEight;
    float len2 = glm::length2(avgNormal);
    if(len2 > 0.0f)
        avgNormal /= (glm::sqrt(len2));//normalize
    else
        avgNormal = glm::vec3(0.0f);

    pVoxels = (char*)mipMapColors.ptr;
    pitch = mipMapColors.pitch;
    slicePitch = mipMapColors.ysize * pitch;

    cuda::VoxColor voxColor;
    Vec4ToVoxColor(&voxColor, avgColor);
    //if(xIndex >= 110 && xIndex <= 112 && yIndex == 7 && zIndex == 8)
    //    printf("[%d %d %d]=[%f %f %f %f]\n", xIndex-7, yIndex-7, zIndex-7, avgColor.x, avgColor.y, avgColor.z, avgColor.w);
    reinterpret_cast<cuda::VoxColor*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = voxColor;
    //reinterpret_cast<cuda::VoxColor*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = cuda::VoxColor(xIndex, yIndex, zIndex, 0.0f);

    pVoxels = (char*)mipMapNormals.ptr;
    pitch = mipMapNormals.pitch;
    slicePitch = mipMapNormals.ysize * pitch;

    reinterpret_cast<glm::vec3*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = avgNormal;
    //reinterpret_cast<glm::vec3*>(pVoxels + (zIndex * slicePitch) + (yIndex * pitch))[xIndex] = glm::vec3(xIndex, yIndex, zIndex);
}

__device__ glm::uint EncodeColorAsUInt(glm::vec4 color)
{
    unsigned int r = static_cast<unsigned int>(color.x * 255.0f);
    unsigned int g = static_cast<unsigned int>(color.y * 255.0f);
    unsigned int b = static_cast<unsigned int>(color.z * 255.0f);
    unsigned int a = static_cast<unsigned int>(color.w * 255.0f);

    glm::uint bits = (r << 24);
    bits |= (g << 16);
    bits |= (b << 8);
    bits |= a;

    return bits;
}

__global__ void ComputeOctTreeNodeConstColor(hipPitchedPtr devVoxels, 
                                             cuda::VoxColor* devOctTreeConstColors,
                                             glm::ivec3 voxOffset,
                                             glm::ivec3 offsetToBrickBorder,
                                             glm::ivec3 voxDim,
                                             glm::uvec3 octTreeChunkDim,
                                             glm::uvec3 octTreeDim,
                                             glm::ivec3 nodeOffset,
                                             glm::uvec3 brickDim)
{
    //compute x, y, z of my voxel in the chunk
    int xIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(xIndex >= octTreeChunkDim.x)
        return;
    
    int yIndex = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(yIndex >= octTreeChunkDim.y)
        return;

    int zIndex = (blockIdx.z * blockDim.z) + threadIdx.z;
    if(zIndex >= octTreeChunkDim.z)
        return;

    //use that to compute node's x, y, z
    int brickX = xIndex + nodeOffset.x;
    int brickY = yIndex + nodeOffset.y;
    int brickZ = zIndex + nodeOffset.z;

    //compute x,y,z of first element of this node's brick
    int brickSampleX = (brickX * brickDim.x) - voxOffset.x + offsetToBrickBorder.x;
    int brickSampleY = (brickY * brickDim.y) - voxOffset.y + offsetToBrickBorder.y;
    int brickSampleZ = (brickZ * brickDim.z) - voxOffset.z + offsetToBrickBorder.z;

    if(brickSampleX < 0)
    {
        //use first available element
        //printf("brickSimpleX < 0 (%d => %d)", brickSampleX, ((brickX+1) * brickDim.x) - voxOffset.x + offsetToBrickBorder.x);
        brickSampleX = 0;
    }
    
    if(brickSampleY < 0)
    {
        //use first available element
        //printf("brickSimpleY < 0 (%d => %d)", brickSampleY, ((brickY+1) * brickDim.y) - voxOffset.y + offsetToBrickBorder.y);
        brickSampleY = 0;
    }
    
    if(brickSampleZ < 0)
    {
        //use first available element
        //printf("brickSimpleZ < 0 (%d => %d)", brickSampleZ, ((brickZ+1) * brickDim.z) - voxOffset.z + offsetToBrickBorder.z);
        brickSampleZ = 0;
    }

    if(brickSampleX >= voxDim.x)
    {
        //use last element from previous brick
        brickSampleX = voxDim.x - 1;
    }
    
    if(brickSampleY >= voxDim.y)
    {
        //use last element from previous brick
        brickSampleY = voxDim.y - 1;
    }

    if(brickSampleZ >= voxDim.z)
    {
        //use last element from previous brick
        brickSampleZ = voxDim.z - 1;
    }

    char* devPtrColors = (char*)devVoxels.ptr;
    size_t voxelsPitch = devVoxels.pitch;
    size_t voxelsSlicePitch = voxelsPitch * devVoxels.ysize;

    char* sliceVoxels = devPtrColors + (brickSampleZ * voxelsSlicePitch);
    cuda::VoxColor* voxelColors = (cuda::VoxColor*)(sliceVoxels + (brickSampleY * voxelsPitch));
    cuda::VoxColor constVal = voxelColors[brickSampleX];

    //if(brickX == 4 && brickY == 0 && brickZ == 0 && 
    //   octTreeDim.x == 8u && octTreeDim.y == 8u && octTreeDim.z == 8u)
    //{
    //    printf("constVal[%d %d %d] = vox[%d %d %d] = %f %f %f %f\n", 
    //           brickX, brickY, brickZ,
    //           brickSampleX, brickSampleY, brickSampleZ,
    //           constVal.x, constVal.y, constVal.z, constVal.w);

    //}
    devOctTreeConstColors[(brickZ * octTreeDim.x * octTreeDim.y)
                          + (brickY * octTreeDim.x)
                          + brickX] = constVal;
}

__device__ bool VEC4_EQUAL(const glm::vec4& v1, const glm::vec4& v2)
{
    const float epsilon = 0.001f;
    glm::vec4 diff = v1 - v2;

    return (diff.x > -epsilon && diff.x < epsilon) &&
           (diff.y > -epsilon && diff.y < epsilon) &&
           (diff.z > -epsilon && diff.z < epsilon) &&
           (diff.w > -epsilon && diff.w < epsilon);
}

__device__ bool VEC4_EQUAL(const uchar4& v1, const uchar4& v2)
{
    return v1.x == v2.x && v1.y == v2.y && v1.z == v2.z && v1.w == v2.w;
}

__device__ void ComputeNodeType(int xIndex,
                                int yIndex,
                                int zIndex,
                                int brickX,
                                int brickY, 
                                int brickZ,
                                //int brickSampleX,
                                //int brickSampleY,
                                //int brickSampleZ,
                                hipPitchedPtr devVoxels,
                                glm::ivec3 voxDim,
                                glm::uint* devOctTreeNodes,
                                cuda::VoxColor* devOctTreeConstColors,
                                glm::uvec3 octTreeDim)
{
    if(xIndex < 0 || xIndex >= voxDim.x ||
       yIndex < 0 || yIndex >= voxDim.y ||
       zIndex < 0 || zIndex >= voxDim.z)
    {
        printf("Shit! xyzIndex=%d %d %d\n", xIndex, yIndex, zIndex);
        return;
    }

    //if(brickSampleX < 0 || brickSampleX >= voxDim.x ||
    //   brickSampleY < 0 || brickSampleY >= voxDim.y ||
    //   brickSampleZ < 0 || brickSampleZ >= voxDim.z)
    //{
    //    printf("SOB! xyzIndex=%d %d %d brickSampleXYZ=%d %d %d\n", 
    //           xIndex, yIndex, zIndex, 
    //           brickSampleX, brickSampleY, brickSampleZ);
    //    return;
    //}

    if(brickX < 0 || brickX >= octTreeDim.x ||
       brickY < 0 || brickY >= octTreeDim.y ||
       brickZ < 0 || brickZ >= octTreeDim.z)
    {
        printf("WTF? xyzIndex=%d %d %d brickXYZ=%d %d %d\n", 
               xIndex, yIndex, zIndex,
               brickX, brickY, brickZ);
        return;
    }

    char* devPtrColors = (char*)devVoxels.ptr;
    size_t voxelsPitch = devVoxels.pitch;
    size_t voxelsSlicePitch = voxelsPitch * devVoxels.ysize;

    //char* sliceVoxels = devPtrColors + (brickSampleZ * voxelsSlicePitch);
    //cuda::VoxColor* voxelColors = (cuda::VoxColor*)(sliceVoxels + (brickSampleY * voxelsPitch));
    //cuda::VoxColor constVal = voxelColors[brickSampleX];
    cuda::VoxColor constVal = devOctTreeConstColors[(brickZ * octTreeDim.x * octTreeDim.y)
                                               + (brickY * octTreeDim.x)
                                               + brickX];

    char* sliceVoxels = devPtrColors + (zIndex * voxelsSlicePitch);
    cuda::VoxColor* voxelColors = (cuda::VoxColor*)(sliceVoxels + (yIndex * voxelsPitch));
    cuda::VoxColor checkVal = voxelColors[xIndex];


    //if(xIndex == 12 &&
    //   yIndex == 110 &&
    //   zIndex == 100 &&
    //   octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
    //{
    //    printf("vox[%d %d %d] = %f %f %f %f == %f %f %f %f -> %d (node=%d %d %d)\n", 
    //           xIndex, yIndex, zIndex,
    //           checkVal.x, checkVal.y, checkVal.z, checkVal.w, 
    //           constVal.x, constVal.y, constVal.z, constVal.w,
    //           VEC4_FLOATS_EQUAL(constVal, checkVal) == false ? 1 : 0,
    //           brickX, brickY, brickZ);

    //}

    if(VEC4_EQUAL(constVal, checkVal) == false)
    {
        //if(brickX == 8u && brickY == 1u && brickZ == 31u && octTreeDim.x == 32u)
        /*if(xIndex == 127 &&
            yIndex == 30 &&
            zIndex == 209 &&
            octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        {
            printf("Node[%d %d %d]==1 voxel[%d %d %d] (%f %f %f %f)=(%f %f %f %f)\n",
                    brickX, brickY, brickZ,
                    xIndex, yIndex, zIndex,
                    constVal.x,
                    constVal.y,
                    constVal.z,
                    constVal.w,
                    checkVal.x,
                    checkVal.y,
                    checkVal.z,
                    checkVal.w);
        }*/
        
        devOctTreeNodes[(brickZ * octTreeDim.x * octTreeDim.y)
                        + (brickY * octTreeDim.x)
                        + brickX] = 1u;//non-const node
        //if(xIndex == 29u && yIndex == 16u && zIndex == 20u)// && octTreeDim.x == 4u)
        //    printf("[%d %d %d]=%f %f %f %f == %f %f %f %f [%d %d %d]=[%d]=%d\n", 
        //       brickSampleX, brickSampleY, brickSampleZ,
        //       constVal.x, constVal.y, constVal.z, constVal.w,
        //       checkVal.x, checkVal.y, checkVal.z, checkVal.w,
        //       brickX, brickY, brickZ,
        //       (brickZ * octTreeDim.x * octTreeDim.y) + (brickY * octTreeDim.x) + brickX,
        //       devOctTreeNodes[(brickZ * octTreeDim.x * octTreeDim.y)
        //                + (brickY * octTreeDim.x)
        //                + brickX]);
    }
}

__device__ void HandleZBorder(int xIndex,
                              int yIndex,
                              int zIndex,
                              int brickX,
                              int brickY, 
                              int brickZ,
                              int brickBaseX,
                              int brickBaseY,
                              int brickBaseZ,
                              //glm::ivec3 brickSampleOffset,
                              glm::uvec3 brickDim,
                              hipPitchedPtr devVoxels,
                              glm::ivec3 voxDim,
                              glm::uint* devOctTreeNodes,
                              cuda::VoxColor* devOctTreeConstColors,
                              glm::uvec3 octTreeDim)
{
    //check if z is one past end of prev brick
    if(zIndex == brickBaseZ && brickZ > 0)
    {
        int borderBrickZ = brickZ - 1;
        
        /*if(xIndex == 127 &&
            yIndex == 30 &&
            zIndex == 209 &&
            octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        {
            printf("In HandleZBorder subZ Node[%d %d %d]==1 voxel[%d %d %d]\n",
                brickX, brickY, borderBrickZ,
                xIndex, yIndex, zIndex);
        }*/
        ComputeNodeType(xIndex, yIndex, zIndex,
                        brickX, brickY, borderBrickZ,
                        //brickBaseX + brickSampleOffset.x, 
                        //brickBaseY + brickSampleOffset.y, 
                        //borderBrickBaseZ + brickSampleOffset.z,
                        devVoxels,
                        voxDim,
                        devOctTreeNodes,
                        devOctTreeConstColors,
                        octTreeDim);
    }
    else if(zIndex == (brickBaseZ + brickDim.z - 1) && brickZ < (octTreeDim.z - 1))//one before start of next brick
    {
        int borderBrickZ = brickZ + 1;

        /*if(xIndex == 127 &&
            yIndex == 30 &&
            zIndex == 209 &&
            octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        {
            printf("In HandleZBorder addZ Node[%d %d %d]==1 voxel[%d %d %d]\n",
                brickX, brickY, borderBrickZ,
                xIndex, yIndex, zIndex);
        }*/
        ComputeNodeType(xIndex, yIndex, zIndex,
                        brickX, brickY, borderBrickZ,
                        //brickBaseX + brickSampleOffset.x, 
                        //brickBaseY + brickSampleOffset.y, 
                        //borderBrickBaseZ + brickSampleOffset.z,
                        devVoxels,
                        voxDim,
                        devOctTreeNodes,
                        devOctTreeConstColors,
                        octTreeDim);
    }
}

__device__ void HandleYBorder(int xIndex,
                              int yIndex,
                              int zIndex,
                              int brickX,
                              int brickY, 
                              int brickZ,
                              int brickBaseX,
                              int brickBaseY,
                              int brickBaseZ,
                              //glm::ivec3 brickSampleOffset,
                              glm::uvec3 brickDim,
                              hipPitchedPtr devVoxels,
                              glm::ivec3 voxDim,
                              glm::uint* devOctTreeNodes,
                              cuda::VoxColor* devOctTreeConstColors,
                              glm::uvec3 octTreeDim)
{
    //check if y is one past end of prev brick
    if(yIndex == brickBaseY && brickY > 0)
    {
        int borderBrickY = brickY - 1;
        int borderBrickBaseY = brickBaseY - brickDim.y;

        /*if(xIndex == 127 &&
            yIndex == 30 &&
            zIndex == 209 &&
            octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        {
            printf("In HandleYBorder sub Node[%d %d %d]==1 voxel[%d %d %d]\n",
                brickX, borderBrickY, brickZ,
                xIndex, yIndex, zIndex);
        }*/

        ComputeNodeType(xIndex, yIndex, zIndex,
                        brickX, borderBrickY, brickZ,
                        //brickBaseX + brickSampleOffset.x, 
                        //borderBrickBaseY + brickSampleOffset.y, 
                        //brickBaseZ + brickSampleOffset.z,
                        devVoxels,
                        voxDim,
                        devOctTreeNodes,
                        devOctTreeConstColors,
                        octTreeDim);

        if((zIndex == brickBaseZ && brickZ > 0) || 
           (zIndex == (brickBaseZ + brickDim.z - 1) && brickZ < (octTreeDim.z - 1)))
        {
            HandleZBorder(xIndex, yIndex, zIndex,
                          brickX, borderBrickY, brickZ,
                          brickBaseX, borderBrickBaseY, brickBaseZ,
                          //brickSampleOffset,
                          brickDim,
                          devVoxels,
                          voxDim,
                          devOctTreeNodes,
                          devOctTreeConstColors,
                          octTreeDim);
        }
    }
    else if(yIndex == (brickBaseY + brickDim.y - 1) && brickY < (octTreeDim.y - 1))//one before start of next brick
    {
        int borderBrickY = brickY + 1;
        int borderBrickBaseY = brickBaseY + brickDim.y;

        /*if(xIndex == 127 &&
            yIndex == 30 &&
            zIndex == 209 &&
            octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        {
            printf("In HandleYBorder add Node[%d %d %d]==1 voxel[%d %d %d]\n",
                brickX, borderBrickY, brickZ,
                xIndex, yIndex, zIndex);
        }*/
                
        ComputeNodeType(xIndex, yIndex, zIndex,
                        brickX, borderBrickY, brickZ,
                        //brickBaseX + brickSampleOffset.x, 
                        //borderBrickBaseY + brickSampleOffset.y, 
                        //brickBaseZ + brickSampleOffset.z,
                        devVoxels,
                        voxDim,
                        devOctTreeNodes,
                        devOctTreeConstColors,
                        octTreeDim);

        if((zIndex == brickBaseZ && brickZ > 0) || 
           (zIndex == (brickBaseZ + brickDim.z - 1) && brickZ < (octTreeDim.z - 1)))
        {
            HandleZBorder(xIndex, yIndex, zIndex,
                          brickX, borderBrickY, brickZ,
                          brickBaseX, borderBrickBaseY, brickBaseZ,
                          //brickSampleOffset,
                          brickDim,
                          devVoxels,
                          voxDim,
                          devOctTreeNodes,
                          devOctTreeConstColors,
                          octTreeDim);
        }
    }
}

__device__ void HandleXBorder(int xIndex,
                              int yIndex,
                              int zIndex,
                              int brickX,
                              int brickY, 
                              int brickZ,
                              int brickBaseX,
                              int brickBaseY,
                              int brickBaseZ,
                              //glm::ivec3 brickSampleOffset,
                              glm::uvec3 brickDim,
                              hipPitchedPtr devVoxels,
                              glm::ivec3 voxDim,
                              glm::uint* devOctTreeNodes,
                              cuda::VoxColor* devOctTreeConstColors,
                              glm::uvec3 octTreeDim)
{
    //if(xIndex == 15u && yIndex == 0u && zIndex == 0u && voxDim.x == 18u && octTreeDim.x == 8u)
    //{
    //    printf("WTF %d %d %d\n", xIndex, brickBaseX, (brickBaseX + brickDim.x - 1));
    //}
    //if xIndex is one voxel past the end of the previous brick in X dim
    if((xIndex == brickBaseX && brickX > 0))
    {
        int borderBrickX = brickX - 1;
        int borderBrickBaseX = brickBaseX - brickDim.x;

        /*if(xIndex == 127 &&
            yIndex == 30 &&
            zIndex == 209 &&
            octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        {
            printf("In HandleXBorder sub Node[%d %d %d]==1 voxel[%d %d %d]\n",
                borderBrickX, brickY, brickZ,
                xIndex, yIndex, zIndex);
        }*/

        ComputeNodeType(xIndex, yIndex, zIndex,
                        borderBrickX, brickY, brickZ,
                        //borderBrickBaseX + brickSampleOffset.x,
                        //brickBaseY + brickSampleOffset.y, 
                        //brickBaseZ + brickSampleOffset.z,
                        devVoxels,
                        voxDim,
                        devOctTreeNodes,
                        devOctTreeConstColors,
                        octTreeDim);
        
        if((yIndex == brickBaseY && brickY > 0) || 
           (yIndex == (brickBaseY + brickDim.y - 1) && brickY < (octTreeDim.y - 1)))
        {
            HandleYBorder(xIndex, yIndex, zIndex,
                          borderBrickX, brickY, brickZ,
                          borderBrickBaseX, brickBaseY, brickBaseZ,
                          //brickSampleOffset,
                          brickDim,
                          devVoxels,
                          voxDim,
                          devOctTreeNodes,
                          devOctTreeConstColors,
                          octTreeDim);
        }

        if((zIndex == brickBaseZ && brickZ > 0) || 
           (zIndex == (brickBaseZ + brickDim.z - 1) && brickZ < (octTreeDim.z - 1)))
        {
            //if(xIndex == 12 &&
            //   yIndex == 110 &&
            //   zIndex == 100 &&
            //   octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
            //{
            //    printf("In HandleXBorder, about to HandleZBorder node=%d %d %d base=%d %d %d\n",
            //           borderBrickX, brickY, brickZ,
            //           borderBrickBaseX, brickBaseY, brickBaseZ);
            //}
            HandleZBorder(xIndex, yIndex, zIndex,
                          borderBrickX, brickY, brickZ,
                          borderBrickBaseX, brickBaseY, brickBaseZ,
                          //brickSampleOffset,
                          brickDim,
                          devVoxels,
                          voxDim,
                          devOctTreeNodes,
                          devOctTreeConstColors,
                          octTreeDim);
        }
    }
    //if xIndex is one voxel before start of next brick
    else if(xIndex == (brickBaseX + brickDim.x - 1) && brickX < (octTreeDim.x - 1))
    {
        int borderBrickX = brickX + 1;
        int borderBrickBaseX = brickBaseX + brickDim.x;
        
        /*if(xIndex == 127 &&
            yIndex == 30 &&
            zIndex == 209 &&
            octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        {
            printf("In HandleXBorder add Node[%d %d %d]==1 voxel[%d %d %d]\n",
                borderBrickX, brickY, brickZ,
                xIndex, yIndex, zIndex);
        }*/

        ComputeNodeType(xIndex, yIndex, zIndex,
                        borderBrickX, brickY, brickZ,
                        //borderBrickBaseX + brickSampleOffset.x,
                        //brickBaseY + brickSampleOffset.y, 
                        //brickBaseZ + brickSampleOffset.z,
                        devVoxels,
                        voxDim,
                        devOctTreeNodes,
                        devOctTreeConstColors,
                        octTreeDim);
        
        if((yIndex == brickBaseY && brickY > 0) || 
           (yIndex == (brickBaseY + brickDim.y - 1) && brickY < (octTreeDim.y - 1)))
        {
            HandleYBorder(xIndex, yIndex, zIndex,
                          borderBrickX, brickY, brickZ,
                          borderBrickBaseX, brickBaseY, brickBaseZ,
                          //brickSampleOffset,
                          brickDim,
                          devVoxels,
                          voxDim,
                          devOctTreeNodes,
                          devOctTreeConstColors,
                          octTreeDim);
        }

        if((zIndex == brickBaseZ && brickZ > 0) ||
           (zIndex == (brickBaseZ + brickDim.z - 1) && brickZ < (octTreeDim.z - 1)))
        {
            HandleZBorder(xIndex, yIndex, zIndex,
                          borderBrickX, brickY, brickZ,
                          borderBrickBaseX, brickBaseY, brickBaseZ,
                          //brickSampleOffset,
                          brickDim,
                          devVoxels,
                          voxDim,
                          devOctTreeNodes,
                          devOctTreeConstColors,
                          octTreeDim);
        }
    }
}

__global__ void ComputeOctTreeNodeType(hipPitchedPtr devVoxels, 
                                       glm::uint* devOctTreeNodes,
                                       cuda::VoxColor* devOctTreeConstColors,
                                       glm::ivec3 voxOffset,
                                       glm::ivec3 offsetToBrickBorder,
                                       //glm::ivec3 brickSampleOffset,
                                       glm::ivec3 voxDim,
                                       glm::ivec3 fullVoxDim,
                                       glm::uvec3 octTreeDim,
                                       glm::uvec3 brickDim,
                                       int xOffset)
{
    //compute x, y, z of my voxel in the chunk
    int xIndex = (blockIdx.x * blockDim.x) + threadIdx.x + xOffset;
    if(xIndex >= voxDim.x)
        return;
    
    int yIndex = (blockIdx.y * blockDim.y) + threadIdx.y;
    if(yIndex >= voxDim.y)
        return;

    int zIndex = (blockIdx.z * blockDim.z) + threadIdx.z;
    if(zIndex >= voxDim.z)
        return;

    glm::ivec3 fullVoxXYZ(xIndex + voxOffset.x - offsetToBrickBorder.x,
                          yIndex + voxOffset.y - offsetToBrickBorder.y,
                          zIndex + voxOffset.z - offsetToBrickBorder.z);

    //if(xIndex == 12 &&
    //   yIndex == 110 &&
    //   zIndex == 100 &&
    //   octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
    //{
    //    printf("vox=[%d %d %d]->[%d %d %d]\n", 
    //               xIndex, yIndex, zIndex, 
    //               fullVoxXYZ.x, fullVoxXYZ.y, fullVoxXYZ.z);
    //}
    //use that to compute node's x, y, z
    if(fullVoxXYZ.x < -1 || fullVoxXYZ.x > fullVoxDim.x ||
       fullVoxXYZ.y < -1 || fullVoxXYZ.y > fullVoxDim.y ||
       fullVoxXYZ.z < -1 || fullVoxXYZ.z > fullVoxDim.z)
    {
        //if(octTreeDim.x == 8u && octTreeDim.y == 8u && octTreeDim.z == 8u)
        //{
        //    printf("skipping vox=[%d %d %d]->[%d %d %d]\n", 
        //           xIndex, yIndex, zIndex, 
        //           fullVoxXYZ.x, fullVoxXYZ.y, fullVoxXYZ.z);
        //}
        return;//this means we are in the extra border area
    }

    int brickX = glm::min(static_cast<int>(static_cast<float>(fullVoxXYZ.x) / static_cast<float>(brickDim.x)),
                          static_cast<int>(octTreeDim.x - 1u));
    int brickY = glm::min(static_cast<int>(static_cast<float>(fullVoxXYZ.y) / static_cast<float>(brickDim.y)),
                          static_cast<int>(octTreeDim.y - 1u));;
    int brickZ = glm::min(static_cast<int>(static_cast<float>(fullVoxXYZ.z) / static_cast<float>(brickDim.z)),
                          static_cast<int>(octTreeDim.z - 1u));

    //compute x,y,z of first element of this node's brick
    int brickBaseX = (brickX * brickDim.x) - voxOffset.x + offsetToBrickBorder.x;
    int brickBaseY = (brickY * brickDim.y) - voxOffset.y + offsetToBrickBorder.y;
    int brickBaseZ = (brickZ * brickDim.z) - voxOffset.z + offsetToBrickBorder.z;

    /*if(octTreeDim.x == 32u && xIndex == 127 && yIndex == 30 && zIndex == 209)
    {
        printf("Base Node[%d %d %d]==[%d %d %d] voxel[%d %d %d]\n",
                brickX, brickY, brickZ,
                fullVoxXYZ.x, fullVoxXYZ.y, fullVoxXYZ.z,
                xIndex, yIndex, zIndex);
    }*/
    
    ComputeNodeType(xIndex, yIndex, zIndex,
                    brickX, brickY, brickZ,
                    //brickBaseX + brickSampleOffset.x,
                    //brickBaseY + brickSampleOffset.y,
                    //brickBaseZ + brickSampleOffset.z,
                    devVoxels,
                    voxDim,
                    devOctTreeNodes,
                    devOctTreeConstColors,
                    octTreeDim);
    //else if(brickX == -1 && brickY == -1 && brickZ == -1)
    //{
    //    printf("BrickXYZ=%d %d %d\n    xyzIndex=%d %d %d\n    brickBaseXYZ=%d %d %d\n    brickBaseXYZ+brickDim.x-1=%d %d %d\n", 
    //           brickX, brickY, brickZ, 
    //           xIndex, yIndex, zIndex, 
    //           brickBaseX, brickBaseY, brickBaseZ,
    //           brickBaseX + brickDim.x - 1, 
    //           brickBaseY + brickDim.y - 1, 
    //           brickBaseZ + brickDim.z - 1);
    //}

    //if(xIndex == 12 &&
    //   yIndex == 110 &&
    //   zIndex == 100 &&
    //   octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
    //{
    //    printf("minBorderX=%d maxBorderX=%d minBorderZ=%d maxBorderZ=%d\n", 
    //               brickBaseX, 
    //               (brickBaseX + brickDim.x - 1),
    //               brickBaseZ,
    //               (brickBaseZ + brickDim.z - 1));
    //}

    if((xIndex == brickBaseX && brickX > 0) || 
       (xIndex == (brickBaseX + brickDim.x - 1) && brickX < (octTreeDim.x - 1)))
    {
        //if(xIndex == 12 &&
        //   yIndex == 110 &&
        //   zIndex == 100 &&
        //   octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        //{
        //    printf("About to HandleXBorder\n");
        //}
        HandleXBorder(xIndex, yIndex, zIndex,
                      brickX, brickY, brickZ,
                      brickBaseX, brickBaseY, brickBaseZ,
                      //brickSampleOffset,
                      brickDim,
                      devVoxels,
                      voxDim,
                      devOctTreeNodes,
                      devOctTreeConstColors,
                      octTreeDim);
    }
    
    if((yIndex == brickBaseY && brickY > 0) || 
       (yIndex == (brickBaseY + brickDim.y - 1) && brickY < (octTreeDim.y - 1)))
    {
        HandleYBorder(xIndex, yIndex, zIndex,
                      brickX, brickY, brickZ,
                      brickBaseX, brickBaseY, brickBaseZ,
                      //brickSampleOffset,
                      brickDim,
                      devVoxels,
                      voxDim,
                      devOctTreeNodes,
                      devOctTreeConstColors,
                      octTreeDim);
    }
    
    if((zIndex == brickBaseZ && brickZ > 0) ||
       (zIndex == (brickBaseZ + brickDim.z - 1) && brickZ < (octTreeDim.z - 1)))
    {
        //if(xIndex == 12 &&
        //   yIndex == 110 &&
        //   zIndex == 100 &&
        //   octTreeDim.x == 32u && octTreeDim.y == 32u && octTreeDim.z == 32u)
        //{
        //        printf("About to HandleZBorder\n");
        //}
        //printf("HandleZBorder %d %d %d\n", xIndex, yIndex, zIndex);
        HandleZBorder(xIndex, yIndex, zIndex,
                      brickX, brickY, brickZ,
                      brickBaseX, brickBaseY, brickBaseZ,
                      //brickSampleOffset,
                      brickDim,
                      devVoxels,
                      voxDim,
                      devOctTreeNodes,
                      devOctTreeConstColors,
                      octTreeDim);
    }
}                                                                              

//void cpuTest(const glm::vec3* verts, 
//             int numVerts,
//             const glm::vec3& p,
//             const glm::vec3& deltaP,
//             int *voxels,
//             const glm::uvec3& voxDim)
//{
//    std::vector< glm::vec3 > edges(numVerts);
//    //std::vector< glm::vec3 > edgeNormals(numVerts);
//    std::vector< glm::vec3 > normals( numVerts / 3);
//    
//    for(int i = 0, int j = 0; i < numVerts; i+=3, ++j)
//    {
//        edges[i] = glm::vec3(verts[i+1] - verts[i]);
//        edges[i+1] = glm::vec3(verts[i+2] - verts[i+1]);
//        edges[i+2] = glm::vec3(verts[i] - verts[i+2]);
//
//        const glm::vec3& p0p1 = edges[i];
//        glm::vec3 p0p2 = verts[i+2] - verts[i];
//        normals[j] = glm::normalize(glm::cross(p0p1, p0p2));
//
//        //edgeNormals[i] = glm::cross(normals[j], edges[i]);
//        //edgeNormals[i+1] = glm::cross(normals[j], edges[i+1]);
//        //edgeNormals[i+2] = glm::cross(normals[j], edges[i+2]);
//    };
//
//    //int index = 0;
//    //for(int tri = 0; tri < 8; ++tri)
//    for(int tri = 0; tri < 2; ++tri)
//    {
//        glm::uvec3 minVox;
//        glm::uvec3 maxVox;
//        int vindex = tri * 3;
//        ComputeTriangleVoxelBounds(&verts[vindex], p, deltaP, voxDim, minVox, maxVox);
//        glm::vec3 uvw(0.0f, 0.0f, 0.0f);
//
//        const glm::vec3& normal = normals[tri];
//        //int thinProjAxis1, thinProjAxis2, thinOtherAxis;
//        //if(thinVoxelization)
//        //{
//        //    glm::vec3 normAbs = glm::abs(normal);
//        //    if(normAbs.z > normAbs.x)
//        //    {
//        //        if(normAbs.z > normAbs.y)
//        //        {
//        //            //z is dominant axis, so test projection onto xy axis
//        //            thinProjAxis1 = X_AXIS;
//        //            thinProjAxis2 = Y_AXIS;
//        //            thinOtherAxis = Z_AXIS;
//        //        }
//        //        else
//        //        {
//        //            //y is dominant axis, so test projection onto zx axis
//        //            thinProjAxis1 = Z_AXIS;
//        //            thinProjAxis2 = X_AXIS;
//        //            thinOtherAxis = Y_AXIS;
//        //        }
//        //    }
//        //    else if(normAbs.x > normAbs.y)
//        //    {
//        //        //x is dominant axis, so test projection onto yz axis
//        //        thinProjAxis1 = Y_AXIS;
//        //        thinProjAxis2 = Z_AXIS;
//        //        thinOtherAxis = X_AXIS;
//        //    }
//        //    else
//        //    {
//        //        //y is dominant axis, so test projection onto zx axis
//        //        thinProjAxis1 = Z_AXIS;
//        //        thinProjAxis2 = X_AXIS;
//        //        thinOtherAxis = Y_AXIS;
//        //    }
//        //}
//
//        for(glm::uint u = minVox.x; u <= maxVox.x; ++u)
//        {
//            uvw.x = (float)u;
//            for(glm::uint v = minVox.y; v <= maxVox.y; ++v)
//            {
//                uvw.y = (float)v;
//                for(glm::uint w = minVox.z; w <= maxVox.z; ++w)
//                {
//                    glm::uint voxelIndex = (w * voxDim.x * voxDim.y) + (v * voxDim.x) + u;
//                    if(voxels[voxelIndex] == 1)
//                        continue;
//
//                    uvw.z = (float)w;
//                    glm::vec3 voxMin = p;// + (uvw * deltaP);
//                    if(TrianglePlaneOverlapsVoxel(verts[vindex], normals[tri], p, deltaP, uvw))
//                    {
//                        ////if(thinVoxelization)
//                        //{
//                        //    //if(!TriangleOverlapsVoxelOnAxis(&verts[vindex], 
//                        //    //                                &edges[vindex], 
//                        //    //                                normals[tri], 
//                        //    //                                p, deltaP*0.5f, uvw,
//                        //    //                                thinProjAxis1, thinProjAxis2, thinOtherAxis))
//                        //    //{
//                        //    //    continue;
//                        //    //}
//                        //    if(!TriangleOverlapsVoxel(&verts[vindex], 
//                        //                              &edges[vindex], 
//                        //                              &edgeNormals[vindex],
//                        //                              normals[tri], 
//                        //                              voxMin, deltaP * 0.5f, uvw))
//                        //    {
//                        //        continue;
//                        //    }
//                        //}
//                        //else 
//                        if(!TriangleOverlapsVoxel(&verts[vindex], 
//                                                       &edges[vindex], 
//                                                       //&edgeNormals[vindex],
//                                                       normals[tri], 
//                                                       voxMin, deltaP, uvw))
//                        {
//                            continue;
//                        }
//                    }
//                    else
//                        continue;
//
//                    voxels[voxelIndex] = 1;
//                }
//            }
//        }
//    }
//}

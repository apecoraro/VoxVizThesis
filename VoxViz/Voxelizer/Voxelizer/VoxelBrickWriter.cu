#include "hip/hip_runtime.h"
#include "VoxelBrickWriter.h"

#include <iostream>
#include <sstream>
#include <direct.h>

#include <osg/Image>
#include <osg/Texture2D>
#include <osg/Texture3D>
#include <osg/GraphicsContext>
#include <osgViewer/GraphicsWindow>

using namespace cuda;

VoxelBrickWriter::VoxelBrickWriter() :
    _binary(false),
    _compressed(false)
{
}

VoxelBrickWriter::VoxelBrickWriter(const VoxelBrickWriter& copy) :
    _binary(copy._binary),
    _compressed(copy._compressed),
    _brickMap(copy._brickMap),
    _storedBrickMap(copy._storedBrickMap),
    _outputFileName(copy._outputFileName),
    _outputPartialBricksDir(copy._outputPartialBricksDir)
{
    _voxFile.swap(const_cast<VoxelBrickWriter&>(copy)._voxFile);
}

VoxelBrickWriter::~VoxelBrickWriter()
{
    //deleteOpenGLContext();
    if(_voxFile.is_open())
    {
        endBricksFile();
    }
}

VoxelBrickWriter& VoxelBrickWriter::operator=(const VoxelBrickWriter& rhs)
{
    if(this != &rhs)
    {
        _binary = rhs._binary;
        _compressed = rhs._compressed;
        _brickMap = rhs._brickMap;
        _storedBrickMap = rhs._storedBrickMap;
        _outputFileName = rhs._outputFileName;
        _outputPartialBricksDir = rhs._outputPartialBricksDir;
        _voxFile.swap(const_cast<VoxelBrickWriter&>(rhs)._voxFile);
    }

    return *this;
}

static bool StartTextBricksFile(const std::string& outFileName,
                               std::ofstream& voxFile)
{
    voxFile.open(outFileName, std::fstream::out);
    if(!voxFile.is_open())
    {
        return false;
    }
    
    voxFile << "<Bricks>" << std::endl;

    return true;
}

bool VoxelBrickWriter::startBricksFile(const std::string& outputDirPartialBricksDir, const std::string& fileName, bool binary, bool compressed)
{
    _binary = binary;
    _compressed = compressed;
    if(_compressed && !createOpenGLContext())
        return false;

    _outputPartialBricksDir = outputDirPartialBricksDir;
    if(mkdir(_outputPartialBricksDir.c_str()) != 0 && errno != EEXIST)
        return false;

    if(!_binary)
    {
        _outputFileName = fileName;
        _outputFileName += ".gvt";
        return StartTextBricksFile(_outputFileName, _voxFile);
    }
    else
    {
        _outputFileName = fileName;
        _outputFileName += ".gvb";
        _voxFile.open(_outputFileName, std::ios_base::out | std::ios_base::binary);
        if(!_voxFile.is_open())
            return false;

        int compressed = _compressed ? 1 : 0;
        _voxFile.write((char*)&compressed, sizeof(compressed));

        return !_voxFile.fail();
    }
}

static bool EndTextBricksFile(std::ofstream& voxFile)
{
    voxFile << "</Bricks>" << std::endl;

    voxFile.close();

    return !voxFile.fail();
}

bool VoxelBrickWriter::endBricksFile()
{
    rmdir(_outputPartialBricksDir.c_str());
    if(!_binary)
    {
        return EndTextBricksFile(_voxFile);
    }
    else
    {
        _voxFile.close();
        return !_voxFile.fail();
    }
}

void CopyPartialBrick(VoxelBrickWriter::BrickData& brickData,//brick storage
                      size_t brickX, size_t brickY, size_t brickZ,//offset in brick to start
                      size_t xOffset, size_t yOffset, size_t zOffset, //offset in mipmap to start
                      size_t xSize, size_t ySize, size_t zSize,//size of data without border
                      size_t brickDimX, size_t brickDimY,// size_t brickDimZ,//size of brick without border
                      const cuda::VoxelColors& voxelColors, //mipmap colors
                      const cuda::VoxelNormals& voxelNormals,//mipmap normals
                      size_t voxDimX, size_t voxDimY, size_t voxDimZ)//dim of mipmap
{
    size_t borderVoxels = 1u;

    size_t xEnd = xOffset + xSize;
    if(xOffset != 0u)//if it is not on left side
    {
        xOffset -= borderVoxels;
        brickX -= borderVoxels;
    }
    if(xEnd < voxDimX) //if it is not on right side
    {
        xEnd += borderVoxels;
    }
    else if(xEnd > voxDimX)
        xEnd = voxDimX;

    size_t yEnd = yOffset + ySize;
    if(yOffset != 0u)//if it is not on left side
    {
        yOffset -= borderVoxels;
        brickY -= borderVoxels;
    }
    if(yEnd < voxDimY) //if it is not on right side
    {
        yEnd += borderVoxels;
    }
    else if(yEnd > voxDimY)
        yEnd = voxDimY;

    size_t zEnd = zOffset + zSize;
    if(zOffset != 0u)//if it is not on left side
    {
        zOffset -= borderVoxels;
        brickZ -= borderVoxels;
    }
    if(zEnd < voxDimZ) //if it is not on right side
    {
        zEnd += borderVoxels;
    }
    else if(zEnd > voxDimZ)
        zEnd = voxDimZ;

    if(!brickData.brickDataStoredInMemory()
        && !brickData.loadPartialBrickFiles())
    {
        std::cerr << "Failed to open partial brick files"
                  << std::endl;
        return;
    }
    size_t brickZIndex = brickZ;
    for(size_t z = zOffset; z < zEnd; ++z, ++brickZIndex)
    {
        size_t brickYIndex = brickY;
        for(size_t y = yOffset; y < yEnd; ++y, ++brickYIndex)
        {
            const cuda::VoxColor* pReadColors = &voxelColors[(z * voxDimY * voxDimX) + (y * voxDimX) + xOffset];
            const glm::vec3* pReadNormals = &voxelNormals[(z * voxDimY * voxDimX) + (y * voxDimX) + xOffset];
            size_t writeIndex = (brickZIndex * brickDimY * brickDimX) + (brickYIndex * brickDimX) + brickX;
            memcpy(&brickData.colors[writeIndex], pReadColors, sizeof(cuda::VoxColor) * (xEnd - xOffset));
            memcpy(&brickData.normals[writeIndex], pReadNormals, sizeof(glm::vec3) * (xEnd - xOffset));
        }
    }
    
    if(!brickData.brickDataStoredInMemory()
        && !brickData.dumpMemoryToPartialBrickFiles())
    {
        std::cerr << "Failed to dump partial brick files"
                  << std::endl;
        return;
    }
    brickData.totalVoxelsStored += ((xEnd - xOffset) * (yEnd - yOffset) * (zEnd - zOffset));
}

static bool WriteTextBrick(std::ofstream& voxFile,
                           size_t xOffset, size_t yOffset, size_t zOffset,
                           size_t xSize, size_t ySize, size_t zSize,
                           size_t brickDimX, size_t brickDimY, size_t brickDimZ,
                           const cuda::VoxelColors& voxelColors,
                           const cuda::VoxelNormals& voxelNormals)
{
    size_t borderVoxels = 1u;

    size_t brickBorderX(0u);
    size_t brickBorderY(0u);
    size_t brickBorderZ(0u);
    
    size_t xEnd = xOffset + brickDimX;
    if(xOffset != 0u && xEnd < xSize)//if it is middle brick
    {
        brickBorderX = borderVoxels;
        xEnd += borderVoxels;
        xOffset -= borderVoxels;
    }
    else if(xOffset == 0u && xEnd < xSize)//if it is left brick
    {
        xEnd += (borderVoxels << 1);
    }
    else if(xOffset != 0u) // must be right side brick
    {
        brickBorderX = borderVoxels;
        xOffset -= (borderVoxels << 1);
    }

    brickDimX = xEnd - xOffset;

    size_t yEnd = yOffset + brickDimY;
    if(yOffset != 0u && yEnd < ySize)//if it is middle brick
    {
        brickBorderY = borderVoxels;
        yEnd += borderVoxels;
        yOffset -= borderVoxels;
    }
    else if(yOffset == 0u && yEnd < ySize)//if it is left brick
    {
        yEnd += (borderVoxels << 1);
    }
    else if(yOffset != 0u) // must be right side brick
    {
        brickBorderY = borderVoxels;
        yOffset -= (borderVoxels << 1);
    }

    brickDimY = (yEnd - yOffset);

    size_t zEnd = zOffset + brickDimZ;
    if(zOffset != 0u && zEnd < zSize)//if it is middle brick
    {
        brickBorderZ = borderVoxels;
        zEnd += borderVoxels;
        zOffset -= borderVoxels;
    }
    else if(zOffset == 0u && zEnd < zSize)//if it is left brick
    {
        zEnd += (borderVoxels << 1);
    }
    else if(zOffset != 0u) // must be right side brick
    {
        brickBorderZ = borderVoxels;
        zOffset -= (borderVoxels << 1);
    }

    brickDimZ = (zEnd - zOffset);

    voxFile << " <Brick "
            << "X=\"" << xOffset << "\" "
            << "Y=\"" << yOffset << "\" "
            << "Z=\"" << zOffset << "\" "
            << "DimX=\"" << brickDimX << "\" "
            << "DimY=\"" << brickDimY << "\" "
            << "DimZ=\"" << brickDimZ << "\" "
            << "BorderX=\"" << brickBorderX << "\" "
            << "BorderY=\"" << brickBorderY << "\" "
            << "BorderZ=\"" << brickBorderZ << "\" "
            << "/>"
            << std::endl;

    voxFile << "  <BrickColors>" << std::endl;
    for(size_t z = zOffset; z < (brickDimZ + zOffset); ++z)
    {
        for(size_t y = yOffset; y < (brickDimY + yOffset); ++y)
        {
            voxFile << "    <Row Y=\"" << y << "\" Z=\"" << z << "\">" << std::endl;
            for(size_t x = xOffset; x < (brickDimX + xOffset); ++x)
            {
                const cuda::VoxColor& color = voxelColors[(z * ySize * xSize) + (y * xSize) + x];
                voxFile << "(" << color.x << "," << color.y << "," << color.z << "," << color.w << ") ";
            }
            voxFile << std::endl << "    </Row>" << std::endl;
        }
    }
    voxFile << "  </BrickColors>" << std::endl;

    voxFile << "  <BrickGradients>" << std::endl;
    for(size_t z = zOffset; z < (brickDimZ + zOffset); ++z)
    {
        for(size_t y = yOffset; y < (brickDimY + yOffset); ++y)
        {
            voxFile << "    <Row YZ=\"" << y << " " << z << "\">" << std::endl;
            for(size_t x = xOffset; x < (brickDimX + xOffset); ++x)
            {
                const glm::vec3& normal = voxelNormals[(z * ySize * xSize) + (y * xSize) + x];
                voxFile << "(" << normal.x << "," << normal.y << "," << normal.z << ") ";
            }
            voxFile << std::endl << "    </Row>" << std::endl;
        }
    }
    voxFile << "  </BrickGradients>" << std::endl;

    voxFile << " </Brick>" << std::endl;

    return !voxFile.fail();
}

class OpenGLContext 
{
public:
    OpenGLContext() {}

    bool init()
    {
        osgViewer::GraphicsWindow window;
        osg::ref_ptr<osg::GraphicsContext::Traits> traits = new osg::GraphicsContext::Traits;
        traits->x = 0;
        traits->y = 0;
        traits->width = 1;
        traits->height = 1;
        traits->windowDecoration = false;
        traits->doubleBuffer = false;
        traits->sharedContext = 0;
        traits->pbuffer = true;

        _gc = osg::GraphicsContext::createGraphicsContext(traits.get());

        if (!_gc)
        {
            //osg::notify(osg::NOTICE)<<"Failed to create pbuffer, failing back to normal graphics window."<<std::endl;
            
            traits->pbuffer = false;
            _gc = osg::GraphicsContext::createGraphicsContext(traits.get());
        }

        if (_gc.valid()) 
        {
            _gc->realize();
            _gc->makeCurrent();
            /*if (dynamic_cast<osgViewer::GraphicsWindow*>(_gc.get()))
            {
                std::cout<<"Realized graphics window for OpenGL operations."<<std::endl;
            }
            else
            {
                std::cout<<"Realized pbuffer for OpenGL operations."<<std::endl;
            }*/
            return true;
        }

        return false;
    }
    
    bool valid() const { return _gc.valid() && _gc->isRealized(); }
    
private:
    osg::ref_ptr<osg::GraphicsContext> _gc;
};

static OpenGLContext* s_pOpenGLCtx = nullptr;

bool VoxelBrickWriter::createOpenGLContext()
{
    if(s_pOpenGLCtx == nullptr)
    {
        s_pOpenGLCtx = new OpenGLContext();
        return s_pOpenGLCtx->init() && s_pOpenGLCtx->valid();
    }

    return true;
}

void VoxelBrickWriter::deleteOpenGLContext()
{
    if(s_pOpenGLCtx != nullptr)
    {
        delete s_pOpenGLCtx;
        s_pOpenGLCtx = nullptr;
    }
}

static bool CompressImage(osg::Texture3D* pTexture3D)
{
    OpenGLContext& context = *s_pOpenGLCtx;

    osg::ref_ptr<osg::State> spState = new osg::State;
        
    osg::ref_ptr<osg::Image> spImage = pTexture3D->getImage();
    if (spImage.valid() && 
        (spImage->getPixelFormat()==GL_RGB || spImage->getPixelFormat()==GL_RGBA) &&
        (spImage->s()>=4 && spImage->t()>=4 && spImage->r()>=4))
    { 
        // get OpenGL driver to create texture from spImage.
        pTexture3D->apply(*spState);

        spImage->readImageFromCurrentTexture(0, false);

        pTexture3D->setInternalFormatMode(osg::Texture::USE_IMAGE_DATA_FORMAT);

        return true;
    }

    return false;
}

static bool DeCompress(osg::Texture2D* pTexture2D)
{
    OpenGLContext& context = *s_pOpenGLCtx;

    osg::ref_ptr<osg::State> spState = new osg::State;
        
    osg::ref_ptr<osg::Image> spImage = pTexture2D->getImage();
    if (spImage.valid())
    { 
        // get OpenGL driver to create texture from spImage.
        pTexture2D->apply(*spState);

        unsigned char* pNewData = (unsigned char*)malloc(sizeof(unsigned char) * 4 
                                                         * spImage->s() 
                                                         * spImage->t());
        glGetTexImage(GL_TEXTURE_2D, 0, GL_RGBA, GL_UNSIGNED_BYTE, pNewData);

        spImage->setImage(spImage->s(), spImage->t(), 1,
                          GL_RGBA8, GL_RGBA, GL_UNSIGNED_BYTE,
                          pNewData,
                          osg::Image::USE_MALLOC_FREE);

        return true;
    }

    return false;
}

//static void Scale3DImage(cuda::VoxelColors& colors,
//                         cuda::VoxelNormals& normals,
//                         glm::uvec3& curDim,
//                         size_t scaleX, size_t scaleY, size_t scaleZ,
//                         float& scalePctX, float& scalePctY, float& scalePctZ)
//{
//    cuda::VoxelColors scaledColors(scaleX * scaleY * scaleZ);
//    cuda::VoxelNormals scaledNormals(scaleX * scaleY * scaleZ);
//    
//    scalePctX = static_cast<float>(scaleX) / static_cast<float>(curDim.x);
//    scalePctY = static_cast<float>(scaleY) / static_cast<float>(curDim.y);
//    scalePctZ = static_cast<float>(scaleZ) / static_cast<float>(curDim.z);
//
//    float invScalePctX = 1.0f / scalePctX;
//    float invScalePctY = 1.0f / scalePctY;
//    float invScalePctZ = 1.0f / scalePctZ;
//
//    for(size_t z = 0; z < scaleZ; ++z)
//    {
//        size_t nearZ = static_cast<size_t>(std::floor((z * invScalePctZ) + 0.49f));
//        if(nearZ >= curDim.z)
//            nearZ = curDim.z-1;
//
//        for(size_t y = 0; y < scaleY; ++y)
//        {
//            size_t nearY = static_cast<size_t>(std::floor((y * invScalePctY) + 0.49f));
//            if(nearY >= curDim.y)
//                nearY = curDim.y-1;
//
//            for(size_t x = 0; x < scaleX; ++x)
//            {
//                size_t nearX = static_cast<size_t>(std::floor((x * invScalePctX) + 0.49f));
//                if(nearX >= curDim.x)
//                    nearX = curDim.x-1;
//
//                size_t nearIndex = (nearZ * curDim.y * curDim.x) 
//                                   + (nearY * curDim.x) 
//                                   + nearX;
//
//                scaledColors[(z * scaleY * scaleX) + (y * scaleX) + x] = 
//                    colors[nearIndex];
//
//                scaledNormals[(z * scaleY * scaleX) + (y * scaleX) + x] =
//                    normals[nearIndex];
//            }
//        }
//    }
//
//    scaledColors.swap(colors);
//    scaledNormals.swap(normals);
//
//    curDim.x = scaleX;
//    curDim.y = scaleY;
//    curDim.z = scaleZ;
//}

static bool WriteUByteBinaryBrick(std::ofstream& voxFile,
                                  size_t xOffset, size_t yOffset, size_t zOffset,
                                  size_t brickDimX, size_t brickDimY, size_t brickDimZ,
                                  const cuda::VoxelColors& voxelColors,
                                  const cuda::VoxelNormals& voxelNormals,
                                  size_t xSize, size_t ySize)
{   
    std::vector<osg::Vec4ub> brickColorsOut;
    struct Vec3ub
    {
        unsigned char x;
        unsigned char y;
        unsigned char z;
    };
    std::vector<Vec3ub> brickNormalsOut;

    brickColorsOut.reserve(brickDimX * brickDimY * brickDimZ);
    brickNormalsOut.reserve(brickDimX * brickDimY * brickDimZ);

    for(size_t z = zOffset; z < (brickDimZ + zOffset); ++z)
    {
        for(size_t y = yOffset; y < (brickDimY + yOffset); ++y)
        {
            const cuda::VoxColor* pColorsRow = &voxelColors[(z * ySize * xSize) + (y * xSize) + xOffset];
            brickColorsOut.resize(brickColorsOut.size() + brickDimX);
                
            const glm::vec3* pNormalsRow = &voxelNormals[(z * ySize * xSize) + (y * xSize) + xOffset];
            brickNormalsOut.resize(brickNormalsOut.size() + brickDimX);
            for(size_t x = 0; x < brickDimX; ++x)
            {
                const cuda::VoxColor& color = pColorsRow[x];
                osg::Vec4ub& ubColor = brickColorsOut[brickColorsOut.size() - brickDimX + x];
                if(sizeof(color) == sizeof(ubColor))
                {
                    memcpy(&ubColor, &color, sizeof(ubColor));
                }
                else
                {
                    ubColor.r() = static_cast<unsigned char>(color.x * 255.0f);
                    ubColor.g() = static_cast<unsigned char>(color.y * 255.0f);
                    ubColor.b() = static_cast<unsigned char>(color.z * 255.0f);
                    ubColor.a() = static_cast<unsigned char>(color.w * 255.0f);
                }

                const glm::vec3& normal = pNormalsRow[x];
                Vec3ub& ubNormal = brickNormalsOut[brickNormalsOut.size() - brickDimX + x];

                if(sizeof(normal) == sizeof(ubNormal))
                {
                    memcpy(&ubNormal, &normal, sizeof(ubNormal));
                }
                else
                {
                    //map to between zero and one
                    float nX = ((normal.x + 1.0f) * 0.5f);
                    float nY = ((normal.y + 1.0f) * 0.5f);
                    float nZ = ((normal.z + 1.0f) * 0.5f);
                    ubNormal.x = static_cast<unsigned char>(nX * 255.0f);
                    ubNormal.y = static_cast<unsigned char>(nY * 255.0f);
                    ubNormal.z = static_cast<unsigned char>(nZ * 255.0f);
                }
            }
        }
    }

    int compressedImage = 0;
    voxFile.write((char*)&compressedImage, sizeof(compressedImage));
    
    unsigned int dataSize = brickColorsOut.size() * sizeof(*brickColorsOut.data());
    voxFile.write((char*)&dataSize, sizeof(dataSize));
    voxFile.write((const char*)brickColorsOut.data(), dataSize);

    voxFile.write((char*)&compressedImage, sizeof(compressedImage));

    dataSize = brickNormalsOut.size() * sizeof(*brickNormalsOut.data());
    voxFile.write((char*)&dataSize, sizeof(dataSize));
    voxFile.write((const char*)brickNormalsOut.data(), dataSize);

    return !voxFile.fail();
}

static bool WriteCompressedBinaryBrick(std::ofstream& voxFile,
                                       size_t xOffset, size_t yOffset, size_t zOffset,
                                       size_t brickDimX, size_t brickDimY, size_t brickDimZ,
                                       const cuda::VoxelColors& voxelColors,
                                       const cuda::VoxelNormals& voxelNormals,
                                       size_t xSize, size_t ySize)
{   
    cuda::VoxelColors brickColorsOut;
    cuda::VoxelNormals brickNormalsOut;

    brickColorsOut.reserve(brickDimX * brickDimY * brickDimZ);
    brickNormalsOut.reserve(brickDimX * brickDimY * brickDimZ);

    for(size_t z = zOffset; z < (brickDimZ + zOffset); ++z)
    {
        for(size_t y = yOffset; y < (brickDimY + yOffset); ++y)
        {
            const cuda::VoxColor* pColorsRow = &voxelColors[(z * ySize * xSize) + (y * xSize) + xOffset];
            brickColorsOut.insert(brickColorsOut.end(), pColorsRow, pColorsRow + brickDimX);
                
            const glm::vec3* pNormalsRow = &voxelNormals[(z * ySize * xSize) + (y * xSize) + xOffset];
            brickNormalsOut.insert(brickNormalsOut.end(), pNormalsRow, pNormalsRow + brickDimX);
            for(size_t x = 0; x < brickDimX; ++x)
            {
                glm::vec3& normal = brickNormalsOut[brickNormalsOut.size() - brickDimX + x];
                //map to between zero and one
                float nX = ((normal.x + 1.0f) * 0.5f);
                float nY = ((normal.y + 1.0f) * 0.5f);
                float nZ = ((normal.z + 1.0f) * 0.5f);
                normal.x = nX;
                normal.y = nY;
                normal.z = nZ;
            }
        }
    }

    if(brickColorsOut.size() > 0)
    {
        osg::ref_ptr<osg::Image> spImage = new osg::Image();
        if(sizeof(cuda::VoxColor) == 4)
        {
            spImage->setImage(brickDimX, brickDimY, brickDimZ, 
                              GL_RGBA8, GL_RGBA, GL_UNSIGNED_BYTE, 
                              (unsigned char*)&brickColorsOut.front(), osg::Image::NO_DELETE);
        }
        else
        {
            spImage->setImage(brickDimX, brickDimY, brickDimZ, 
                              GL_RGBA32F_ARB, GL_RGBA, GL_FLOAT, 
                              (unsigned char*)&brickColorsOut.front(), osg::Image::NO_DELETE);
        }

        osg::ref_ptr<osg::Texture3D> spTexture = new osg::Texture3D(spImage);
        
        spTexture->setInternalFormatMode(osg::Texture::USE_S3TC_DXT5_COMPRESSION);

        spTexture->setResizeNonPowerOfTwoHint(false);
        spTexture->setUnRefImageDataAfterApply(false);
        //no need to generate mip maps
        spTexture->setUseHardwareMipMapGeneration(false);
        //this prevents cpu side mip map
        spTexture->setFilter(osg::Texture::MIN_FILTER, osg::Texture::NEAREST);

        unsigned int before = spImage->getTotalSizeInBytes();

        if(!CompressImage(spTexture.get()))
            return false;

        int compressedImage = 1;
        if(spImage->getInternalTextureFormat() != GL_COMPRESSED_RGBA_S3TC_DXT5_EXT)
        {
            std::cerr << "Brick colors compression failed " 
                      << GL_COMPRESSED_RGBA_S3TC_DXT5_EXT << " != " 
                      << spImage->getInternalTextureFormat() << std::endl;
            compressedImage = 0;
        }

        unsigned int after = spImage->getTotalSizeInBytes();

        voxFile.write((char*)&compressedImage, sizeof(compressedImage));
        voxFile.write((char*)&after, sizeof(after));
        voxFile.write((const char*)spImage->data(), after);
    }

    if(brickNormalsOut.size() > 0)
    {
        osg::ref_ptr<osg::Image> spImage = new osg::Image();
        if(sizeof(cuda::VoxNorm) == 4)
        {
            spImage->setImage(brickDimX, brickDimY, brickDimZ, 
                              GL_RGB8, GL_RGB, GL_UNSIGNED_BYTE, 
                              (unsigned char*)&brickNormalsOut.front(), osg::Image::NO_DELETE);
        }
        else
        {
            spImage->setImage(brickDimX, brickDimY, brickDimZ, 
                              GL_RGB32F_ARB, GL_RGB, GL_FLOAT, 
                              (unsigned char*)&brickNormalsOut.front(), osg::Image::NO_DELETE);
        }

        osg::ref_ptr<osg::Texture3D> spTexture = new osg::Texture3D(spImage);
        
        spTexture->setInternalFormatMode(osg::Texture::USE_S3TC_DXT1_COMPRESSION);

        spTexture->setResizeNonPowerOfTwoHint(false);
        spTexture->setUnRefImageDataAfterApply(false);
        //no need to generate mip maps
        spTexture->setUseHardwareMipMapGeneration(false);
        //this prevents cpu side mip maps
        spTexture->setFilter(osg::Texture::MIN_FILTER, osg::Texture::NEAREST);

        unsigned int before = spImage->getTotalSizeInBytes();

        if(!CompressImage(spTexture.get()))
            return false;

        int compressedImage = 1;
        if(spImage->getInternalTextureFormat() != GL_COMPRESSED_RGB_S3TC_DXT1_EXT)
        {
            std::cerr << "Brick gradient compression failed " 
                      << GL_COMPRESSED_RGB_S3TC_DXT1_EXT 
                      << " != " << spImage->getInternalTextureFormat() << std::endl;
            compressedImage = 0;
        }

        unsigned int after = spImage->getTotalSizeInBytes();
        
        voxFile.write((char*)&compressedImage, sizeof(compressedImage));
        voxFile.write((char*)&after, sizeof(after));
        voxFile.write((const char*)spImage->data(), after);
    }

    return !voxFile.fail();
}

static bool WriteBinaryBrick(std::ofstream& voxFile,
                             bool compressed,
                             size_t xOffset, size_t yOffset, size_t zOffset,
                             size_t xSize, size_t ySize, size_t zSize,
                             size_t brickDimX, size_t brickDimY, size_t brickDimZ,
                             const cuda::VoxelColors& voxelColors,
                             const cuda::VoxelNormals& voxelNormals)
{
    //bricks must contain border brick data  for correct interpolation.
    unsigned int borderVoxels = 1u;

    unsigned int brickBorderX(0u);
    unsigned int brickBorderY(0u);
    unsigned int brickBorderZ(0u);
    
    size_t xEnd = xOffset + brickDimX;
    if(xOffset != 0u && xEnd < xSize)//if it is middle brick
    {
        brickBorderX = borderVoxels;
        xEnd += borderVoxels;
        xOffset -= borderVoxels;
    }
    else if(xOffset == 0u && xEnd < xSize)//if it is left brick
    {
        xEnd += (borderVoxels << 1);
    }
    else if(xOffset != 0u) // must be right side brick
    {
        brickBorderX = borderVoxels;
        xOffset -= (borderVoxels << 1);
    }

    brickDimX = (xEnd - xOffset);

    size_t yEnd = yOffset + brickDimY;
    if(yOffset != 0u && yEnd < ySize)//if it is middle brick
    {
        brickBorderY = borderVoxels;
        yEnd += borderVoxels;
        yOffset -= borderVoxels;
    }
    else if(yOffset == 0u && yEnd < ySize)//if it is left brick
    {
        yEnd += (borderVoxels << 1);
    }
    else if(yOffset != 0u) // must be right side brick
    {
        brickBorderY = borderVoxels;
        yOffset -= (borderVoxels << 1);
    }

    brickDimY = (yEnd - yOffset);

    size_t zEnd = zOffset + brickDimZ;
    if(zOffset != 0u && zEnd < zSize)//if it is middle brick
    {
        brickBorderZ = borderVoxels;
        zEnd += borderVoxels;
        zOffset -= borderVoxels;
    }
    else if(zOffset == 0u && zEnd < zSize)//if it is left brick
    {
        zEnd += (borderVoxels << 1);
    }
    else if(zOffset != 0u) // must be right side brick
    {
        brickBorderZ = borderVoxels;
        zOffset -= (borderVoxels << 1);
    }

    brickDimZ = (zEnd - zOffset);

    //write brick dims
    voxFile.write((char*)&brickDimX, sizeof(brickDimX));
    voxFile.write((char*)&brickDimY, sizeof(brickDimY));
    voxFile.write((char*)&brickDimZ, sizeof(brickDimZ));
    //write the border
    voxFile.write((char*)&brickBorderX, sizeof(brickBorderX));
    voxFile.write((char*)&brickBorderY, sizeof(brickBorderY));
    voxFile.write((char*)&brickBorderZ, sizeof(brickBorderZ));

    if(compressed)
        return WriteCompressedBinaryBrick(voxFile, 
                                          xOffset, yOffset, zOffset,
                                          brickDimX, brickDimY, brickDimZ,
                                          voxelColors, voxelNormals,
                                          xSize, ySize);
    else
        return WriteUByteBinaryBrick(voxFile,
                                     xOffset, yOffset, zOffset,
                                     brickDimX, brickDimY, brickDimZ,
                                     voxelColors, voxelNormals,
                                     xSize, ySize);
}

bool VoxelBrickWriter::writeBrick(size_t nodeX, size_t nodeY, size_t nodeZ,
                                  size_t xOffset, size_t yOffset, size_t zOffset,
                                  size_t xSize, size_t ySize, size_t zSize,
                                  size_t brickDimX, size_t brickDimY, size_t brickDimZ,
                                  const cuda::VoxelColors& voxelColors,
                                  const cuda::VoxelNormals& voxelNormals)
{
    BrickID brickID(nodeX, nodeY, nodeZ);
#ifdef __DEBUG__
    size_t before = _brickMap.size();
#endif
    _brickMap[brickID] = _brickMap.size();
#ifdef __DEBUG__
    size_t after = _brickMap.size();
    if(before == after)
        return false;
#endif

    if(!_binary)
    {
        return WriteTextBrick(_voxFile,
                              xOffset, yOffset, zOffset,
                              xSize, ySize, zSize,
                              brickDimX, brickDimY, brickDimZ,
                              voxelColors,
                              voxelNormals);
    }
    else
    {
        return WriteBinaryBrick(_voxFile,
                                _compressed,
                                xOffset, yOffset, zOffset,
                                xSize, ySize, zSize,
                                brickDimX, brickDimY, brickDimZ,
                                voxelColors,
                                voxelNormals);
    }
}

static size_t s_totalPartialBricksAllocatedBytes = 0;
static size_t s_maxPartialBricksAllocatedBytes = 1200000000;//1.75GB

VoxelBrickWriter::BrickData::~BrickData()
{
    if(this->colors.size() > 0)
        s_totalPartialBricksAllocatedBytes -= (this->colors.size() * sizeof(cuda::VoxColor));
    if(this->normals.size() > 0)
        s_totalPartialBricksAllocatedBytes -= (this->normals.size() * sizeof(glm::vec3));
}

bool VoxelBrickWriter::BrickData::init(const std::string& partialBrickDataDir,
                                       size_t nodeX, size_t nodeY, size_t nodeZ)
{
    if(s_totalPartialBricksAllocatedBytes < s_maxPartialBricksAllocatedBytes)
    {
        this->storeBrickDataInFiles = false;
        this->colors.resize(this->brickDimX *
                            this->brickDimY *
                            this->brickDimZ);

        this->normals.resize(this->brickDimX *
                             this->brickDimY *
                             this->brickDimZ);

        s_totalPartialBricksAllocatedBytes += (this->colors.size() * sizeof(cuda::VoxColor));
        s_totalPartialBricksAllocatedBytes += (this->normals.size() * sizeof(glm::vec3));
    }
    else
    {
        this->storeBrickDataInFiles = true;
        std::stringstream colorsFilePath;
        colorsFilePath << partialBrickDataDir << "/colors" << nodeX << "_" << nodeY << "_" << nodeZ << ".voxb";
    
        partialBrickColorsFilePath = colorsFilePath.str();

        std::ofstream partialBrickColorsFile(partialBrickColorsFilePath,
                                             std::ios_base::out | std::ios_base::binary);

        size_t fileSize = sizeof(cuda::VoxColor) * 
                            (this->brickDimX *
                             this->brickDimY *
                             this->brickDimZ);
        int zero = 0;
        partialBrickColorsFile.seekp(fileSize - sizeof(zero));
        partialBrickColorsFile.write((char*)&zero, sizeof(zero));
        
        std::stringstream normalsFilePath;
        normalsFilePath << partialBrickDataDir << "/normals" << nodeX << "_" << nodeY << "_" << nodeZ << ".voxb";
    
        partialBrickNormalsFilePath = normalsFilePath.str();

        std::ofstream partialBrickNormalsFile(partialBrickNormalsFilePath,
                                              std::ios_base::out | std::ios_base::binary);

        fileSize = sizeof(glm::vec3) * 
                            (this->brickDimX *
                             this->brickDimY *
                             this->brickDimZ);

        partialBrickNormalsFile.seekp(fileSize - sizeof(zero));
        partialBrickNormalsFile.write((char*)&zero, sizeof(zero));

        if(partialBrickColorsFile.is_open() == false
            || partialBrickNormalsFile.is_open() == false
            || partialBrickColorsFile.fail()
            || partialBrickColorsFile.bad()
            || partialBrickNormalsFile.fail()
            || partialBrickNormalsFile.bad())
        {
            return false;
        }
    }

    return true;
}

bool VoxelBrickWriter::BrickData::dumpMemoryToPartialBrickFiles()
{
    std::ofstream partialBrickColorsFile(partialBrickColorsFilePath,
                                         std::ios_base::in | std::ios_base::out | std::ios_base::binary);

    size_t dataSize = sizeof(cuda::VoxColor) * 
                        (this->brickDimX *
                        this->brickDimY *
                        this->brickDimZ);

    partialBrickColorsFile.write((const char*)this->colors.data(),
                                        dataSize);
    {
        //free memory
        s_totalPartialBricksAllocatedBytes -= (this->colors.size() * sizeof(cuda::VoxColor));
        cuda::VoxelColors empty;
        empty.swap(this->colors);
    }

    std::ofstream partialBrickNormalsFile(partialBrickNormalsFilePath,
                                          std::ios_base::in | std::ios_base::out | std::ios_base::binary);
   
    dataSize = sizeof(glm::vec3) * 
                        (this->brickDimX *
                        this->brickDimY *
                        this->brickDimZ);

    partialBrickColorsFile.write((const char*)this->normals.data(),
                                        dataSize);

    {
        s_totalPartialBricksAllocatedBytes -= (this->normals.size() * sizeof(glm::vec3));
        //free memory
        cuda::VoxelNormals empty;
        empty.swap(this->normals);
    }

    if(partialBrickColorsFile.is_open() == false
        || partialBrickNormalsFile.is_open() == false
        || partialBrickColorsFile.fail()
        || partialBrickColorsFile.bad()
        || partialBrickNormalsFile.fail()
        || partialBrickNormalsFile.bad())
    {
        partialBrickNormalsFile.close();
        partialBrickColorsFile.close();
        return false;
    }

    partialBrickNormalsFile.close();
    partialBrickColorsFile.close();

    return true;
}

bool VoxelBrickWriter::BrickData::loadPartialBrickFiles()
{
    this->colors.resize(this->brickDimX *
                        this->brickDimY *
                        this->brickDimZ);
    s_totalPartialBricksAllocatedBytes += (this->colors.size() * sizeof(cuda::VoxColor));

    std::ifstream colorsFile;
    colorsFile.open(partialBrickColorsFilePath.c_str(),
                    std::ios_base::in | std::ios_base::binary);
    if(!colorsFile.is_open())
        return false;

    colorsFile.read((char*)this->colors.data(),
                    colors.size() * sizeof(cuda::VoxColor));
    colorsFile.close();

    this->normals.resize(this->brickDimX *
                         this->brickDimY *
                         this->brickDimZ);
    s_totalPartialBricksAllocatedBytes += (this->normals.size() * sizeof(glm::vec3));

    std::ifstream normalsFile;
    normalsFile.open(partialBrickNormalsFilePath.c_str(),
                     std::ios_base::in | std::ios_base::binary);
    if(!normalsFile.is_open())
        return false;

    normalsFile.read((char*)normals.data(),
                     normals.size() * sizeof(glm::vec3));
    normalsFile.close();

    return true;
}

bool VoxelBrickWriter::storePartialBrick(size_t nodeX, size_t nodeY, size_t nodeZ,
                                         size_t brickX, size_t brickY, size_t brickZ,
                                         size_t xOffset, size_t yOffset, size_t zOffset,
                                         size_t xSize, size_t ySize, size_t zSize,
                                         size_t brickDimX, size_t brickDimY, size_t brickDimZ,
                                         const cuda::VoxelColors& voxelColors,
                                         const cuda::VoxelNormals& voxelNormals,
                                         size_t voxDimX, size_t voxDimY, size_t voxDimZ)
{
    BrickID brickID(nodeX, nodeY, nodeZ);

    BrickDataMap::iterator brickDataItr = _storedBrickMap.find(brickID);
    if(brickDataItr == _storedBrickMap.end())
    {
        size_t borderVoxels(2u);
        std::pair<BrickDataMap::iterator, bool> ret = 
            _storedBrickMap.insert(std::make_pair(brickID, 
                                                  BrickData(brickDimX + borderVoxels, 
                                                            brickDimY + borderVoxels, 
                                                            brickDimZ + borderVoxels)));
        if(ret.second == false)
            return false;
        brickDataItr = ret.first;
        if(!brickDataItr->second.init(_outputPartialBricksDir,
                                  nodeX, nodeY, nodeZ))
            return false;                             
    }

    CopyPartialBrick(brickDataItr->second,
                     brickX, brickY, brickZ,
                     xOffset, yOffset, zOffset,
                     xSize, ySize, zSize,
                     brickDataItr->second.brickDimX,
                     brickDataItr->second.brickDimY,
                     //brickDataItr->second.brickDimZ,
                     voxelColors, voxelNormals,
                     voxDimX, voxDimY, voxDimZ);
    return true;
}

static inline bool VEC4_EQUAL(const glm::vec4& v1, const glm::vec4& v2)
{
    static float epsilon = 0.001f;
    glm::vec4 diff = v1 - v2;

    return (diff.x > -epsilon && diff.x < epsilon) &&
           (diff.y > -epsilon && diff.y < epsilon) &&
           (diff.z > -epsilon && diff.z < epsilon) &&
           (diff.w > -epsilon && diff.w < epsilon);
}

static inline bool VEC4_EQUAL(const uchar4& v1, const uchar4& v2)
{
    return v1.x == v2.x && v1.y == v2.y && v1.z == v2.z && v1.w == v2.w;
}

static bool ValidateNode(size_t xOffset, size_t yOffset, size_t zOffset,
                         size_t xSize, size_t ySize, size_t zSize,
                         const glm::uvec3& brickDim,
                         const cuda::VoxelColors& voxelColors,
                         glm::uint type)
{
    size_t startX = xOffset;
    if(xOffset != 0)
        startX -= 1;

    size_t endX = xOffset + brickDim.x;
    if(endX != xSize)
        endX += 1;

    size_t startY = yOffset;
    if(yOffset != 0)
        startY -= 1;

    size_t endY = yOffset + brickDim.y;
    if(endY != ySize)
        endY += 1;

    size_t startZ = zOffset;
    if(zOffset != 0)
        startZ -= 1;

    size_t endZ = zOffset + brickDim.z;
    if(endZ != zSize)
        endZ += 1;

    const cuda::VoxColor& constColor = voxelColors[(zOffset * xSize * ySize) + (yOffset * xSize) + xOffset];
    for(size_t z = startZ; z < endZ; ++z)
    {
        for(size_t y = startY; y < endY; ++y)
        {
            for(size_t x = startX; x < endX; ++x)
            {
                const cuda::VoxColor& curColor = voxelColors[(z * xSize * ySize) + (y * xSize) + x];
                if(VEC4_EQUAL(curColor, constColor) == false)
                {
                    return type == 1u;
                }
            }
        }
    }

    return type == 0u || type == 2u;
}

bool VoxelBrickWriter::writeCompletedStoredBricks(const unsigned int* pOctTreeNodes,
                                         size_t octTreeDimX,
                                         size_t octTreeDimY)
{
    bool retVal = true;
    std::vector<BrickID> completedBricks;
    for(BrickDataMap::iterator itr = _storedBrickMap.begin();
        itr != _storedBrickMap.end();
        ++itr)
    {
        size_t octTreeNodeIndex = (itr->first.z * octTreeDimY * octTreeDimX)
                                + (itr->first.y * octTreeDimY)
                                + itr->first.x;
        if(itr->second.isComplete())
        {
#ifdef __DEBUG__
            if(!itr->second.brickDataStoredInMemory())
            {
                if(!itr->second.loadPartialBrickFiles())
                {
                    std::cerr << "Failed to load partial brick files." << std::endl;
                }
            }

            if(!ValidateNode(1, 1, 1,
                           itr->second.brickDimX,
                           itr->second.brickDimY,
                           itr->second.brickDimZ,
                           glm::uvec3(itr->second.brickDimX - 2u,
                           itr->second.brickDimY - 2u,
                           itr->second.brickDimZ - 2u),
                           itr->second.colors,
                           pOctTreeNodes[octTreeNodeIndex]))
            {
                 std::cerr << "Failed to validate node." << std::endl;
            }
#endif
            if(pOctTreeNodes[octTreeNodeIndex] == 1u)
            {
#ifndef __DEBUG__
                if(!itr->second.brickDataStoredInMemory())
                {
                    if(!itr->second.loadPartialBrickFiles())
                    {
                        std::cerr << "Failed to load partial brick files." << std::endl;
                    }
                }
#endif
                if(!writeBrick(itr->first.x, itr->first.y, itr->first.z,
                                1, 1, 1,
                                itr->second.brickDimX,
                                itr->second.brickDimY,
                                itr->second.brickDimZ,
                                itr->second.brickDimX - 2u,
                                itr->second.brickDimY - 2u,
                                itr->second.brickDimZ - 2u,
                                itr->second.colors,
                                itr->second.normals))
                {
                    std::cerr << "Error writing stored brick." << std::endl;
                    retVal = false;
                }
            }
            std::remove(itr->second.partialBrickColorsFilePath.c_str());
            std::remove(itr->second.partialBrickNormalsFilePath.c_str());
            completedBricks.push_back(itr->first);
        }
    }

    //erase any bricks that have been completed 
    //(i.e. all voxels have been examined and stored)
    for(std::vector<BrickID>::iterator itr = completedBricks.begin();
        itr != completedBricks.end();
        ++itr)
    {
        _storedBrickMap.erase(*itr);
    }

    return retVal;
}

bool VoxelBrickWriter::ConvertImageToRGBA8(osg::Image* pImage)
{
    osg::ref_ptr<osg::Texture2D> spTexture = new osg::Texture2D(pImage);
    spTexture->setInternalFormatMode(osg::Texture::USE_IMAGE_DATA_FORMAT);
    spTexture->setResizeNonPowerOfTwoHint(false);
    spTexture->setUnRefImageDataAfterApply(false);
    //no need to generate mip maps
    spTexture->setUseHardwareMipMapGeneration(false);
    //this prevents cpu side mip maps
    spTexture->setFilter(osg::Texture::MIN_FILTER, osg::Texture::NEAREST);

    unsigned int before = pImage->getTotalSizeInBytes();

    if(!DeCompress(spTexture.get()))
        return false;

    unsigned int after = pImage->getTotalSizeInBytes();
    GLint fmt = pImage->getInternalTextureFormat();
    return fmt == GL_RGBA8;
}

bool VoxelBrickWriter::ExportToFileHeader(size_t voxDimX, size_t voxDimY, size_t voxDimZ,
                                          const std::string& headerFileName,
                                          bool formatIsUByte,
                                          const std::vector<std::string>& voxelColorFiles,
                                          const std::vector<glm::uvec3>& voxelColorStartRanges,
                                          const std::vector<glm::uvec3>& voxelColorEndRanges)
{
    std::ofstream voxFile;
    voxFile.open(headerFileName, std::fstream::out);
    if(!voxFile.is_open())
    {
        return false;
    }

    voxFile << "VOXEL_HEADER" << std::endl;
    voxFile << "POSITION " 
            << -1.0f * static_cast<float>(voxDimX >> 1) << " "
            << -1.0f * static_cast<float>(voxDimY >> 1) << " "
            << -1.0f * static_cast<float>(voxDimZ >> 1) << std::endl;
    voxFile << "ORIENTATION 0 0 1 0" << std::endl;
    voxFile << "SCALE 1" << std::endl;
    voxFile << "DIMENSIONS "
            << voxDimX << " "
            << voxDimY << " "
            << voxDimZ << std::endl;
    voxFile << "TYPE COLORS" << std::endl;
    voxFile << "FORMAT " << (formatIsUByte ? "GL_RGBA8" : "GL_RGBF32") << std::endl;
    voxFile << "VOXEL_HEADER_END" << std::endl;
    voxFile << "VOXEL_SUB_IMAGE_FILES" << std::endl;
    for(size_t i = 0; i < voxelColorFiles.size(); ++i)
    {
        const std::string& voxelBinaryFile = voxelColorFiles.at(i);
        const glm::uvec3& subRangeStart = voxelColorStartRanges.at(i);
        const glm::uvec3& subRangeEnd = voxelColorEndRanges.at(i);

        voxFile << "VOXEL_SUB_IMAGE_FILE " 
                << subRangeStart.x << " " << subRangeStart.y << " " << subRangeStart.z << " "
                << subRangeEnd.x << " " << subRangeEnd.y << " " << subRangeEnd.z << " "
                << voxelBinaryFile
                << std::endl;
    }
    voxFile.close();

    return true;
}

bool VoxelBrickWriter::ExportToFile(const cuda::VoxelColors& voxelColors,
                                    const cuda::VoxelNormals& voxelNormals,
                                    bool convertToUByte,
                                    size_t voxDimX, size_t voxDimY, size_t voxDimZ,
                                    size_t xStart, size_t yStart, size_t zStart,
                                    size_t xEnd, size_t yEnd, size_t zEnd,
                                    const std::string& outputFileName)
{    
    std::ofstream voxColors;
    voxColors.open(outputFileName, std::ios_base::out | std::ios_base::binary);
    if(voxColors.is_open() == false)
        return false;
    int formatIsUByte = convertToUByte ? 1 : 0;
    voxColors.write((const char*)&formatIsUByte, sizeof(formatIsUByte));
    size_t xSize = xEnd - xStart;
    size_t ySize = yEnd - yStart;
    size_t zSize = zEnd - zStart;
    unsigned int dataSize;
    if(convertToUByte)
        dataSize = sizeof(osg::Vec4ub)
                      * xSize * ySize * zSize;
    else
        dataSize = sizeof(cuda::VoxelColors::value_type)
                      * xSize * ySize * zSize;
    voxColors.write((const char*)&dataSize, sizeof(dataSize));
    for(size_t z = zStart; z < zEnd; ++z)
    {
        for(size_t y = yStart; y < yEnd; ++y)
        {
            if(!convertToUByte || sizeof(cuda::VoxelColors::value_type) == 4)
            {
                size_t rowSize = xSize * sizeof(cuda::VoxelColors::value_type);
                size_t index = (z * voxDimX * voxDimY) + (y * voxDimX) + xStart;
                voxColors.write((const char*)(&voxelColors[index]),
                                rowSize);
            }
            else
            {
                std::vector<osg::Vec4ub> colorsRow;
                colorsRow.reserve(xSize);
                for(size_t x = xStart; x < xEnd; ++x)
                {
                    colorsRow.resize(colorsRow.size() + 1);
                    osg::Vec4ub& color = colorsRow.back();
                    size_t index = (z * voxDimX * voxDimY) + (y * voxDimX) + x;
                    const cuda::VoxelColors::value_type& fColor = voxelColors.at(index);
                    color.r() = static_cast<unsigned char>(fColor.x * 255.0f);
                    color.g() = static_cast<unsigned char>(fColor.y * 255.0f);
                    color.b() = static_cast<unsigned char>(fColor.z * 255.0f);
                    color.a() = static_cast<unsigned char>(fColor.w * 255.0f);
                }
                voxColors.write((const char*)&colorsRow[0], sizeof(osg::Vec4ub) * xSize);
            }
        }
    }
    voxColors.close();
    
    return true;
}